// RUN: %run_test hipify "%s" "%t" %hipify_args 4 --skip-excluded-preprocessor-conditional-blocks --experimental --roc --use-hip-data-types %clang_args -ferror-limit=500

// CHECK: #include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
// CHECK: #include "hip/hip_complex.h"
#include "hip/hip_complex.h"
#include <stdio.h>
// CHECK: #include "rocsparse.h"
#include "hipsparse.h"
// CHECK-NOT: #include "rocsparse.h"

int main() {
  printf("18. cuSPARSE API to rocSPARSE API synthetic test\n");

  // CHECK: _rocsparse_handle *handle = nullptr;
  // CHECK-NEXT: rocsparse_handle handle_t;
  cusparseContext *handle = nullptr;
  hipsparseHandle_t handle_t;

  // CHECK: _rocsparse_mat_descr *matDescr = nullptr;
  // CHECK-NEXT: rocsparse_mat_descr matDescr_t, matDescr_t_2, matDescr_A, matDescr_C;
  cusparseMatDescr *matDescr = nullptr;
  hipsparseMatDescr_t matDescr_t, matDescr_t_2, matDescr_A, matDescr_C;

  // CHECK: _rocsparse_color_info *colorInfo = nullptr;
  // CHECK-NEXT: rocsparse_color_info colorInfo_t;
  cusparseColorInfo *colorInfo = nullptr;
  hipsparseColorInfo_t colorInfo_t;

  // CHECK: rocsparse_operation sparseOperation_t;
  // CHECK-NEXT: rocsparse_operation OPERATION_NON_TRANSPOSE = rocsparse_operation_none;
  // CHECK-NEXT: rocsparse_operation OPERATION_TRANSPOSE = rocsparse_operation_transpose;
  // CHECK-NEXT: rocsparse_operation OPERATION_CONJUGATE_TRANSPOSE = rocsparse_operation_conjugate_transpose;
  hipsparseOperation_t sparseOperation_t;
  hipsparseOperation_t OPERATION_NON_TRANSPOSE = HIPSPARSE_OPERATION_NON_TRANSPOSE;
  hipsparseOperation_t OPERATION_TRANSPOSE = HIPSPARSE_OPERATION_TRANSPOSE;
  hipsparseOperation_t OPERATION_CONJUGATE_TRANSPOSE = HIPSPARSE_OPERATION_CONJUGATE_TRANSPOSE;

  // CHECK: rocsparse_index_base indexBase_t;
  // CHECK-NEXT: rocsparse_index_base INDEX_BASE_ZERO = rocsparse_index_base_zero;
  // CHECK-NEXT: rocsparse_index_base INDEX_BASE_ONE = rocsparse_index_base_one;
  hipsparseIndexBase_t indexBase_t;
  hipsparseIndexBase_t INDEX_BASE_ZERO = HIPSPARSE_INDEX_BASE_ZERO;
  hipsparseIndexBase_t INDEX_BASE_ONE = HIPSPARSE_INDEX_BASE_ONE;

  // CHECK: rocsparse_matrix_type matrixType_t;
  // CHECK-NEXT: rocsparse_matrix_type MATRIX_TYPE_GENERAL = rocsparse_matrix_type_general;
  // CHECK-NEXT: rocsparse_matrix_type MATRIX_TYPE_SYMMETRIC = rocsparse_matrix_type_symmetric;
  // CHECK-NEXT: rocsparse_matrix_type MATRIX_TYPE_HERMITIAN = rocsparse_matrix_type_hermitian;
  // CHECK-NEXT: rocsparse_matrix_type MATRIX_TYPE_TRIANGULAR = rocsparse_matrix_type_triangular;
  hipsparseMatrixType_t matrixType_t;
  hipsparseMatrixType_t MATRIX_TYPE_GENERAL = HIPSPARSE_MATRIX_TYPE_GENERAL;
  hipsparseMatrixType_t MATRIX_TYPE_SYMMETRIC = HIPSPARSE_MATRIX_TYPE_SYMMETRIC;
  hipsparseMatrixType_t MATRIX_TYPE_HERMITIAN = HIPSPARSE_MATRIX_TYPE_HERMITIAN;
  hipsparseMatrixType_t MATRIX_TYPE_TRIANGULAR = HIPSPARSE_MATRIX_TYPE_TRIANGULAR;

  // CHECK: rocsparse_diag_type diagType_t;
  // CHECK-NEXT: rocsparse_diag_type DIAG_TYPE_NON_UNIT = rocsparse_diag_type_non_unit;
  // CHECK-NEXT: rocsparse_diag_type DIAG_TYPE_UNIT = rocsparse_diag_type_unit;
  hipsparseDiagType_t diagType_t;
  hipsparseDiagType_t DIAG_TYPE_NON_UNIT = HIPSPARSE_DIAG_TYPE_NON_UNIT;
  hipsparseDiagType_t DIAG_TYPE_UNIT = HIPSPARSE_DIAG_TYPE_UNIT;

  // CHECK: rocsparse_fill_mode fillMode_t;
  // CHECK-NEXT: rocsparse_fill_mode FILL_MODE_LOWER = rocsparse_fill_mode_lower;
  // CHECK-NEXT: rocsparse_fill_mode FILL_MODE_UPPER = rocsparse_fill_mode_upper;
  hipsparseFillMode_t fillMode_t;
  hipsparseFillMode_t FILL_MODE_LOWER = HIPSPARSE_FILL_MODE_LOWER;
  hipsparseFillMode_t FILL_MODE_UPPER = HIPSPARSE_FILL_MODE_UPPER;

  // CHECK: rocsparse_action action_t, copyValues;
  // CHECK-NEXT: rocsparse_action ACTION_SYMBOLIC = rocsparse_action_symbolic;
  // CHECK-NEXT: rocsparse_action ACTION_NUMERIC = rocsparse_action_numeric;
  hipsparseAction_t action_t, copyValues;
  hipsparseAction_t ACTION_SYMBOLIC = HIPSPARSE_ACTION_SYMBOLIC;
  hipsparseAction_t ACTION_NUMERIC = HIPSPARSE_ACTION_NUMERIC;

  // CHECK: rocsparse_direction direction_t;
  // CHECK-NEXT: rocsparse_direction DIRECTION_ROW = rocsparse_direction_row;
  // CHECK-NEXT: rocsparse_direction DIRECTION_COLUMN = rocsparse_direction_column;
  hipsparseDirection_t direction_t;
  hipsparseDirection_t DIRECTION_ROW = HIPSPARSE_DIRECTION_ROW;
  hipsparseDirection_t DIRECTION_COLUMN = HIPSPARSE_DIRECTION_COLUMN;

  // CHECK: rocsparse_solve_policy solvePolicy_t;
  // CHECK-NEXT: rocsparse_solve_policy SOLVE_POLICY_NO_LEVEL = rocsparse_solve_policy_auto;
  // CHECK-NEXT: rocsparse_solve_policy SOLVE_POLICY_USE_LEVEL = rocsparse_solve_policy_auto;
  hipsparseSolvePolicy_t solvePolicy_t;
  hipsparseSolvePolicy_t SOLVE_POLICY_NO_LEVEL = HIPSPARSE_SOLVE_POLICY_NO_LEVEL;
  hipsparseSolvePolicy_t SOLVE_POLICY_USE_LEVEL = HIPSPARSE_SOLVE_POLICY_USE_LEVEL;

  // CHECK: rocsparse_pointer_mode pointerMode_t;
  // CHECK-NEXT: rocsparse_pointer_mode POINTER_MODE_HOST = rocsparse_pointer_mode_host;
  // CHECK-NEXT: rocsparse_pointer_mode POINTER_MODE_DEVICE = rocsparse_pointer_mode_device;
  hipsparsePointerMode_t pointerMode_t;
  hipsparsePointerMode_t POINTER_MODE_HOST = HIPSPARSE_POINTER_MODE_HOST;
  hipsparsePointerMode_t POINTER_MODE_DEVICE = HIPSPARSE_POINTER_MODE_DEVICE;

  // CHECK: rocsparse_status status_t;
  // CHECK-NEXT: rocsparse_status STATUS_SUCCESS = rocsparse_status_success;
  // CHECK-NEXT: rocsparse_status STATUS_NOT_INITIALIZED = rocsparse_status_not_initialized;
  // CHECK-NEXT: rocsparse_status STATUS_ALLOC_FAILED = rocsparse_status_memory_error;
  // CHECK-NEXT: rocsparse_status STATUS_INVALID_VALUE = rocsparse_status_invalid_value;
  // CHECK-NEXT: rocsparse_status STATUS_ARCH_MISMATCH = rocsparse_status_arch_mismatch;
  // CHECK-NEXT: rocsparse_status STATUS_INTERNAL_ERROR = rocsparse_status_internal_error;
  // CHECK-NEXT: rocsparse_status STATUS_ZERO_PIVOT = rocsparse_status_zero_pivot;
  hipsparseStatus_t status_t;
  hipsparseStatus_t STATUS_SUCCESS = HIPSPARSE_STATUS_SUCCESS;
  hipsparseStatus_t STATUS_NOT_INITIALIZED = HIPSPARSE_STATUS_NOT_INITIALIZED;
  hipsparseStatus_t STATUS_ALLOC_FAILED = HIPSPARSE_STATUS_ALLOC_FAILED;
  hipsparseStatus_t STATUS_INVALID_VALUE = HIPSPARSE_STATUS_INVALID_VALUE;
  hipsparseStatus_t STATUS_ARCH_MISMATCH = HIPSPARSE_STATUS_ARCH_MISMATCH;
  hipsparseStatus_t STATUS_INTERNAL_ERROR = HIPSPARSE_STATUS_INTERNAL_ERROR;
  hipsparseStatus_t STATUS_ZERO_PIVOT = HIPSPARSE_STATUS_ZERO_PIVOT;

  // CHECK: hipStream_t stream_t;
  hipStream_t stream_t;

  int iVal = 0;
  int batchCount = 0;
  int m = 0;
  int n = 0;
  int mb = 0;
  int nb = 0;
  int nnza = 0;
  int nnzb = 0;
  int nnzc = 0;
  int nnzPerRow = 0;
  int nnzPerCol = 0;
  int innz = 0;
  int lda = 0;
  int blockDim = 0;
  int csrSortedRowPtr = 0;
  int csrSortedColInd = 0;
  int cscRowIndA = 0;
  int cscColPtrA = 0;
  int csrRowPtrA = 0;
  int csrColIndA = 0;
  int ncolors = 0;
  int coloring = 0;
  int reordering = 0;
  int bscRowInd = 0;
  int bsrRowPtrA = 0;
  int bsrRowPtrC = 0;
  int csrRowPtrC = 0;
  int bscColPtr = 0;
  int bsrColIndA = 0;
  int bsrColIndC = 0;
  int csrColIndC = 0;
  int rowBlockDim = 0;
  int rowBlockDimA = 0;
  int colBlockDimA = 0;
  int rowBlockDimC = 0;
  int colBlockDim = 0;
  int colBlockDimC = 0;
  int bsrSortedRowPtr = 0;
  int bsrSortedRowPtrC = 0;
  int bsrSortedColInd = 0;
  int bsrSortedColIndC = 0;
  int bufferSizeInBytes = 0;
  int nnzTotalDevHostPtr = 0;
  int nnzPerRowCol = 0;
  int userEllWidth = 0;
  int64_t size = 0;
  int64_t nnz = 0;
  int64_t rows = 0;
  int64_t cols = 0;
  int64_t ellCols = 0;
  int64_t ellBlockSize = 0;
  int64_t batchStride = 0;
  int64_t offsetsBatchStride = 0;
  int64_t columnsValuesBatchStride = 0;
  int64_t ld = 0;
  void *indices = nullptr;
  void *values = nullptr;
  void *cooRowInd = nullptr;
  int icooRowInd = 0;
  void *cscRowInd = nullptr;
  void *csrColInd = nullptr;
  void *cooColInd = nullptr;
  void *ellColInd = nullptr;
  void *cooValues = nullptr;
  void *csrValues = nullptr;
  void *cscValues = nullptr;
  void *ellValue = nullptr;
  void *csrRowOffsets = nullptr;
  void *cscColOffsets = nullptr;
  void *cooRows = nullptr;
  int icooRows = 0;
  void *cooColumns = nullptr;
  int icooColumns = 0;
  void *data = nullptr;
  void *alpha = nullptr;
  void *beta = nullptr;
  void *pBuffer = nullptr;
  int *P = nullptr;
  void *tempBuffer = nullptr;
  void *c_coeff = nullptr;
  void *s_coeff = nullptr;
  size_t dataSize = 0;
  size_t bufferSize = 0;
  double dfractionToColor = 0.f;
  float ffractionToColor = 0.f;
  double bsrValA = 0.f;
  double csrValA = 0.f;
  float fcsrValA = 0.f;
  double csrValC = 0.f;
  float fcsrValC = 0.f;
  float csrSortedVal = 0.f;
  float cscSortedVal = 0.f;
  float csrSortedValA = 0.f;
  double dcsrSortedVal = 0.f;
  double dcscSortedVal = 0.f;
  double dcsrSortedValA = 0.f;
  double dbsrSortedVal = 0.f;
  double dbsrSortedValA = 0.f;
  double dbsrSortedValC = 0.f;
  float fbsrSortedVal = 0.f;
  float fbsrSortedValA = 0.f;
  float fbsrSortedValC = 0.f;
  float fcsrSortedValC = 0.f;
  double dcsrSortedValC = 0.f;
  double percentage = 0.f;
  float fpercentage = 0.f;
  double dthreshold = 0.f;
  float fthreshold = 0.f;
  double dtol = 0.f;
  float ftol = 0.f;
  double dbscVal = 0.f;
  float fbscVal = 0.f;
  double dA = 0.f;
  float fA = 0.f;

  // CHECK: rocsparse_mat_info prune_info;
  pruneInfo_t prune_info;

  // TODO: should be rocsparse_double_complex
  // TODO: add to TypeOverloads cuDoubleComplex -> rocsparse_double_complex under a new option --sparse
  // CHECK: rocblas_double_complex dcomplex, dcomplexA, dComplexbsrSortedValA, dComplexbsrSortedValC, dComplexcsrSortedValA, dComplexcsrSortedValC, dcomplextol, dComplexbsrSortedVal, dComplexbscVal, dComplexcscSortedVal;
  hipDoubleComplex dcomplex, dcomplexA, dComplexbsrSortedValA, dComplexbsrSortedValC, dComplexcsrSortedValA, dComplexcsrSortedValC, dcomplextol, dComplexbsrSortedVal, dComplexbscVal, dComplexcscSortedVal;

  // TODO: should be rocsparse_double_complex
  // TODO: add to TypeOverloads cuComplex -> rocsparse_float_complex under a new option --sparse
  // CHECK: rocblas_float_complex complex, complexA, complexbsrValA, complexbsrSortedValC, complexcsrSortedValA, complexcsrSortedValC, complextol, complexbsrSortedVal, complexbscVal, complexcscSortedVal;
  hipComplex complex, complexA, complexbsrValA, complexbsrSortedValC, complexcsrSortedValA, complexcsrSortedValC, complextol, complexbsrSortedVal, complexbscVal, complexcscSortedVal;

  // CHECK: rocsparse_operation opA, opB;
  hipsparseOperation_t opA, opB;

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseCreate(cusparseHandle_t* handle);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_create_handle(rocsparse_handle* handle);
  // CHECK: status_t = rocsparse_create_handle(&handle_t);
  status_t = hipsparseCreate(&handle_t);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseDestroy(cusparseHandle_t handle);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_destroy_handle(rocsparse_handle handle);
  // CHECK: status_t = rocsparse_destroy_handle(handle_t);
  status_t = hipsparseDestroy(handle_t);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseSetStream(cusparseHandle_t handle, cudaStream_t streamId);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_set_stream(rocsparse_handle handle, hipStream_t stream);
  // CHECK: status_t = rocsparse_set_stream(handle_t, stream_t);
  status_t = hipsparseSetStream(handle_t, stream_t);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseGetStream(cusparseHandle_t handle, cudaStream_t* streamId);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_get_stream(rocsparse_handle handle, hipStream_t* stream);
  // CHECK: status_t = rocsparse_get_stream(handle_t, &stream_t);
  status_t = hipsparseGetStream(handle_t, &stream_t);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseSetPointerMode(cusparseHandle_t handle, cusparsePointerMode_t mode);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_set_pointer_mode(rocsparse_handle handle, rocsparse_pointer_mode pointer_mode);
  // CHECK: status_t = rocsparse_set_pointer_mode(handle_t, pointerMode_t);
  status_t = hipsparseSetPointerMode(handle_t, pointerMode_t);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseGetPointerMode(cusparseHandle_t handle, cusparsePointerMode_t* mode);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_get_pointer_mode(rocsparse_handle handle, rocsparse_pointer_mode* pointer_mode);
  // CHECK: status_t = rocsparse_get_pointer_mode(handle_t, &pointerMode_t);
  status_t = hipsparseGetPointerMode(handle_t, &pointerMode_t);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseGetVersion(cusparseHandle_t handle, int* version);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_get_version(rocsparse_handle handle, int* version);
  // CHECK: status_t = rocsparse_get_version(handle_t, &iVal);
  status_t = hipsparseGetVersion(handle_t, &iVal);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseCreateMatDescr(cusparseMatDescr_t* descrA);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_create_mat_descr(rocsparse_mat_descr* descr);
  // CHECK: status_t = rocsparse_create_mat_descr(&matDescr_t);
  status_t = hipsparseCreateMatDescr(&matDescr_t);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseDestroyMatDescr(cusparseMatDescr_t descrA);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_destroy_mat_descr(rocsparse_mat_descr descr);
  // CHECK: status_t = rocsparse_destroy_mat_descr(matDescr_t);
  status_t = hipsparseDestroyMatDescr(matDescr_t);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseSetMatIndexBase(cusparseMatDescr_t descrA, cusparseIndexBase_t base);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_set_mat_index_base(rocsparse_mat_descr descr, rocsparse_index_base base);
  // CHECK: status_t = rocsparse_set_mat_index_base(matDescr_t, indexBase_t);
  status_t = hipsparseSetMatIndexBase(matDescr_t, indexBase_t);

  // CUDA: cusparseIndexBase_t CUSPARSEAPI cusparseGetMatIndexBase(const cusparseMatDescr_t descrA);
  // ROC: ROCSPARSE_EXPORT rocsparse_index_base rocsparse_get_mat_index_base(const rocsparse_mat_descr descr);
  // CHECK: indexBase_t = rocsparse_get_mat_index_base(matDescr_t);
  indexBase_t = hipsparseGetMatIndexBase(matDescr_t);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseSetMatType(cusparseMatDescr_t descrA, cusparseMatrixType_t type);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_set_mat_type(rocsparse_mat_descr descr, rocsparse_matrix_type type);
  // CHECK: status_t = rocsparse_set_mat_type(matDescr_t, matrixType_t);
  status_t = hipsparseSetMatType(matDescr_t, matrixType_t);

  // CUDA: cusparseMatrixType_t CUSPARSEAPI cusparseGetMatType(const cusparseMatDescr_t descrA);
  // ROC: ROCSPARSE_EXPORT rocsparse_matrix_type rocsparse_get_mat_type(const rocsparse_mat_descr descr);
  // CHECK: matrixType_t = rocsparse_get_mat_type(matDescr_t);
  matrixType_t = hipsparseGetMatType(matDescr_t);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseSetMatFillMode(cusparseMatDescr_t descrA, cusparseFillMode_t fillMode);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_set_mat_fill_mode(rocsparse_mat_descr descr, rocsparse_fill_mode fill_mode);
  // CHECK: status_t = rocsparse_set_mat_fill_mode(matDescr_t, fillMode_t);
  status_t = hipsparseSetMatFillMode(matDescr_t, fillMode_t);

  // CUDA: cusparseFillMode_t CUSPARSEAPI cusparseGetMatFillMode(const cusparseMatDescr_t descrA);
  // ROC: ROCSPARSE_EXPORT rocsparse_fill_mode rocsparse_get_mat_fill_mode(const rocsparse_mat_descr descr);
  // CHECK: fillMode_t = rocsparse_get_mat_fill_mode(matDescr_t);
  fillMode_t = hipsparseGetMatFillMode(matDescr_t);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseSetMatDiagType(cusparseMatDescr_t descrA, cusparseDiagType_t diagType);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_set_mat_diag_type(rocsparse_mat_descr descr, rocsparse_diag_type diag_type);
  // CHECK: status_t = rocsparse_set_mat_diag_type(matDescr_t, diagType_t);
  status_t = hipsparseSetMatDiagType(matDescr_t, diagType_t);

  // CUDA: cusparseDiagType_t CUSPARSEAPI cusparseGetMatDiagType(const cusparseMatDescr_t descrA);
  // ROC: ROCSPARSE_EXPORT rocsparse_diag_type rocsparse_get_mat_diag_type(const rocsparse_mat_descr descr);
  // CHECK: diagType_t = rocsparse_get_mat_diag_type(matDescr_t);
  diagType_t = hipsparseGetMatDiagType(matDescr_t);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseCreateColorInfo(cusparseColorInfo_t* info);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_create_color_info(rocsparse_color_info* info);
  // CHECK: status_t = rocsparse_create_color_info(&colorInfo_t);
  status_t = hipsparseCreateColorInfo(&colorInfo_t);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseDestroyColorInfo(cusparseColorInfo_t info);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_destroy_color_info(rocsparse_color_info info);
  // CHECK: status_t = rocsparse_destroy_color_info(colorInfo_t);
  status_t = hipsparseDestroyColorInfo(colorInfo_t);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseZcsrcolor(cusparseHandle_t handle, int m, int nnz, const cusparseMatDescr_t descrA, const cuDoubleComplex* csrSortedValA, const int* csrSortedRowPtrA, const int* csrSortedColIndA, const double* fractionToColor, int* ncolors, int* coloring, int* reordering, const cusparseColorInfo_t info);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_zcsrcolor(rocsparse_handle handle, rocsparse_int m, rocsparse_int nnz, const rocsparse_mat_descr descr, const rocsparse_double_complex* csr_val, const rocsparse_int* csr_row_ptr, const rocsparse_int* csr_col_ind, const double* fraction_to_color, rocsparse_int* ncolors, rocsparse_int* coloring, rocsparse_int* reordering, rocsparse_mat_info info);
  // CHECK: status_t = rocsparse_zcsrcolor(handle_t, m, innz, matDescr_t, &dcomplex, &csrRowPtrA, &csrColIndA, &dfractionToColor, &ncolors, &coloring, &reordering, colorInfo_t);
  status_t = hipsparseZcsrcolor(handle_t, m, innz, matDescr_t, &dcomplex, &csrRowPtrA, &csrColIndA, &dfractionToColor, &ncolors, &coloring, &reordering, colorInfo_t);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseCcsrcolor(cusparseHandle_t handle, int m, int nnz, const cusparseMatDescr_t descrA, const cuComplex* csrSortedValA, const int* csrSortedRowPtrA, const int* csrSortedColIndA, const float* fractionToColor, int* ncolors, int* coloring, int* reordering, const cusparseColorInfo_t info);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_ccsrcolor(rocsparse_handle handle, rocsparse_int m, rocsparse_int nnz, const rocsparse_mat_descr descr, const rocsparse_float_complex* csr_val, const rocsparse_int* csr_row_ptr, const rocsparse_int* csr_col_ind, const float* fraction_to_color, rocsparse_int* ncolors, rocsparse_int* coloring, rocsparse_int* reordering, rocsparse_mat_info info);
  // CHECK: status_t = rocsparse_ccsrcolor(handle_t, m, innz, matDescr_t, &complex, &csrRowPtrA, &csrColIndA, &ffractionToColor, &ncolors, &coloring, &reordering, colorInfo_t);
  status_t = hipsparseCcsrcolor(handle_t, m, innz, matDescr_t, &complex, &csrRowPtrA, &csrColIndA, &ffractionToColor, &ncolors, &coloring, &reordering, colorInfo_t);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseDcsrcolor(cusparseHandle_t handle, int m, int nnz, const cusparseMatDescr_t descrA, const double* csrSortedValA, const int* csrSortedRowPtrA, const int* csrSortedColIndA, const double* fractionToColor, int* ncolors, int* coloring, int* reordering, const cusparseColorInfo_t info);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_dcsrcolor(rocsparse_handle handle, rocsparse_int m, rocsparse_int nnz, const rocsparse_mat_descr descr, const double* csr_val, const rocsparse_int* csr_row_ptr, const rocsparse_int* csr_col_ind, const double* fraction_to_color, rocsparse_int* ncolors, rocsparse_int* coloring, rocsparse_int* reordering, rocsparse_mat_info info);
  // CHECK: status_t = rocsparse_dcsrcolor(handle_t, m, innz, matDescr_t, &csrValA, &csrRowPtrA, &csrColIndA, &dfractionToColor, &ncolors, &coloring, &reordering, colorInfo_t);
  status_t = hipsparseDcsrcolor(handle_t, m, innz, matDescr_t, &csrValA, &csrRowPtrA, &csrColIndA, &dfractionToColor, &ncolors, &coloring, &reordering, colorInfo_t);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseScsrcolor(cusparseHandle_t handle, int m, int nnz, const cusparseMatDescr_t descrA, const float* csrSortedValA, const int* csrSortedRowPtrA, const int* csrSortedColIndA, const float* fractionToColor, int* ncolors, int* coloring, int* reordering, const cusparseColorInfo_t info);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_scsrcolor(rocsparse_handle handle, rocsparse_int m, rocsparse_int nnz, const rocsparse_mat_descr descr, const float* csr_val, const rocsparse_int* csr_row_ptr, const rocsparse_int* csr_col_ind, const float* fraction_to_color, rocsparse_int* ncolors, rocsparse_int* coloring, rocsparse_int* reordering, rocsparse_mat_info info);
  // CHECK: status_t = rocsparse_scsrcolor(handle_t, m, innz, matDescr_t, &csrSortedValA, &csrRowPtrA, &csrColIndA, &ffractionToColor, &ncolors, &coloring, &reordering, colorInfo_t);
  status_t = hipsparseScsrcolor(handle_t, m, innz, matDescr_t, &csrSortedValA, &csrRowPtrA, &csrColIndA, &ffractionToColor, &ncolors, &coloring, &reordering, colorInfo_t);

  // CUDA:cusparseStatus_t CUSPARSEAPI cusparseZgebsr2gebsr(cusparseHandle_t handle, cusparseDirection_t dirA, int mb, int nb, int nnzb, const cusparseMatDescr_t descrA, const cuDoubleComplex* bsrSortedValA, const int* bsrSortedRowPtrA, const int* bsrSortedColIndA, int rowBlockDimA, int colBlockDimA, const cusparseMatDescr_t descrC, cuDoubleComplex* bsrSortedValC, int* bsrSortedRowPtrC, int* bsrSortedColIndC, int rowBlockDimC, int colBlockDimC, void* pBuffer);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_zgebsr2gebsr(rocsparse_handle handle, rocsparse_direction dir, rocsparse_int mb, rocsparse_int nb, rocsparse_int nnzb, const rocsparse_mat_descr descr_A, const rocsparse_double_complex* bsr_val_A, const rocsparse_int* bsr_row_ptr_A, const rocsparse_int* bsr_col_ind_A, rocsparse_int row_block_dim_A, rocsparse_int col_block_dim_A, const rocsparse_mat_descr descr_C, rocsparse_double_complex* bsr_val_C, rocsparse_int* bsr_row_ptr_C, rocsparse_int* bsr_col_ind_C, rocsparse_int row_block_dim_C, rocsparse_int col_block_dim_C, void* temp_buffer);
  // CHECK: status_t = rocsparse_zgebsr2gebsr(handle_t, direction_t, mb, nb, nnzb, matDescr_t, &dcomplex, &bsrRowPtrA, &bsrColIndA, rowBlockDimA, colBlockDimA, matDescr_C, &dComplexbsrSortedValC, &bsrSortedRowPtrC, &bsrSortedColIndC, bsrRowPtrC, bsrColIndC, tempBuffer);
  status_t = hipsparseZgebsr2gebsr(handle_t, direction_t, mb, nb, nnzb, matDescr_t, &dcomplex, &bsrRowPtrA, &bsrColIndA, rowBlockDimA, colBlockDimA, matDescr_C, &dComplexbsrSortedValC, &bsrSortedRowPtrC, &bsrSortedColIndC, bsrRowPtrC, bsrColIndC, tempBuffer);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseZgebsr2gebsr_bufferSize(cusparseHandle_t handle, cusparseDirection_t dirA, int mb, int nb, int nnzb, const cusparseMatDescr_t descrA, const cuDoubleComplex* bsrSortedValA, const int* bsrSortedRowPtrA, const int* bsrSortedColIndA, int rowBlockDimA, int colBlockDimA, int rowBlockDimC,int colBlockDimC, int* pBufferSizeInBytes);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_zgebsr2gebsr_buffer_size(rocsparse_handle handle, rocsparse_direction dir, rocsparse_int mb, rocsparse_int nb, rocsparse_int nnzb, const rocsparse_mat_descr descr_A, const rocsparse_double_complex* bsr_val_A, const rocsparse_int* bsr_row_ptr_A, const rocsparse_int* bsr_col_ind_A, rocsparse_int row_block_dim_A, rocsparse_int col_block_dim_A, rocsparse_int row_block_dim_C, rocsparse_int col_block_dim_C, size_t* buffer_size);
  // CHECK: status_t = rocsparse_zgebsr2gebsr_buffer_size(handle_t, direction_t, mb, nb, nnzb, matDescr_t, &dcomplex, &bsrRowPtrA, &bsrColIndA, rowBlockDimA, colBlockDimA, rowBlockDimC, colBlockDimC, &bufferSizeInBytes);
  status_t = hipsparseZgebsr2gebsr_bufferSize(handle_t, direction_t, mb, nb, nnzb, matDescr_t, &dcomplex, &bsrRowPtrA, &bsrColIndA, rowBlockDimA, colBlockDimA, rowBlockDimC, colBlockDimC, &bufferSizeInBytes);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseCgebsr2gebsr(cusparseHandle_t handle, cusparseDirection_t dirA, int mb, int nb, int nnzb, const cusparseMatDescr_t descrA, const cuComplex* bsrSortedValA, const int* bsrSortedRowPtrA, const int* bsrSortedColIndA, int rowBlockDimA, int colBlockDimA, const cusparseMatDescr_t descrC, cuComplex* bsrSortedValC, int* bsrSortedRowPtrC, int* bsrSortedColIndC, int rowBlockDimC, int colBlockDimC, void* pBuffer);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_cgebsr2gebsr(rocsparse_handle handle, rocsparse_direction dir, rocsparse_int mb, rocsparse_int nb, rocsparse_int nnzb, const rocsparse_mat_descr descr_A, const rocsparse_float_complex* bsr_val_A, const rocsparse_int* bsr_row_ptr_A, const rocsparse_int* bsr_col_ind_A, rocsparse_int row_block_dim_A, rocsparse_int col_block_dim_A, const rocsparse_mat_descr descr_C, rocsparse_float_complex* bsr_val_C, rocsparse_int* bsr_row_ptr_C, rocsparse_int* bsr_col_ind_C, rocsparse_int row_block_dim_C, rocsparse_int col_block_dim_C, void* temp_buffer);
  // CHECK: status_t = rocsparse_cgebsr2gebsr(handle_t, direction_t, mb, nb, nnzb, matDescr_t, &complex, &bsrRowPtrA, &bsrColIndA, rowBlockDimA, colBlockDimA, matDescr_C, &complexbsrSortedValC, &bsrSortedRowPtrC, &bsrSortedColIndC, rowBlockDimC, colBlockDimC, tempBuffer);
  status_t = hipsparseCgebsr2gebsr(handle_t, direction_t, mb, nb, nnzb, matDescr_t, &complex, &bsrRowPtrA, &bsrColIndA, rowBlockDimA, colBlockDimA, matDescr_C, &complexbsrSortedValC, &bsrSortedRowPtrC, &bsrSortedColIndC, rowBlockDimC, colBlockDimC, tempBuffer);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseCgebsr2gebsr_bufferSize(cusparseHandle_t handle, cusparseDirection_t dirA, int mb, int nb, int nnzb, const cusparseMatDescr_t descrA, const cuComplex* bsrSortedValA, const int* bsrSortedRowPtrA, const int* bsrSortedColIndA, int rowBlockDimA, int colBlockDimA, int rowBlockDimC, int colBlockDimC, int* pBufferSizeInBytes);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_cgebsr2gebsr_buffer_size(rocsparse_handle handle, rocsparse_direction dir, rocsparse_int mb, rocsparse_int nb, rocsparse_int nnzb, const rocsparse_mat_descr descr_A, const rocsparse_float_complex* bsr_val_A, const rocsparse_int* bsr_row_ptr_A, const rocsparse_int* bsr_col_ind_A, rocsparse_int row_block_dim_A, rocsparse_int col_block_dim_A, rocsparse_int row_block_dim_C, rocsparse_int col_block_dim_C, size_t* buffer_size);
  // CHECK: status_t = rocsparse_cgebsr2gebsr_buffer_size(handle_t, direction_t, mb, nb, nnzb, matDescr_t, &complex, &bsrRowPtrA, &bsrColIndA, rowBlockDimA, colBlockDimA, rowBlockDimC, colBlockDimC, &bufferSizeInBytes);
  status_t = hipsparseCgebsr2gebsr_bufferSize(handle_t, direction_t, mb, nb, nnzb, matDescr_t, &complex, &bsrRowPtrA, &bsrColIndA, rowBlockDimA, colBlockDimA, rowBlockDimC, colBlockDimC, &bufferSizeInBytes);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseDgebsr2gebsr(cusparseHandle_t handle, cusparseDirection_t dirA, int mb, int nb, int nnzb, const cusparseMatDescr_t descrA, const double* bsrSortedValA, const int* bsrSortedRowPtrA, const int* bsrSortedColIndA, int rowBlockDimA, int colBlockDimA, const cusparseMatDescr_t descrC, double* bsrSortedValC, int* bsrSortedRowPtrC, int* bsrSortedColIndC, int rowBlockDimC, int colBlockDimC, void* pBuffer);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_dgebsr2gebsr(rocsparse_handle handle, rocsparse_direction dir, rocsparse_int mb, rocsparse_int nb, rocsparse_int nnzb, const rocsparse_mat_descr descr_A, const double* bsr_val_A, const rocsparse_int* bsr_row_ptr_A, const rocsparse_int* bsr_col_ind_A, rocsparse_int row_block_dim_A, rocsparse_int col_block_dim_A, const rocsparse_mat_descr descr_C, double* bsr_val_C, rocsparse_int* bsr_row_ptr_C, rocsparse_int* bsr_col_ind_C, rocsparse_int row_block_dim_C, rocsparse_int col_block_dim_C, void* temp_buffer);
  // CHECK: status_t = rocsparse_dgebsr2gebsr(handle_t, direction_t, mb, nb, nnzb, matDescr_t, &dbsrSortedValA, &bsrRowPtrA, &bsrColIndA, rowBlockDimA, colBlockDimA, matDescr_C, &dbsrSortedValC, &bsrSortedRowPtrC, &bsrSortedColIndC, rowBlockDimC, colBlockDimC, tempBuffer);
  status_t = hipsparseDgebsr2gebsr(handle_t, direction_t, mb, nb, nnzb, matDescr_t, &dbsrSortedValA, &bsrRowPtrA, &bsrColIndA, rowBlockDimA, colBlockDimA, matDescr_C, &dbsrSortedValC, &bsrSortedRowPtrC, &bsrSortedColIndC, rowBlockDimC, colBlockDimC, tempBuffer);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseDgebsr2gebsr_bufferSize(cusparseHandle_t handle, cusparseDirection_t dirA, int mb, int nb, int nnzb, const cusparseMatDescr_t descrA, const double* bsrSortedValA, const int* bsrSortedRowPtrA, const int* bsrSortedColIndA, int rowBlockDimA, int colBlockDimA, int rowBlockDimC, int colBlockDimC, int* pBufferSizeInBytes);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_dgebsr2gebsr_buffer_size(rocsparse_handle handle, rocsparse_direction dir, rocsparse_int mb, rocsparse_int nb, rocsparse_int nnzb, const rocsparse_mat_descr descr_A, const double* bsr_val_A, const rocsparse_int* bsr_row_ptr_A, const rocsparse_int* bsr_col_ind_A, rocsparse_int row_block_dim_A, rocsparse_int col_block_dim_A, rocsparse_int row_block_dim_C, rocsparse_int col_block_dim_C, size_t* buffer_size);
  // CHECK: status_t = rocsparse_dgebsr2gebsr_buffer_size(handle_t, direction_t, mb, nb, nnzb, matDescr_t, &dbsrSortedValA, &bsrRowPtrA, &bsrColIndA, rowBlockDimA, colBlockDimA, rowBlockDimC, colBlockDimC, &bufferSizeInBytes);
  status_t = hipsparseDgebsr2gebsr_bufferSize(handle_t, direction_t, mb, nb, nnzb, matDescr_t, &dbsrSortedValA, &bsrRowPtrA, &bsrColIndA, rowBlockDimA, colBlockDimA, rowBlockDimC, colBlockDimC, &bufferSizeInBytes);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseSgebsr2gebsr(cusparseHandle_t handle, cusparseDirection_t dirA, int mb, int nb, int nnzb, const cusparseMatDescr_t descrA, const float* bsrSortedValA, const int* bsrSortedRowPtrA, const int* bsrSortedColIndA, int rowBlockDimA, int colBlockDimA, const cusparseMatDescr_t descrC, float* bsrSortedValC, int* bsrSortedRowPtrC, int* bsrSortedColIndC, int rowBlockDimC, int colBlockDimC, void* pBuffer);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_sgebsr2gebsr(rocsparse_handle handle, rocsparse_direction dir, rocsparse_int mb, rocsparse_int nb, rocsparse_int nnzb, const rocsparse_mat_descr descr_A, const float* bsr_val_A, const rocsparse_int* bsr_row_ptr_A, const rocsparse_int* bsr_col_ind_A, rocsparse_int row_block_dim_A, rocsparse_int col_block_dim_A, const rocsparse_mat_descr descr_C, float* bsr_val_C, rocsparse_int* bsr_row_ptr_C, rocsparse_int* bsr_col_ind_C, rocsparse_int row_block_dim_C, rocsparse_int col_block_dim_C, void* temp_buffer);
  // CHECK: status_t = rocsparse_sgebsr2gebsr(handle_t, direction_t, mb, nb, nnzb, matDescr_t, &fbsrSortedValA, &bsrRowPtrA, &bsrColIndA, rowBlockDimA, colBlockDimA, matDescr_C, &fbsrSortedValC, &bsrSortedRowPtrC, &bsrSortedColIndC, rowBlockDimC, colBlockDimC, tempBuffer);
  status_t = hipsparseSgebsr2gebsr(handle_t, direction_t, mb, nb, nnzb, matDescr_t, &fbsrSortedValA, &bsrRowPtrA, &bsrColIndA, rowBlockDimA, colBlockDimA, matDescr_C, &fbsrSortedValC, &bsrSortedRowPtrC, &bsrSortedColIndC, rowBlockDimC, colBlockDimC, tempBuffer);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseSgebsr2gebsr_bufferSize(cusparseHandle_t handle, cusparseDirection_t dirA, int mb, int nb, int nnzb, const cusparseMatDescr_t descrA, const float* bsrSortedValA, const int* bsrSortedRowPtrA, const int* bsrSortedColIndA, int rowBlockDimA, int colBlockDimA, int rowBlockDimC, int colBlockDimC, int* pBufferSizeInBytes);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_sgebsr2gebsr_buffer_size(rocsparse_handle handle, rocsparse_direction dir, rocsparse_int mb, rocsparse_int nb, rocsparse_int nnzb, const rocsparse_mat_descr descr_A, const float* bsr_val_A, const rocsparse_int* bsr_row_ptr_A, const rocsparse_int* bsr_col_ind_A, rocsparse_int row_block_dim_A, rocsparse_int col_block_dim_A, rocsparse_int row_block_dim_C, rocsparse_int col_block_dim_C, size_t* buffer_size);
  // CHECK: status_t = rocsparse_sgebsr2gebsr_buffer_size(handle_t, direction_t, mb, nb, nnzb, matDescr_t, &fbsrSortedValA, &bsrRowPtrA, &bsrColIndA, rowBlockDimA, colBlockDimA, rowBlockDimC, colBlockDimC, &bufferSizeInBytes);
  status_t = hipsparseSgebsr2gebsr_bufferSize(handle_t, direction_t, mb, nb, nnzb, matDescr_t, &fbsrSortedValA, &bsrRowPtrA, &bsrColIndA, rowBlockDimA, colBlockDimA, rowBlockDimC, colBlockDimC, &bufferSizeInBytes);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseXgebsr2gebsrNnz(cusparseHandle_t handle, cusparseDirection_t dirA, int mb, int nb, int nnzb, const cusparseMatDescr_t descrA, const int* bsrSortedRowPtrA, const int* bsrSortedColIndA, int rowBlockDimA, int colBlockDimA, const cusparseMatDescr_t descrC, int* bsrSortedRowPtrC, int rowBlockDimC, int colBlockDimC, int* nnzTotalDevHostPtr, void* pBuffer);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_gebsr2gebsr_nnz(rocsparse_handle handle, rocsparse_direction dir, rocsparse_int mb, rocsparse_int nb, rocsparse_int nnzb, const rocsparse_mat_descr descr_A, const rocsparse_int* bsr_row_ptr_A, const rocsparse_int* bsr_col_ind_A, rocsparse_int row_block_dim_A, rocsparse_int col_block_dim_A, const rocsparse_mat_descr descr_C, rocsparse_int* bsr_row_ptr_C, rocsparse_int row_block_dim_C, rocsparse_int col_block_dim_C, rocsparse_int* nnz_total_dev_host_ptr, void* temp_buffer);
  // CHECK: status_t = rocsparse_gebsr2gebsr_nnz(handle_t, direction_t, mb, nb, nnzb, matDescr_t, &bsrRowPtrA, &bsrColIndA, rowBlockDimA, colBlockDimA, matDescr_C, &bsrSortedRowPtrC, rowBlockDimC, colBlockDimC, &nnzTotalDevHostPtr, &tempBuffer);
  status_t = hipsparseXgebsr2gebsrNnz(handle_t, direction_t, mb, nb, nnzb, matDescr_t, &bsrRowPtrA, &bsrColIndA, rowBlockDimA, colBlockDimA, matDescr_C, &bsrSortedRowPtrC, rowBlockDimC, colBlockDimC, &nnzTotalDevHostPtr, &tempBuffer);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseZgebsr2csr(cusparseHandle_t handle, cusparseDirection_t dirA, int mb, int nb, const cusparseMatDescr_t descrA, const cuDoubleComplex* bsrSortedValA, const int* bsrSortedRowPtrA, const int* bsrSortedColIndA, int rowBlockDim, int colBlockDim, const cusparseMatDescr_t descrC, cuDoubleComplex* csrSortedValC, int* csrSortedRowPtrC, int* csrSortedColIndC);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_zgebsr2csr(rocsparse_handle handle, rocsparse_direction dir, rocsparse_int mb, rocsparse_int nb, const rocsparse_mat_descr bsr_descr, const rocsparse_double_complex* bsr_val, const rocsparse_int* bsr_row_ptr, const rocsparse_int* bsr_col_ind, rocsparse_int row_block_dim, rocsparse_int col_block_dim, const rocsparse_mat_descr csr_descr, rocsparse_double_complex* csr_val, rocsparse_int* csr_row_ptr, rocsparse_int* csr_col_ind);
  // CHECK: status_t = rocsparse_zgebsr2csr(handle_t, direction_t, mb, nb, matDescr_A, &dComplexbsrSortedValA, &bsrRowPtrA, &bsrColIndA, rowBlockDimA, colBlockDimA, matDescr_C, &dComplexbsrSortedValC, &csrRowPtrC, &csrColIndC);
  status_t = hipsparseZgebsr2csr(handle_t, direction_t, mb, nb, matDescr_A, &dComplexbsrSortedValA, &bsrRowPtrA, &bsrColIndA, rowBlockDimA, colBlockDimA, matDescr_C, &dComplexbsrSortedValC, &csrRowPtrC, &csrColIndC);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseCgebsr2csr(cusparseHandle_t handle, cusparseDirection_t dirA, int mb, int nb, const cusparseMatDescr_t descrA, const cuComplex* bsrSortedValA, const int* bsrSortedRowPtrA, const int* bsrSortedColIndA, int rowBlockDim, int colBlockDim, const cusparseMatDescr_t descrC, cuComplex* csrSortedValC, int* csrSortedRowPtrC, int* csrSortedColIndC);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_cgebsr2csr(rocsparse_handle handle, rocsparse_direction dir, rocsparse_int mb, rocsparse_int nb, const rocsparse_mat_descr bsr_descr, const rocsparse_float_complex* bsr_val, const rocsparse_int* bsr_row_ptr, const rocsparse_int* bsr_col_ind, rocsparse_int row_block_dim, rocsparse_int col_block_dim, const rocsparse_mat_descr csr_descr, rocsparse_float_complex* csr_val, rocsparse_int* csr_row_ptr, rocsparse_int* csr_col_ind);
  // CHECK: status_t = rocsparse_cgebsr2csr(handle_t, direction_t, mb, nb, matDescr_A, &complexbsrValA, &bsrRowPtrA, &bsrColIndA, rowBlockDimA, colBlockDimA, matDescr_C, &complexbsrSortedValC, &csrRowPtrC, &csrColIndC);
  status_t = hipsparseCgebsr2csr(handle_t, direction_t, mb, nb, matDescr_A, &complexbsrValA, &bsrRowPtrA, &bsrColIndA, rowBlockDimA, colBlockDimA, matDescr_C, &complexbsrSortedValC, &csrRowPtrC, &csrColIndC);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseDgebsr2csr(cusparseHandle_t handle, cusparseDirection_t dirA, int mb, int nb, const cusparseMatDescr_t descrA, const double* bsrSortedValA, const int* bsrSortedRowPtrA, const int* bsrSortedColIndA, int rowBlockDim, int colBlockDim, const cusparseMatDescr_t descrC, double* csrSortedValC, int* csrSortedRowPtrC, int* csrSortedColIndC);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_dgebsr2csr(rocsparse_handle handle, rocsparse_direction dir, rocsparse_int mb, rocsparse_int nb, const rocsparse_mat_descr bsr_descr, const double* bsr_val, const rocsparse_int* bsr_row_ptr, const rocsparse_int* bsr_col_ind, rocsparse_int row_block_dim, rocsparse_int col_block_dim, const rocsparse_mat_descr csr_descr, double* csr_val, rocsparse_int* csr_row_ptr, rocsparse_int* csr_col_ind);
  // CHECK: status_t = rocsparse_dgebsr2csr(handle_t, direction_t, mb, nb, matDescr_A, &bsrValA, &bsrRowPtrA, &bsrColIndA, rowBlockDimA, colBlockDimA, matDescr_C, &csrValC, &csrRowPtrC, &csrColIndC);
  status_t = hipsparseDgebsr2csr(handle_t, direction_t, mb, nb, matDescr_A, &bsrValA, &bsrRowPtrA, &bsrColIndA, rowBlockDimA, colBlockDimA, matDescr_C, &csrValC, &csrRowPtrC, &csrColIndC);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseSgebsr2csr(cusparseHandle_t handle, cusparseDirection_t dirA, int mb, int nb, const cusparseMatDescr_t descrA, const float* bsrSortedValA, const int* bsrSortedRowPtrA, const int* bsrSortedColIndA, int rowBlockDim, int colBlockDim, const cusparseMatDescr_t descrC, float* csrSortedValC, int* csrSortedRowPtrC, int* csrSortedColIndC);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_sgebsr2csr(rocsparse_handle handle, rocsparse_direction dir, rocsparse_int mb, rocsparse_int nb, const rocsparse_mat_descr bsr_descr, const float* bsr_val, const rocsparse_int* bsr_row_ptr, const rocsparse_int* bsr_col_ind, rocsparse_int row_block_dim, rocsparse_int col_block_dim, const rocsparse_mat_descr csr_descr, float* csr_val, rocsparse_int* csr_row_ptr, rocsparse_int* csr_col_ind);
  // CHECK: status_t = rocsparse_sgebsr2csr(handle_t, direction_t, mb, nb, matDescr_A, &fbsrSortedValA, &bsrRowPtrA, &bsrColIndA, rowBlockDimA, colBlockDimA, matDescr_C, &fcsrSortedValC, &csrRowPtrC, &csrColIndC);
  status_t = hipsparseSgebsr2csr(handle_t, direction_t, mb, nb, matDescr_A, &fbsrSortedValA, &bsrRowPtrA, &bsrColIndA, rowBlockDimA, colBlockDimA, matDescr_C, &fcsrSortedValC, &csrRowPtrC, &csrColIndC);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseZbsr2csr(cusparseHandle_t handle, cusparseDirection_t dirA, int mb, int nb, const cusparseMatDescr_t descrA, const cuDoubleComplex* bsrSortedValA, const int* bsrSortedRowPtrA, const int* bsrSortedColIndA, int blockDim, const cusparseMatDescr_t descrC, cuDoubleComplex* csrSortedValC, int* csrSortedRowPtrC, int* csrSortedColIndC);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_zbsr2csr(rocsparse_handle handle, rocsparse_direction dir, rocsparse_int mb, rocsparse_int nb, const rocsparse_mat_descr bsr_descr, const rocsparse_double_complex* bsr_val, const rocsparse_int* bsr_row_ptr, const rocsparse_int* bsr_col_ind, rocsparse_int block_dim, const rocsparse_mat_descr csr_descr, rocsparse_double_complex* csr_val, rocsparse_int* csr_row_ptr, rocsparse_int* csr_col_ind);
  // CHECK: status_t = rocsparse_zbsr2csr(handle_t, direction_t, mb, nb, matDescr_A, &dComplexbsrSortedValA, &bsrRowPtrA, &bsrColIndA, blockDim, matDescr_C, &dComplexbsrSortedValC, &csrRowPtrC, &csrColIndC);
  status_t = hipsparseZbsr2csr(handle_t, direction_t, mb, nb, matDescr_A, &dComplexbsrSortedValA, &bsrRowPtrA, &bsrColIndA, blockDim, matDescr_C, &dComplexbsrSortedValC, &csrRowPtrC, &csrColIndC);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseCbsr2csr(cusparseHandle_t handle, cusparseDirection_t dirA, int mb, int nb, const cusparseMatDescr_t descrA, const cuComplex* bsrSortedValA, const int* bsrSortedRowPtrA, const int* bsrSortedColIndA, int blockDim, const cusparseMatDescr_t descrC, cuComplex* csrSortedValC, int* csrSortedRowPtrC, int* csrSortedColIndC);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_cbsr2csr(rocsparse_handle handle, rocsparse_direction dir, rocsparse_int mb, rocsparse_int nb, const rocsparse_mat_descr bsr_descr, const rocsparse_float_complex* bsr_val, const rocsparse_int* bsr_row_ptr, const rocsparse_int* bsr_col_ind, rocsparse_int block_dim, const rocsparse_mat_descr csr_descr, rocsparse_float_complex* csr_val, rocsparse_int* csr_row_ptr, rocsparse_int* csr_col_ind);
  // CHECK: status_t = rocsparse_cbsr2csr(handle_t, direction_t, mb, nb, matDescr_A, &complexbsrValA, &bsrRowPtrA, &bsrColIndA, blockDim, matDescr_C, &complexbsrSortedValC, &csrRowPtrC, &csrColIndC);
  status_t = hipsparseCbsr2csr(handle_t, direction_t, mb, nb, matDescr_A, &complexbsrValA, &bsrRowPtrA, &bsrColIndA, blockDim, matDescr_C, &complexbsrSortedValC, &csrRowPtrC, &csrColIndC);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseDbsr2csr(cusparseHandle_t handle, cusparseDirection_t dirA, int mb, int nb, const cusparseMatDescr_t descrA, const double* bsrSortedValA, const int* bsrSortedRowPtrA, const int* bsrSortedColIndA, int blockDim, const cusparseMatDescr_t descrC, double* csrSortedValC, int* csrSortedRowPtrC, int* csrSortedColIndC);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_dbsr2csr(rocsparse_handle handle, rocsparse_direction dir, rocsparse_int mb, rocsparse_int nb, const rocsparse_mat_descr bsr_descr, const double* bsr_val, const rocsparse_int* bsr_row_ptr, const rocsparse_int* bsr_col_ind, rocsparse_int block_dim, const rocsparse_mat_descr csr_descr, double* csr_val, rocsparse_int* csr_row_ptr, rocsparse_int* csr_col_ind);
  // CHECK: status_t = rocsparse_dbsr2csr(handle_t, direction_t, mb, nb, matDescr_A, &bsrValA, &bsrRowPtrA, &bsrColIndA, blockDim, matDescr_C, &csrValC, &csrRowPtrC, &csrColIndC);
  status_t = hipsparseDbsr2csr(handle_t, direction_t, mb, nb, matDescr_A, &bsrValA, &bsrRowPtrA, &bsrColIndA, blockDim, matDescr_C, &csrValC, &csrRowPtrC, &csrColIndC);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseSbsr2csr(cusparseHandle_t handle, cusparseDirection_t dirA, int mb, int nb, const cusparseMatDescr_t descrA, const float* bsrSortedValA, const int* bsrSortedRowPtrA, const int* bsrSortedColIndA, int blockDim, const cusparseMatDescr_t descrC, float* csrSortedValC, int* csrSortedRowPtrC, int* csrSortedColIndC);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_sbsr2csr(rocsparse_handle handle, rocsparse_direction dir, rocsparse_int mb, rocsparse_int nb, const rocsparse_mat_descr bsr_descr, const float* bsr_val, const rocsparse_int* bsr_row_ptr, const rocsparse_int* bsr_col_ind, rocsparse_int block_dim, const rocsparse_mat_descr csr_descr, float* csr_val, rocsparse_int* csr_row_ptr, rocsparse_int* csr_col_ind);
  // CHECK: status_t = rocsparse_sbsr2csr(handle_t, direction_t, mb, nb, matDescr_A, &fbsrSortedValA, &bsrRowPtrA, &bsrColIndA, blockDim, matDescr_C, &fcsrSortedValC, &csrRowPtrC, &csrColIndC);
  status_t = hipsparseSbsr2csr(handle_t, direction_t, mb, nb, matDescr_A, &fbsrSortedValA, &bsrRowPtrA, &bsrColIndA, blockDim, matDescr_C, &fcsrSortedValC, &csrRowPtrC, &csrColIndC);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseXcoosortByColumn(cusparseHandle_t handle, int m, int n, int nnz, int* cooRowsA, int* cooColsA, int* P, void* pBuffer);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_coosort_by_column(rocsparse_handle handle, rocsparse_int m, rocsparse_int n, rocsparse_int nnz, rocsparse_int* coo_row_ind, rocsparse_int* coo_col_ind, rocsparse_int* perm, void* temp_buffer);
  // CHECK: status_t = rocsparse_coosort_by_column(handle_t, m, n, innz, &icooRows, &icooColumns, P, pBuffer);
  status_t = hipsparseXcoosortByColumn(handle_t, m, n, innz, &icooRows, &icooColumns, P, pBuffer);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseXcoosortByRow(cusparseHandle_t handle, int m, int n, int nnz, int* cooRowsA, int* cooColsA, int* P, void* pBuffer);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_coosort_by_row(rocsparse_handle handle, rocsparse_int m, rocsparse_int n, rocsparse_int nnz, rocsparse_int* coo_row_ind, rocsparse_int* coo_col_ind, rocsparse_int* perm, void* temp_buffer);
  // CHECK: status_t = rocsparse_coosort_by_row(handle_t, m, n, innz, &icooRows, &icooColumns, P, pBuffer);
  status_t = hipsparseXcoosortByRow(handle_t, m, n, innz, &icooRows, &icooColumns, P, pBuffer);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseXcoosort_bufferSizeExt(cusparseHandle_t handle, int m, int n, int nnz, const int* cooRowsA, const int* cooColsA, size_t* pBufferSizeInBytes);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_coosort_buffer_size(rocsparse_handle handle, rocsparse_int m, rocsparse_int n, rocsparse_int nnz, const rocsparse_int* coo_row_ind, const rocsparse_int* coo_col_ind, size_t* buffer_size);
  // CHECK: status_t = rocsparse_coosort_buffer_size(handle_t, m, n, innz, &icooRows, &icooColumns, &bufferSize);
  status_t = hipsparseXcoosort_bufferSizeExt(handle_t, m, n, innz, &icooRows, &icooColumns, &bufferSize);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseXcscsort(cusparseHandle_t handle, int m, int n, int nnz, const cusparseMatDescr_t descrA, const int* cscColPtrA, int* cscRowIndA, int* P, void* pBuffer);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_cscsort(rocsparse_handle handle, rocsparse_int m, rocsparse_int n, rocsparse_int nnz, const rocsparse_mat_descr descr, const rocsparse_int* csc_col_ptr, rocsparse_int* csc_row_ind, rocsparse_int* perm, void* temp_buffer);
  // CHECK: status_t = rocsparse_cscsort(handle_t, m, n, innz, matDescr_A, &cscColPtrA, &cscRowIndA, P, pBuffer);
  status_t = hipsparseXcscsort(handle_t, m, n, innz, matDescr_A, &cscColPtrA, &cscRowIndA, P, pBuffer);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseXcscsort_bufferSizeExt(cusparseHandle_t handle, int m, int n, int nnz, const int* cscColPtrA, const int* cscRowIndA, size_t* pBufferSizeInBytes);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_cscsort_buffer_size(rocsparse_handle handle, rocsparse_int m, rocsparse_int n, rocsparse_int nnz, const rocsparse_int* csc_col_ptr, const rocsparse_int* csc_row_ind, size_t* buffer_size);
  // CHECK: status_t = rocsparse_cscsort_buffer_size(handle_t, m, n, innz, &cscColPtrA, &cscRowIndA, &bufferSize);
  status_t = hipsparseXcscsort_bufferSizeExt(handle_t, m, n, innz, &cscColPtrA, &cscRowIndA, &bufferSize);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseXcsrsort(cusparseHandle_t handle, int m, int n, int nnz, const cusparseMatDescr_t descrA, const int* csrRowPtrA, int* csrColIndA, int* P, void* pBuffer);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_csrsort(rocsparse_handle handle, rocsparse_int m, rocsparse_int n, rocsparse_int nnz, const rocsparse_mat_descr descr, const rocsparse_int* csr_row_ptr, rocsparse_int* csr_col_ind, rocsparse_int* perm, void* temp_buffer);
  // CHECK: status_t = rocsparse_csrsort(handle_t, m, n, innz, matDescr_A, &cscRowIndA, &cscColPtrA, P, pBuffer);
  status_t = hipsparseXcsrsort(handle_t, m, n, innz, matDescr_A, &cscRowIndA, &cscColPtrA, P, pBuffer);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseXcsrsort_bufferSizeExt(cusparseHandle_t handle, int m, int n, int nnz, const int* csrRowPtrA, const int* csrColIndA, size_t* pBufferSizeInBytes);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_csrsort_buffer_size(rocsparse_handle handle, rocsparse_int m, rocsparse_int n, rocsparse_int nnz, const rocsparse_int* csr_row_ptr, const rocsparse_int* csr_col_ind, size_t* buffer_size);
  // CHECK: status_t = rocsparse_csrsort_buffer_size(handle_t, m, n, innz, &cscRowIndA, &cscColPtrA, &bufferSize);
  status_t = hipsparseXcsrsort_bufferSizeExt(handle_t, m, n, innz, &cscRowIndA, &cscColPtrA, &bufferSize);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseCreateIdentityPermutation(cusparseHandle_t handle, int n, int* p);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_create_identity_permutation(rocsparse_handle handle, rocsparse_int n, rocsparse_int* p);
  // CHECK: status_t = rocsparse_create_identity_permutation(handle_t, n, P);
  status_t = hipsparseCreateIdentityPermutation(handle_t, n, P);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseXcoo2csr(cusparseHandle_t handle, const int* cooRowInd, int nnz, int m, int* csrSortedRowPtr, cusparseIndexBase_t idxBase);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_coo2csr(rocsparse_handle handle, const rocsparse_int* coo_row_ind, rocsparse_int nnz, rocsparse_int m, rocsparse_int* csr_row_ptr, rocsparse_index_base idx_base);
  // CHECK: status_t = rocsparse_coo2csr(handle_t, &icooRowInd, nnz, m, &csrRowPtrA, indexBase_t);
  status_t = hipsparseXcoo2csr(handle_t, &icooRowInd, nnz, m, &csrRowPtrA, indexBase_t);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseZcsr2gebsr(cusparseHandle_t handle, cusparseDirection_t dirA, int m, int n, const cusparseMatDescr_t descrA, const cuDoubleComplex* csrSortedValA, const int* csrSortedRowPtrA, const int* csrSortedColIndA, const cusparseMatDescr_t descrC, cuDoubleComplex* bsrSortedValC, int* bsrSortedRowPtrC, int* bsrSortedColIndC, int rowBlockDim, int colBlockDim, void* pBuffer);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_zcsr2gebsr(rocsparse_handle handle, rocsparse_direction dir, rocsparse_int m, rocsparse_int n, const rocsparse_mat_descr csr_descr, const rocsparse_double_complex* csr_val, const rocsparse_int* csr_row_ptr, const rocsparse_int* csr_col_ind, const rocsparse_mat_descr bsr_descr, rocsparse_double_complex* bsr_val, rocsparse_int* bsr_row_ptr, rocsparse_int* bsr_col_ind, rocsparse_int row_block_dim, rocsparse_int col_block_dim, void* temp_buffer);
  // CHECK: status_t = rocsparse_zcsr2gebsr(handle_t, direction_t, m, n, matDescr_A, &dComplexcsrSortedValA, &csrRowPtrA, &csrColIndA, matDescr_C, &dComplexcsrSortedValC, &bsrSortedRowPtrC, &bsrSortedColIndC, rowBlockDimA, colBlockDimA, pBuffer);
  status_t = hipsparseZcsr2gebsr(handle_t, direction_t, m, n, matDescr_A, &dComplexcsrSortedValA, &csrRowPtrA, &csrColIndA, matDescr_C, &dComplexcsrSortedValC, &bsrSortedRowPtrC, &bsrSortedColIndC, rowBlockDimA, colBlockDimA, pBuffer);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseCcsr2gebsr(cusparseHandle_t handle, cusparseDirection_t dirA, int m, int n, const cusparseMatDescr_t descrA, const cuComplex* csrSortedValA, const int* csrSortedRowPtrA, const int* csrSortedColIndA, const cusparseMatDescr_t descrC, cuComplex* bsrSortedValC, int* bsrSortedRowPtrC, int* bsrSortedColIndC, int rowBlockDim, int colBlockDim, void* pBuffer);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_ccsr2gebsr(rocsparse_handle handle, rocsparse_direction dir, rocsparse_int m, rocsparse_int n, const rocsparse_mat_descr csr_descr, const rocsparse_float_complex* csr_val, const rocsparse_int* csr_row_ptr, const rocsparse_int* csr_col_ind, const rocsparse_mat_descr bsr_descr, rocsparse_float_complex* bsr_val, rocsparse_int* bsr_row_ptr, rocsparse_int* bsr_col_ind, rocsparse_int row_block_dim, rocsparse_int col_block_dim, void* temp_buffer);
  // CHECK: status_t = rocsparse_ccsr2gebsr(handle_t, direction_t, m, n, matDescr_A, &complexcsrSortedValA, &csrRowPtrA, &csrColIndA, matDescr_C, &complexcsrSortedValC, &bsrSortedRowPtrC, &bsrSortedColIndC, rowBlockDimA, colBlockDimA, pBuffer);
  status_t = hipsparseCcsr2gebsr(handle_t, direction_t, m, n, matDescr_A, &complexcsrSortedValA, &csrRowPtrA, &csrColIndA, matDescr_C, &complexcsrSortedValC, &bsrSortedRowPtrC, &bsrSortedColIndC, rowBlockDimA, colBlockDimA, pBuffer);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseDcsr2gebsr(cusparseHandle_t handle, cusparseDirection_t dirA, int m, int n, const cusparseMatDescr_t descrA, const double* csrSortedValA, const int* csrSortedRowPtrA, const int* csrSortedColIndA, const cusparseMatDescr_t descrC, double* bsrSortedValC, int* bsrSortedRowPtrC, int* bsrSortedColIndC, int rowBlockDim, int colBlockDim, void* pBuffer);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_dcsr2gebsr(rocsparse_handle handle, rocsparse_direction dir, rocsparse_int m, rocsparse_int n, const rocsparse_mat_descr csr_descr, const double* csr_val, const rocsparse_int* csr_row_ptr, const rocsparse_int* csr_col_ind, const rocsparse_mat_descr bsr_descr, double* bsr_val, rocsparse_int* bsr_row_ptr, rocsparse_int* bsr_col_ind, rocsparse_int row_block_dim, rocsparse_int col_block_dim, void* temp_buffer);
  // CHECK: status_t = rocsparse_dcsr2gebsr(handle_t, direction_t, m, n, matDescr_A, &dcsrSortedValA, &csrRowPtrA, &csrColIndA, matDescr_C, &dcsrSortedValC, &bsrSortedRowPtrC, &bsrSortedColIndC, rowBlockDimA, colBlockDimA, pBuffer);
  status_t = hipsparseDcsr2gebsr(handle_t, direction_t, m, n, matDescr_A, &dcsrSortedValA, &csrRowPtrA, &csrColIndA, matDescr_C, &dcsrSortedValC, &bsrSortedRowPtrC, &bsrSortedColIndC, rowBlockDimA, colBlockDimA, pBuffer);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseScsr2gebsr(cusparseHandle_t handle, cusparseDirection_t dirA, int m, int n, const cusparseMatDescr_t descrA, const float* csrSortedValA, const int* csrSortedRowPtrA, const int* csrSortedColIndA, const cusparseMatDescr_t descrC, float* bsrSortedValC, int* bsrSortedRowPtrC, int* bsrSortedColIndC, int rowBlockDim, int colBlockDim, void* pBuffer);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_scsr2gebsr(rocsparse_handle handle, rocsparse_direction dir, rocsparse_int m, rocsparse_int n, const rocsparse_mat_descr csr_descr, const float* csr_val, const rocsparse_int* csr_row_ptr, const rocsparse_int* csr_col_ind, const rocsparse_mat_descr bsr_descr, float* bsr_val, rocsparse_int* bsr_row_ptr, rocsparse_int* bsr_col_ind, rocsparse_int row_block_dim, rocsparse_int col_block_dim, void* temp_buffer);
  // CHECK: status_t = rocsparse_scsr2gebsr(handle_t, direction_t, m, n, matDescr_A, &csrSortedValA, &csrRowPtrA, &csrColIndA, matDescr_C, &fcsrSortedValC, &bsrSortedRowPtrC, &bsrSortedColIndC, rowBlockDimA, colBlockDimA, pBuffer);
  status_t = hipsparseScsr2gebsr(handle_t, direction_t, m, n, matDescr_A, &csrSortedValA, &csrRowPtrA, &csrColIndA, matDescr_C, &fcsrSortedValC, &bsrSortedRowPtrC, &bsrSortedColIndC, rowBlockDimA, colBlockDimA, pBuffer);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseXcsr2gebsrNnz(cusparseHandle_t handle, cusparseDirection_t dirA, int m, int n, const cusparseMatDescr_t descrA, const int* csrSortedRowPtrA, const int* csrSortedColIndA, const cusparseMatDescr_t descrC, int* bsrSortedRowPtrC, int rowBlockDim, int colBlockDim, int* nnzTotalDevHostPtr, void* pBuffer);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_csr2gebsr_nnz(rocsparse_handle handle, rocsparse_direction dir, rocsparse_int m, rocsparse_int n, const rocsparse_mat_descr csr_descr, const rocsparse_int* csr_row_ptr, const rocsparse_int* csr_col_ind, const rocsparse_mat_descr bsr_descr, rocsparse_int* bsr_row_ptr, rocsparse_int row_block_dim, rocsparse_int col_block_dim, rocsparse_int* bsr_nnz_devhost, void* temp_buffer);
  // CHECK: status_t = rocsparse_csr2gebsr_nnz(handle_t, direction_t, m, n, matDescr_A, &csrRowPtrA, &csrColIndA, matDescr_C, &bsrSortedRowPtrC, rowBlockDimA, colBlockDimA, &nnzTotalDevHostPtr, pBuffer);
  status_t = hipsparseXcsr2gebsrNnz(handle_t, direction_t, m, n, matDescr_A, &csrRowPtrA, &csrColIndA, matDescr_C, &bsrSortedRowPtrC, rowBlockDimA, colBlockDimA, &nnzTotalDevHostPtr, pBuffer);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseZcsr2gebsr_bufferSize(cusparseHandle_t handle, cusparseDirection_t dirA, int m, int n, const cusparseMatDescr_t descrA, const cuDoubleComplex* csrSortedValA, const int* csrSortedRowPtrA, const int* csrSortedColIndA, int rowBlockDim, int colBlockDim, int* pBufferSizeInBytes);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_zcsr2gebsr_buffer_size(rocsparse_handle handle, rocsparse_direction dir, rocsparse_int m, rocsparse_int n, const rocsparse_mat_descr csr_descr, const rocsparse_double_complex* csr_val, const rocsparse_int* csr_row_ptr, const rocsparse_int* csr_col_ind, rocsparse_int row_block_dim, rocsparse_int col_block_dim, size_t* buffer_size);
  // CHECK: status_t = rocsparse_zcsr2gebsr_buffer_size(handle_t, direction_t, m, n, matDescr_A, &dComplexcsrSortedValA, &csrRowPtrA, &csrColIndA, rowBlockDimA, colBlockDimA, &bufferSizeInBytes);
  status_t = hipsparseZcsr2gebsr_bufferSize(handle_t, direction_t, m, n, matDescr_A, &dComplexcsrSortedValA, &csrRowPtrA, &csrColIndA, rowBlockDimA, colBlockDimA, &bufferSizeInBytes);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseCcsr2gebsr_bufferSize(cusparseHandle_t handle, cusparseDirection_t dirA, int m, int n, const cusparseMatDescr_t descrA, const cuComplex* csrSortedValA, const int* csrSortedRowPtrA, const int* csrSortedColIndA, int rowBlockDim, int colBlockDim, int* pBufferSizeInBytes);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_ccsr2gebsr_buffer_size(rocsparse_handle handle, rocsparse_direction dir, rocsparse_int m, rocsparse_int n, const rocsparse_mat_descr csr_descr, const rocsparse_float_complex* csr_val, const rocsparse_int* csr_row_ptr, const rocsparse_int* csr_col_ind, rocsparse_int row_block_dim, rocsparse_int col_block_dim, size_t* buffer_size);
 // CHECK: status_t = rocsparse_ccsr2gebsr_buffer_size(handle_t, direction_t, m, n, matDescr_A, &complexcsrSortedValA, &csrRowPtrA, &csrColIndA, rowBlockDimA, colBlockDimA, &bufferSizeInBytes);
  status_t = hipsparseCcsr2gebsr_bufferSize(handle_t, direction_t, m, n, matDescr_A, &complexcsrSortedValA, &csrRowPtrA, &csrColIndA, rowBlockDimA, colBlockDimA, &bufferSizeInBytes);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseDcsr2gebsr_bufferSize(cusparseHandle_t handle, cusparseDirection_t dirA, int m, int n, const cusparseMatDescr_t descrA, const double* csrSortedValA, const int* csrSortedRowPtrA, const int* csrSortedColIndA, int rowBlockDim, int colBlockDim, int* pBufferSizeInBytes);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_dcsr2gebsr_buffer_size(rocsparse_handle handle, rocsparse_direction dir, rocsparse_int m, rocsparse_int n, const rocsparse_mat_descr csr_descr, const double* csr_val, const rocsparse_int* csr_row_ptr, const rocsparse_int* csr_col_ind, rocsparse_int row_block_dim, rocsparse_int col_block_dim, size_t* buffer_size);
  // CHECK: status_t = rocsparse_dcsr2gebsr_buffer_size(handle_t, direction_t, m, n, matDescr_A, &dcsrSortedValA, &csrRowPtrA, &csrColIndA, rowBlockDimA, colBlockDimA, &bufferSizeInBytes);
  status_t = hipsparseDcsr2gebsr_bufferSize(handle_t, direction_t, m, n, matDescr_A, &dcsrSortedValA, &csrRowPtrA, &csrColIndA, rowBlockDimA, colBlockDimA, &bufferSizeInBytes);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseScsr2gebsr_bufferSize(cusparseHandle_t handle, cusparseDirection_t dirA, int m, int n, const cusparseMatDescr_t descrA, const float* csrSortedValA, const int* csrSortedRowPtrA, const int* csrSortedColIndA, int rowBlockDim, int colBlockDim, int* pBufferSizeInBytes);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_scsr2gebsr_buffer_size(rocsparse_handle handle, rocsparse_direction dir, rocsparse_int m, rocsparse_int n, const rocsparse_mat_descr csr_descr, const float* csr_val, const rocsparse_int* csr_row_ptr, const rocsparse_int* csr_col_ind, rocsparse_int row_block_dim, rocsparse_int col_block_dim, size_t* buffer_size);
  // CHECK: status_t = rocsparse_scsr2gebsr_buffer_size(handle_t, direction_t, m, n, matDescr_A, &csrSortedValA, &csrRowPtrA, &csrColIndA, rowBlockDimA, colBlockDimA, &bufferSizeInBytes);
  status_t = hipsparseScsr2gebsr_bufferSize(handle_t, direction_t, m, n, matDescr_A, &csrSortedValA, &csrRowPtrA, &csrColIndA, rowBlockDimA, colBlockDimA, &bufferSizeInBytes);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseZcsr2bsr(cusparseHandle_t handle, cusparseDirection_t dirA, int m, int n, const cusparseMatDescr_t descrA, const cuDoubleComplex* csrSortedValA, const int* csrSortedRowPtrA, const int* csrSortedColIndA, int blockDim, const cusparseMatDescr_t descrC, cuDoubleComplex* bsrSortedValC, int* bsrSortedRowPtrC, int* bsrSortedColIndC);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_zcsr2bsr(rocsparse_handle handle, rocsparse_direction dir, rocsparse_int m, rocsparse_int n, const rocsparse_mat_descr csr_descr, const rocsparse_double_complex* csr_val, const rocsparse_int* csr_row_ptr, const rocsparse_int* csr_col_ind, rocsparse_int block_dim, const rocsparse_mat_descr bsr_descr, rocsparse_double_complex* bsr_val, rocsparse_int* bsr_row_ptr, rocsparse_int* bsr_col_ind);
  // CHECK: status_t = rocsparse_zcsr2bsr(handle_t, direction_t, m, n, matDescr_A, &dComplexcsrSortedValA, &csrRowPtrA, &csrColIndA, blockDim, matDescr_C, &dComplexcsrSortedValC, &bsrSortedRowPtrC, &bsrSortedColIndC);
  status_t = hipsparseZcsr2bsr(handle_t, direction_t, m, n, matDescr_A, &dComplexcsrSortedValA, &csrRowPtrA, &csrColIndA, blockDim, matDescr_C, &dComplexcsrSortedValC, &bsrSortedRowPtrC, &bsrSortedColIndC);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseCcsr2bsr(cusparseHandle_t handle, cusparseDirection_t dirA, int m, int n, const cusparseMatDescr_t descrA, const cuComplex* csrSortedValA, const int* csrSortedRowPtrA, const int* csrSortedColIndA, int blockDim, const cusparseMatDescr_t descrC, cuComplex* bsrSortedValC, int* bsrSortedRowPtrC, int* bsrSortedColIndC);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_ccsr2bsr(rocsparse_handle handle, rocsparse_direction dir, rocsparse_int m, rocsparse_int n, const rocsparse_mat_descr csr_descr, const rocsparse_float_complex* csr_val, const rocsparse_int* csr_row_ptr, const rocsparse_int* csr_col_ind,rocsparse_int block_dim, const rocsparse_mat_descr bsr_descr, rocsparse_float_complex* bsr_val, rocsparse_int* bsr_row_ptr, rocsparse_int* bsr_col_ind);
  // CHECK: status_t = rocsparse_ccsr2bsr(handle_t, direction_t, m, n, matDescr_A, &complexcsrSortedValA, &csrRowPtrA, &csrColIndA, blockDim, matDescr_C, &complexcsrSortedValC, &bsrSortedRowPtrC, &bsrSortedColIndC);
  status_t = hipsparseCcsr2bsr(handle_t, direction_t, m, n, matDescr_A, &complexcsrSortedValA, &csrRowPtrA, &csrColIndA, blockDim, matDescr_C, &complexcsrSortedValC, &bsrSortedRowPtrC, &bsrSortedColIndC);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseDcsr2bsr(cusparseHandle_t handle, cusparseDirection_t dirA, int m, int n, const cusparseMatDescr_t descrA, const double* csrSortedValA, const int* csrSortedRowPtrA, const int* csrSortedColIndA, int blockDim, const cusparseMatDescr_t descrC, double* bsrSortedValC, int* bsrSortedRowPtrC, int* bsrSortedColIndC);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_dcsr2bsr(rocsparse_handle handle, rocsparse_direction dir, rocsparse_int m, rocsparse_int n, const rocsparse_mat_descr csr_descr, const double* csr_val, const rocsparse_int* csr_row_ptr, const rocsparse_int* csr_col_ind, rocsparse_int block_dim, const rocsparse_mat_descr bsr_descr, double* bsr_val, rocsparse_int* bsr_row_ptr, rocsparse_int* bsr_col_ind);
  // CHECK: status_t = rocsparse_dcsr2bsr(handle_t, direction_t, m, n, matDescr_A, &dcsrSortedValA, &csrRowPtrA, &csrColIndA, blockDim, matDescr_C, &dcsrSortedValC, &bsrSortedRowPtrC, &bsrSortedColIndC);
  status_t = hipsparseDcsr2bsr(handle_t, direction_t, m, n, matDescr_A, &dcsrSortedValA, &csrRowPtrA, &csrColIndA, blockDim, matDescr_C, &dcsrSortedValC, &bsrSortedRowPtrC, &bsrSortedColIndC);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseScsr2bsr(cusparseHandle_t handle, cusparseDirection_t dirA, int m, int n, const cusparseMatDescr_t descrA, const float* csrSortedValA, const int* csrSortedRowPtrA, const int* csrSortedColIndA, int blockDim, const cusparseMatDescr_t descrC, float* bsrSortedValC, int* bsrSortedRowPtrC, int* bsrSortedColIndC);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_scsr2bsr(rocsparse_handle handle, rocsparse_direction dir, rocsparse_int m, rocsparse_int n, const rocsparse_mat_descr csr_descr, const float* csr_val, const rocsparse_int* csr_row_ptr, const rocsparse_int* csr_col_ind, rocsparse_int block_dim, const rocsparse_mat_descr bsr_descr, float* bsr_val, rocsparse_int* bsr_row_ptr, rocsparse_int* bsr_col_ind);
  // CHECK: status_t = rocsparse_scsr2bsr(handle_t, direction_t, m, n, matDescr_A, &csrSortedValA, &csrRowPtrA, &csrColIndA, blockDim, matDescr_C, &fcsrSortedValC, &bsrSortedRowPtrC, &bsrSortedColIndC);
  status_t = hipsparseScsr2bsr(handle_t, direction_t, m, n, matDescr_A, &csrSortedValA, &csrRowPtrA, &csrColIndA, blockDim, matDescr_C, &fcsrSortedValC, &bsrSortedRowPtrC, &bsrSortedColIndC);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseXcsr2bsrNnz(cusparseHandle_t handle, cusparseDirection_t dirA, int m, int n, const cusparseMatDescr_t descrA, const int* csrSortedRowPtrA, const int* csrSortedColIndA, int blockDim, const cusparseMatDescr_t descrC, int* bsrSortedRowPtrC, int* nnzTotalDevHostPtr);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_csr2bsr_nnz(rocsparse_handle handle, rocsparse_direction dir, rocsparse_int m, rocsparse_int n, const rocsparse_mat_descr csr_descr, const rocsparse_int* csr_row_ptr, const rocsparse_int* csr_col_ind, rocsparse_int block_dim, const rocsparse_mat_descr bsr_descr, rocsparse_int* bsr_row_ptr, rocsparse_int* bsr_nnz);
  // CHECK: status_t = rocsparse_csr2bsr_nnz(handle_t, direction_t, m, n, matDescr_A, &csrRowPtrA, &csrColIndA, blockDim, matDescr_C, &bsrSortedRowPtrC, &nnzTotalDevHostPtr);
  status_t = hipsparseXcsr2bsrNnz(handle_t, direction_t, m, n, matDescr_A, &csrRowPtrA, &csrColIndA, blockDim, matDescr_C, &bsrSortedRowPtrC, &nnzTotalDevHostPtr);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseZgebsr2gebsc(cusparseHandle_t handle, int mb, int nb, int nnzb, const cuDoubleComplex* bsrSortedVal, const int* bsrSortedRowPtr, const int* bsrSortedColInd, int rowBlockDim, int colBlockDim, cuDoubleComplex* bscVal, int* bscRowInd, int* bscColPtr, cusparseAction_t copyValues, cusparseIndexBase_t idxBase, void* pBuffer);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_zgebsr2gebsc(rocsparse_handle handle, rocsparse_int mb, rocsparse_int nb, rocsparse_int nnzb, const rocsparse_double_complex* bsr_val, const rocsparse_int* bsr_row_ptr, const rocsparse_int* bsr_col_ind, rocsparse_int row_block_dim, rocsparse_int col_block_dim, rocsparse_double_complex* bsc_val, rocsparse_int* bsc_row_ind, rocsparse_int* bsc_col_ptr, rocsparse_action copy_values, rocsparse_index_base idx_base, void* temp_buffer);
  // CHECK: status_t = rocsparse_zgebsr2gebsc(handle_t, mb, nb, nnzb, &dComplexbsrSortedVal, &bsrSortedRowPtr, &bsrSortedColInd, rowBlockDim, colBlockDim, &dComplexbscVal, &bscRowInd, &bscColPtr, copyValues, indexBase_t, pBuffer);
  status_t = hipsparseZgebsr2gebsc(handle_t, mb, nb, nnzb, &dComplexbsrSortedVal, &bsrSortedRowPtr, &bsrSortedColInd, rowBlockDim, colBlockDim, &dComplexbscVal, &bscRowInd, &bscColPtr, copyValues, indexBase_t, pBuffer);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseCgebsr2gebsc(cusparseHandle_t handle, int mb, int nb, int nnzb, const cuComplex* bsrSortedVal, const int* bsrSortedRowPtr, const int* bsrSortedColInd, int rowBlockDim, int colBlockDim, cuComplex* bscVal, int* bscRowInd, int* bscColPtr, cusparseAction_t copyValues, cusparseIndexBase_t idxBase, void* pBuffer);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_cgebsr2gebsc(rocsparse_handle handle, rocsparse_int mb, rocsparse_int nb, rocsparse_int nnzb, const rocsparse_float_complex* bsr_val, const rocsparse_int* bsr_row_ptr, const rocsparse_int* bsr_col_ind, rocsparse_int row_block_dim, rocsparse_int col_block_dim, rocsparse_float_complex* bsc_val, rocsparse_int* bsc_row_ind, rocsparse_int* bsc_col_ptr, rocsparse_action copy_values, rocsparse_index_base idx_base, void* temp_buffer);
  // CHECK: status_t = rocsparse_cgebsr2gebsc(handle_t, mb, nb, nnzb, &complexbsrSortedVal, &bsrSortedRowPtr, &bsrSortedColInd, rowBlockDim, colBlockDim, &complexbscVal, &bscRowInd, &bscColPtr, copyValues, indexBase_t, pBuffer);
  status_t = hipsparseCgebsr2gebsc(handle_t, mb, nb, nnzb, &complexbsrSortedVal, &bsrSortedRowPtr, &bsrSortedColInd, rowBlockDim, colBlockDim, &complexbscVal, &bscRowInd, &bscColPtr, copyValues, indexBase_t, pBuffer);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseDgebsr2gebsc(cusparseHandle_t handle, int mb, int nb, int nnzb, const double* bsrSortedVal, const int* bsrSortedRowPtr, const int* bsrSortedColInd, int rowBlockDim, int colBlockDim, double* bscVal, int* bscRowInd, int* bscColPtr, cusparseAction_t copyValues, cusparseIndexBase_t idxBase, void* pBuffer);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_dgebsr2gebsc(rocsparse_handle handle, rocsparse_int mb, rocsparse_int nb, rocsparse_int nnzb, const double* bsr_val, const rocsparse_int* bsr_row_ptr, const rocsparse_int* bsr_col_ind, rocsparse_int row_block_dim, rocsparse_int col_block_dim, double* bsc_val, rocsparse_int* bsc_row_ind, rocsparse_int* bsc_col_ptr, rocsparse_action copy_values, rocsparse_index_base idx_base, void* temp_buffer);
  // CHECK: status_t = rocsparse_dgebsr2gebsc(handle_t, mb, nb, nnzb, &dbsrSortedVal, &bsrSortedRowPtr, &bsrSortedColInd, rowBlockDim, colBlockDim, &dbscVal, &bscRowInd, &bscColPtr, copyValues, indexBase_t, pBuffer);
  status_t = hipsparseDgebsr2gebsc(handle_t, mb, nb, nnzb, &dbsrSortedVal, &bsrSortedRowPtr, &bsrSortedColInd, rowBlockDim, colBlockDim, &dbscVal, &bscRowInd, &bscColPtr, copyValues, indexBase_t, pBuffer);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseSgebsr2gebsc(cusparseHandle_t handle, int mb, int nb, int nnzb, const float* bsrSortedVal, const int* bsrSortedRowPtr, const int* bsrSortedColInd, int rowBlockDim, int colBlockDim, float* bscVal, int* bscRowInd, int* bscColPtr, cusparseAction_t copyValues, cusparseIndexBase_t idxBase, void* pBuffer);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_sgebsr2gebsc(rocsparse_handle handle, rocsparse_int mb, rocsparse_int nb, rocsparse_int nnzb, const float* bsr_val, const rocsparse_int* bsr_row_ptr, const rocsparse_int* bsr_col_ind, rocsparse_int row_block_dim, rocsparse_int col_block_dim, float* bsc_val, rocsparse_int* bsc_row_ind, rocsparse_int* bsc_col_ptr, rocsparse_action copy_values, rocsparse_index_base idx_base, void* temp_buffer);
  // CHECK: status_t = rocsparse_sgebsr2gebsc(handle_t, mb, nb, nnzb, &fbsrSortedVal, &bsrSortedRowPtr, &bsrSortedColInd, rowBlockDim, colBlockDim, &fbscVal, &bscRowInd, &bscColPtr, copyValues, indexBase_t, pBuffer);
  status_t = hipsparseSgebsr2gebsc(handle_t, mb, nb, nnzb, &fbsrSortedVal, &bsrSortedRowPtr, &bsrSortedColInd, rowBlockDim, colBlockDim, &fbscVal, &bscRowInd, &bscColPtr, copyValues, indexBase_t, pBuffer);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseZgebsr2gebsc_bufferSize(cusparseHandle_t handle, int mb, int nb, int nnzb, const cuDoubleComplex* bsrSortedVal, const int* bsrSortedRowPtr, const int* bsrSortedColInd, int rowBlockDim, int colBlockDim, int* pBufferSizeInBytes);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_zgebsr2gebsc_buffer_size(rocsparse_handle handle, rocsparse_int mb, rocsparse_int nb, rocsparse_int nnzb, const rocsparse_double_complex* bsr_val, const rocsparse_int* bsr_row_ptr, const rocsparse_int* bsr_col_ind, rocsparse_int row_block_dim, rocsparse_int col_block_dim, size_t* p_buffer_size);
  // CHECK: status_t = rocsparse_zgebsr2gebsc_buffer_size(handle_t, mb, nb, nnzb, &dComplexbsrSortedVal, &bsrSortedRowPtr, &bsrSortedColInd, rowBlockDim, colBlockDim, &bufferSizeInBytes);
  status_t = hipsparseZgebsr2gebsc_bufferSize(handle_t, mb, nb, nnzb, &dComplexbsrSortedVal, &bsrSortedRowPtr, &bsrSortedColInd, rowBlockDim, colBlockDim, &bufferSizeInBytes);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseCgebsr2gebsc_bufferSize(cusparseHandle_t handle, int mb, int nb, int nnzb, const cuComplex* bsrSortedVal, const int* bsrSortedRowPtr, const int* bsrSortedColInd, int rowBlockDim, int colBlockDim, int* pBufferSizeInBytes);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_cgebsr2gebsc_buffer_size(rocsparse_handle handle, rocsparse_int mb, rocsparse_int nb, rocsparse_int nnzb, const rocsparse_float_complex* bsr_val, const rocsparse_int* bsr_row_ptr, const rocsparse_int* bsr_col_ind, rocsparse_int row_block_dim, rocsparse_int col_block_dim, size_t* p_buffer_size);
  // CHECK: status_t = rocsparse_cgebsr2gebsc_buffer_size(handle_t, mb, nb, nnzb, &complexbsrSortedVal, &bsrSortedRowPtr, &bsrSortedColInd, rowBlockDim, colBlockDim, &bufferSizeInBytes);
  status_t = hipsparseCgebsr2gebsc_bufferSize(handle_t, mb, nb, nnzb, &complexbsrSortedVal, &bsrSortedRowPtr, &bsrSortedColInd, rowBlockDim, colBlockDim, &bufferSizeInBytes);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseDgebsr2gebsc_bufferSize(cusparseHandle_t handle, int mb, int nb, int nnzb, const double* bsrSortedVal, const int* bsrSortedRowPtr, const int* bsrSortedColInd, int rowBlockDim, int colBlockDim, int* pBufferSizeInBytes);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_dgebsr2gebsc_buffer_size(rocsparse_handle handle, rocsparse_int mb, rocsparse_int nb, rocsparse_int nnzb, const double* bsr_val, const rocsparse_int* bsr_row_ptr, const rocsparse_int* bsr_col_ind, rocsparse_int row_block_dim, rocsparse_int col_block_dim, size_t* p_buffer_size);
  // CHECK: status_t = rocsparse_dgebsr2gebsc_buffer_size(handle_t, mb, nb, nnzb, &dbsrSortedVal, &bsrSortedRowPtr, &bsrSortedColInd, rowBlockDim, colBlockDim, &bufferSizeInBytes);
  status_t = hipsparseDgebsr2gebsc_bufferSize(handle_t, mb, nb, nnzb, &dbsrSortedVal, &bsrSortedRowPtr, &bsrSortedColInd, rowBlockDim, colBlockDim, &bufferSizeInBytes);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseSgebsr2gebsc_bufferSize(cusparseHandle_t handle, int mb, int nb, int nnzb, const float* bsrSortedVal, const int* bsrSortedRowPtr, const int* bsrSortedColInd, int rowBlockDim, int colBlockDim, int* pBufferSizeInBytes);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_sgebsr2gebsc_buffer_size(rocsparse_handle handle, rocsparse_int mb, rocsparse_int nb, rocsparse_int nnzb, const float* bsr_val, const rocsparse_int* bsr_row_ptr, const rocsparse_int* bsr_col_ind, rocsparse_int row_block_dim, rocsparse_int col_block_dim, size_t* p_buffer_size);
  // CHECK: status_t = rocsparse_sgebsr2gebsc_buffer_size(handle_t, mb, nb, nnzb, &fbsrSortedVal, &bsrSortedRowPtr, &bsrSortedColInd, rowBlockDim, colBlockDim, &bufferSizeInBytes);
  status_t = hipsparseSgebsr2gebsc_bufferSize(handle_t, mb, nb, nnzb, &fbsrSortedVal, &bsrSortedRowPtr, &bsrSortedColInd, rowBlockDim, colBlockDim, &bufferSizeInBytes);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseXcsr2coo(cusparseHandle_t handle, const int* csrSortedRowPtr, int nnz, int m, int* cooRowInd, cusparseIndexBase_t idxBase);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_csr2coo(rocsparse_handle handle, const rocsparse_int* csr_row_ptr, rocsparse_int nnz, rocsparse_int m, rocsparse_int* coo_row_ind, rocsparse_index_base idx_base);
  // CHECK: status_t = rocsparse_csr2coo(handle_t, &csrSortedRowPtr, nnz, m, &icooRowInd, indexBase_t);
  status_t = hipsparseXcsr2coo(handle_t, &csrSortedRowPtr, nnz, m, &icooRowInd, indexBase_t);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseZnnz(cusparseHandle_t handle, cusparseDirection_t dirA, int m, int n, const cusparseMatDescr_t descrA, const cuDoubleComplex* A, int lda, int* nnzPerRowCol, int* nnzTotalDevHostPtr);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_znnz(rocsparse_handle handle, rocsparse_direction dir, rocsparse_int m, rocsparse_int n, const rocsparse_mat_descr descr, const rocsparse_double_complex* A, rocsparse_int ld, rocsparse_int* nnz_per_row_columns, rocsparse_int* nnz_total_dev_host_ptr);
  // CHECK: status_t = rocsparse_znnz(handle_t, direction_t, m, n, matDescr_A, &dcomplexA, lda, &nnzPerRowCol, &nnzTotalDevHostPtr);
  status_t = hipsparseZnnz(handle_t, direction_t, m, n, matDescr_A, &dcomplexA, lda, &nnzPerRowCol, &nnzTotalDevHostPtr);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseCnnz(cusparseHandle_t handle, cusparseDirection_t dirA, int m, int n, const cusparseMatDescr_t descrA, const cuComplex* A, int lda, int* nnzPerRowCol, int* nnzTotalDevHostPtr);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_cnnz(rocsparse_handle handle, rocsparse_direction dir, rocsparse_int m, rocsparse_int n, const rocsparse_mat_descr descr, const rocsparse_float_complex* A, rocsparse_int ld, rocsparse_int* nnz_per_row_columns, rocsparse_int* nnz_total_dev_host_ptr);
  // CHECK: status_t = rocsparse_cnnz(handle_t, direction_t, m, n, matDescr_A, &complexA, lda, &nnzPerRowCol, &nnzTotalDevHostPtr);
  status_t = hipsparseCnnz(handle_t, direction_t, m, n, matDescr_A, &complexA, lda, &nnzPerRowCol, &nnzTotalDevHostPtr);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseDnnz(cusparseHandle_t handle, cusparseDirection_t dirA, int m, int n, const cusparseMatDescr_t descrA, const double* A, int lda, int* nnzPerRowCol, int* nnzTotalDevHostPtr);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_dnnz(rocsparse_handle handle, rocsparse_direction dir, rocsparse_int m, rocsparse_int n, const rocsparse_mat_descr descr, const double* A, rocsparse_int ld, rocsparse_int* nnz_per_row_columns, rocsparse_int* nnz_total_dev_host_ptr);
  // CHECK: status_t = rocsparse_dnnz(handle_t, direction_t, m, n, matDescr_A, &dA, lda, &nnzPerRowCol, &nnzTotalDevHostPtr);
  status_t = hipsparseDnnz(handle_t, direction_t, m, n, matDescr_A, &dA, lda, &nnzPerRowCol, &nnzTotalDevHostPtr);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseSnnz(cusparseHandle_t handle, cusparseDirection_t dirA, int m, int n, const cusparseMatDescr_t descrA, const float* A, int lda, int* nnzPerRowCol, int* nnzTotalDevHostPtr);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_snnz(rocsparse_handle handle, rocsparse_direction dir, rocsparse_int m, rocsparse_int n, const rocsparse_mat_descr descr, const float* A, rocsparse_int ld, rocsparse_int* nnz_per_row_columns, rocsparse_int* nnz_total_dev_host_ptr);
  // CHECK: status_t = rocsparse_snnz(handle_t, direction_t, m, n, matDescr_A, &fA, lda, &nnzPerRowCol, &nnzTotalDevHostPtr);
  status_t = hipsparseSnnz(handle_t, direction_t, m, n, matDescr_A, &fA, lda, &nnzPerRowCol, &nnzTotalDevHostPtr);

#if CUDA_VERSION >= 8000
  // CHECK: hipDataType dataType_t;
  // TODO: [#899] There should be rocsparse_datatype
  // CHECK-NEXT: hipDataType dataType;
  hipDataType dataType_t;
  hipDataType dataType;

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseZcsr2csr_compress(cusparseHandle_t handle, int m, int n, const cusparseMatDescr_t descrA, const cuDoubleComplex* csrSortedValA, const int* csrSortedColIndA, const int* csrSortedRowPtrA, int nnzA, const int* nnzPerRow, cuDoubleComplex* csrSortedValC, int* csrSortedColIndC, int* csrSortedRowPtrC, cuDoubleComplex tol);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_zcsr2csr_compress(rocsparse_handle handle, rocsparse_int m, rocsparse_int n, const rocsparse_mat_descr descr_A, const rocsparse_double_complex* csr_val_A, const rocsparse_int* csr_row_ptr_A, const rocsparse_int* csr_col_ind_A, rocsparse_int nnz_A, const rocsparse_int* nnz_per_row, rocsparse_double_complex* csr_val_C, rocsparse_int* csr_row_ptr_C, rocsparse_int* csr_col_ind_C, rocsparse_double_complex tol);
  // CHECK: status_t = rocsparse_zcsr2csr_compress(handle_t, m, n, matDescr_A, &dComplexcsrSortedValA, &csrColIndA, &csrRowPtrA, nnza, &nnzPerRow, &dComplexcsrSortedValC, &csrColIndC, &csrRowPtrC, dcomplextol);
  status_t = hipsparseZcsr2csr_compress(handle_t, m, n, matDescr_A, &dComplexcsrSortedValA, &csrColIndA, &csrRowPtrA, nnza, &nnzPerRow, &dComplexcsrSortedValC, &csrColIndC, &csrRowPtrC, dcomplextol);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseCcsr2csr_compress(cusparseHandle_t handle, int m, int n, const cusparseMatDescr_t descrA, const cuComplex* csrSortedValA, const int* csrSortedColIndA, const int* csrSortedRowPtrA, int nnzA, const int* nnzPerRow, cuComplex* csrSortedValC, int* csrSortedColIndC, int* csrSortedRowPtrC, cuComplex tol);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_ccsr2csr_compress(rocsparse_handle handle, rocsparse_int m, rocsparse_int n, const rocsparse_mat_descr descr_A, const rocsparse_float_complex* csr_val_A, const rocsparse_int* csr_row_ptr_A, const rocsparse_int* csr_col_ind_A, rocsparse_int nnz_A, const rocsparse_int* nnz_per_row, rocsparse_float_complex* csr_val_C, rocsparse_int* csr_row_ptr_C, rocsparse_int* csr_col_ind_C, rocsparse_float_complex tol);
  // CHECK: status_t = rocsparse_ccsr2csr_compress(handle_t, m, n, matDescr_A, &complexcsrSortedValA, &csrColIndA, &csrRowPtrA, nnza, &nnzPerRow, &complexcsrSortedValC, &csrColIndC, &csrRowPtrC, complextol);
  status_t = hipsparseCcsr2csr_compress(handle_t, m, n, matDescr_A, &complexcsrSortedValA, &csrColIndA, &csrRowPtrA, nnza, &nnzPerRow, &complexcsrSortedValC, &csrColIndC, &csrRowPtrC, complextol);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseDcsr2csr_compress(cusparseHandle_t handle, int m, int n, const cusparseMatDescr_t descrA, const double* csrSortedValA, const int* csrSortedColIndA, const int* csrSortedRowPtrA, int nnzA, const int* nnzPerRow, double* csrSortedValC, int* csrSortedColIndC, int* csrSortedRowPtrC, double tol);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_dcsr2csr_compress(rocsparse_handle handle, rocsparse_int m, rocsparse_int n, const rocsparse_mat_descr descr_A, const double* csr_val_A, const rocsparse_int* csr_row_ptr_A, const rocsparse_int* csr_col_ind_A, rocsparse_int nnz_A, const rocsparse_int* nnz_per_row, double* csr_val_C, rocsparse_int* csr_row_ptr_C, rocsparse_int* csr_col_ind_C, double tol);
  // CHECK: status_t = rocsparse_dcsr2csr_compress(handle_t, m, n, matDescr_A, &dcsrSortedValA, &csrColIndA, &csrRowPtrA, nnza, &nnzPerRow, &dcsrSortedValC, &csrColIndC, &csrRowPtrC, dtol);
  status_t = hipsparseDcsr2csr_compress(handle_t, m, n, matDescr_A, &dcsrSortedValA, &csrColIndA, &csrRowPtrA, nnza, &nnzPerRow, &dcsrSortedValC, &csrColIndC, &csrRowPtrC, dtol);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseScsr2csr_compress(cusparseHandle_t handle, int m, int n, const cusparseMatDescr_t descrA, const float* csrSortedValA, const int* csrSortedColIndA, const int* csrSortedRowPtrA, int nnzA, const int* nnzPerRow, float* csrSortedValC, int* csrSortedColIndC, int* csrSortedRowPtrC, float tol);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_scsr2csr_compress(rocsparse_handle handle, rocsparse_int m, rocsparse_int n, const rocsparse_mat_descr descr_A, const float* csr_val_A, const rocsparse_int* csr_row_ptr_A, const rocsparse_int* csr_col_ind_A, rocsparse_int nnz_A, const rocsparse_int* nnz_per_row, float* csr_val_C, rocsparse_int* csr_row_ptr_C, rocsparse_int* csr_col_ind_C, float tol);
  // CHECK: status_t = rocsparse_scsr2csr_compress(handle_t, m, n, matDescr_A, &csrSortedValA, &csrColIndA, &csrRowPtrA, nnza, &nnzPerRow, &fcsrSortedValC, &csrColIndC, &csrRowPtrC, ftol);
  status_t = hipsparseScsr2csr_compress(handle_t, m, n, matDescr_A, &csrSortedValA, &csrColIndA, &csrRowPtrA, nnza, &nnzPerRow, &fcsrSortedValC, &csrColIndC, &csrRowPtrC, ftol);

  // CUDA: CUSPARSE_DEPRECATED cusparseStatus_t CUSPARSEAPI cusparseZnnz_compress(cusparseHandle_t handle, int m, const cusparseMatDescr_t descr, const cuDoubleComplex* csrSortedValA, const int* csrSortedRowPtrA, int* nnzPerRow, int* nnzC, cuDoubleComplex tol);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_znnz_compress(rocsparse_handle handle, rocsparse_int m, const rocsparse_mat_descr descr_A, const rocsparse_double_complex* csr_val_A, const rocsparse_int* csr_row_ptr_A, rocsparse_int* nnz_per_row, rocsparse_int* nnz_C, rocsparse_double_complex tol);
  // CHECK: status_t = rocsparse_znnz_compress(handle_t, m, matDescr_A, &dComplexcsrSortedValA, &csrRowPtrA, &nnzPerRow, &nnzc, dcomplextol);
  status_t = hipsparseZnnz_compress(handle_t, m, matDescr_A, &dComplexcsrSortedValA, &csrRowPtrA, &nnzPerRow, &nnzc, dcomplextol);

  // CUDA: CUSPARSE_DEPRECATED cusparseStatus_t CUSPARSEAPI cusparseCnnz_compress(cusparseHandle_t handle, int m, const cusparseMatDescr_t descr, const cuComplex* csrSortedValA, const int* csrSortedRowPtrA, int* nnzPerRow, int* nnzC, cuComplex tol);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_cnnz_compress(rocsparse_handle handle, rocsparse_int m, const rocsparse_mat_descr descr_A, const rocsparse_float_complex* csr_val_A, const rocsparse_int* csr_row_ptr_A, rocsparse_int* nnz_per_row, rocsparse_int* nnz_C, rocsparse_float_complex tol);
  // CHECK: status_t = rocsparse_cnnz_compress(handle_t, m, matDescr_A, &complexcsrSortedValA, &csrRowPtrA, &nnzPerRow, &nnzc, complextol);
  status_t = hipsparseCnnz_compress(handle_t, m, matDescr_A, &complexcsrSortedValA, &csrRowPtrA, &nnzPerRow, &nnzc, complextol);

  // CUDA: CUSPARSE_DEPRECATED cusparseStatus_t CUSPARSEAPI cusparseDnnz_compress(cusparseHandle_t handle, int m, const cusparseMatDescr_t descr, const double* csrSortedValA, const int* csrSortedRowPtrA, int* nnzPerRow, int* nnzC, double tol);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_dnnz_compress(rocsparse_handle handle, rocsparse_int m, const rocsparse_mat_descr descr_A, const double* csr_val_A, const rocsparse_int* csr_row_ptr_A, rocsparse_int* nnz_per_row, rocsparse_int* nnz_C, double tol);
  // CHECK: status_t = rocsparse_dnnz_compress(handle_t, m, matDescr_A, &dcsrSortedValA, &csrRowPtrA, &nnzPerRow, &nnzc, dtol);
  status_t = hipsparseDnnz_compress(handle_t, m, matDescr_A, &dcsrSortedValA, &csrRowPtrA, &nnzPerRow, &nnzc, dtol);

  // CUDA: CUSPARSE_DEPRECATED cusparseStatus_t CUSPARSEAPI cusparseSnnz_compress(cusparseHandle_t handle, int m, const cusparseMatDescr_t descr, const float* csrSortedValA, const int* csrSortedRowPtrA, int* nnzPerRow, int* nnzC, float tol);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_snnz_compress(rocsparse_handle handle, rocsparse_int m, const rocsparse_mat_descr descr_A, const float* csr_val_A, const rocsparse_int* csr_row_ptr_A, rocsparse_int* nnz_per_row, rocsparse_int* nnz_C, float tol);
  // CHECK: status_t = rocsparse_snnz_compress(handle_t, m, matDescr_A, &csrSortedValA, &csrRowPtrA, &nnzPerRow, &nnzc, ftol);
  status_t = hipsparseSnnz_compress(handle_t, m, matDescr_A, &csrSortedValA, &csrRowPtrA, &nnzPerRow, &nnzc, ftol);
#endif

#if CUDA_VERSION >= 8000 && CUDA_VERSION < 12000
  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseCopyMatDescr(cusparseMatDescr_t dest, const cusparseMatDescr_t src);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_copy_mat_descr(rocsparse_mat_descr dest, const rocsparse_mat_descr src);
  // CHECK: status_t = rocsparse_copy_mat_descr(matDescr_t, matDescr_t_2);
  status_t = cusparseCopyMatDescr(matDescr_t, matDescr_t_2);
#endif

#if CUDA_VERSION >= 9000
  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseDpruneCsr2csrByPercentage(cusparseHandle_t handle, int m, int n, int nnzA, const cusparseMatDescr_t descrA, const double* csrSortedValA, const int* csrSortedRowPtrA, const int* csrSortedColIndA, float percentage, const cusparseMatDescr_t descrC, double* csrSortedValC, const int* csrSortedRowPtrC, int* csrSortedColIndC, pruneInfo_t info, void* pBuffer);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_dprune_csr2csr_by_percentage(rocsparse_handle handle, rocsparse_int m, rocsparse_int n, rocsparse_int nnz_A, const rocsparse_mat_descr csr_descr_A, const double* csr_val_A, const rocsparse_int* csr_row_ptr_A, const rocsparse_int* csr_col_ind_A, double percentage, const rocsparse_mat_descr csr_descr_C, double* csr_val_C, const rocsparse_int* csr_row_ptr_C, rocsparse_int* csr_col_ind_C, rocsparse_mat_info info, void* temp_buffer);
  // CHECK: status_t = rocsparse_dprune_csr2csr_by_percentage(handle_t, m, n, nnz, matDescr_A, &csrValA, &csrRowPtrA, &csrColIndA, percentage, matDescr_C, &csrValC, &csrRowPtrC, &csrColIndC, prune_info, pBuffer);
  status_t = hipsparseDpruneCsr2csrByPercentage(handle_t, m, n, nnz, matDescr_A, &csrValA, &csrRowPtrA, &csrColIndA, percentage, matDescr_C, &csrValC, &csrRowPtrC, &csrColIndC, prune_info, pBuffer);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseSpruneCsr2csrByPercentage(cusparseHandle_t handle, int m, int n, int nnzA, const cusparseMatDescr_t descrA, const float* csrSortedValA, const int* csrSortedRowPtrA, const int* csrSortedColIndA, float percentage, const cusparseMatDescr_t descrC, float* csrSortedValC, const int* csrSortedRowPtrC, int* csrSortedColIndC, pruneInfo_t info, void* pBuffer);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_sprune_csr2csr_by_percentage(rocsparse_handle handle, rocsparse_int m, rocsparse_int n, rocsparse_int nnz_A, const rocsparse_mat_descr csr_descr_A, const float* csr_val_A, const rocsparse_int* csr_row_ptr_A, const rocsparse_int* csr_col_ind_A, float percentage, const rocsparse_mat_descr csr_descr_C, float* csr_val_C, const rocsparse_int* csr_row_ptr_C, rocsparse_int* csr_col_ind_C, rocsparse_mat_info info, void* temp_buffer);
  // CHECK: status_t = rocsparse_sprune_csr2csr_by_percentage(handle_t, m, n, nnz, matDescr_A, &fcsrValA, &csrRowPtrA, &csrColIndA, percentage, matDescr_C, &fcsrValC, &csrRowPtrC, &csrColIndC, prune_info, pBuffer);
  status_t = hipsparseSpruneCsr2csrByPercentage(handle_t, m, n, nnz, matDescr_A, &fcsrValA, &csrRowPtrA, &csrColIndA, percentage, matDescr_C, &fcsrValC, &csrRowPtrC, &csrColIndC, prune_info, pBuffer);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseDpruneCsr2csrNnzByPercentage(cusparseHandle_t handle, int m, int n, int nnzA, const cusparseMatDescr_t descrA, const double* csrSortedValA, const int* csrSortedRowPtrA, const int* csrSortedColIndA, float percentage, const cusparseMatDescr_t descrC, int* csrSortedRowPtrC, int* nnzTotalDevHostPtr, pruneInfo_t info, void* pBuffer);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_dprune_csr2csr_nnz_by_percentage(rocsparse_handle handle, rocsparse_int m, rocsparse_int n, rocsparse_int nnz_A, const rocsparse_mat_descr csr_descr_A, const double* csr_val_A, const rocsparse_int* csr_row_ptr_A, const rocsparse_int* csr_col_ind_A, double percentage, const rocsparse_mat_descr csr_descr_C, rocsparse_int* csr_row_ptr_C, rocsparse_int* nnz_total_dev_host_ptr, rocsparse_mat_info info, void* temp_buffer);
  // CHECK: status_t = rocsparse_dprune_csr2csr_nnz_by_percentage(handle_t, m, n, nnz, matDescr_A, &dbsrSortedValA, &csrRowPtrA, &csrColIndA, percentage, matDescr_C, &csrRowPtrC, &csrColIndC, prune_info, pBuffer);
  status_t = hipsparseDpruneCsr2csrNnzByPercentage(handle_t, m, n, nnz, matDescr_A, &dbsrSortedValA, &csrRowPtrA, &csrColIndA, percentage, matDescr_C, &csrRowPtrC, &csrColIndC, prune_info, pBuffer);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseSpruneCsr2csrNnzByPercentage(cusparseHandle_t handle, int m, int n, int nnzA, const cusparseMatDescr_t descrA, const float* csrSortedValA, const int* csrSortedRowPtrA, const int* csrSortedColIndA, float percentage, const cusparseMatDescr_t descrC, int* csrSortedRowPtrC, int* nnzTotalDevHostPtr, pruneInfo_t info, void* pBuffer);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_sprune_csr2csr_nnz_by_percentage(rocsparse_handle handle, rocsparse_int m, rocsparse_int n, rocsparse_int nnz_A, const rocsparse_mat_descr csr_descr_A, const float* csr_val_A, const rocsparse_int* csr_row_ptr_A, const rocsparse_int* csr_col_ind_A, float percentage, const rocsparse_mat_descr csr_descr_C, rocsparse_int* csr_row_ptr_C, rocsparse_int* nnz_total_dev_host_ptr, rocsparse_mat_info info, void* temp_buffer);
  // CHECK: status_t = rocsparse_sprune_csr2csr_nnz_by_percentage(handle_t, m, n, nnz, matDescr_A, &csrSortedValA, &csrRowPtrA, &csrColIndA, percentage, matDescr_C, &csrRowPtrC, &csrColIndC, prune_info, pBuffer);
  status_t = hipsparseSpruneCsr2csrNnzByPercentage(handle_t, m, n, nnz, matDescr_A, &csrSortedValA, &csrRowPtrA, &csrColIndA, percentage, matDescr_C, &csrRowPtrC, &csrColIndC, prune_info, pBuffer);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseDpruneCsr2csrByPercentage_bufferSizeExt(cusparseHandle_t handle, int m, int n, int nnzA, const cusparseMatDescr_t descrA, const double* csrSortedValA, const int* csrSortedRowPtrA, const int* csrSortedColIndA, float percentage, const cusparseMatDescr_t descrC, const double* csrSortedValC, const int* csrSortedRowPtrC, const int* csrSortedColIndC, pruneInfo_t info, size_t* pBufferSizeInBytes);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_dprune_csr2csr_by_percentage_buffer_size(rocsparse_handle handle, rocsparse_int m, rocsparse_int n, rocsparse_int nnz_A, const rocsparse_mat_descr csr_descr_A, const double* csr_val_A, const rocsparse_int* csr_row_ptr_A, const rocsparse_int* csr_col_ind_A, double percentage, const rocsparse_mat_descr csr_descr_C, const double* csr_val_C, const rocsparse_int* csr_row_ptr_C, const rocsparse_int* csr_col_ind_C, rocsparse_mat_info info, size_t* buffer_size);
  // CHECK: status_t = rocsparse_dprune_csr2csr_by_percentage_buffer_size(handle_t, m, n, nnz, matDescr_A, &dbsrSortedValA, &csrRowPtrA, &csrColIndA, percentage, matDescr_C, &dbsrSortedValC, &csrRowPtrC, &csrColIndC, prune_info, &bufferSize);
  status_t = hipsparseDpruneCsr2csrByPercentage_bufferSizeExt(handle_t, m, n, nnz, matDescr_A, &dbsrSortedValA, &csrRowPtrA, &csrColIndA, percentage, matDescr_C, &dbsrSortedValC, &csrRowPtrC, &csrColIndC, prune_info, &bufferSize);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseSpruneCsr2csrByPercentage_bufferSizeExt(cusparseHandle_t handle, int m, int n, int nnzA, const cusparseMatDescr_t descrA, const float* csrSortedValA, const int* csrSortedRowPtrA, const int* csrSortedColIndA, float percentage, const cusparseMatDescr_t descrC, const float* csrSortedValC, const int* csrSortedRowPtrC, const int* csrSortedColIndC, pruneInfo_t info, size_t* pBufferSizeInBytes);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_sprune_csr2csr_by_percentage_buffer_size(rocsparse_handle handle, rocsparse_int m, rocsparse_int n, rocsparse_int nnz_A, const rocsparse_mat_descr csr_descr_A, const float* csr_val_A, const rocsparse_int* csr_row_ptr_A, const rocsparse_int* csr_col_ind_A, float percentage, const rocsparse_mat_descr csr_descr_C, const float* csr_val_C, const rocsparse_int* csr_row_ptr_C, const rocsparse_int* csr_col_ind_C, rocsparse_mat_info info, size_t* buffer_size);
  // CHECK: status_t = rocsparse_sprune_csr2csr_by_percentage_buffer_size(handle_t, m, n, nnz, matDescr_A, &fbsrSortedValA, &csrRowPtrA, &csrColIndA, percentage, matDescr_C, &fbsrSortedValC, &csrRowPtrC, &csrColIndC, prune_info, &bufferSize);
  status_t = hipsparseSpruneCsr2csrByPercentage_bufferSizeExt(handle_t, m, n, nnz, matDescr_A, &fbsrSortedValA, &csrRowPtrA, &csrColIndA, percentage, matDescr_C, &fbsrSortedValC, &csrRowPtrC, &csrColIndC, prune_info, &bufferSize);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseDpruneCsr2csr(cusparseHandle_t handle, int m, int n, int nnzA, const cusparseMatDescr_t descrA, const double* csrSortedValA, const int* csrSortedRowPtrA, const int* csrSortedColIndA, const double* threshold, const cusparseMatDescr_t descrC, double* csrSortedValC, const int* csrSortedRowPtrC, int* csrSortedColIndC, void* pBuffer);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_dprune_csr2csr(rocsparse_handle handle, rocsparse_int m, rocsparse_int n, rocsparse_int nnz_A, const rocsparse_mat_descr csr_descr_A, const double* csr_val_A, const rocsparse_int* csr_row_ptr_A, const rocsparse_int* csr_col_ind_A, const double* threshold, const rocsparse_mat_descr csr_descr_C, double* csr_val_C, const rocsparse_int* csr_row_ptr_C, rocsparse_int* csr_col_ind_C, void* temp_buffer);
  // CHECK: status_t = rocsparse_dprune_csr2csr(handle_t, m, n, nnz, matDescr_A, &dbsrSortedValA, &csrRowPtrA, &csrColIndA, &dthreshold, matDescr_C, &dbsrSortedValC, &csrRowPtrC, &csrColIndC, pBuffer);
  status_t = hipsparseDpruneCsr2csr(handle_t, m, n, nnz, matDescr_A, &dbsrSortedValA, &csrRowPtrA, &csrColIndA, &dthreshold, matDescr_C, &dbsrSortedValC, &csrRowPtrC, &csrColIndC, pBuffer);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseSpruneCsr2csr(cusparseHandle_t handle, int m, int n, int nnzA, const cusparseMatDescr_t descrA, const float* csrSortedValA, const int* csrSortedRowPtrA, const int* csrSortedColIndA, const float* threshold, const cusparseMatDescr_t descrC, float* csrSortedValC, const int* csrSortedRowPtrC, int* csrSortedColIndC, void* pBuffer);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_sprune_csr2csr(rocsparse_handle handle, rocsparse_int m, rocsparse_int n, rocsparse_int nnz_A, const rocsparse_mat_descr csr_descr_A, const float* csr_val_A, const rocsparse_int* csr_row_ptr_A, const rocsparse_int* csr_col_ind_A, const float* threshold, const rocsparse_mat_descr csr_descr_C, float* csr_val_C, const rocsparse_int* csr_row_ptr_C, rocsparse_int* csr_col_ind_C, void* temp_buffer);
  // CHECK: status_t = rocsparse_sprune_csr2csr(handle_t, m, n, nnz, matDescr_A, &fbsrSortedValA, &csrRowPtrA, &csrColIndA, &fthreshold, matDescr_C, &fbsrSortedValC, &csrRowPtrC, &csrColIndC, pBuffer);
  status_t = hipsparseSpruneCsr2csr(handle_t, m, n, nnz, matDescr_A, &fbsrSortedValA, &csrRowPtrA, &csrColIndA, &fthreshold, matDescr_C, &fbsrSortedValC, &csrRowPtrC, &csrColIndC, pBuffer);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseDpruneCsr2csrNnz(cusparseHandle_t handle, int m, int n, int nnzA, const cusparseMatDescr_t descrA, const double* csrSortedValA, const int* csrSortedRowPtrA, const int* csrSortedColIndA, const double* threshold, const cusparseMatDescr_t descrC, int* csrSortedRowPtrC, int* nnzTotalDevHostPtr, void* pBuffer);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_dprune_csr2csr_nnz(rocsparse_handle handle, rocsparse_int m, rocsparse_int n, rocsparse_int nnz_A, const rocsparse_mat_descr csr_descr_A, const double* csr_val_A, const rocsparse_int* csr_row_ptr_A, const rocsparse_int* csr_col_ind_A, const double* threshold, const rocsparse_mat_descr csr_descr_C, rocsparse_int* csr_row_ptr_C, rocsparse_int* nnz_total_dev_host_ptr, void* temp_buffer);
  // CHECK: status_t = rocsparse_dprune_csr2csr_nnz(handle_t, m, n, nnz, matDescr_A, &dbsrSortedValA, &csrRowPtrA, &csrColIndA, &dthreshold, matDescr_C, &csrRowPtrC, &nnzTotalDevHostPtr, pBuffer);
  status_t = hipsparseDpruneCsr2csrNnz(handle_t, m, n, nnz, matDescr_A, &dbsrSortedValA, &csrRowPtrA, &csrColIndA, &dthreshold, matDescr_C, &csrRowPtrC, &nnzTotalDevHostPtr, pBuffer);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseSpruneCsr2csrNnz(cusparseHandle_t handle, int m, int n, int nnzA, const cusparseMatDescr_t descrA, const float* csrSortedValA, const int* csrSortedRowPtrA, const int* csrSortedColIndA, const float* threshold, const cusparseMatDescr_t descrC, int* csrSortedRowPtrC, int* nnzTotalDevHostPtr, void* pBuffer);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_sprune_csr2csr_nnz(rocsparse_handle handle, rocsparse_int m, rocsparse_int n, rocsparse_int nnz_A, const rocsparse_mat_descr csr_descr_A, const float* csr_val_A, const rocsparse_int* csr_row_ptr_A, const rocsparse_int* csr_col_ind_A, const float* threshold, const rocsparse_mat_descr csr_descr_C, rocsparse_int* csr_row_ptr_C, rocsparse_int* nnz_total_dev_host_ptr, void* temp_buffer);
  // CHECK: status_t = rocsparse_sprune_csr2csr_nnz(handle_t, m, n, nnz, matDescr_A, &csrSortedValA, &csrRowPtrA, &csrColIndA, &fthreshold, matDescr_C, &csrRowPtrC, &nnzTotalDevHostPtr, pBuffer);
  status_t = hipsparseSpruneCsr2csrNnz(handle_t, m, n, nnz, matDescr_A, &csrSortedValA, &csrRowPtrA, &csrColIndA, &fthreshold, matDescr_C, &csrRowPtrC, &nnzTotalDevHostPtr, pBuffer);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseDpruneCsr2csr_bufferSizeExt(cusparseHandle_t handle, int m, int n, int nnzA, const cusparseMatDescr_t descrA, const double* csrSortedValA, const int* csrSortedRowPtrA, const int* csrSortedColIndA, const double* threshold, const cusparseMatDescr_t descrC, const double* csrSortedValC, const int* csrSortedRowPtrC, const int* csrSortedColIndC, size_t* pBufferSizeInBytes);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_dprune_csr2csr_buffer_size(rocsparse_handle handle, rocsparse_int m, rocsparse_int n, rocsparse_int nnz_A, const rocsparse_mat_descr csr_descr_A, const double* csr_val_A, const rocsparse_int* csr_row_ptr_A, const rocsparse_int* csr_col_ind_A, const double* threshold, const rocsparse_mat_descr csr_descr_C, const double* csr_val_C, const rocsparse_int* csr_row_ptr_C, const rocsparse_int* csr_col_ind_C, size_t* buffer_size);
  // CHECK: status_t = rocsparse_dprune_csr2csr_buffer_size(handle_t, m, n, nnz, matDescr_A, &dbsrSortedValA, &csrRowPtrA, &csrColIndA, &dthreshold, matDescr_C, &dbsrSortedValC, &csrRowPtrC, &csrColIndC, &bufferSize);
  status_t = hipsparseDpruneCsr2csr_bufferSizeExt(handle_t, m, n, nnz, matDescr_A, &dbsrSortedValA, &csrRowPtrA, &csrColIndA, &dthreshold, matDescr_C, &dbsrSortedValC, &csrRowPtrC, &csrColIndC, &bufferSize);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseSpruneCsr2csr_bufferSizeExt(cusparseHandle_t handle, int m, int n, int nnzA, const cusparseMatDescr_t descrA, const float* csrSortedValA, const int* csrSortedRowPtrA, const int* csrSortedColIndA, const float* threshold, const cusparseMatDescr_t descrC, const float* csrSortedValC, const int* csrSortedRowPtrC, const int* csrSortedColIndC, size_t* pBufferSizeInBytes);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_sprune_csr2csr_buffer_size(rocsparse_handle handle, rocsparse_int m, rocsparse_int n, rocsparse_int nnz_A, const rocsparse_mat_descr csr_descr_A, const float* csr_val_A, const rocsparse_int* csr_row_ptr_A, const rocsparse_int* csr_col_ind_A, const float* threshold, const rocsparse_mat_descr csr_descr_C, const float* csr_val_C, const rocsparse_int* csr_row_ptr_C, const rocsparse_int* csr_col_ind_C, size_t* buffer_size);
  // CHECK: status_t = rocsparse_sprune_csr2csr_buffer_size(handle_t, m, n, nnz, matDescr_A, &fbsrSortedValA, &csrRowPtrA, &csrColIndA, &fthreshold, matDescr_C, &fbsrSortedValC, &csrRowPtrC, &csrColIndC, &bufferSize);
  status_t = hipsparseSpruneCsr2csr_bufferSizeExt(handle_t, m, n, nnz, matDescr_A, &fbsrSortedValA, &csrRowPtrA, &csrColIndA, &fthreshold, matDescr_C, &fbsrSortedValC, &csrRowPtrC, &csrColIndC, &bufferSize);

  // CUDA: CUSPARSE_DEPRECATED cusparseStatus_t CUSPARSEAPI cusparseDpruneDense2csrByPercentage(cusparseHandle_t handle, int m, int n, const double* A, int lda, float percentage, const cusparseMatDescr_t descrC, double* csrSortedValC, const int* csrSortedRowPtrC,int* csrSortedColIndC, pruneInfo_t info, void* pBuffer);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_dprune_dense2csr_by_percentage(rocsparse_handle handle, rocsparse_int m, rocsparse_int n, const double* A, rocsparse_int lda, double percentage, const rocsparse_mat_descr descr, double* csr_val, const rocsparse_int* csr_row_ptr, rocsparse_int* csr_col_ind, rocsparse_mat_info info, void* temp_buffer);
  // CHECK: status_t = rocsparse_dprune_dense2csr_by_percentage(handle_t, m, n, &dA, lda, percentage, matDescr_C, &dcsrSortedValC, &csrRowPtrC, &csrColIndC, prune_info, pBuffer);
  status_t = hipsparseDpruneDense2csrByPercentage(handle_t, m, n, &dA, lda, percentage, matDescr_C, &dcsrSortedValC, &csrRowPtrC, &csrColIndC, prune_info, pBuffer);

  // CUDA: CUSPARSE_DEPRECATED cusparseStatus_t CUSPARSEAPI cusparseSpruneDense2csrByPercentage(cusparseHandle_t handle, int m, int n, const float* A, int lda, float percentage, const cusparseMatDescr_t descrC, float* csrSortedValC, const int* csrSortedRowPtrC, int* csrSortedColIndC, pruneInfo_t info, void* pBuffer);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_sprune_dense2csr_by_percentage(rocsparse_handle handle, rocsparse_int m, rocsparse_int n, const float* A, rocsparse_int lda, float percentage, const rocsparse_mat_descr descr, float* csr_val, const rocsparse_int* csr_row_ptr, rocsparse_int* csr_col_ind, rocsparse_mat_info info, void* temp_buffer);
  // CHECK: status_t = rocsparse_sprune_dense2csr_by_percentage(handle_t, m, n, &fA, lda, fpercentage, matDescr_C, &fcsrSortedValC, &csrRowPtrC, &csrColIndC, prune_info, pBuffer);
  status_t = hipsparseSpruneDense2csrByPercentage(handle_t, m, n, &fA, lda, fpercentage, matDescr_C, &fcsrSortedValC, &csrRowPtrC, &csrColIndC, prune_info, pBuffer);

  // CUDA: CUSPARSE_DEPRECATED cusparseStatus_t CUSPARSEAPI cusparseDpruneDense2csrNnzByPercentage(cusparseHandle_t handle, int m, int n, const double* A, int lda, float percentage, const cusparseMatDescr_t descrC, int* csrRowPtrC, int* nnzTotalDevHostPtr, pruneInfo_t info, void* pBuffer);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_dprune_dense2csr_nnz_by_percentage(rocsparse_handle handle, rocsparse_int m, rocsparse_int n, const double* A, rocsparse_int lda, double percentage, const rocsparse_mat_descr descr, rocsparse_int* csr_row_ptr, rocsparse_int* nnz_total_dev_host_ptr, rocsparse_mat_info info, void* temp_buffer);
  // CHECK: status_t = rocsparse_dprune_dense2csr_nnz_by_percentage(handle_t, m, n, &dA, lda, fpercentage, matDescr_C, &csrRowPtrC, &nnzTotalDevHostPtr, prune_info, pBuffer);
  status_t = hipsparseDpruneDense2csrNnzByPercentage(handle_t, m, n, &dA, lda, fpercentage, matDescr_C, &csrRowPtrC, &nnzTotalDevHostPtr, prune_info, pBuffer);

  // CUDA: CUSPARSE_DEPRECATED cusparseStatus_t CUSPARSEAPI cusparseSpruneDense2csrNnzByPercentage(cusparseHandle_t handle, int m, int n, const float* A, int lda, float percentage, const cusparseMatDescr_t descrC, int* csrRowPtrC, int* nnzTotalDevHostPtr, pruneInfo_t info, void* pBuffer);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_sprune_dense2csr_nnz_by_percentage(rocsparse_handle handle, rocsparse_int m, rocsparse_int n, const float* A, rocsparse_int lda, float percentage, const rocsparse_mat_descr descr, rocsparse_int* csr_row_ptr, rocsparse_int* nnz_total_dev_host_ptr, rocsparse_mat_info info, void* temp_buffer);
  // CHECK: status_t = rocsparse_sprune_dense2csr_nnz_by_percentage(handle_t, m, n, &fA, lda, fpercentage, matDescr_C, &csrRowPtrC, &nnzTotalDevHostPtr, prune_info, pBuffer);
  status_t = hipsparseSpruneDense2csrNnzByPercentage(handle_t, m, n, &fA, lda, fpercentage, matDescr_C, &csrRowPtrC, &nnzTotalDevHostPtr, prune_info, pBuffer);

  // CUDA: CUSPARSE_DEPRECATED cusparseStatus_t CUSPARSEAPI cusparseDpruneDense2csrByPercentage_bufferSizeExt(cusparseHandle_t handle, int m, int n, const double* A, int lda, float percentage, const cusparseMatDescr_t descrC, const double* csrSortedValC, const int* csrSortedRowPtrC, const int* csrSortedColIndC, pruneInfo_t info, size_t* pBufferSizeInBytes);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_dprune_dense2csr_by_percentage_buffer_size(rocsparse_handle handle, rocsparse_int m, rocsparse_int n, const double* A, rocsparse_int lda, double percentage, const rocsparse_mat_descr descr, const double* csr_val, const rocsparse_int* csr_row_ptr, const rocsparse_int* csr_col_ind, rocsparse_mat_info info, size_t* buffer_size);
  // CHECK: status_t = rocsparse_dprune_dense2csr_by_percentage_buffer_size(handle_t, m, n, &dA, lda, fpercentage, matDescr_C, &dcsrSortedValC, &csrRowPtrC, &csrColIndC, prune_info, &bufferSize);
  status_t = hipsparseDpruneDense2csrByPercentage_bufferSizeExt(handle_t, m, n, &dA, lda, fpercentage, matDescr_C, &dcsrSortedValC, &csrRowPtrC, &csrColIndC, prune_info, &bufferSize);

  // CUDA: CUSPARSE_DEPRECATED cusparseStatus_t CUSPARSEAPI cusparseSpruneDense2csrByPercentage_bufferSizeExt(cusparseHandle_t handle, int m, int n, const float* A, int lda, float percentage, const cusparseMatDescr_t descrC, const float* csrSortedValC, const int* csrSortedRowPtrC, const int* csrSortedColIndC, pruneInfo_t info, size_t* pBufferSizeInBytes);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_sprune_dense2csr_by_percentage_buffer_size(rocsparse_handle handle, rocsparse_int m, rocsparse_int n, const float* A, rocsparse_int lda, float percentage, const rocsparse_mat_descr descr, const float* csr_val, const rocsparse_int* csr_row_ptr, const rocsparse_int* csr_col_ind, rocsparse_mat_info info, size_t* buffer_size);
  // CHECK: status_t = rocsparse_sprune_dense2csr_by_percentage_buffer_size(handle_t, m, n, &fA, lda, fpercentage, matDescr_C, &fcsrSortedValC, &csrRowPtrC, &csrColIndC, prune_info, &bufferSize);
  status_t = hipsparseSpruneDense2csrByPercentage_bufferSizeExt(handle_t, m, n, &fA, lda, fpercentage, matDescr_C, &fcsrSortedValC, &csrRowPtrC, &csrColIndC, prune_info, &bufferSize);

  // CUDA: CUSPARSE_DEPRECATED cusparseStatus_t CUSPARSEAPI cusparseDpruneDense2csr(cusparseHandle_t handle, int m, int n, const double* A, int lda, const double* threshold, const cusparseMatDescr_t descrC, double* csrSortedValC, const int* csrSortedRowPtrC, int* csrSortedColIndC, void* pBuffer);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_dprune_dense2csr(rocsparse_handle handle, rocsparse_int m, rocsparse_int n, const double* A, rocsparse_int lda, const double* threshold, const rocsparse_mat_descr descr, double* csr_val, const rocsparse_int* csr_row_ptr, rocsparse_int* csr_col_ind, void* temp_buffer);
  // CHECK: status_t = rocsparse_dprune_dense2csr(handle_t, m, n, &dA, lda, &dthreshold, matDescr_C, &dcsrSortedValC, &csrRowPtrC, &csrColIndC, pBuffer);
  status_t = hipsparseDpruneDense2csr(handle_t, m, n, &dA, lda, &dthreshold, matDescr_C, &dcsrSortedValC, &csrRowPtrC, &csrColIndC, pBuffer);

  // CUDA: CUSPARSE_DEPRECATED cusparseStatus_t CUSPARSEAPI cusparseSpruneDense2csr(cusparseHandle_t handle, int m, int n, const float* A, int lda, const float* threshold, const cusparseMatDescr_t descrC, float* csrSortedValC, const int* csrSortedRowPtrC, int* csrSortedColIndC, void* pBuffer);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_sprune_dense2csr(rocsparse_handle handle, rocsparse_int m, rocsparse_int n, const float* A, rocsparse_int lda, const float* threshold, const rocsparse_mat_descr descr, float* csr_val, const rocsparse_int* csr_row_ptr, rocsparse_int* csr_col_ind, void* temp_buffer);
  // CHECK: status_t = rocsparse_sprune_dense2csr(handle_t, m, n, &fA, lda, &fthreshold, matDescr_C, &fcsrSortedValC, &csrRowPtrC, &csrColIndC, pBuffer);
  status_t = hipsparseSpruneDense2csr(handle_t, m, n, &fA, lda, &fthreshold, matDescr_C, &fcsrSortedValC, &csrRowPtrC, &csrColIndC, pBuffer);

  // CUDA: CUSPARSE_DEPRECATED cusparseStatus_t CUSPARSEAPI cusparseDpruneDense2csrNnz(cusparseHandle_t handle, int m, int n, const double* A, int lda, const double* threshold, const cusparseMatDescr_t descrC, int* csrSortedRowPtrC, int* nnzTotalDevHostPtr, void* pBuffer);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_dprune_dense2csr_nnz(rocsparse_handle handle, rocsparse_int m, rocsparse_int n, const double* A, rocsparse_int lda, const double* threshold, const rocsparse_mat_descr descr, rocsparse_int* csr_row_ptr, rocsparse_int* nnz_total_dev_host_ptr, void* temp_buffer);
  // CHECK: status_t = rocsparse_dprune_dense2csr_nnz(handle_t, m, n, &dA, lda, &dthreshold, matDescr_C, &csrRowPtrC, &nnzTotalDevHostPtr, pBuffer);
  status_t = hipsparseDpruneDense2csrNnz(handle_t, m, n, &dA, lda, &dthreshold, matDescr_C, &csrRowPtrC, &nnzTotalDevHostPtr, pBuffer);

  // CUDA: CUSPARSE_DEPRECATED cusparseStatus_t CUSPARSEAPI cusparseSpruneDense2csrNnz(cusparseHandle_t handle, int m, int n, const float* A, int lda, const float* threshold, const cusparseMatDescr_t descrC, int* csrRowPtrC, int* nnzTotalDevHostPtr, void* pBuffer);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_sprune_dense2csr_nnz(rocsparse_handle handle, rocsparse_int m, rocsparse_int n, const float* A, rocsparse_int lda, const float* threshold, const rocsparse_mat_descr descr, rocsparse_int* csr_row_ptr, rocsparse_int* nnz_total_dev_host_ptr, void* temp_buffer);
  // CHECK: status_t = rocsparse_sprune_dense2csr_nnz(handle_t, m, n, &fA, lda, &fthreshold, matDescr_C, &csrRowPtrC, &nnzTotalDevHostPtr, pBuffer);
  status_t = hipsparseSpruneDense2csrNnz(handle_t, m, n, &fA, lda, &fthreshold, matDescr_C, &csrRowPtrC, &nnzTotalDevHostPtr, pBuffer);

  // CUDA: CUSPARSE_DEPRECATED cusparseStatus_t CUSPARSEAPI cusparseDpruneDense2csr_bufferSizeExt(cusparseHandle_t handle, int m, int n, const double* A, int lda, const double* threshold, const cusparseMatDescr_t descrC, const double* csrSortedValC, const int* csrSortedRowPtrC, const int* csrSortedColIndC, size_t* pBufferSizeInBytes);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_dprune_dense2csr_buffer_size(rocsparse_handle handle, rocsparse_int m, rocsparse_int n, const double* A, rocsparse_int lda, const double* threshold, const rocsparse_mat_descr descr, const double* csr_val, const rocsparse_int* csr_row_ptr, const rocsparse_int* csr_col_ind, size_t* buffer_size);
  // CHECK: status_t = rocsparse_dprune_dense2csr_buffer_size(handle_t, m, n, &dA, lda, &dthreshold, matDescr_C, &dcsrSortedValC, &csrRowPtrC, &csrColIndC, &bufferSize);
  status_t = hipsparseDpruneDense2csr_bufferSizeExt(handle_t, m, n, &dA, lda, &dthreshold, matDescr_C, &dcsrSortedValC, &csrRowPtrC, &csrColIndC, &bufferSize);

  // CUDA: CUSPARSE_DEPRECATED cusparseStatus_t CUSPARSEAPI cusparseSpruneDense2csr_bufferSizeExt(cusparseHandle_t handle, int m, int n, const float* A, int lda, const float* threshold, const cusparseMatDescr_t descrC, const float* csrSortedValC, const int* csrSortedRowPtrC, const int* csrSortedColIndC, size_t* pBufferSizeInBytes);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_sprune_dense2csr_buffer_size(rocsparse_handle handle, rocsparse_int m, rocsparse_int n, const float* A, rocsparse_int lda, const float* threshold, const rocsparse_mat_descr descr, const float* csr_val, const rocsparse_int* csr_row_ptr, const rocsparse_int* csr_col_ind, size_t* buffer_size);
  // CHECK: status_t = rocsparse_sprune_dense2csr_buffer_size(handle_t, m, n, &fA, lda, &fthreshold, matDescr_C, &fcsrSortedValC, &csrRowPtrC, &csrColIndC, &bufferSize);
  status_t = hipsparseSpruneDense2csr_bufferSizeExt(handle_t, m, n, &fA, lda, &fthreshold, matDescr_C, &fcsrSortedValC, &csrRowPtrC, &csrColIndC, &bufferSize);
#endif

#if (CUDA_VERSION >= 10010 && CUDA_VERSION < 11000 && !defined(_WIN32)) || CUDA_VERSION >= 11000
  // CHECK: _rocsparse_spmat_descr *spMatDescr = nullptr;
  // CHECK-NEXT: rocsparse_spmat_descr spMatDescr_t, matC;
  cusparseSpMatDescr *spMatDescr = nullptr;
  hipsparseSpMatDescr_t spMatDescr_t, matC;

  // CHECK: _rocsparse_dnmat_descr *dnMatDescr = nullptr;
  // CHECK-NEXT: rocsparse_dnmat_descr dnMatDescr_t, matA, matB;
  cusparseDnMatDescr *dnMatDescr = nullptr;
  hipsparseDnMatDescr_t dnMatDescr_t, matA, matB;

  // CHECK: rocsparse_indextype indexType_t;
  // CHECK-NEXT: rocsparse_indextype csrRowOffsetsType;
  // CHECK-NEXT: rocsparse_indextype cscColOffsetsType;
  // CHECK-NEXT: rocsparse_indextype cscRowIndType;
  // CHECK-NEXT: rocsparse_indextype csrColIndType;
  // CHECK-NEXT: rocsparse_indextype ellIdxType;
  // CHECK-NEXT: rocsparse_indextype INDEX_16U = rocsparse_indextype_u16;
  // CHECK-NEXT: rocsparse_indextype INDEX_32I = rocsparse_indextype_i32;
  // CHECK-NEXT: rocsparse_indextype INDEX_64I = rocsparse_indextype_i64;
  hipsparseIndexType_t indexType_t;
  hipsparseIndexType_t csrRowOffsetsType;
  hipsparseIndexType_t cscColOffsetsType;
  hipsparseIndexType_t cscRowIndType;
  hipsparseIndexType_t csrColIndType;
  hipsparseIndexType_t ellIdxType;
  hipsparseIndexType_t INDEX_16U = HIPSPARSE_INDEX_16U;
  hipsparseIndexType_t INDEX_32I = HIPSPARSE_INDEX_32I;
  hipsparseIndexType_t INDEX_64I = HIPSPARSE_INDEX_64I;

  // CHECK: rocsparse_format format_t;
  // CHECK-NEXT: rocsparse_format FORMAT_CSR = rocsparse_format_csr;
  // CHECK-NEXT: rocsparse_format FORMAT_CSC = rocsparse_format_csc;
  // CHECK-NEXT: rocsparse_format FORMAT_CSO = rocsparse_format_coo;
  hipsparseFormat_t format_t;
  hipsparseFormat_t FORMAT_CSR = HIPSPARSE_FORMAT_CSR;
  hipsparseFormat_t FORMAT_CSC = HIPSPARSE_FORMAT_CSC;
  hipsparseFormat_t FORMAT_CSO = HIPSPARSE_FORMAT_COO;

  // CHECK: rocsparse_order order_t;
  // CHECK-NEXT: rocsparse_order ORDER_COL = rocsparse_order_row;
  // CHECK-NEXT: rocsparse_order ORDER_ROW = rocsparse_order_column;
  hipsparseOrder_t order_t;
  hipsparseOrder_t ORDER_COL = HIPSPARSE_ORDER_COL;
  hipsparseOrder_t ORDER_ROW = HIPSPARSE_ORDER_ROW;

  // CHECK: rocsparse_spmm_alg spMMAlg_t;
  hipsparseSpMMAlg_t spMMAlg_t;

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseCreateCoo(cusparseSpMatDescr_t* spMatDescr, int64_t ows, int64_t cols, int64_t nnz, void* cooRowInd, void* cooColInd, void* cooValues, cusparseIndexType_t cooIdxType, cusparseIndexBase_t idxBase, cudaDataType valueType);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_create_coo_descr(rocsparse_spmat_descr* descr, int64_t rows, int64_t cols, int64_t nnz, void* coo_row_ind, void* coo_col_ind, void* coo_val, rocsparse_indextype idx_type, rocsparse_index_base idx_base, rocsparse_datatype data_type);
  // CHECK: status_t = rocsparse_create_coo_descr(&spMatDescr_t, rows, cols, nnz, cooRowInd, cooColInd, cooValues, indexType_t, indexBase_t, dataType);
  status_t = hipsparseCreateCoo(&spMatDescr_t, rows, cols, nnz, cooRowInd, cooColInd, cooValues, indexType_t, indexBase_t, dataType);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseDestroySpMat(cusparseConstSpMatDescr_t spMatDescr);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_destroy_spmat_descr(rocsparse_spmat_descr descr);
  // CHECK: status_t = rocsparse_destroy_spmat_descr(spMatDescr_t);
  status_t = hipsparseDestroySpMat(spMatDescr_t);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseCooGet(cusparseSpMatDescr_t spMatDescr, int64_t* rows, int64_t* cols, int64_t* nnz, void** cooRowInd, void** cooColInd, void** cooValues, cusparseIndexType_t* idxType, cusparseIndexBase_t* idxBase, cudaDataType* valueType);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_coo_get(const rocsparse_spmat_descr descr, int64_t* rows, int64_t* cols, int64_t* nnz, void** coo_row_ind, void** coo_col_ind, void** coo_val, rocsparse_indextype* idx_type, rocsparse_index_base* idx_base, rocsparse_datatype* data_type);
  // CHECK: status_t = rocsparse_coo_get(spMatDescr_t, &rows, &cols, &nnz, &cooRowInd, &cooColInd, &cooValues, &indexType_t, &indexBase_t, &dataType);
  status_t = hipsparseCooGet(spMatDescr_t, &rows, &cols, &nnz, &cooRowInd, &cooColInd, &cooValues, &indexType_t, &indexBase_t, &dataType);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseSpMatGetFormat(cusparseConstSpMatDescr_t spMatDescr, cusparseFormat_t* format);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_spmat_get_format(const rocsparse_spmat_descr descr, rocsparse_format* format);
  // CHECK: status_t = rocsparse_spmat_get_format(spMatDescr_t, &format_t);
  status_t = hipsparseSpMatGetFormat(spMatDescr_t, &format_t);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseSpMatGetIndexBase(cusparseConstSpMatDescr_t spMatDescr, cusparseIndexBase_t* idxBase);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_spmat_get_index_base(const rocsparse_spmat_descr descr, rocsparse_index_base* idx_base);
  // CHECK: status_t = rocsparse_spmat_get_index_base(spMatDescr_t, &indexBase_t);
  status_t = hipsparseSpMatGetIndexBase(spMatDescr_t, &indexBase_t);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseCreateDnMat(cusparseDnMatDescr_t* dnMatDescr, int64_t rows, int64_t cols, int64_t ld, void* values, cudaDataType valueType, cusparseOrder_t order);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_create_dnmat_descr(rocsparse_dnmat_descr* descr, int64_t rows, int64_t cols, int64_t ld, void* values, rocsparse_datatype data_type, rocsparse_order order);
  // CHECK: status_t = rocsparse_create_dnmat_descr(&dnMatDescr_t, rows, cols, ld, values, dataType, order_t);
  status_t = hipsparseCreateDnMat(&dnMatDescr_t, rows, cols, ld, values, dataType, order_t);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseDestroyDnMat(cusparseConstDnMatDescr_t dnMatDescr);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_destroy_dnmat_descr(rocsparse_dnmat_descr descr);
  // CHECK: status_t = rocsparse_destroy_dnmat_descr(dnMatDescr_t);
  status_t = hipsparseDestroyDnMat(dnMatDescr_t);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseDnMatGet(cusparseDnMatDescr_t dnMatDescr, int64_t* rows, int64_t* cols, int64_t* ld, void** values, cudaDataType* type, cusparseOrder_t* order);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_dnmat_get(const rocsparse_dnmat_descr descr, int64_t* rows, int64_t* cols, int64_t* ld, void** values, rocsparse_datatype* data_type, rocsparse_order* order);
  // CHECK: status_t = rocsparse_dnmat_get(dnMatDescr_t, &rows, &cols, &ld, &values, &dataType, &order_t);
  status_t = hipsparseDnMatGet(dnMatDescr_t, &rows, &cols, &ld, &values, &dataType, &order_t);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseDnMatGetStridedBatch(cusparseConstDnMatDescr_t dnMatDescr, int* batchCount, int64_t* batchStride);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_dnmat_get_strided_batch(rocsparse_dnmat_descr descr, int* batch_count, int64_t* batch_stride);
  // CHECK: status_t = rocsparse_dnmat_get_strided_batch(dnMatDescr_t, &batchCount, &batchStride);
  status_t = hipsparseDnMatGetStridedBatch(dnMatDescr_t, &batchCount, &batchStride);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseDnMatSetStridedBatch(cusparseDnMatDescr_t dnMatDescr, int batchCount, int64_t batchStride);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_dnmat_set_strided_batch(rocsparse_dnmat_descr descr, int batch_count, int64_t batch_stride);
  // CHECK: status_t = rocsparse_dnmat_set_strided_batch(dnMatDescr_t, batchCount, batchStride);
  status_t = hipsparseDnMatSetStridedBatch(dnMatDescr_t, batchCount, batchStride);
#endif

#if CUDA_VERSION >= 10020
  // CHECK: rocsparse_status STATUS_NOT_SUPPORTED = rocsparse_status_not_implemented;
  hipsparseStatus_t STATUS_NOT_SUPPORTED = HIPSPARSE_STATUS_NOT_SUPPORTED;
#endif

#if (CUDA_VERSION >= 10020 && CUDA_VERSION < 11000 && !defined(_WIN32)) || CUDA_VERSION >= 11000
  // CHECK: _rocsparse_spvec_descr *spVecDescr = nullptr;
  // CHECK-NEXT: rocsparse_spvec_descr spVecDescr_t;
  cusparseSpVecDescr *spVecDescr = nullptr;
  hipsparseSpVecDescr_t spVecDescr_t;

  // CHECK: _rocsparse_dnvec_descr *dnVecDescr = nullptr;
  // CHECK-NEXT: rocsparse_dnvec_descr dnVecDescr_t, vecX, vecY;
  cusparseDnVecDescr *dnVecDescr = nullptr;
  hipsparseDnVecDescr_t dnVecDescr_t, vecX, vecY;

  // CHECK: rocsparse_spmv_alg spMVAlg_t;
  hipsparseSpMVAlg_t spMVAlg_t;

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseCreateSpVec(cusparseSpVecDescr_t* spVecDescr, int64_t size, int64_t nnz, void* indices, void* values, cusparseIndexType_t idxType, cusparseIndexBase_t idxBase, cudaDataType valueType);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_create_spvec_descr(rocsparse_spvec_descr* descr, int64_t size, int64_t nnz, void* indices, void* values, rocsparse_indextype idx_type, rocsparse_index_base idx_base, rocsparse_datatype data_type);
  // CHECK: status_t = rocsparse_create_spvec_descr(&spVecDescr_t, size, nnz, indices, values, indexType_t, indexBase_t, dataType);
  status_t = hipsparseCreateSpVec(&spVecDescr_t, size, nnz, indices, values, indexType_t, indexBase_t, dataType);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseDestroySpVec(cusparseConstSpVecDescr_t spVecDescr);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_destroy_spvec_descr(rocsparse_spvec_descr descr);
  // CHECK: status_t = rocsparse_destroy_spvec_descr(spVecDescr_t);
  status_t = hipsparseDestroySpVec(spVecDescr_t);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseSpVecGet(cusparseSpVecDescr_t spVecDescr, int64_t* size, int64_t* nnz, void** indices, void** values, cusparseIndexType_t* idxType, cusparseIndexBase_t* idxBase, cudaDataType* valueType);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_spvec_get(const rocsparse_spvec_descr descr, int64_t* size, int64_t* nnz, void** indices, void** values, rocsparse_indextype* idx_type, rocsparse_index_base* idx_base, rocsparse_datatype* data_type);
  // CHECK: status_t = rocsparse_spvec_get(spVecDescr_t, &size, &nnz, &indices, &values, &indexType_t, &indexBase_t, &dataType);
  status_t = hipsparseSpVecGet(spVecDescr_t, &size, &nnz, &indices, &values, &indexType_t, &indexBase_t, &dataType);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseSpVecGetIndexBase(cusparseConstSpVecDescr_t spVecDescr, cusparseIndexBase_t* idxBase);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_spvec_get_index_base(const rocsparse_spvec_descr descr, rocsparse_index_base* idx_base);
  // CHECK: status_t = rocsparse_spvec_get_index_base(spVecDescr_t, &indexBase_t);
  status_t = hipsparseSpVecGetIndexBase(spVecDescr_t, &indexBase_t);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseSpVecGetValues(cusparseSpVecDescr_t spVecDescr, void** values);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_spvec_get_values(const rocsparse_spvec_descr descr, void** values);
  // CHECK: status_t = rocsparse_spvec_get_values(spVecDescr_t, &values);
  status_t = hipsparseSpVecGetValues(spVecDescr_t, &values);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseSpVecSetValues(cusparseSpVecDescr_t spVecDescr, void* values);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_spvec_set_values(rocsparse_spvec_descr descr, void* values);
  // CHECK: status_t = rocsparse_spvec_set_values(spVecDescr_t, values);
  status_t = hipsparseSpVecSetValues(spVecDescr_t, values);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseCreateCsr(cusparseSpMatDescr_t* spMatDescr, int64_t rows, int64_t cols, int64_t nnz, void* csrRowOffsets, void* csrColInd, void* csrValues, cusparseIndexType_t csrRowOffsetsType, cusparseIndexType_t csrColIndType, cusparseIndexBase_t idxBase, cudaDataType valueType);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_create_csr_descr(rocsparse_spmat_descr* descr, int64_t rows, int64_t cols, int64_t nnz, void* csr_row_ptr, void* csr_col_ind, void* csr_val, rocsparse_indextype row_ptr_type, rocsparse_indextype col_ind_type, rocsparse_index_base idx_base, rocsparse_datatype data_type);
  // CHECK: status_t = rocsparse_create_csr_descr(&spMatDescr_t, rows, cols, nnz, csrRowOffsets, csrColInd, csrValues, csrRowOffsetsType, csrColIndType, indexBase_t, dataType);
  status_t = hipsparseCreateCsr(&spMatDescr_t, rows, cols, nnz, csrRowOffsets, csrColInd, csrValues, csrRowOffsetsType, csrColIndType, indexBase_t, dataType);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseCsrGet(cusparseSpMatDescr_t spMatDescr, int64_t* rows, int64_t* cols, int64_t* nnz, void** csrRowOffsets, void** csrColInd, void** csrValues, cusparseIndexType_t* csrRowOffsetsType, cusparseIndexType_t* csrColIndType, cusparseIndexBase_t* idxBase, cudaDataType* valueType);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_csr_get(const rocsparse_spmat_descr descr, int64_t* rows, int64_t* cols, int64_t* nnz, void** csr_row_ptr, void** csr_col_ind, void** csr_val, rocsparse_indextype* row_ptr_type, rocsparse_indextype* col_ind_type, rocsparse_index_base* idx_base, rocsparse_datatype* data_type);
  // CHECK: status_t = rocsparse_csr_get(spMatDescr_t, &rows, &cols, &nnz, &csrRowOffsets, &csrColInd, &csrValues, &csrRowOffsetsType, &csrColIndType, &indexBase_t, &dataType);
  status_t = hipsparseCsrGet(spMatDescr_t, &rows, &cols, &nnz, &csrRowOffsets, &csrColInd, &csrValues, &csrRowOffsetsType, &csrColIndType, &indexBase_t, &dataType);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseSpMatGetValues(cusparseSpMatDescr_t spMatDescr, void** values);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_spmat_get_values(rocsparse_spmat_descr descr, void** values);
  // CHECK: status_t = rocsparse_spmat_get_values(spMatDescr_t, &values);
  status_t = hipsparseSpMatGetValues(spMatDescr_t, &values);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseSpMatSetValues(cusparseSpMatDescr_t spMatDescr, void* values);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_spmat_set_values(rocsparse_spmat_descr descr, void* values);
  // CHECK: status_t = rocsparse_spmat_set_values(spMatDescr_t, values);
  status_t = hipsparseSpMatSetValues(spMatDescr_t, values);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseSpMatGetStridedBatch(cusparseConstSpMatDescr_t spMatDescr, int* batchCount);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_spmat_get_strided_batch(rocsparse_spmat_descr descr, int* batch_count);
  // CHECK: status_t = rocsparse_spmat_get_strided_batch(spMatDescr_t, &batchCount);
  status_t = hipsparseSpMatGetStridedBatch(spMatDescr_t, &batchCount);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseCreateDnVec(cusparseDnVecDescr_t* dnVecDescr, int64_t size, void* values, cudaDataType valueType);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_create_dnvec_descr(rocsparse_dnvec_descr* descr, int64_t size, void* values, rocsparse_datatype data_type);
  // CHECK: status_t = rocsparse_create_dnvec_descr(&dnVecDescr_t, size, values, dataType);
  status_t = hipsparseCreateDnVec(&dnVecDescr_t, size, values, dataType);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseDestroyDnVec(cusparseConstDnVecDescr_t dnVecDescr);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_destroy_dnvec_descr(rocsparse_dnvec_descr descr);
  // CHECK: status_t = rocsparse_destroy_dnvec_descr(dnVecDescr_t);
  status_t = hipsparseDestroyDnVec(dnVecDescr_t);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseDnVecGet(cusparseDnVecDescr_t dnVecDescr, int64_t* size, void** values, cudaDataType* valueType);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_dnvec_get(const rocsparse_dnvec_descr descr, int64_t* size, void** values, rocsparse_datatype* data_type);
  // CHECK: status_t = rocsparse_dnvec_get(dnVecDescr_t, &size, &values, &dataType);
  status_t = hipsparseDnVecGet(dnVecDescr_t, &size, &values, &dataType);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseDnVecGetValues(cusparseDnVecDescr_t dnVecDescr, void** values);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_dnvec_get_values(const rocsparse_dnvec_descr descr, void** values);
  // CHECK: status_t = rocsparse_dnvec_get_values(dnVecDescr_t, &values);
  status_t = hipsparseDnVecGetValues(dnVecDescr_t, &values);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseDnVecSetValues(cusparseDnVecDescr_t dnVecDescr, void* values);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_dnvec_set_values(rocsparse_dnvec_descr descr, void* values);
  // CHECK: status_t = rocsparse_dnvec_set_values(dnVecDescr_t, values);
  status_t = hipsparseDnVecSetValues(dnVecDescr_t, values);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseDnMatGetValues(cusparseDnMatDescr_t dnMatDescr, void** values);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_dnmat_get_values(const rocsparse_dnmat_descr descr, void** values);
  // CHECK: status_t = rocsparse_dnmat_get_values(dnMatDescr_t, &values);
  status_t = hipsparseDnMatGetValues(dnMatDescr_t, &values);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseDnMatSetValues(cusparseDnMatDescr_t dnMatDescr, void* values);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_dnmat_set_values(rocsparse_dnmat_descr descr, void* values);
  // CHECK: status_t = rocsparse_dnmat_set_values(dnMatDescr_t, values);
  status_t = hipsparseDnMatSetValues(dnMatDescr_t, values);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseSpMV(cusparseHandle_t handle, cusparseOperation_t opA, const void* alpha, cusparseConstSpMatDescr_t matA, cusparseConstDnVecDescr_t vecX, const void* beta, cusparseDnVecDescr_t vecY, cudaDataType computeType, cusparseSpMVAlg_t alg, void* externalBuffer);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_spmv(rocsparse_handle handle, rocsparse_operation trans, const void* alpha, const rocsparse_spmat_descr mat, const rocsparse_dnvec_descr x, const void* beta, const rocsparse_dnvec_descr y, rocsparse_datatype compute_type, rocsparse_spmv_alg alg, size_t* buffer_size, void* temp_buffer);
  // CHECK: status_t = rocsparse_spmv(handle_t, opA, alpha, spMatDescr_t, vecX, beta, vecY, dataType, spMVAlg_t, tempBuffer);
  status_t = hipsparseSpMV(handle_t, opA, alpha, spMatDescr_t, vecX, beta, vecY, dataType, spMVAlg_t, tempBuffer);
#endif

#if (CUDA_VERSION >= 10020 && CUDA_VERSION < 11000 && !defined(_WIN32)) || (CUDA_VERSION >= 11000 && CUDA_VERSION < 12000)
  // CHECK: rocsparse_format FORMAT_COO_AOS = rocsparse_format_coo_aos;
  cusparseFormat_t FORMAT_COO_AOS = CUSPARSE_FORMAT_COO_AOS;

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseCreateCooAoS(cusparseSpMatDescr_t* spMatDescr, int64_t rows, int64_t cols, int64_t nnz, void* cooInd, void* cooValues, cusparseIndexType_t cooIdxType, cusparseIndexBase_t idxBase, cudaDataType valueType);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_create_coo_aos_descr(rocsparse_spmat_descr* descr, int64_t rows, int64_t cols, int64_t nnz, void* coo_ind, void* coo_val, rocsparse_indextype idx_type, rocsparse_index_base idx_base, rocsparse_datatype data_type);
  // CHECK: status_t = rocsparse_create_coo_aos_descr(&spMatDescr_t, rows, cols, nnz, cooColInd, cooValues, indexType_t, indexBase_t, dataType);
  status_t = cusparseCreateCooAoS(&spMatDescr_t, rows, cols, nnz, cooColInd, cooValues, indexType_t, indexBase_t, dataType);

  // CUDA: CUSPARSE_DEPRECATED(cusparseCooGet) cusparseStatus_t CUSPARSEAPI cusparseCooAoSGet(cusparseSpMatDescr_t spMatDescr, int64_t* rows, int64_t* cols, int64_t* nnz, void** cooInd, void** cooValues, cusparseIndexType_t* idxType, cusparseIndexBase_t* idxBase, cudaDataType* valueType);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_coo_aos_get(const rocsparse_spmat_descr descr, int64_t* rows, int64_t* cols, int64_t* nnz, void** coo_ind, void** coo_val, rocsparse_indextype* idx_type, rocsparse_index_base* idx_base, rocsparse_datatype* data_type);
  // CHECK: status_t = rocsparse_coo_aos_get(spMatDescr_t, &rows, &cols, &nnz, &cooColInd, &cooValues, &indexType_t, &indexBase_t, &dataType);
  status_t = cusparseCooAoSGet(spMatDescr_t, &rows, &cols, &nnz, &cooColInd, &cooValues, &indexType_t, &indexBase_t, &dataType);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseSpMatSetStridedBatch(cusparseSpMatDescr_t spMatDescr, int batchCount);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_spmat_set_strided_batch(rocsparse_spmat_descr descr, int batch_count);
  // CHECK: status_t = rocsparse_spmat_set_strided_batch(spMatDescr_t, batchCount);
  status_t = cusparseSpMatSetStridedBatch(spMatDescr_t, batchCount);
#endif

#if CUDA_VERSION < 11000
  // CHECK: _rocsparse_hyb_mat *hybMat = nullptr;
  // CHECK-NEXT: rocsparse_hyb_mat hybMat_t;
  cusparseHybMat *hybMat = nullptr;
  cusparseHybMat_t hybMat_t;

  // CHECK: rocsparse_hyb_partition hybPartition_t;
  // CHECK-NEXT: rocsparse_hyb_partition HYB_PARTITION_AUTO = rocsparse_hyb_partition_auto;
  // CHECK-NEXT: rocsparse_hyb_partition HYB_PARTITION_USER = rocsparse_hyb_partition_user;
  // CHECK-NEXT: rocsparse_hyb_partition HYB_PARTITION_MAX = rocsparse_hyb_partition_max;
  cusparseHybPartition_t hybPartition_t;
  cusparseHybPartition_t HYB_PARTITION_AUTO = CUSPARSE_HYB_PARTITION_AUTO;
  cusparseHybPartition_t HYB_PARTITION_USER = CUSPARSE_HYB_PARTITION_USER;
  cusparseHybPartition_t HYB_PARTITION_MAX = CUSPARSE_HYB_PARTITION_MAX;

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseCreateHybMat(cusparseHybMat_t* hybA);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_create_hyb_mat(rocsparse_hyb_mat* hyb);
  // CHECK: status_t = rocsparse_create_hyb_mat(&hybMat_t);
  status_t = cusparseCreateHybMat(&hybMat_t);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseDestroyHybMat(cusparseHybMat_t hybA);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_destroy_hyb_mat(rocsparse_hyb_mat hyb);
  // CHECK: status_t = rocsparse_destroy_hyb_mat(hybMat_t);
  status_t = cusparseDestroyHybMat(hybMat_t);

  // CUDA: CUSPARSE_DEPRECATED cusparseStatus_t CUSPARSEAPI cusparseZcsr2hyb(cusparseHandle_t handle, int m, int n, const cusparseMatDescr_t descrA, const cuDoubleComplex* csrSortedValA, const int* csrSortedRowPtrA, const int* csrSortedColIndA, cusparseHybMat_t hybA, int userEllWidth, cusparseHybPartition_t   partitionType);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_zcsr2hyb(rocsparse_handle handle, rocsparse_int m, rocsparse_int n, const rocsparse_mat_descr descr, const rocsparse_double_complex* csr_val, const rocsparse_int* csr_row_ptr, const rocsparse_int* csr_col_ind, rocsparse_hyb_mat hyb, rocsparse_int user_ell_width, rocsparse_hyb_partition partition_type);
  // CHECK: status_t = rocsparse_zcsr2hyb(handle_t, m, n, matDescr_A, &dComplexcsrSortedValA, &csrRowPtrA, &csrColIndA, hybMat_t, userEllWidth, hybPartition_t);
  status_t = cusparseZcsr2hyb(handle_t, m, n, matDescr_A, &dComplexcsrSortedValA, &csrRowPtrA, &csrColIndA, hybMat_t, userEllWidth, hybPartition_t);

  // CUDA: CUSPARSE_DEPRECATED cusparseStatus_t CUSPARSEAPI cusparseCcsr2hyb(cusparseHandle_t handle, int m, int n, const cusparseMatDescr_t descrA, const cuComplex* csrSortedValA, const int* csrSortedRowPtrA, const int* csrSortedColIndA, cusparseHybMat_t hybA, int userEllWidth, cusparseHybPartition_t partitionType);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_ccsr2hyb(rocsparse_handle handle, rocsparse_int m, rocsparse_int n, const rocsparse_mat_descr descr, const rocsparse_float_complex* csr_val, const rocsparse_int* csr_row_ptr, const rocsparse_int* csr_col_ind, rocsparse_hyb_mat hyb, rocsparse_int user_ell_width, rocsparse_hyb_partition partition_type);
  // CHECK: status_t = rocsparse_ccsr2hyb(handle_t, m, n, matDescr_A, &complexcsrSortedValA, &csrRowPtrA, &csrColIndA, hybMat_t, userEllWidth, hybPartition_t);
  status_t = cusparseCcsr2hyb(handle_t, m, n, matDescr_A, &complexcsrSortedValA, &csrRowPtrA, &csrColIndA, hybMat_t, userEllWidth, hybPartition_t);

  // CUDA: CUSPARSE_DEPRECATED cusparseStatus_t CUSPARSEAPI cusparseDcsr2hyb(cusparseHandle_t handle, int m, int n, const cusparseMatDescr_t descrA, const double* csrSortedValA, const int* csrSortedRowPtrA, const int* csrSortedColIndA, cusparseHybMat_t hybA, int userEllWidth, cusparseHybPartition_t partitionType);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_dcsr2hyb(rocsparse_handle handle, rocsparse_int m, rocsparse_int n, const rocsparse_mat_descr descr, const double* csr_val, const rocsparse_int* csr_row_ptr, const rocsparse_int* csr_col_ind, rocsparse_hyb_mat hyb, rocsparse_int user_ell_width, rocsparse_hyb_partition partition_type);
  // CHECK: status_t = rocsparse_dcsr2hyb(handle_t, m, n, matDescr_A, &dcsrSortedValA, &csrRowPtrA, &csrColIndA, hybMat_t, userEllWidth, hybPartition_t);
  status_t = cusparseDcsr2hyb(handle_t, m, n, matDescr_A, &dcsrSortedValA, &csrRowPtrA, &csrColIndA, hybMat_t, userEllWidth, hybPartition_t);

  // CUDA: CUSPARSE_DEPRECATED cusparseStatus_t CUSPARSEAPI cusparseScsr2hyb(cusparseHandle_t handle, int m, int n, const cusparseMatDescr_t descrA, const float* csrSortedValA, const int* csrSortedRowPtrA, const int* csrSortedColIndA, cusparseHybMat_t hybA, int userEllWidth, cusparseHybPartition_t partitionType);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_scsr2hyb(rocsparse_handle handle, rocsparse_int m, rocsparse_int n, const rocsparse_mat_descr descr, const float* csr_val, const rocsparse_int* csr_row_ptr, const rocsparse_int* csr_col_ind, rocsparse_hyb_mat hyb, rocsparse_int user_ell_width, rocsparse_hyb_partition partition_type);
  // CHECK: status_t = rocsparse_scsr2hyb(handle_t, m, n, matDescr_A, &csrSortedValA, &csrRowPtrA, &csrColIndA, hybMat_t, userEllWidth, hybPartition_t);
  status_t = cusparseScsr2hyb(handle_t, m, n, matDescr_A, &csrSortedValA, &csrRowPtrA, &csrColIndA, hybMat_t, userEllWidth, hybPartition_t);
#endif

#if CUDA_VERSION >= 11000
  // CHECK: rocsparse_spgemm_alg spGEMMAlg_t;
  // CHECK-NEXT: rocsparse_spgemm_alg SPGEMM_DEFAULT = rocsparse_spgemm_alg_default;
  hipsparseSpGEMMAlg_t spGEMMAlg_t;
  hipsparseSpGEMMAlg_t SPGEMM_DEFAULT = HIPSPARSE_SPGEMM_DEFAULT;

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseCsrSetPointers(cusparseSpMatDescr_t spMatDescr, void* csrRowOffsets, void* csrColInd, void* csrValues);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_csr_set_pointers(rocsparse_spmat_descr descr, void* csr_row_ptr, void* csr_col_ind, void* csr_val);
  // CHECK: status_t = rocsparse_csr_set_pointers(spMatDescr_t, csrRowOffsets, csrColInd, csrValues);
  status_t = hipsparseCsrSetPointers(spMatDescr_t, csrRowOffsets, csrColInd, csrValues);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseSpMatGetSize(cusparseConstSpMatDescr_t spMatDescr, int64_t* rows, int64_t* cols, int64_t* nnz);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_spmat_get_size(rocsparse_spmat_descr descr, int64_t* rows, int64_t* cols, int64_t* nnz);
  // CHECK: status_t = rocsparse_spmat_get_size(spMatDescr_t, &rows, &cols, &nnz);
  status_t = hipsparseSpMatGetSize(spMatDescr_t, &rows, &cols, &nnz);
#endif

#if CUDA_VERSION >= 11000 && CUSPARSE_VERSION >= 11100
  // CHECK: rocsparse_spmm_alg SPMM_ALG_DEFAULT = rocsparse_spmm_alg_default;
  // CHECK-NEXT: rocsparse_spmm_alg SPMM_COO_ALG1 = rocsparse_spmm_alg_coo_segmented;
  // CHECK-NEXT: rocsparse_spmm_alg SPMM_COO_ALG2 = rocsparse_spmm_alg_coo_atomic;
  // CHECK-NEXT: rocsparse_spmm_alg SPMM_COO_ALG3 = rocsparse_spmm_alg_coo_segmented_atomic;
  // CHECK-NEXT: rocsparse_spmm_alg SPMM_CSR_ALG1 = rocsparse_spmm_alg_csr;
  // CHECK-NEXT: rocsparse_spmm_alg SPMM_CSR_ALG2 = rocsparse_spmm_alg_csr_row_split;
  hipsparseSpMMAlg_t SPMM_ALG_DEFAULT = HIPSPARSE_SPMM_ALG_DEFAULT;
  hipsparseSpMMAlg_t SPMM_COO_ALG1 = HIPSPARSE_SPMM_COO_ALG1;
  hipsparseSpMMAlg_t SPMM_COO_ALG2 = HIPSPARSE_SPMM_COO_ALG2;
  hipsparseSpMMAlg_t SPMM_COO_ALG3 = HIPSPARSE_SPMM_COO_ALG3;
  hipsparseSpMMAlg_t SPMM_CSR_ALG1 = HIPSPARSE_SPMM_CSR_ALG1;
  hipsparseSpMMAlg_t SPMM_CSR_ALG2 = HIPSPARSE_SPMM_CSR_ALG2;

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseCooSetStridedBatch(cusparseSpMatDescr_t spMatDescr, int batchCount, int64_t batchStride);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_coo_set_strided_batch(rocsparse_spmat_descr descr, int batch_count, int64_t batch_stride);
  // CHECK: status_t = rocsparse_coo_set_strided_batch(spMatDescr_t, batchCount, batchStride);
  status_t = hipsparseCooSetStridedBatch(spMatDescr_t, batchCount, batchStride);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseCsrSetStridedBatch(cusparseSpMatDescr_t spMatDescr, int batchCount, int64_t offsetsBatchStride, int64_t columnsValuesBatchStride);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_csr_set_strided_batch(rocsparse_spmat_descr descr, int batch_count, int64_t offsets_batch_stride, int64_t columns_values_batch_stride);
  // CHECK: status_t = rocsparse_csr_set_strided_batch(spMatDescr_t, batchCount, offsetsBatchStride, columnsValuesBatchStride);
  status_t = hipsparseCsrSetStridedBatch(spMatDescr_t, batchCount, offsetsBatchStride, columnsValuesBatchStride);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseRot(cusparseHandle_t handle, const void* c_coeff, const void* s_coeff, cusparseSpVecDescr_t vecX, cusparseDnVecDescr_t vecY);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_rot(rocsparse_handle handle, const void* c, const void* s, rocsparse_spvec_descr x, rocsparse_dnvec_descr y);
  // CHECK: status_t = rocsparse_rot(handle_t, c_coeff, s_coeff, spVecDescr_t, vecY);
  status_t = hipsparseRot(handle_t, c_coeff, s_coeff, spVecDescr_t, vecY);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseScatter(cusparseHandle_t handle, cusparseConstSpVecDescr_t vecX, cusparseDnVecDescr_t vecY);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_scatter(rocsparse_handle handle, const rocsparse_spvec_descr x, rocsparse_dnvec_descr y);
  // CHECK: status_t = rocsparse_scatter(handle_t, spVecDescr_t, vecY);
  status_t = hipsparseScatter(handle_t, spVecDescr_t, vecY);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseGather(cusparseHandle_t handle, cusparseConstDnVecDescr_t vecY, cusparseSpVecDescr_t vecX);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_gather(rocsparse_handle handle, const rocsparse_dnvec_descr y, rocsparse_spvec_descr x);
  // CHECK: status_t = rocsparse_gather(handle_t, vecY, spVecDescr_t);
  status_t = hipsparseGather(handle_t, vecY, spVecDescr_t);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseAxpby(cusparseHandle_t handle, const void* alpha, cusparseConstSpVecDescr_t vecX, const void* beta, cusparseDnVecDescr_t vecY);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_axpby(rocsparse_handle handle, const void* alpha, const rocsparse_spvec_descr x, const void* beta, rocsparse_dnvec_descr y);
  // CHECK: status_t = rocsparse_axpby(handle_t, alpha, spVecDescr_t, beta, vecY);
  status_t = hipsparseAxpby(handle_t, alpha, spVecDescr_t, beta, vecY);
#endif

#if CUDA_VERSION >= 11010 && CUSPARSE_VERSION >= 11300
  // CHECK: rocsparse_sparse_to_dense_alg sparseToDenseAlg_t;
  // CHECK-NEXT: rocsparse_sparse_to_dense_alg SPARSETODENSE_ALG_DEFAULT = rocsparse_sparse_to_dense_alg_default;
  hipsparseSparseToDenseAlg_t sparseToDenseAlg_t;
  hipsparseSparseToDenseAlg_t SPARSETODENSE_ALG_DEFAULT = HIPSPARSE_SPARSETODENSE_ALG_DEFAULT;

  // CHECK: rocsparse_dense_to_sparse_alg denseToSparseAlg_t;
  // CHECK-NEXT: rocsparse_dense_to_sparse_alg DENSETOSPARSE_ALG_DEFAULT = rocsparse_dense_to_sparse_alg_default;
  hipsparseDenseToSparseAlg_t denseToSparseAlg_t;
  hipsparseDenseToSparseAlg_t DENSETOSPARSE_ALG_DEFAULT = HIPSPARSE_DENSETOSPARSE_ALG_DEFAULT;

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseCreateCsc(cusparseSpMatDescr_t* spMatDescr, int64_t rows, int64_t cols, int64_t nnz, void* cscColOffsets, void* cscRowInd, void* cscValues, cusparseIndexType_t cscColOffsetsType, cusparseIndexType_t cscRowIndType, cusparseIndexBase_t idxBase, cudaDataType valueType);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_create_csc_descr(rocsparse_spmat_descr* descr, int64_t rows, int64_t cols, int64_t nnz, void* csc_col_ptr, void* csc_row_ind, void* csc_val, rocsparse_indextype col_ptr_type, rocsparse_indextype row_ind_type, rocsparse_index_base idx_base, rocsparse_datatype data_type);
  // CHECK: status_t = rocsparse_create_csc_descr(&spMatDescr_t, rows, cols, nnz, cscColOffsets, cscRowInd, cscValues, cscColOffsetsType, csrColIndType, indexBase_t, dataType);
  status_t = hipsparseCreateCsc(&spMatDescr_t, rows, cols, nnz, cscColOffsets, cscRowInd, cscValues, cscColOffsetsType, csrColIndType, indexBase_t, dataType);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseCooSetPointers(cusparseSpMatDescr_t spMatDescr, void* cooRows, void* cooColumns, void* cooValues);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_coo_set_pointers(rocsparse_spmat_descr descr, void* coo_row_ind, void* coo_col_ind, void* coo_val);
  // CHECK: status_t = rocsparse_coo_set_pointers(spMatDescr_t, cooRows, cooColumns, cooValues);
  status_t = hipsparseCooSetPointers(spMatDescr_t, cooRows, cooColumns, cooValues);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseCscSetPointers(cusparseSpMatDescr_t spMatDescr, void* cscColOffsets, void* cscRowInd, void* cscValues);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_csc_set_pointers(rocsparse_spmat_descr descr, void* csc_col_ptr, void* csc_row_ind, void* csc_val);
  // CHECK: status_t = rocsparse_csc_set_pointers(spMatDescr_t, cscColOffsets, cscRowInd, cscValues);
  status_t = hipsparseCscSetPointers(spMatDescr_t, cscColOffsets, cscRowInd, cscValues);
#endif

#if CUDA_VERSION >= 11020 && CUSPARSE_VERSION >= 11400
  // CHECK: rocsparse_format FORMAT_BLOCKED_ELL = rocsparse_format_bell;
  hipsparseFormat_t FORMAT_BLOCKED_ELL = HIPSPARSE_FORMAT_BLOCKED_ELL;

  // CHECK: rocsparse_spmv_alg SPMV_ALG_DEFAULT = rocsparse_spmv_alg_default;
  // CHECK-NEXT: rocsparse_spmv_alg SPMV_COO_ALG1 = rocsparse_spmv_alg_coo;
  // CHECK-NEXT: rocsparse_spmv_alg SPMV_COO_ALG2 = rocsparse_spmv_alg_coo_atomic;
  // CHECK-NEXT: rocsparse_spmv_alg SPMV_CSR_ALG1 = rocsparse_spmv_alg_csr_adaptive;
  // CHECK-NEXT: rocsparse_spmv_alg SPMV_CSR_ALG2 = rocsparse_spmv_alg_csr_stream;
  hipsparseSpMVAlg_t SPMV_ALG_DEFAULT = HIPSPARSE_SPMV_ALG_DEFAULT;
  hipsparseSpMVAlg_t SPMV_COO_ALG1 = HIPSPARSE_SPMV_COO_ALG1;
  hipsparseSpMVAlg_t SPMV_COO_ALG2 = HIPSPARSE_SPMV_COO_ALG2;
  hipsparseSpMVAlg_t SPMV_CSR_ALG1 = HIPSPARSE_SPMV_CSR_ALG1;
  hipsparseSpMVAlg_t SPMV_CSR_ALG2 = HIPSPARSE_SPMV_CSR_ALG2;

  // CHECK: rocsparse_spmm_alg SPMM_CSR_ALG3 = rocsparse_spmm_alg_csr_merge;
  // CHECK-NEXT: rocsparse_spmm_alg SPMM_BLOCKED_ELL_ALG1 = rocsparse_spmm_alg_bell;
  hipsparseSpMMAlg_t SPMM_CSR_ALG3 = HIPSPARSE_SPMM_CSR_ALG3;
  hipsparseSpMMAlg_t SPMM_BLOCKED_ELL_ALG1 = HIPSPARSE_SPMM_BLOCKED_ELL_ALG1;

  // CHECK: rocsparse_sddmm_alg sDDMMAlg_t;
  // CHECK-NEXT: rocsparse_sddmm_alg SDDMM_ALG_DEFAULT = rocsparse_sddmm_alg_default;
  hipsparseSDDMMAlg_t sDDMMAlg_t;
  hipsparseSDDMMAlg_t SDDMM_ALG_DEFAULT = HIPSPARSE_SDDMM_ALG_DEFAULT;

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseCreateBlockedEll(cusparseSpMatDescr_t* spMatDescr, int64_t rows, int64_t cols, int64_t ellBlockSize, int64_t ellCols, void* ellColInd, void* ellValue, cusparseIndexType_t ellIdxType, cusparseIndexBase_t idxBase, cudaDataType valueType);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_create_bell_descr(rocsparse_spmat_descr* descr, int64_t rows, int64_t cols, rocsparse_direction ell_block_dir, int64_t ell_block_dim, int64_t ell_cols, void* ell_col_ind, void* ell_val, rocsparse_indextype idx_type, rocsparse_index_base idx_base, rocsparse_datatype data_type);
  // CHECK: status_t = rocsparse_create_bell_descr(&spMatDescr_t, rows, cols, ellBlockSize, ellCols, ellColInd, ellValue, ellIdxType, indexBase_t, dataType);
  status_t = hipsparseCreateBlockedEll(&spMatDescr_t, rows, cols, ellBlockSize, ellCols, ellColInd, ellValue, ellIdxType, indexBase_t, dataType);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseBlockedEllGet(cusparseSpMatDescr_t spMatDescr, int64_t* rows, int64_t* cols, int64_t* ellBlockSize, int64_t* ellCols, void** ellColInd, void** ellValue, cusparseIndexType_t* ellIdxType, cusparseIndexBase_t* idxBase, cudaDataType* valueType);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_bell_get(const rocsparse_spmat_descr descr, int64_t* rows, int64_t* cols, rocsparse_direction* ell_block_dir, int64_t* ell_block_dim, int64_t* ell_cols, void** ell_col_ind, void** ell_val, rocsparse_indextype* idx_type, rocsparse_index_base* idx_base, rocsparse_datatype* data_type);
  // CHECK: status_t = rocsparse_bell_get(spMatDescr_t, &rows, &cols, &ellBlockSize, &ellCols, &ellColInd, &ellValue, &ellIdxType, &indexBase_t, &dataType);
  status_t = hipsparseBlockedEllGet(spMatDescr_t, &rows, &cols, &ellBlockSize, &ellCols, &ellColInd, &ellValue, &ellIdxType, &indexBase_t, &dataType);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseSDDMM_preprocess(cusparseHandle_t handle, cusparseOperation_t opA, cusparseOperation_t opB, const void* alpha, cusparseConstDnMatDescr_t matA, cusparseConstDnMatDescr_t matB, const void* beta, cusparseSpMatDescr_t matC, cudaDataType computeType, cusparseSDDMMAlg_t alg, void* externalBuffer);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_sddmm_preprocess(rocsparse_handle handle, rocsparse_operation opA, rocsparse_operation opB, const void* alpha, const rocsparse_dnmat_descr A, const rocsparse_dnmat_descr B, const void* beta, rocsparse_spmat_descr C, rocsparse_datatype compute_type, rocsparse_sddmm_alg alg, void* temp_buffer);
  // CHECK: status_t = rocsparse_sddmm_preprocess(handle_t, opA, opB, alpha, matA, matB, beta, matC, dataType, sDDMMAlg_t, tempBuffer);
  status_t = hipsparseSDDMM_preprocess(handle_t, opA, opB, alpha, matA, matB, beta, matC, dataType, sDDMMAlg_t, tempBuffer);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseSDDMM_bufferSize(cusparseHandle_t handle, cusparseOperation_t opA, cusparseOperation_t opB, const void* alpha, cusparseConstDnMatDescr_t matA, cusparseConstDnMatDescr_t matB, const void* beta, cusparseSpMatDescr_t matC, cudaDataType computeType, cusparseSDDMMAlg_t alg, size_t* bufferSize);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_sddmm_buffer_size(rocsparse_handle handle, rocsparse_operation opA, rocsparse_operation opB, const void* alpha, const rocsparse_dnmat_descr A, const rocsparse_dnmat_descr B, const void* beta, rocsparse_spmat_descr C, rocsparse_datatype compute_type, rocsparse_sddmm_alg alg, size_t* buffer_size);
  // CHECK: status_t = rocsparse_sddmm_buffer_size(handle_t, opA, opB, alpha, matA, matB, beta, matC, dataType, sDDMMAlg_t, &bufferSize);
  status_t = hipsparseSDDMM_bufferSize(handle_t, opA, opB, alpha, matA, matB, beta, matC, dataType, sDDMMAlg_t, &bufferSize);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseSDDMM(cusparseHandle_t handle, cusparseOperation_t opA, cusparseOperation_t opB, const void* alpha, cusparseConstDnMatDescr_t matA, cusparseConstDnMatDescr_t matB, const void* beta, cusparseSpMatDescr_t matC, cudaDataType computeType, cusparseSDDMMAlg_t alg, void* externalBuffer);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_sddmm(rocsparse_handle handle, rocsparse_operation opA, rocsparse_operation opB, const void* alpha, const rocsparse_dnmat_descr A, const rocsparse_dnmat_descr B, const void* beta, rocsparse_spmat_descr C, rocsparse_datatype compute_type, rocsparse_sddmm_alg alg, void* temp_buffer);
  // CHECK: status_t = rocsparse_sddmm(handle_t, opA, opB, alpha, matA, matB, beta, matC, dataType, sDDMMAlg_t, tempBuffer);
  status_t = hipsparseSDDMM(handle_t, opA, opB, alpha, matA, matB, beta, matC, dataType, sDDMMAlg_t, tempBuffer);
#endif

#if CUDA_VERSION >= 11030
  // CHECK: rocsparse_spmat_attribute spMatAttribute_t;
  // CHECK-NEXT: rocsparse_spmat_attribute SPMAT_FILL_MODE = rocsparse_spmat_fill_mode;
  // CHECK-NEXT: rocsparse_spmat_attribute SPMAT_DIAG_TYPE = rocsparse_spmat_diag_type;
  hipsparseSpMatAttribute_t spMatAttribute_t;
  hipsparseSpMatAttribute_t SPMAT_FILL_MODE = HIPSPARSE_SPMAT_FILL_MODE;
  hipsparseSpMatAttribute_t SPMAT_DIAG_TYPE = HIPSPARSE_SPMAT_DIAG_TYPE;

  // CHECK: rocsparse_spsv_alg spSVAlg_t;
  // CHECK-NEXT: rocsparse_spsv_alg SPSV_ALG_DEFAULT = rocsparse_spsv_alg_default;
  hipsparseSpSVAlg_t spSVAlg_t;
  hipsparseSpSVAlg_t SPSV_ALG_DEFAULT = HIPSPARSE_SPSV_ALG_DEFAULT;

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseSpMatGetAttribute(cusparseConstSpMatDescr_t spMatDescr, cusparseSpMatAttribute_t attribute, void* data, size_t dataSize);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_spmat_get_attribute(rocsparse_spmat_descr descr, rocsparse_spmat_attribute attribute, void* data, size_t data_size);
  // CHECK: status_t = rocsparse_spmat_get_attribute(spMatDescr_t, spMatAttribute_t, &data, dataSize);
  status_t = hipsparseSpMatGetAttribute(spMatDescr_t, spMatAttribute_t, &data, dataSize);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseSpMatSetAttribute(cusparseSpMatDescr_t spMatDescr, cusparseSpMatAttribute_t attribute, void* data, size_t dataSize);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_spmat_set_attribute(rocsparse_spmat_descr descr, rocsparse_spmat_attribute attribute, const void* data, size_t data_size);
  // CHECK: status_t = rocsparse_spmat_set_attribute(spMatDescr_t, spMatAttribute_t, &data, dataSize);
  status_t = hipsparseSpMatSetAttribute(spMatDescr_t, spMatAttribute_t, &data, dataSize);
#endif

#if CUDA_VERSION >= 11030 && CUSPARSE_VERSION >= 11600
  // CHECK: rocsparse_spsm_alg spSMAlg_t;
  // CHECK-NEXT: rocsparse_spsm_alg SPSM_ALG_DEFAULT = rocsparse_spsm_alg_default;
  hipsparseSpSMAlg_t spSMAlg_t;
  hipsparseSpSMAlg_t SPSM_ALG_DEFAULT = HIPSPARSE_SPSM_ALG_DEFAULT;
#endif

#if CUDA_VERSION < 12000
  // CUDA: CUSPARSE_DEPRECATED(cusparseSparseToDense) cusparseStatus_t CUSPARSEAPI cusparseZcsc2dense(cusparseHandle_t handle, int m, int n, const cusparseMatDescr_t descrA, const cuDoubleComplex* cscSortedValA, const int* cscSortedRowIndA, const int* cscSortedColPtrA, cuDoubleComplex* A, int lda);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_zcsc2dense(rocsparse_handle handle, rocsparse_int m, rocsparse_int n, const rocsparse_mat_descr descr, const rocsparse_double_complex* csc_val, const rocsparse_int* csc_col_ptr, const rocsparse_int* csc_row_ind, rocsparse_double_complex* A, rocsparse_int ld);
  // CHECK: status_t = rocsparse_zcsc2dense(handle_t, m, n, matDescr_A, &dComplexcscSortedVal, &csrSortedRowPtr, &csrSortedColInd, &dcomplexA, lda);
  status_t = cusparseZcsc2dense(handle_t, m, n, matDescr_A, &dComplexcscSortedVal, &csrSortedRowPtr, &csrSortedColInd, &dcomplexA, lda);

  // CUDA: CUSPARSE_DEPRECATED(cusparseSparseToDense) cusparseStatus_t CUSPARSEAPI cusparseCcsc2dense(cusparseHandle_t handle, int m, int n, const cusparseMatDescr_t descrA, const cuComplex* cscSortedValA, const int* cscSortedRowIndA, const int* cscSortedColPtrA, cuComplex* A, int lda);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_ccsc2dense(rocsparse_handle handle, rocsparse_int m, rocsparse_int n, const rocsparse_mat_descr descr, const rocsparse_float_complex* csc_val, const rocsparse_int* csc_col_ptr, const rocsparse_int* csc_row_ind, rocsparse_float_complex* A, rocsparse_int ld);
  // CHECK: status_t = rocsparse_ccsc2dense(handle_t, m, n, matDescr_A, &complexcscSortedVal, &csrSortedRowPtr, &csrSortedColInd, &complexA, lda);
  status_t = cusparseCcsc2dense(handle_t, m, n, matDescr_A, &complexcscSortedVal, &csrSortedRowPtr, &csrSortedColInd, &complexA, lda);

  // CUDA: CUSPARSE_DEPRECATED(cusparseSparseToDense) cusparseStatus_t CUSPARSEAPI cusparseDcsc2dense(cusparseHandle_t handle, int m, int n, const cusparseMatDescr_t descrA, const double* cscSortedValA, const int* cscSortedRowIndA, const int* cscSortedColPtrA, double* A, int lda);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_dcsc2dense(rocsparse_handle handle, rocsparse_int m, rocsparse_int n, const rocsparse_mat_descr descr, const double* csc_val, const rocsparse_int* csc_col_ptr, const rocsparse_int* csc_row_ind, double* A,  rocsparse_int ld);
  // CHECK: status_t = rocsparse_dcsc2dense(handle_t, m, n, matDescr_A, &dcscSortedVal, &csrSortedRowPtr, &csrSortedColInd, &dA, lda);
  status_t = cusparseDcsc2dense(handle_t, m, n, matDescr_A, &dcscSortedVal, &csrSortedRowPtr, &csrSortedColInd, &dA, lda);

  // CUDA: CUSPARSE_DEPRECATED(cusparseSparseToDense) cusparseStatus_t CUSPARSEAPI cusparseScsc2dense(cusparseHandle_t handle, int m, int n, const cusparseMatDescr_t descrA, const float* cscSortedValA, const int* cscSortedRowIndA, const int* cscSortedColPtrA, float* A, int lda);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_scsc2dense(rocsparse_handle handle, rocsparse_int m, rocsparse_int n, const rocsparse_mat_descr descr, const float* csc_val, const rocsparse_int* csc_col_ptr, const rocsparse_int* csc_row_ind, float* A, rocsparse_int ld);
  // CHECK: status_t = rocsparse_scsc2dense(handle_t, m, n, matDescr_A, &cscSortedVal, &csrSortedRowPtr, &csrSortedColInd, &fA, lda);
  status_t = cusparseScsc2dense(handle_t, m, n, matDescr_A, &cscSortedVal, &csrSortedRowPtr, &csrSortedColInd, &fA, lda);

  // CUDA: CUSPARSE_DEPRECATED(cusparseSparseToDense) cusparseStatus_t CUSPARSEAPI cusparseZcsr2dense(cusparseHandle_t handle, int m, int n, const cusparseMatDescr_t descrA, const cuDoubleComplex* csrSortedValA, const int* csrSortedRowPtrA, const int* csrSortedColIndA, cuDoubleComplex* A, int lda);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_zcsr2dense(rocsparse_handle handle, rocsparse_int m, rocsparse_int n, const rocsparse_mat_descr descr, const rocsparse_double_complex* csr_val, const rocsparse_int* csr_row_ptr, const rocsparse_int* csr_col_ind, rocsparse_double_complex* A, rocsparse_int ld);
  // CHECK: status_t = rocsparse_zcsr2dense(handle_t, m, n, matDescr_A, &dComplexcscSortedVal, &csrSortedRowPtr, &csrSortedColInd, &dcomplexA, lda);
  status_t = cusparseZcsr2dense(handle_t, m, n, matDescr_A, &dComplexcscSortedVal, &csrSortedRowPtr, &csrSortedColInd, &dcomplexA, lda);

  // CUDA: CUSPARSE_DEPRECATED(cusparseSparseToDense) cusparseStatus_t CUSPARSEAPI cusparseCcsr2dense(cusparseHandle_t handle, int m, int n, const cusparseMatDescr_t descrA, const cuComplex* csrSortedValA, const int* csrSortedRowPtrA, const int* csrSortedColIndA, cuComplex* A, int lda);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_ccsr2dense(rocsparse_handle handle, rocsparse_int m, rocsparse_int n, const rocsparse_mat_descr descr, const rocsparse_float_complex* csr_val, const rocsparse_int* csr_row_ptr, const rocsparse_int* csr_col_ind,rocsparse_float_complex* A, rocsparse_int ld);
  // CHECK: status_t = rocsparse_ccsr2dense(handle_t, m, n, matDescr_A, &complexcscSortedVal, &csrSortedRowPtr, &csrSortedColInd, &complexA, lda);
  status_t = cusparseCcsr2dense(handle_t, m, n, matDescr_A, &complexcscSortedVal, &csrSortedRowPtr, &csrSortedColInd, &complexA, lda);

  // CUDA: CUSPARSE_DEPRECATED(cusparseSparseToDense) cusparseStatus_t CUSPARSEAPI cusparseDcsr2dense(cusparseHandle_t handle, int m, int n, const cusparseMatDescr_t descrA, const double* csrSortedValA, const int* csrSortedRowPtrA, const int* csrSortedColIndA, double* A, int lda);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_dcsr2dense(rocsparse_handle handle, rocsparse_int m, rocsparse_int n, const rocsparse_mat_descr descr, const double* csr_val, const rocsparse_int* csr_row_ptr, const rocsparse_int* csr_col_ind, double* A, rocsparse_int ld);
  // CHECK: status_t = rocsparse_dcsr2dense(handle_t, m, n, matDescr_A, &dcscSortedVal, &csrSortedRowPtr, &csrSortedColInd, &dA, lda);
  status_t = cusparseDcsr2dense(handle_t, m, n, matDescr_A, &dcscSortedVal, &csrSortedRowPtr, &csrSortedColInd, &dA, lda);

  // CUDA: CUSPARSE_DEPRECATED(cusparseSparseToDense) cusparseStatus_t CUSPARSEAPI cusparseScsr2dense(cusparseHandle_t handle, int m, int n, const cusparseMatDescr_t descrA, const float* csrSortedValA, const int* csrSortedRowPtrA, const int* csrSortedColIndA, float* A, int lda);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_scsr2dense(rocsparse_handle handle, rocsparse_int m, rocsparse_int n, const rocsparse_mat_descr descr, const float* csr_val, const rocsparse_int* csr_row_ptr, const rocsparse_int* csr_col_ind, float* A, rocsparse_int ld);
  // CHECK: status_t = rocsparse_scsr2dense(handle_t, m, n, matDescr_A, &cscSortedVal, &csrSortedRowPtr, &csrSortedColInd, &fA, lda);
  status_t = cusparseScsr2dense(handle_t, m, n, matDescr_A, &cscSortedVal, &csrSortedRowPtr, &csrSortedColInd, &fA, lda);

  // CUDA: CUSPARSE_DEPRECATED(cusparseDenseToSparse) cusparseStatus_t CUSPARSEAPI cusparseZdense2csc(cusparseHandle_t handle, int m, int n, const cusparseMatDescr_t descrA, const cuDoubleComplex* A, int lda, const int* nnzPerCol, cuDoubleComplex* cscSortedValA, int* cscSortedRowIndA, int* cscSortedColPtrA);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_zdense2csc(rocsparse_handle handle, rocsparse_int m, rocsparse_int n, const rocsparse_mat_descr descr, const rocsparse_double_complex* A, rocsparse_int ld, const rocsparse_int* nnz_per_columns, rocsparse_double_complex* csc_val, rocsparse_int* csc_col_ptr, rocsparse_int* csc_row_ind);
  // CHECK: status_t = rocsparse_zdense2csc(handle_t, m, n, matDescr_A, &dcomplexA, lda, &nnzPerCol, &dComplexcscSortedVal, &csrSortedRowPtr, &csrSortedColInd);
  status_t = cusparseZdense2csc(handle_t, m, n, matDescr_A, &dcomplexA, lda, &nnzPerCol, &dComplexcscSortedVal, &csrSortedRowPtr, &csrSortedColInd);

  // CUDA: CUSPARSE_DEPRECATED(cusparseDenseToSparse) cusparseStatus_t CUSPARSEAPI cusparseCdense2csc(cusparseHandle_t handle, int m, int n, const cusparseMatDescr_t descrA, const cuComplex* A, int lda, const int* nnzPerCol, cuComplex* cscSortedValA, int* cscSortedRowIndA, int* cscSortedColPtrA);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_cdense2csc(rocsparse_handle handle, rocsparse_int m, rocsparse_int n, const rocsparse_mat_descr descr, const rocsparse_float_complex* A, rocsparse_int ld, const rocsparse_int* nnz_per_columns, rocsparse_float_complex* csc_val, rocsparse_int* csc_col_ptr, rocsparse_int* csc_row_ind);
  // CHECK: status_t = rocsparse_cdense2csc(handle_t, m, n, matDescr_A, &complexA, lda, &nnzPerCol, &complexcscSortedVal, &csrSortedRowPtr, &csrSortedColInd);
  status_t = cusparseCdense2csc(handle_t, m, n, matDescr_A, &complexA, lda, &nnzPerCol, &complexcscSortedVal, &csrSortedRowPtr, &csrSortedColInd);

  // CUDA: CUSPARSE_DEPRECATED(cusparseDenseToSparse) cusparseStatus_t CUSPARSEAPI cusparseDdense2csc(cusparseHandle_t handle, int m, int n, const cusparseMatDescr_t descrA, const double* A, int lda, const int* nnzPerCol, double* cscSortedValA, int* cscSortedRowIndA, int* cscSortedColPtrA);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_ddense2csc(rocsparse_handle handle, rocsparse_int m, rocsparse_int n, const rocsparse_mat_descr descr, const double* A, rocsparse_int ld, const rocsparse_int* nnz_per_columns, double* csc_val, rocsparse_int* csc_col_ptr, rocsparse_int* csc_row_ind);
  // CHECK: status_t = rocsparse_ddense2csc(handle_t, m, n, matDescr_A, &dA, lda, &nnzPerCol, &dcscSortedVal, &csrSortedRowPtr, &csrSortedColInd);
  status_t = cusparseDdense2csc(handle_t, m, n, matDescr_A, &dA, lda, &nnzPerCol, &dcscSortedVal, &csrSortedRowPtr, &csrSortedColInd);

  // CUDA: CUSPARSE_DEPRECATED(cusparseDenseToSparse) cusparseStatus_t CUSPARSEAPI cusparseSdense2csc(cusparseHandle_t handle, int m, int n, const cusparseMatDescr_t descrA, const float* A, int lda, const int* nnzPerCol, float* cscSortedValA, int* cscSortedRowIndA, int* cscSortedColPtrA);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_sdense2csc(rocsparse_handle handle, rocsparse_int m, rocsparse_int n, const rocsparse_mat_descr descr, const float* A, rocsparse_int ld, const rocsparse_int* nnz_per_columns, float* csc_val, rocsparse_int* csc_col_ptr, rocsparse_int* csc_row_ind);
  // CHECK: status_t = rocsparse_sdense2csc(handle_t, m, n, matDescr_A, &fA, lda, &nnzPerCol, &cscSortedVal, &csrSortedRowPtr, &csrSortedColInd);
  status_t = cusparseSdense2csc(handle_t, m, n, matDescr_A, &fA, lda, &nnzPerCol, &cscSortedVal, &csrSortedRowPtr, &csrSortedColInd);

  // CUDA: CUSPARSE_DEPRECATED(cusparseDenseToSparse) cusparseStatus_t CUSPARSEAPI cusparseZdense2csr(cusparseHandle_t handle, int m, int n, const cusparseMatDescr_t descrA, const cuDoubleComplex* A, int lda, const int* nnzPerRow, cuDoubleComplex* csrSortedValA, int* csrSortedRowPtrA, int* csrSortedColIndA);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_zdense2csr(rocsparse_handle handle, rocsparse_int m, rocsparse_int n, const rocsparse_mat_descr descr, const rocsparse_double_complex* A, rocsparse_int ld, const rocsparse_int* nnz_per_rows, rocsparse_double_complex* csr_val, rocsparse_int* csr_row_ptr, rocsparse_int* csr_col_ind);
  // CHECK: status_t = rocsparse_zdense2csr(handle_t, m, n, matDescr_A, &dcomplexA, lda, &nnzPerRow, &dComplexcsrSortedValA, &csrRowPtrA, &csrColIndA);
  status_t = cusparseZdense2csr(handle_t, m, n, matDescr_A, &dcomplexA, lda, &nnzPerRow, &dComplexcsrSortedValA, &csrRowPtrA, &csrColIndA);

  // CUDA: CUSPARSE_DEPRECATED(cusparseDenseToSparse) cusparseStatus_t CUSPARSEAPI cusparseCdense2csr(cusparseHandle_t handle, int m, int n, const cusparseMatDescr_t descrA, const cuComplex* A, int lda, const int* nnzPerRow, cuComplex* csrSortedValA, int* csrSortedRowPtrA, int* csrSortedColIndA);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_cdense2csr(rocsparse_handle handle, rocsparse_int m, rocsparse_int n, const rocsparse_mat_descr descr, const rocsparse_float_complex* A, rocsparse_int ld, const rocsparse_int* nnz_per_rows, rocsparse_float_complex* csr_val, rocsparse_int* csr_row_ptr, rocsparse_int* csr_col_ind);
  // CHECK: status_t = rocsparse_cdense2csr(handle_t, m, n, matDescr_A, &complexA, lda, &nnzPerRow, &complexcsrSortedValA, &csrRowPtrA, &csrColIndA);
  status_t = cusparseCdense2csr(handle_t, m, n, matDescr_A, &complexA, lda, &nnzPerRow, &complexcsrSortedValA, &csrRowPtrA, &csrColIndA);

  // CUDA: CUSPARSE_DEPRECATED(cusparseDenseToSparse) cusparseStatus_t CUSPARSEAPI cusparseDdense2csr(cusparseHandle_t handle, int m, int n, const cusparseMatDescr_t descrA, const double* A, int lda, const int* nnzPerRow, double* csrSortedValA, int* csrSortedRowPtrA, int* csrSortedColIndA);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_ddense2csr(rocsparse_handle handle, rocsparse_int m, rocsparse_int n, const rocsparse_mat_descr descr, const double* A, rocsparse_int ld, const rocsparse_int* nnz_per_rows, double* csr_val, rocsparse_int* csr_row_ptr, rocsparse_int* csr_col_ind);
  // CHECK: status_t = rocsparse_ddense2csr(handle_t, m, n, matDescr_A, &dA, lda, &nnzPerRow, &dcsrSortedValA, &csrRowPtrA, &csrColIndA);
  status_t = cusparseDdense2csr(handle_t, m, n, matDescr_A, &dA, lda, &nnzPerRow, &dcsrSortedValA, &csrRowPtrA, &csrColIndA);

  // CUDA: CUSPARSE_DEPRECATED(cusparseDenseToSparse) cusparseStatus_t CUSPARSEAPI cusparseSdense2csr(cusparseHandle_t handle, int m, int n, const cusparseMatDescr_t descrA, const float* A, int lda, const int* nnzPerRow, float* csrSortedValA, int* csrSortedRowPtrA, int* csrSortedColIndA);
  // ROC: ROCSPARSE_EXPORT rocsparse_status rocsparse_sdense2csr(rocsparse_handle handle, rocsparse_int m, rocsparse_int n, const rocsparse_mat_descr descr, const float* A, rocsparse_int ld, const rocsparse_int* nnz_per_rows, float* csr_val, rocsparse_int* csr_row_ptr, rocsparse_int* csr_col_ind);
  // CHECK: status_t = rocsparse_sdense2csr(handle_t, m, n, matDescr_A, &fA, lda, &nnzPerRow, &csrSortedValA, &csrRowPtrA, &csrColIndA);
  status_t = cusparseSdense2csr(handle_t, m, n, matDescr_A, &fA, lda, &nnzPerRow, &csrSortedValA, &csrRowPtrA, &csrColIndA);
#endif

#if CUDA_VERSION >= 12010 && CUSPARSE_VERSION >= 12100
  // CHECK: rocsparse_spmv_alg SPMV_SELL_ALG1 = rocsparse_spmv_alg_ell;
  hipsparseSpMVAlg_t SPMV_SELL_ALG1 = CUSPARSE_SPMV_SELL_ALG1;

  // CHECK: rocsparse_format FORMAT_BSR = rocsparse_format_bsr;
  // CHECK-NEXT: rocsparse_format FORMAT_SLICED_ELLPACK = rocsparse_format_ell;
  hipsparseFormat_t FORMAT_BSR = CUSPARSE_FORMAT_BSR;
  hipsparseFormat_t FORMAT_SLICED_ELLPACK = CUSPARSE_FORMAT_SLICED_ELLPACK;
#endif

  return 0;
}
