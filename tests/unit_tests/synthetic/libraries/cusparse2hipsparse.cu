// RUN: %run_test hipify "%s" "%t" %hipify_args 3 --skip-excluded-preprocessor-conditional-blocks --experimental --use-hip-data-types %clang_args -ferror-limit=500

// CHECK: #include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
// CHECK: #include "hip/hip_complex.h"
#include "hip/hip_complex.h"
#include <stdio.h>
// CHECK: #include "hipsparse.h"
#include "hipsparse.h"
// CHECK-NOT: #include "hipsparse.h"

int main() {
  printf("17. cuSPARSE API to hipSPARSE API synthetic test\n");

  // CHECK: hipsparseHandle_t handle_t;
  hipsparseHandle_t handle_t;

  // CHECK: hipsparseMatDescr_t matDescr_t, matDescr_t_2, matDescr_A, matDescr_C;
  hipsparseMatDescr_t matDescr_t, matDescr_t_2, matDescr_A, matDescr_C;

  // CHECK: hipsparseColorInfo_t colorInfo_t;
  hipsparseColorInfo_t colorInfo_t;

  // CHECK: hipsparseOperation_t sparseOperation_t;
  // CHECK-NEXT: hipsparseOperation_t OPERATION_NON_TRANSPOSE = HIPSPARSE_OPERATION_NON_TRANSPOSE;
  // CHECK-NEXT: hipsparseOperation_t OPERATION_TRANSPOSE = HIPSPARSE_OPERATION_TRANSPOSE;
  // CHECK-NEXT: hipsparseOperation_t OPERATION_CONJUGATE_TRANSPOSE = HIPSPARSE_OPERATION_CONJUGATE_TRANSPOSE;
  hipsparseOperation_t sparseOperation_t;
  hipsparseOperation_t OPERATION_NON_TRANSPOSE = HIPSPARSE_OPERATION_NON_TRANSPOSE;
  hipsparseOperation_t OPERATION_TRANSPOSE = HIPSPARSE_OPERATION_TRANSPOSE;
  hipsparseOperation_t OPERATION_CONJUGATE_TRANSPOSE = HIPSPARSE_OPERATION_CONJUGATE_TRANSPOSE;

  // CHECK: hipsparseIndexBase_t indexBase_t;
  // CHECK-NEXT: hipsparseIndexBase_t INDEX_BASE_ZERO = HIPSPARSE_INDEX_BASE_ZERO;
  // CHECK-NEXT: hipsparseIndexBase_t INDEX_BASE_ONE = HIPSPARSE_INDEX_BASE_ONE;
  hipsparseIndexBase_t indexBase_t;
  hipsparseIndexBase_t INDEX_BASE_ZERO = HIPSPARSE_INDEX_BASE_ZERO;
  hipsparseIndexBase_t INDEX_BASE_ONE = HIPSPARSE_INDEX_BASE_ONE;

  // CHECK: hipsparseMatrixType_t matrixType_t;
  // CHECK-NEXT: hipsparseMatrixType_t MATRIX_TYPE_GENERAL = HIPSPARSE_MATRIX_TYPE_GENERAL;
  // CHECK-NEXT: hipsparseMatrixType_t MATRIX_TYPE_SYMMETRIC = HIPSPARSE_MATRIX_TYPE_SYMMETRIC;
  // CHECK-NEXT: hipsparseMatrixType_t MATRIX_TYPE_HERMITIAN = HIPSPARSE_MATRIX_TYPE_HERMITIAN;
  // CHECK-NEXT: hipsparseMatrixType_t MATRIX_TYPE_TRIANGULAR = HIPSPARSE_MATRIX_TYPE_TRIANGULAR;
  hipsparseMatrixType_t matrixType_t;
  hipsparseMatrixType_t MATRIX_TYPE_GENERAL = HIPSPARSE_MATRIX_TYPE_GENERAL;
  hipsparseMatrixType_t MATRIX_TYPE_SYMMETRIC = HIPSPARSE_MATRIX_TYPE_SYMMETRIC;
  hipsparseMatrixType_t MATRIX_TYPE_HERMITIAN = HIPSPARSE_MATRIX_TYPE_HERMITIAN;
  hipsparseMatrixType_t MATRIX_TYPE_TRIANGULAR = HIPSPARSE_MATRIX_TYPE_TRIANGULAR;

  // CHECK: hipsparseDiagType_t diagType_t;
  // CHECK-NEXT: hipsparseDiagType_t DIAG_TYPE_NON_UNIT = HIPSPARSE_DIAG_TYPE_NON_UNIT;
  // CHECK-NEXT: hipsparseDiagType_t DIAG_TYPE_UNIT = HIPSPARSE_DIAG_TYPE_UNIT;
  hipsparseDiagType_t diagType_t;
  hipsparseDiagType_t DIAG_TYPE_NON_UNIT = HIPSPARSE_DIAG_TYPE_NON_UNIT;
  hipsparseDiagType_t DIAG_TYPE_UNIT = HIPSPARSE_DIAG_TYPE_UNIT;

  // CHECK: hipsparseFillMode_t fillMode_t;
  // CHECK-NEXT: hipsparseFillMode_t FILL_MODE_LOWER = HIPSPARSE_FILL_MODE_LOWER;
  // CHECK-NEXT: hipsparseFillMode_t FILL_MODE_UPPER = HIPSPARSE_FILL_MODE_UPPER;
  hipsparseFillMode_t fillMode_t;
  hipsparseFillMode_t FILL_MODE_LOWER = HIPSPARSE_FILL_MODE_LOWER;
  hipsparseFillMode_t FILL_MODE_UPPER = HIPSPARSE_FILL_MODE_UPPER;

  // CHECK: hipsparseAction_t action_t, copyValues;
  // CHECK-NEXT: hipsparseAction_t ACTION_SYMBOLIC = HIPSPARSE_ACTION_SYMBOLIC;
  // CHECK-NEXT: hipsparseAction_t ACTION_NUMERIC = HIPSPARSE_ACTION_NUMERIC;
  hipsparseAction_t action_t, copyValues;
  hipsparseAction_t ACTION_SYMBOLIC = HIPSPARSE_ACTION_SYMBOLIC;
  hipsparseAction_t ACTION_NUMERIC = HIPSPARSE_ACTION_NUMERIC;

  // CHECK: hipsparseDirection_t direction_t;
  // CHECK-NEXT: hipsparseDirection_t DIRECTION_ROW = HIPSPARSE_DIRECTION_ROW;
  // CHECK-NEXT: hipsparseDirection_t DIRECTION_COLUMN = HIPSPARSE_DIRECTION_COLUMN;
  hipsparseDirection_t direction_t;
  hipsparseDirection_t DIRECTION_ROW = HIPSPARSE_DIRECTION_ROW;
  hipsparseDirection_t DIRECTION_COLUMN = HIPSPARSE_DIRECTION_COLUMN;

  // CHECK: hipsparseSolvePolicy_t solvePolicy_t;
  // CHECK-NEXT: hipsparseSolvePolicy_t SOLVE_POLICY_NO_LEVEL = HIPSPARSE_SOLVE_POLICY_NO_LEVEL;
  // CHECK-NEXT: hipsparseSolvePolicy_t SOLVE_POLICY_USE_LEVEL = HIPSPARSE_SOLVE_POLICY_USE_LEVEL;
  hipsparseSolvePolicy_t solvePolicy_t;
  hipsparseSolvePolicy_t SOLVE_POLICY_NO_LEVEL = HIPSPARSE_SOLVE_POLICY_NO_LEVEL;
  hipsparseSolvePolicy_t SOLVE_POLICY_USE_LEVEL = HIPSPARSE_SOLVE_POLICY_USE_LEVEL;

  // CHECK: hipsparsePointerMode_t pointerMode_t;
  // CHECK-NEXT: hipsparsePointerMode_t POINTER_MODE_HOST = HIPSPARSE_POINTER_MODE_HOST;
  // CHECK-NEXT: hipsparsePointerMode_t POINTER_MODE_DEVICE = HIPSPARSE_POINTER_MODE_DEVICE;
  hipsparsePointerMode_t pointerMode_t;
  hipsparsePointerMode_t POINTER_MODE_HOST = HIPSPARSE_POINTER_MODE_HOST;
  hipsparsePointerMode_t POINTER_MODE_DEVICE = HIPSPARSE_POINTER_MODE_DEVICE;

  // CHECK: hipsparseStatus_t status_t;
  // CHECK-NEXT: hipsparseStatus_t STATUS_SUCCESS = HIPSPARSE_STATUS_SUCCESS;
  // CHECK-NEXT: hipsparseStatus_t STATUS_NOT_INITIALIZED = HIPSPARSE_STATUS_NOT_INITIALIZED;
  // CHECK-NEXT: hipsparseStatus_t STATUS_ALLOC_FAILED = HIPSPARSE_STATUS_ALLOC_FAILED;
  // CHECK-NEXT: hipsparseStatus_t STATUS_INVALID_VALUE = HIPSPARSE_STATUS_INVALID_VALUE;
  // CHECK-NEXT: hipsparseStatus_t STATUS_ARCH_MISMATCH = HIPSPARSE_STATUS_ARCH_MISMATCH;
  // CHECK-NEXT: hipsparseStatus_t STATUS_MAPPING_ERROR = HIPSPARSE_STATUS_MAPPING_ERROR;
  // CHECK-NEXT: hipsparseStatus_t STATUS_EXECUTION_FAILED = HIPSPARSE_STATUS_EXECUTION_FAILED;
  // CHECK-NEXT: hipsparseStatus_t STATUS_INTERNAL_ERROR = HIPSPARSE_STATUS_INTERNAL_ERROR;
  // CHECK-NEXT: hipsparseStatus_t STATUS_MATRIX_TYPE_NOT_SUPPORTED = HIPSPARSE_STATUS_MATRIX_TYPE_NOT_SUPPORTED;
  // CHECK-NEXT: hipsparseStatus_t STATUS_ZERO_PIVOT = HIPSPARSE_STATUS_ZERO_PIVOT;
  hipsparseStatus_t status_t;
  hipsparseStatus_t STATUS_SUCCESS = HIPSPARSE_STATUS_SUCCESS;
  hipsparseStatus_t STATUS_NOT_INITIALIZED = HIPSPARSE_STATUS_NOT_INITIALIZED;
  hipsparseStatus_t STATUS_ALLOC_FAILED = HIPSPARSE_STATUS_ALLOC_FAILED;
  hipsparseStatus_t STATUS_INVALID_VALUE = HIPSPARSE_STATUS_INVALID_VALUE;
  hipsparseStatus_t STATUS_ARCH_MISMATCH = HIPSPARSE_STATUS_ARCH_MISMATCH;
  hipsparseStatus_t STATUS_MAPPING_ERROR = HIPSPARSE_STATUS_MAPPING_ERROR;
  hipsparseStatus_t STATUS_EXECUTION_FAILED = HIPSPARSE_STATUS_EXECUTION_FAILED;
  hipsparseStatus_t STATUS_INTERNAL_ERROR = HIPSPARSE_STATUS_INTERNAL_ERROR;
  hipsparseStatus_t STATUS_MATRIX_TYPE_NOT_SUPPORTED = HIPSPARSE_STATUS_MATRIX_TYPE_NOT_SUPPORTED;
  hipsparseStatus_t STATUS_ZERO_PIVOT = HIPSPARSE_STATUS_ZERO_PIVOT;

  // CHECK: hipStream_t stream_t;
  hipStream_t stream_t;

  int iVal = 0;
  int batchCount = 0;
  int m = 0;
  int n = 0;
  int mb = 0;
  int nb = 0;
  int nnza = 0;
  int nnzb = 0;
  int nnzc = 0;
  int nnzPerRow = 0;
  int nnzPerCol = 0;
  int innz = 0;
  int lda = 0;
  int blockDim = 0;
  int csrSortedRowPtr = 0;
  int csrSortedColInd = 0;
  int cscRowIndA = 0;
  int cscColPtrA = 0;
  int csrRowPtrA = 0;
  int csrColIndA = 0;
  int ncolors = 0;
  int coloring = 0;
  int reordering = 0;
  int bscRowInd = 0;
  int bsrRowPtrA = 0;
  int bsrRowPtrC = 0;
  int csrRowPtrC = 0;
  int bscColPtr = 0;
  int bsrColIndA = 0;
  int bsrColIndC = 0;
  int csrColIndC = 0;
  int rowBlockDim = 0;
  int rowBlockDimA = 0;
  int colBlockDimA = 0;
  int rowBlockDimC = 0;
  int colBlockDim = 0;
  int colBlockDimC = 0;
  int bsrSortedRowPtr = 0;
  int bsrSortedRowPtrC = 0;
  int bsrSortedColInd = 0;
  int bsrSortedColIndC = 0;
  int bufferSizeInBytes = 0;
  int nnzTotalDevHostPtr = 0;
  int nnzPerRowCol = 0;
  int userEllWidth = 0;
  int64_t size = 0;
  int64_t nnz = 0;
  int64_t rows = 0;
  int64_t cols = 0;
  int64_t ellCols = 0;
  int64_t ellBlockSize = 0;
  int64_t batchStride = 0;
  int64_t offsetsBatchStride = 0;
  int64_t columnsValuesBatchStride = 0;
  int64_t ld = 0;
  void *indices = nullptr;
  void *values = nullptr;
  void *cooRowInd = nullptr;
  int icooRowInd = 0;
  void *cscRowInd = nullptr;
  void *csrColInd = nullptr;
  void *cooColInd = nullptr;
  void *ellColInd = nullptr;
  void *cooValues = nullptr;
  void *csrValues = nullptr;
  void *cscValues = nullptr;
  void *ellValue = nullptr;
  void *csrRowOffsets = nullptr;
  void *cscColOffsets = nullptr;
  void *cooRows = nullptr;
  int icooRows = 0;
  void *cooColumns = nullptr;
  int icooColumns = 0;
  void *data = nullptr;
  void *alpha = nullptr;
  void *beta = nullptr;
  void *pBuffer = nullptr;
  int *P = nullptr;
  void *tempBuffer = nullptr;
  void *c_coeff = nullptr;
  void *s_coeff = nullptr;
  size_t dataSize = 0;
  size_t bufferSize = 0;
  double dfractionToColor = 0.f;
  float ffractionToColor = 0.f;
  double bsrValA = 0.f;
  double csrValA = 0.f;
  float fcsrValA = 0.f;
  double csrValC = 0.f;
  float fcsrValC = 0.f;
  float csrSortedVal = 0.f;
  float cscSortedVal = 0.f;
  float csrSortedValA = 0.f;
  double dcsrSortedVal = 0.f;
  double dcscSortedVal = 0.f;
  double dcsrSortedValA = 0.f;
  double dbsrSortedVal = 0.f;
  double dbsrSortedValA = 0.f;
  double dbsrSortedValC = 0.f;
  float fbsrSortedVal = 0.f;
  float fbsrSortedValA = 0.f;
  float fbsrSortedValC = 0.f;
  float fcsrSortedValC = 0.f;
  double dcsrSortedValC = 0.f;
  double percentage = 0.f;
  float fpercentage = 0.f;
  double dthreshold = 0.f;
  float fthreshold = 0.f;
  double dtol = 0.f;
  float ftol = 0.f;
  double dbscVal = 0.f;
  float fbscVal = 0.f;
  double dA = 0.f;
  float fA = 0.f;
  int algo = 0;
  double dds = 0.f;
  double ddl = 0.f;
  double dd = 0.f;
  double ddu = 0.f;
  double ddw = 0.f;
  double dx = 0.f;
  float fds = 0.f;
  float fdl = 0.f;
  float fd = 0.f;
  float fdu = 0.f;
  float fdw = 0.f;
  float fx = 0.f;
  pruneInfo_t prune_info;

  // CHECK: hipDoubleComplex dcomplex, dcomplexA, dComplexbsrSortedValA, dComplexbsrSortedValC, dComplexcsrSortedValA, dComplexcsrSortedValC, dcomplextol, dComplexbsrSortedVal, dComplexbscVal, dComplexcscSortedVal, dcomplexds, dcomplexdl, dcomplexd, dcomplexdu, dcomplexdw, dcomplexx;
  hipDoubleComplex dcomplex, dcomplexA, dComplexbsrSortedValA, dComplexbsrSortedValC, dComplexcsrSortedValA, dComplexcsrSortedValC, dcomplextol, dComplexbsrSortedVal, dComplexbscVal, dComplexcscSortedVal, dcomplexds, dcomplexdl, dcomplexd, dcomplexdu, dcomplexdw, dcomplexx;

  // CHECK: hipComplex complex, complexA, complexbsrValA, complexbsrSortedValC, complexcsrSortedValA, complexcsrSortedValC, complextol, complexbsrSortedVal, complexbscVal, complexcscSortedVal, complexds, complexdl, complexd, complexdu, complexdw, complexx;
  hipComplex complex, complexA, complexbsrValA, complexbsrSortedValC, complexcsrSortedValA, complexcsrSortedValC, complextol, complexbsrSortedVal, complexbscVal, complexcscSortedVal, complexds, complexdl, complexd, complexdu, complexdw, complexx;

  // CHECK: hipsparseOperation_t opA, opB;
  hipsparseOperation_t opA, opB;

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseCreate(cusparseHandle_t* handle);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseCreate(hipsparseHandle_t* handle);
  // CHECK: status_t = hipsparseCreate(&handle_t);
  status_t = hipsparseCreate(&handle_t);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseDestroy(cusparseHandle_t handle);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseDestroy(hipsparseHandle_t handle);
  // CHECK: status_t = hipsparseDestroy(handle_t);
  status_t = hipsparseDestroy(handle_t);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseSetStream(cusparseHandle_t handle, cudaStream_t streamId);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseSetStream(hipsparseHandle_t handle, hipStream_t streamId);
  // CHECK: status_t = hipsparseSetStream(handle_t, stream_t);
  status_t = hipsparseSetStream(handle_t, stream_t);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseGetStream(cusparseHandle_t handle, cudaStream_t* streamId);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseGetStream(hipsparseHandle_t handle, hipStream_t* streamId);
  // CHECK: status_t = hipsparseGetStream(handle_t, &stream_t);
  status_t = hipsparseGetStream(handle_t, &stream_t);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseSetPointerMode(cusparseHandle_t handle, cusparsePointerMode_t mode);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseSetPointerMode(hipsparseHandle_t handle, hipsparsePointerMode_t mode);
  // CHECK: status_t = hipsparseSetPointerMode(handle_t, pointerMode_t);
  status_t = hipsparseSetPointerMode(handle_t, pointerMode_t);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseGetPointerMode(cusparseHandle_t handle, cusparsePointerMode_t* mode);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseGetPointerMode(hipsparseHandle_t handle, hipsparsePointerMode_t* mode);
  // CHECK: status_t = hipsparseGetPointerMode(handle_t, &pointerMode_t);
  status_t = hipsparseGetPointerMode(handle_t, &pointerMode_t);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseGetVersion(cusparseHandle_t handle, int* version);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseGetVersion(hipsparseHandle_t handle, int* version);
  // CHECK: status_t = hipsparseGetVersion(handle_t, &iVal);
  status_t = hipsparseGetVersion(handle_t, &iVal);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseCreateMatDescr(cusparseMatDescr_t* descrA);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseCreateMatDescr(hipsparseMatDescr_t* descrA);
  // CHECK: status_t = hipsparseCreateMatDescr(&matDescr_t);
  status_t = hipsparseCreateMatDescr(&matDescr_t);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseDestroyMatDescr(cusparseMatDescr_t descrA);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseDestroyMatDescr(hipsparseMatDescr_t descrA);
  // CHECK: status_t = hipsparseDestroyMatDescr(matDescr_t);
  status_t = hipsparseDestroyMatDescr(matDescr_t);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseSetMatIndexBase(cusparseMatDescr_t descrA, cusparseIndexBase_t base);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseSetMatIndexBase(hipsparseMatDescr_t descrA, hipsparseIndexBase_t base);
  // CHECK: status_t = hipsparseSetMatIndexBase(matDescr_t, indexBase_t);
  status_t = hipsparseSetMatIndexBase(matDescr_t, indexBase_t);

  // CUDA: cusparseIndexBase_t CUSPARSEAPI cusparseGetMatIndexBase(const cusparseMatDescr_t descrA);
  // HIP: HIPSPARSE_EXPORT hipsparseIndexBase_t hipsparseGetMatIndexBase(const hipsparseMatDescr_t descrA);
  // CHECK: indexBase_t = hipsparseGetMatIndexBase(matDescr_t);
  indexBase_t = hipsparseGetMatIndexBase(matDescr_t);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseSetMatType(cusparseMatDescr_t descrA, cusparseMatrixType_t type);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseSetMatType(hipsparseMatDescr_t descrA, hipsparseMatrixType_t type);
  // CHECK: status_t = hipsparseSetMatType(matDescr_t, matrixType_t);
  status_t = hipsparseSetMatType(matDescr_t, matrixType_t);

  // CUDA: cusparseMatrixType_t CUSPARSEAPI cusparseGetMatType(const cusparseMatDescr_t descrA);
  // HIP: HIPSPARSE_EXPORT hipsparseMatrixType_t hipsparseGetMatType(const hipsparseMatDescr_t descrA);
  // CHECK: matrixType_t = hipsparseGetMatType(matDescr_t);
  matrixType_t = hipsparseGetMatType(matDescr_t);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseSetMatFillMode(cusparseMatDescr_t descrA, cusparseFillMode_t fillMode);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseSetMatFillMode(hipsparseMatDescr_t descrA, hipsparseFillMode_t fillMode);
  // CHECK: status_t = hipsparseSetMatFillMode(matDescr_t, fillMode_t);
  status_t = hipsparseSetMatFillMode(matDescr_t, fillMode_t);

  // CUDA: cusparseFillMode_t CUSPARSEAPI cusparseGetMatFillMode(const cusparseMatDescr_t descrA);
  // HIP: HIPSPARSE_EXPORT hipsparseFillMode_t hipsparseGetMatFillMode(const hipsparseMatDescr_t descrA);
  // CHECK: fillMode_t = hipsparseGetMatFillMode(matDescr_t);
  fillMode_t = hipsparseGetMatFillMode(matDescr_t);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseSetMatDiagType(cusparseMatDescr_t descrA, cusparseDiagType_t diagType);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseSetMatDiagType(hipsparseMatDescr_t descrA, hipsparseDiagType_t diagType);
  // CHECK: status_t = hipsparseSetMatDiagType(matDescr_t, diagType_t);
  status_t = hipsparseSetMatDiagType(matDescr_t, diagType_t);

  // CUDA: cusparseDiagType_t CUSPARSEAPI cusparseGetMatDiagType(const cusparseMatDescr_t descrA);
  // HIP: HIPSPARSE_EXPORT hipsparseDiagType_t hipsparseGetMatDiagType(const hipsparseMatDescr_t descrA);
  // CHECK: diagType_t = hipsparseGetMatDiagType(matDescr_t);
  diagType_t = hipsparseGetMatDiagType(matDescr_t);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseCreateColorInfo(cusparseColorInfo_t* info);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseCreateColorInfo(hipsparseColorInfo_t* info);
  // CHECK: status_t = hipsparseCreateColorInfo(&colorInfo_t);
  status_t = hipsparseCreateColorInfo(&colorInfo_t);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseDestroyColorInfo(cusparseColorInfo_t info);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseDestroyColorInfo(hipsparseColorInfo_t info);
  // CHECK: status_t = hipsparseDestroyColorInfo(colorInfo_t);
  status_t = hipsparseDestroyColorInfo(colorInfo_t);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseZcsrcolor(cusparseHandle_t handle, int m, int nnz, const cusparseMatDescr_t descrA, const cuDoubleComplex* csrSortedValA, const int* csrSortedRowPtrA, const int* csrSortedColIndA, const double* fractionToColor, int* ncolors, int* coloring, int* reordering, const cusparseColorInfo_t info);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseZcsrcolor(hipsparseHandle_t handle, int m, int nnz, const hipsparseMatDescr_t descrA, const hipDoubleComplex* csrValA, const int* csrRowPtrA, const int* csrColIndA, const double* fractionToColor, int* ncolors, int* coloring, int* reordering, hipsparseColorInfo_t info);
  // CHECK: status_t = hipsparseZcsrcolor(handle_t, m, innz, matDescr_t, &dcomplex, &csrRowPtrA, &csrColIndA, &dfractionToColor, &ncolors, &coloring, &reordering, colorInfo_t);
  status_t = hipsparseZcsrcolor(handle_t, m, innz, matDescr_t, &dcomplex, &csrRowPtrA, &csrColIndA, &dfractionToColor, &ncolors, &coloring, &reordering, colorInfo_t);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseCcsrcolor(cusparseHandle_t handle, int m, int nnz, const cusparseMatDescr_t descrA, const cuComplex* csrSortedValA, const int* csrSortedRowPtrA, const int* csrSortedColIndA, const float* fractionToColor, int* ncolors, int* coloring, int* reordering, const cusparseColorInfo_t info);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseCcsrcolor(hipsparseHandle_t handle, int m, int nnz, const hipsparseMatDescr_t descrA, const hipComplex* csrValA, const int* csrRowPtrA, const int* csrColIndA, const float* fractionToColor, int* ncolors, int* coloring, int* reordering, hipsparseColorInfo_t info);
  // CHECK: status_t = hipsparseCcsrcolor(handle_t, m, innz, matDescr_t, &complex, &csrRowPtrA, &csrColIndA, &ffractionToColor, &ncolors, &coloring, &reordering, colorInfo_t);
  status_t = hipsparseCcsrcolor(handle_t, m, innz, matDescr_t, &complex, &csrRowPtrA, &csrColIndA, &ffractionToColor, &ncolors, &coloring, &reordering, colorInfo_t);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseDcsrcolor(cusparseHandle_t handle, int m, int nnz, const cusparseMatDescr_t descrA, const double* csrSortedValA, const int* csrSortedRowPtrA, const int* csrSortedColIndA, const double* fractionToColor, int* ncolors, int* coloring, int* reordering, const cusparseColorInfo_t info);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseDcsrcolor(hipsparseHandle_t handle, int m, int nnz, const hipsparseMatDescr_t descrA, const double* csrValA, const int* csrRowPtrA, const int* csrColIndA, const double* fractionToColor, int* ncolors, int* coloring, int* reordering, hipsparseColorInfo_t info);
  // CHECK: status_t = hipsparseDcsrcolor(handle_t, m, innz, matDescr_t, &csrValA, &csrRowPtrA, &csrColIndA, &dfractionToColor, &ncolors, &coloring, &reordering, colorInfo_t);
  status_t = hipsparseDcsrcolor(handle_t, m, innz, matDescr_t, &csrValA, &csrRowPtrA, &csrColIndA, &dfractionToColor, &ncolors, &coloring, &reordering, colorInfo_t);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseScsrcolor(cusparseHandle_t handle, int m, int nnz, const cusparseMatDescr_t descrA, const float* csrSortedValA, const int* csrSortedRowPtrA, const int* csrSortedColIndA, const float* fractionToColor, int* ncolors, int* coloring, int* reordering, const cusparseColorInfo_t info);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseScsrcolor(hipsparseHandle_t handle, int m, int nnz, const hipsparseMatDescr_t descrA, const float* csrValA, const int* csrRowPtrA, const int* csrColIndA, const float* fractionToColor, int* ncolors, int* coloring, int* reordering, hipsparseColorInfo_t info);
  // CHECK: status_t = hipsparseScsrcolor(handle_t, m, innz, matDescr_t, &csrSortedValA, &csrRowPtrA, &csrColIndA, &ffractionToColor, &ncolors, &coloring, &reordering, colorInfo_t);
  status_t = hipsparseScsrcolor(handle_t, m, innz, matDescr_t, &csrSortedValA, &csrRowPtrA, &csrColIndA, &ffractionToColor, &ncolors, &coloring, &reordering, colorInfo_t);

  // CUDA:cusparseStatus_t CUSPARSEAPI cusparseZgebsr2gebsr(cusparseHandle_t handle, cusparseDirection_t dirA, int mb, int nb, int nnzb, const cusparseMatDescr_t descrA, const cuDoubleComplex* bsrSortedValA, const int* bsrSortedRowPtrA, const int* bsrSortedColIndA, int rowBlockDimA, int colBlockDimA, const cusparseMatDescr_t descrC, cuDoubleComplex* bsrSortedValC, int* bsrSortedRowPtrC, int* bsrSortedColIndC, int rowBlockDimC, int colBlockDimC, void* pBuffer);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseZgebsr2gebsr(hipsparseHandle_t handle, hipsparseDirection_t dirA, int mb, int nb, int nnzb, const hipsparseMatDescr_t descrA, const hipDoubleComplex* bsrValA, const int* bsrRowPtrA, const int* bsrColIndA, int rowBlockDimA, int colBlockDimA, const hipsparseMatDescr_t descrC, hipDoubleComplex* bsrValC, int* bsrRowPtrC, int* bsrColIndC, int rowBlockDimC, int colBlockDimC, void* buffer);
  // CHECK: status_t = hipsparseZgebsr2gebsr(handle_t, direction_t, mb, nb, nnzb, matDescr_t, &dcomplex, &bsrRowPtrA, &bsrColIndA, rowBlockDimA, colBlockDimA, matDescr_C, &dComplexbsrSortedValC, &bsrSortedRowPtrC, &bsrSortedColIndC, bsrRowPtrC, bsrColIndC, tempBuffer);
  status_t = hipsparseZgebsr2gebsr(handle_t, direction_t, mb, nb, nnzb, matDescr_t, &dcomplex, &bsrRowPtrA, &bsrColIndA, rowBlockDimA, colBlockDimA, matDescr_C, &dComplexbsrSortedValC, &bsrSortedRowPtrC, &bsrSortedColIndC, bsrRowPtrC, bsrColIndC, tempBuffer);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseZgebsr2gebsr_bufferSize(cusparseHandle_t handle, cusparseDirection_t dirA, int mb, int nb, int nnzb, const cusparseMatDescr_t descrA, const cuDoubleComplex* bsrSortedValA, const int* bsrSortedRowPtrA, const int* bsrSortedColIndA, int rowBlockDimA, int colBlockDimA, int rowBlockDimC,int colBlockDimC, int* pBufferSizeInBytes);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseZgebsr2gebsr_bufferSize(hipsparseHandle_t handle, hipsparseDirection_t dirA, int mb, int nb, int nnzb, const hipsparseMatDescr_t descrA, const hipDoubleComplex* bsrValA, const int* bsrRowPtrA, const int* bsrColIndA, int rowBlockDimA, int colBlockDimA, int rowBlockDimC, int colBlockDimC, int* bufferSize);
  // CHECK: status_t = hipsparseZgebsr2gebsr_bufferSize(handle_t, direction_t, mb, nb, nnzb, matDescr_t, &dcomplex, &bsrRowPtrA, &bsrColIndA, rowBlockDimA, colBlockDimA, rowBlockDimC, colBlockDimC, &bufferSizeInBytes);
  status_t = hipsparseZgebsr2gebsr_bufferSize(handle_t, direction_t, mb, nb, nnzb, matDescr_t, &dcomplex, &bsrRowPtrA, &bsrColIndA, rowBlockDimA, colBlockDimA, rowBlockDimC, colBlockDimC, &bufferSizeInBytes);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseCgebsr2gebsr(cusparseHandle_t handle, cusparseDirection_t dirA, int mb, int nb, int nnzb, const cusparseMatDescr_t descrA, const cuComplex* bsrSortedValA, const int* bsrSortedRowPtrA, const int* bsrSortedColIndA, int rowBlockDimA, int colBlockDimA, const cusparseMatDescr_t descrC, cuComplex* bsrSortedValC, int* bsrSortedRowPtrC, int* bsrSortedColIndC, int rowBlockDimC, int colBlockDimC, void* pBuffer);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseCgebsr2gebsr(hipsparseHandle_t handle, hipsparseDirection_t dirA, int mb, int nb, int nnzb, const hipsparseMatDescr_t descrA, const hipComplex* bsrValA, const int* bsrRowPtrA, const int* bsrColIndA, int rowBlockDimA, int colBlockDimA, const hipsparseMatDescr_t descrC, hipComplex* bsrValC, int* bsrRowPtrC, int* bsrColIndC, int rowBlockDimC, int colBlockDimC, void* buffer);
  // CHECK: status_t = hipsparseCgebsr2gebsr(handle_t, direction_t, mb, nb, nnzb, matDescr_t, &complex, &bsrRowPtrA, &bsrColIndA, rowBlockDimA, colBlockDimA, matDescr_C, &complexbsrSortedValC, &bsrSortedRowPtrC, &bsrSortedColIndC, rowBlockDimC, colBlockDimC, tempBuffer);
  status_t = hipsparseCgebsr2gebsr(handle_t, direction_t, mb, nb, nnzb, matDescr_t, &complex, &bsrRowPtrA, &bsrColIndA, rowBlockDimA, colBlockDimA, matDescr_C, &complexbsrSortedValC, &bsrSortedRowPtrC, &bsrSortedColIndC, rowBlockDimC, colBlockDimC, tempBuffer);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseCgebsr2gebsr_bufferSize(cusparseHandle_t handle, cusparseDirection_t dirA, int mb, int nb, int nnzb, const cusparseMatDescr_t descrA, const cuComplex* bsrSortedValA, const int* bsrSortedRowPtrA, const int* bsrSortedColIndA, int rowBlockDimA, int colBlockDimA, int rowBlockDimC, int colBlockDimC, int* pBufferSizeInBytes);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseCgebsr2gebsr_bufferSize(hipsparseHandle_t handle, hipsparseDirection_t dirA, int mb, int nb, int nnzb, const hipsparseMatDescr_t descrA, const hipComplex* bsrValA, const int* bsrRowPtrA, const int* bsrColIndA, int rowBlockDimA, int colBlockDimA, int rowBlockDimC, int colBlockDimC, int* bufferSize);
  // CHECK: status_t = hipsparseCgebsr2gebsr_bufferSize(handle_t, direction_t, mb, nb, nnzb, matDescr_t, &complex, &bsrRowPtrA, &bsrColIndA, rowBlockDimA, colBlockDimA, rowBlockDimC, colBlockDimC, &bufferSizeInBytes);
  status_t = hipsparseCgebsr2gebsr_bufferSize(handle_t, direction_t, mb, nb, nnzb, matDescr_t, &complex, &bsrRowPtrA, &bsrColIndA, rowBlockDimA, colBlockDimA, rowBlockDimC, colBlockDimC, &bufferSizeInBytes);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseDgebsr2gebsr(cusparseHandle_t handle, cusparseDirection_t dirA, int mb, int nb, int nnzb, const cusparseMatDescr_t descrA, const double* bsrSortedValA, const int* bsrSortedRowPtrA, const int* bsrSortedColIndA, int rowBlockDimA, int colBlockDimA, const cusparseMatDescr_t descrC, double* bsrSortedValC, int* bsrSortedRowPtrC, int* bsrSortedColIndC, int rowBlockDimC, int colBlockDimC, void* pBuffer);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseDgebsr2gebsr(hipsparseHandle_t handle, hipsparseDirection_t dirA, int mb, int nb, int nnzb, const hipsparseMatDescr_t descrA, const double* bsrValA, const int* bsrRowPtrA, const int* bsrColIndA, int rowBlockDimA, int colBlockDimA, const hipsparseMatDescr_t descrC, double* bsrValC, int* bsrRowPtrC, int* bsrColIndC, int rowBlockDimC, int colBlockDimC, void* buffer);
  // CHECK: status_t = hipsparseDgebsr2gebsr(handle_t, direction_t, mb, nb, nnzb, matDescr_t, &dbsrSortedValA, &bsrRowPtrA, &bsrColIndA, rowBlockDimA, colBlockDimA, matDescr_C, &dbsrSortedValC, &bsrSortedRowPtrC, &bsrSortedColIndC, rowBlockDimC, colBlockDimC, tempBuffer);
  status_t = hipsparseDgebsr2gebsr(handle_t, direction_t, mb, nb, nnzb, matDescr_t, &dbsrSortedValA, &bsrRowPtrA, &bsrColIndA, rowBlockDimA, colBlockDimA, matDescr_C, &dbsrSortedValC, &bsrSortedRowPtrC, &bsrSortedColIndC, rowBlockDimC, colBlockDimC, tempBuffer);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseDgebsr2gebsr_bufferSize(cusparseHandle_t handle, cusparseDirection_t dirA, int mb, int nb, int nnzb, const cusparseMatDescr_t descrA, const double* bsrSortedValA, const int* bsrSortedRowPtrA, const int* bsrSortedColIndA, int rowBlockDimA, int colBlockDimA, int rowBlockDimC, int colBlockDimC, int* pBufferSizeInBytes);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseDgebsr2gebsr_bufferSize(hipsparseHandle_t handle, hipsparseDirection_t dirA, int mb, int nb, int nnzb, const hipsparseMatDescr_t descrA, const double* bsrValA, const int* bsrRowPtrA, const int* bsrColIndA, int rowBlockDimA, int colBlockDimA, int rowBlockDimC, int colBlockDimC, int* bufferSize);
  // CHECK: status_t = hipsparseDgebsr2gebsr_bufferSize(handle_t, direction_t, mb, nb, nnzb, matDescr_t, &dbsrSortedValA, &bsrRowPtrA, &bsrColIndA, rowBlockDimA, colBlockDimA, rowBlockDimC, colBlockDimC, &bufferSizeInBytes);
  status_t = hipsparseDgebsr2gebsr_bufferSize(handle_t, direction_t, mb, nb, nnzb, matDescr_t, &dbsrSortedValA, &bsrRowPtrA, &bsrColIndA, rowBlockDimA, colBlockDimA, rowBlockDimC, colBlockDimC, &bufferSizeInBytes);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseSgebsr2gebsr(cusparseHandle_t handle, cusparseDirection_t dirA, int mb, int nb, int nnzb, const cusparseMatDescr_t descrA, const float* bsrSortedValA, const int* bsrSortedRowPtrA, const int* bsrSortedColIndA, int rowBlockDimA, int colBlockDimA, const cusparseMatDescr_t descrC, float* bsrSortedValC, int* bsrSortedRowPtrC, int* bsrSortedColIndC, int rowBlockDimC, int colBlockDimC, void* pBuffer);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseSgebsr2gebsr(hipsparseHandle_t handle, hipsparseDirection_t dirA, int mb, int nb, int nnzb, const hipsparseMatDescr_t descrA, const float* bsrValA, const int* bsrRowPtrA, const int* bsrColIndA, int rowBlockDimA, int colBlockDimA, const hipsparseMatDescr_t descrC, float* bsrValC, int* bsrRowPtrC, int* bsrColIndC, int rowBlockDimC, int colBlockDimC, void* buffer);
  // CHECK: status_t = hipsparseSgebsr2gebsr(handle_t, direction_t, mb, nb, nnzb, matDescr_t, &fbsrSortedValA, &bsrRowPtrA, &bsrColIndA, rowBlockDimA, colBlockDimA, matDescr_C, &fbsrSortedValC, &bsrSortedRowPtrC, &bsrSortedColIndC, rowBlockDimC, colBlockDimC, tempBuffer);
  status_t = hipsparseSgebsr2gebsr(handle_t, direction_t, mb, nb, nnzb, matDescr_t, &fbsrSortedValA, &bsrRowPtrA, &bsrColIndA, rowBlockDimA, colBlockDimA, matDescr_C, &fbsrSortedValC, &bsrSortedRowPtrC, &bsrSortedColIndC, rowBlockDimC, colBlockDimC, tempBuffer);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseSgebsr2gebsr_bufferSize(cusparseHandle_t handle, cusparseDirection_t dirA, int mb, int nb, int nnzb, const cusparseMatDescr_t descrA, const float* bsrSortedValA, const int* bsrSortedRowPtrA, const int* bsrSortedColIndA, int rowBlockDimA, int colBlockDimA, int rowBlockDimC, int colBlockDimC, int* pBufferSizeInBytes);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseSgebsr2gebsr_bufferSize(hipsparseHandle_t handle, hipsparseDirection_t dirA, int mb, int nb, int nnzb, const hipsparseMatDescr_t descrA, const float* bsrValA, const int* bsrRowPtrA, const int* bsrColIndA, int rowBlockDimA, int colBlockDimA, int rowBlockDimC, int colBlockDimC, int* bufferSize);
  // CHECK: status_t = hipsparseSgebsr2gebsr_bufferSize(handle_t, direction_t, mb, nb, nnzb, matDescr_t, &fbsrSortedValA, &bsrRowPtrA, &bsrColIndA, rowBlockDimA, colBlockDimA, rowBlockDimC, colBlockDimC, &bufferSizeInBytes);
  status_t = hipsparseSgebsr2gebsr_bufferSize(handle_t, direction_t, mb, nb, nnzb, matDescr_t, &fbsrSortedValA, &bsrRowPtrA, &bsrColIndA, rowBlockDimA, colBlockDimA, rowBlockDimC, colBlockDimC, &bufferSizeInBytes);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseXgebsr2gebsrNnz(cusparseHandle_t handle, cusparseDirection_t dirA, int mb, int nb, int nnzb, const cusparseMatDescr_t descrA, const int* bsrSortedRowPtrA, const int* bsrSortedColIndA, int rowBlockDimA, int colBlockDimA, const cusparseMatDescr_t descrC, int* bsrSortedRowPtrC, int rowBlockDimC, int colBlockDimC, int* nnzTotalDevHostPtr, void* pBuffer);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseXgebsr2gebsrNnz(hipsparseHandle_t handle, hipsparseDirection_t dirA, int mb, int nb, int nnzb, const hipsparseMatDescr_t descrA, const int* bsrRowPtrA, const int* bsrColIndA, int rowBlockDimA, int colBlockDimA, const hipsparseMatDescr_t descrC, int* bsrRowPtrC, int rowBlockDimC, int colBlockDimC, int* nnzTotalDevHostPtr, void* buffer);
  // CHECK: status_t = hipsparseXgebsr2gebsrNnz(handle_t, direction_t, mb, nb, nnzb, matDescr_t, &bsrRowPtrA, &bsrColIndA, rowBlockDimA, colBlockDimA, matDescr_C, &bsrSortedRowPtrC, rowBlockDimC, colBlockDimC, &nnzTotalDevHostPtr, &tempBuffer);
  status_t = hipsparseXgebsr2gebsrNnz(handle_t, direction_t, mb, nb, nnzb, matDescr_t, &bsrRowPtrA, &bsrColIndA, rowBlockDimA, colBlockDimA, matDescr_C, &bsrSortedRowPtrC, rowBlockDimC, colBlockDimC, &nnzTotalDevHostPtr, &tempBuffer);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseZgebsr2csr(cusparseHandle_t handle, cusparseDirection_t dirA, int mb, int nb, const cusparseMatDescr_t descrA, const cuDoubleComplex* bsrSortedValA, const int* bsrSortedRowPtrA, const int* bsrSortedColIndA, int rowBlockDim, int colBlockDim, const cusparseMatDescr_t descrC, cuDoubleComplex* csrSortedValC, int* csrSortedRowPtrC, int* csrSortedColIndC);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseZgebsr2csr(hipsparseHandle_t handle, hipsparseDirection_t dirA, int mb, int nb, const hipsparseMatDescr_t descrA, const hipDoubleComplex* bsrValA, const int* bsrRowPtrA, const int* bsrColIndA, int rowBlockDim, int colBlockDim, const hipsparseMatDescr_t descrC, hipDoubleComplex* csrValC, int* csrRowPtrC, int* csrColIndC);
  // CHECK: status_t = hipsparseZgebsr2csr(handle_t, direction_t, mb, nb, matDescr_A, &dComplexbsrSortedValA, &bsrRowPtrA, &bsrColIndA, rowBlockDimA, colBlockDimA, matDescr_C, &dComplexbsrSortedValC, &csrRowPtrC, &csrColIndC);
  status_t = hipsparseZgebsr2csr(handle_t, direction_t, mb, nb, matDescr_A, &dComplexbsrSortedValA, &bsrRowPtrA, &bsrColIndA, rowBlockDimA, colBlockDimA, matDescr_C, &dComplexbsrSortedValC, &csrRowPtrC, &csrColIndC);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseCgebsr2csr(cusparseHandle_t handle, cusparseDirection_t dirA, int mb, int nb, const cusparseMatDescr_t descrA, const cuComplex* bsrSortedValA, const int* bsrSortedRowPtrA, const int* bsrSortedColIndA, int rowBlockDim, int colBlockDim, const cusparseMatDescr_t descrC, cuComplex* csrSortedValC, int* csrSortedRowPtrC, int* csrSortedColIndC);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseCgebsr2csr(hipsparseHandle_t handle, hipsparseDirection_t dirA, int mb, int nb, const hipsparseMatDescr_t descrA, const hipComplex* bsrValA, const int* bsrRowPtrA, const int* bsrColIndA, int rowBlockDim, int colBlockDim, const hipsparseMatDescr_t descrC, hipComplex* csrValC, int* csrRowPtrC, int* csrColIndC);
  // CHECK: status_t = hipsparseCgebsr2csr(handle_t, direction_t, mb, nb, matDescr_A, &complexbsrValA, &bsrRowPtrA, &bsrColIndA, rowBlockDimA, colBlockDimA, matDescr_C, &complexbsrSortedValC, &csrRowPtrC, &csrColIndC);
  status_t = hipsparseCgebsr2csr(handle_t, direction_t, mb, nb, matDescr_A, &complexbsrValA, &bsrRowPtrA, &bsrColIndA, rowBlockDimA, colBlockDimA, matDescr_C, &complexbsrSortedValC, &csrRowPtrC, &csrColIndC);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseDgebsr2csr(cusparseHandle_t handle, cusparseDirection_t dirA, int mb, int nb, const cusparseMatDescr_t descrA, const double* bsrSortedValA, const int* bsrSortedRowPtrA, const int* bsrSortedColIndA, int rowBlockDim, int colBlockDim, const cusparseMatDescr_t descrC, double* csrSortedValC, int* csrSortedRowPtrC, int* csrSortedColIndC);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseDgebsr2csr(hipsparseHandle_t handle, hipsparseDirection_t dirA, int mb, int nb, const hipsparseMatDescr_t descrA, const double* bsrValA, const int* bsrRowPtrA, const int* bsrColIndA, int rowBlockDim, int colBlockDim, const hipsparseMatDescr_t descrC, double* csrValC, int* csrRowPtrC, int* csrColIndC);
  // CHECK: status_t = hipsparseDgebsr2csr(handle_t, direction_t, mb, nb, matDescr_A, &bsrValA, &bsrRowPtrA, &bsrColIndA, rowBlockDimA, colBlockDimA, matDescr_C, &csrValC, &csrRowPtrC, &csrColIndC);
  status_t = hipsparseDgebsr2csr(handle_t, direction_t, mb, nb, matDescr_A, &bsrValA, &bsrRowPtrA, &bsrColIndA, rowBlockDimA, colBlockDimA, matDescr_C, &csrValC, &csrRowPtrC, &csrColIndC);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseSgebsr2csr(cusparseHandle_t handle, cusparseDirection_t dirA, int mb, int nb, const cusparseMatDescr_t descrA, const float* bsrSortedValA, const int* bsrSortedRowPtrA, const int* bsrSortedColIndA, int rowBlockDim, int colBlockDim, const cusparseMatDescr_t descrC, float* csrSortedValC, int* csrSortedRowPtrC, int* csrSortedColIndC);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseSgebsr2csr(hipsparseHandle_t handle, hipsparseDirection_t dirA, int mb, int nb, const hipsparseMatDescr_t descrA, const float* bsrValA, const int* bsrRowPtrA, const int* bsrColIndA, int rowBlockDim, int colBlockDim, const hipsparseMatDescr_t descrC, float* csrValC, int* csrRowPtrC, int* csrColIndC);
  // CHECK: status_t = hipsparseSgebsr2csr(handle_t, direction_t, mb, nb, matDescr_A, &fbsrSortedValA, &bsrRowPtrA, &bsrColIndA, rowBlockDimA, colBlockDimA, matDescr_C, &fcsrSortedValC, &csrRowPtrC, &csrColIndC);
  status_t = hipsparseSgebsr2csr(handle_t, direction_t, mb, nb, matDescr_A, &fbsrSortedValA, &bsrRowPtrA, &bsrColIndA, rowBlockDimA, colBlockDimA, matDescr_C, &fcsrSortedValC, &csrRowPtrC, &csrColIndC);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseZbsr2csr(cusparseHandle_t handle, cusparseDirection_t dirA, int mb, int nb, const cusparseMatDescr_t descrA, const cuDoubleComplex* bsrSortedValA, const int* bsrSortedRowPtrA, const int* bsrSortedColIndA, int blockDim, const cusparseMatDescr_t descrC, cuDoubleComplex* csrSortedValC, int* csrSortedRowPtrC, int* csrSortedColIndC);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseZbsr2csr(hipsparseHandle_t handle, hipsparseDirection_t dirA, int mb, int nb, const hipsparseMatDescr_t descrA, const hipDoubleComplex* bsrValA, const int* bsrRowPtrA, const int* bsrColIndA, int blockDim, const hipsparseMatDescr_t descrC, hipDoubleComplex* csrValC, int* csrRowPtrC, int* csrColIndC);
  // CHECK: status_t = hipsparseZbsr2csr(handle_t, direction_t, mb, nb, matDescr_A, &dComplexbsrSortedValA, &bsrRowPtrA, &bsrColIndA, blockDim, matDescr_C, &dComplexbsrSortedValC, &csrRowPtrC, &csrColIndC);
  status_t = hipsparseZbsr2csr(handle_t, direction_t, mb, nb, matDescr_A, &dComplexbsrSortedValA, &bsrRowPtrA, &bsrColIndA, blockDim, matDescr_C, &dComplexbsrSortedValC, &csrRowPtrC, &csrColIndC);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseCbsr2csr(cusparseHandle_t handle, cusparseDirection_t dirA, int mb, int nb, const cusparseMatDescr_t descrA, const cuComplex* bsrSortedValA, const int* bsrSortedRowPtrA, const int* bsrSortedColIndA, int blockDim, const cusparseMatDescr_t descrC, cuComplex* csrSortedValC, int* csrSortedRowPtrC, int* csrSortedColIndC);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseCbsr2csr(hipsparseHandle_t handle, hipsparseDirection_t dirA, int mb, int nb, const hipsparseMatDescr_t descrA, const hipComplex* bsrValA, const int* bsrRowPtrA, const int* bsrColIndA, int blockDim, const hipsparseMatDescr_t descrC, hipComplex* csrValC, int* csrRowPtrC, int* csrColIndC);
  // CHECK: status_t = hipsparseCbsr2csr(handle_t, direction_t, mb, nb, matDescr_A, &complexbsrValA, &bsrRowPtrA, &bsrColIndA, blockDim, matDescr_C, &complexbsrSortedValC, &csrRowPtrC, &csrColIndC);
  status_t = hipsparseCbsr2csr(handle_t, direction_t, mb, nb, matDescr_A, &complexbsrValA, &bsrRowPtrA, &bsrColIndA, blockDim, matDescr_C, &complexbsrSortedValC, &csrRowPtrC, &csrColIndC);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseDbsr2csr(cusparseHandle_t handle, cusparseDirection_t dirA, int mb, int nb, const cusparseMatDescr_t descrA, const double* bsrSortedValA, const int* bsrSortedRowPtrA, const int* bsrSortedColIndA, int blockDim, const cusparseMatDescr_t descrC, double* csrSortedValC, int* csrSortedRowPtrC, int* csrSortedColIndC);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseDbsr2csr(hipsparseHandle_t handle, hipsparseDirection_t dirA, int mb, int nb, const hipsparseMatDescr_t descrA, const double* bsrValA, const int* bsrRowPtrA, const int* bsrColIndA, int blockDim, const hipsparseMatDescr_t descrC, double* csrValC, int* csrRowPtrC, int* csrColIndC);
  // CHECK: status_t = hipsparseDbsr2csr(handle_t, direction_t, mb, nb, matDescr_A, &bsrValA, &bsrRowPtrA, &bsrColIndA, blockDim, matDescr_C, &csrValC, &csrRowPtrC, &csrColIndC);
  status_t = hipsparseDbsr2csr(handle_t, direction_t, mb, nb, matDescr_A, &bsrValA, &bsrRowPtrA, &bsrColIndA, blockDim, matDescr_C, &csrValC, &csrRowPtrC, &csrColIndC);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseSbsr2csr(cusparseHandle_t handle, cusparseDirection_t dirA, int mb, int nb, const cusparseMatDescr_t descrA, const float* bsrSortedValA, const int* bsrSortedRowPtrA, const int* bsrSortedColIndA, int blockDim, const cusparseMatDescr_t descrC, float* csrSortedValC, int* csrSortedRowPtrC, int* csrSortedColIndC);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseSbsr2csr(hipsparseHandle_t handle, hipsparseDirection_t dirA, int mb, int nb, const hipsparseMatDescr_t descrA, const float* bsrValA, const int* bsrRowPtrA, const int* bsrColIndA, int blockDim, const hipsparseMatDescr_t descrC, float* csrValC, int* csrRowPtrC, int* csrColIndC);
  // CHECK: status_t = hipsparseSbsr2csr(handle_t, direction_t, mb, nb, matDescr_A, &fbsrSortedValA, &bsrRowPtrA, &bsrColIndA, blockDim, matDescr_C, &fcsrSortedValC, &csrRowPtrC, &csrColIndC);
  status_t = hipsparseSbsr2csr(handle_t, direction_t, mb, nb, matDescr_A, &fbsrSortedValA, &bsrRowPtrA, &bsrColIndA, blockDim, matDescr_C, &fcsrSortedValC, &csrRowPtrC, &csrColIndC);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseXcoosortByColumn(cusparseHandle_t handle, int m, int n, int nnz, int* cooRowsA, int* cooColsA, int* P, void* pBuffer);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseXcoosortByColumn(hipsparseHandle_t handle, int m, int n, int nnz, int* cooRows, int* cooCols, int* P, void* pBuffer);
  // CHECK: status_t = hipsparseXcoosortByColumn(handle_t, m, n, innz, &icooRows, &icooColumns, P, pBuffer);
  status_t = hipsparseXcoosortByColumn(handle_t, m, n, innz, &icooRows, &icooColumns, P, pBuffer);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseXcoosortByRow(cusparseHandle_t handle, int m, int n, int nnz, int* cooRowsA, int* cooColsA, int* P, void* pBuffer);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseXcoosortByRow(hipsparseHandle_t handle, int m, int n, int nnz, int* cooRows, int* cooCols, int* P, void* pBuffer);
  // CHECK: status_t = hipsparseXcoosortByRow(handle_t, m, n, innz, &icooRows, &icooColumns, P, pBuffer);
  status_t = hipsparseXcoosortByRow(handle_t, m, n, innz, &icooRows, &icooColumns, P, pBuffer);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseXcoosort_bufferSizeExt(cusparseHandle_t handle, int m, int n, int nnz, const int* cooRowsA, const int* cooColsA, size_t* pBufferSizeInBytes);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseXcoosort_bufferSizeExt(hipsparseHandle_t handle, int m, int n, int nnz, const int* cooRows, const int* cooCols, size_t* pBufferSizeInBytes);
  // CHECK: status_t = hipsparseXcoosort_bufferSizeExt(handle_t, m, n, innz, &icooRows, &icooColumns, &bufferSize);
  status_t = hipsparseXcoosort_bufferSizeExt(handle_t, m, n, innz, &icooRows, &icooColumns, &bufferSize);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseXcscsort(cusparseHandle_t handle, int m, int n, int nnz, const cusparseMatDescr_t descrA, const int* cscColPtrA, int* cscRowIndA, int* P, void* pBuffer);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseXcscsort(hipsparseHandle_t handle, int m, int n, int nnz, const hipsparseMatDescr_t descrA, const int* cscColPtr, int* cscRowInd, int* P, void* pBuffer);
  // CHECK: status_t = hipsparseXcscsort(handle_t, m, n, innz, matDescr_A, &cscColPtrA, &cscRowIndA, P, pBuffer);
  status_t = hipsparseXcscsort(handle_t, m, n, innz, matDescr_A, &cscColPtrA, &cscRowIndA, P, pBuffer);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseXcscsort_bufferSizeExt(cusparseHandle_t handle, int m, int n, int nnz, const int* cscColPtrA, const int* cscRowIndA, size_t* pBufferSizeInBytes);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseXcscsort_bufferSizeExt(hipsparseHandle_t handle, int m, int n, int nnz, const int* cscColPtr, const int* cscRowInd, size_t* pBufferSizeInBytes);
  // CHECK: status_t = hipsparseXcscsort_bufferSizeExt(handle_t, m, n, innz, &cscColPtrA, &cscRowIndA, &bufferSize);
  status_t = hipsparseXcscsort_bufferSizeExt(handle_t, m, n, innz, &cscColPtrA, &cscRowIndA, &bufferSize);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseXcsrsort(cusparseHandle_t handle, int m, int n, int nnz, const cusparseMatDescr_t descrA, const int* csrRowPtrA, int* csrColIndA, int* P, void* pBuffer);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseXcsrsort(hipsparseHandle_t handle, int m, int n, int nnz, const hipsparseMatDescr_t descrA, const int* csrRowPtr, int* csrColInd, int* P, void* pBuffer);
  // CHECK: status_t = hipsparseXcsrsort(handle_t, m, n, innz, matDescr_A, &cscRowIndA, &cscColPtrA, P, pBuffer);
  status_t = hipsparseXcsrsort(handle_t, m, n, innz, matDescr_A, &cscRowIndA, &cscColPtrA, P, pBuffer);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseXcsrsort_bufferSizeExt(cusparseHandle_t handle, int m, int n, int nnz, const int* csrRowPtrA, const int* csrColIndA, size_t* pBufferSizeInBytes);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseXcsrsort_bufferSizeExt(hipsparseHandle_t handle, int m, int n, int nnz, const int* csrRowPtr, const int* csrColInd, size_t* pBufferSizeInBytes);
  // CHECK: status_t = hipsparseXcsrsort_bufferSizeExt(handle_t, m, n, innz, &cscRowIndA, &cscColPtrA, &bufferSize);
  status_t = hipsparseXcsrsort_bufferSizeExt(handle_t, m, n, innz, &cscRowIndA, &cscColPtrA, &bufferSize);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseCreateIdentityPermutation(cusparseHandle_t handle, int n, int* p);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseCreateIdentityPermutation(hipsparseHandle_t handle, int n, int* p);
  // CHECK: status_t = hipsparseCreateIdentityPermutation(handle_t, n, P);
  status_t = hipsparseCreateIdentityPermutation(handle_t, n, P);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseXcoo2csr(cusparseHandle_t handle, const int* cooRowInd, int nnz, int m, int* csrSortedRowPtr, cusparseIndexBase_t idxBase);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseXcoo2csr(hipsparseHandle_t handle, const int* cooRowInd, int nnz, int m, int* csrRowPtr, hipsparseIndexBase_t idxBase);
  // CHECK: status_t = hipsparseXcoo2csr(handle_t, &icooRowInd, nnz, m, &csrRowPtrA, indexBase_t);
  status_t = hipsparseXcoo2csr(handle_t, &icooRowInd, nnz, m, &csrRowPtrA, indexBase_t);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseZcsr2gebsr(cusparseHandle_t handle, cusparseDirection_t dirA, int m, int n, const cusparseMatDescr_t descrA, const cuDoubleComplex* csrSortedValA, const int* csrSortedRowPtrA, const int* csrSortedColIndA, const cusparseMatDescr_t descrC, cuDoubleComplex* bsrSortedValC, int* bsrSortedRowPtrC, int* bsrSortedColIndC, int rowBlockDim, int colBlockDim, void* pBuffer);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseZcsr2gebsr(hipsparseHandle_t handle, hipsparseDirection_t dir, int m, int n, const hipsparseMatDescr_t csr_descr, const hipDoubleComplex* csr_val, const int* csr_row_ptr, const int* csr_col_ind, const hipsparseMatDescr_t bsr_descr, hipDoubleComplex* bsr_val, int* bsr_row_ptr, int* bsr_col_ind, int row_block_dim, int col_block_dim, void* p_buffer);
  // CHECK: status_t = hipsparseZcsr2gebsr(handle_t, direction_t, m, n, matDescr_A, &dcomplex, &csrRowPtrA, &csrColIndA, matDescr_C, &dComplexcsrSortedValC, &bsrSortedRowPtrC, &bsrSortedColIndC, rowBlockDimA, colBlockDimA, pBuffer);
 status_t = hipsparseZcsr2gebsr(handle_t, direction_t, m, n, matDescr_A, &dcomplex, &csrRowPtrA, &csrColIndA, matDescr_C, &dComplexcsrSortedValC, &bsrSortedRowPtrC, &bsrSortedColIndC, rowBlockDimA, colBlockDimA, pBuffer);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseCcsr2gebsr(cusparseHandle_t handle, cusparseDirection_t dirA, int m, int n, const cusparseMatDescr_t descrA, const cuComplex* csrSortedValA, const int* csrSortedRowPtrA, const int* csrSortedColIndA, const cusparseMatDescr_t descrC, cuComplex* bsrSortedValC, int* bsrSortedRowPtrC, int* bsrSortedColIndC, int rowBlockDim, int colBlockDim, void* pBuffer);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseCcsr2gebsr(hipsparseHandle_t handle, hipsparseDirection_t dir, int m, int n, const hipsparseMatDescr_t csr_descr, const hipComplex* csr_val, const int* csr_row_ptr, const int* csr_col_ind, const hipsparseMatDescr_t bsr_descr, hipComplex* bsr_val, int* bsr_row_ptr, int* bsr_col_ind, int row_block_dim, int col_block_dim, void* p_buffer);
  // CHECK: status_t = hipsparseCcsr2gebsr(handle_t, direction_t, m, n, matDescr_A, &complexcsrSortedValA, &csrRowPtrA, &csrColIndA, matDescr_C, &complexcsrSortedValC, &bsrSortedRowPtrC, &bsrSortedColIndC, rowBlockDimA, colBlockDimA, pBuffer);
  status_t = hipsparseCcsr2gebsr(handle_t, direction_t, m, n, matDescr_A, &complexcsrSortedValA, &csrRowPtrA, &csrColIndA, matDescr_C, &complexcsrSortedValC, &bsrSortedRowPtrC, &bsrSortedColIndC, rowBlockDimA, colBlockDimA, pBuffer);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseDcsr2gebsr(cusparseHandle_t handle, cusparseDirection_t dirA, int m, int n, const cusparseMatDescr_t descrA, const double* csrSortedValA, const int* csrSortedRowPtrA, const int* csrSortedColIndA, const cusparseMatDescr_t descrC, double* bsrSortedValC, int* bsrSortedRowPtrC, int* bsrSortedColIndC, int rowBlockDim, int colBlockDim, void* pBuffer);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseDcsr2gebsr(hipsparseHandle_t handle, hipsparseDirection_t dir, int m, int n, const hipsparseMatDescr_t csr_descr, const double* csr_val, const int* csr_row_ptr, const int* csr_col_ind, const hipsparseMatDescr_t bsr_descr, double* bsr_val, int* bsr_row_ptr, int* bsr_col_ind, int row_block_dim, int col_block_dim, void* p_buffer);
  // CHECK: status_t = hipsparseDcsr2gebsr(handle_t, direction_t, m, n, matDescr_A, &dcsrSortedValA, &csrRowPtrA, &csrColIndA, matDescr_C, &dcsrSortedValC, &bsrSortedRowPtrC, &bsrSortedColIndC, rowBlockDimA, colBlockDimA, pBuffer);
  status_t = hipsparseDcsr2gebsr(handle_t, direction_t, m, n, matDescr_A, &dcsrSortedValA, &csrRowPtrA, &csrColIndA, matDescr_C, &dcsrSortedValC, &bsrSortedRowPtrC, &bsrSortedColIndC, rowBlockDimA, colBlockDimA, pBuffer);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseScsr2gebsr(cusparseHandle_t handle, cusparseDirection_t dirA, int m, int n, const cusparseMatDescr_t descrA, const float* csrSortedValA, const int* csrSortedRowPtrA, const int* csrSortedColIndA, const cusparseMatDescr_t descrC, float* bsrSortedValC, int* bsrSortedRowPtrC, int* bsrSortedColIndC, int rowBlockDim, int colBlockDim, void* pBuffer);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseScsr2gebsr(hipsparseHandle_t handle, hipsparseDirection_t dir, int m, int n, const hipsparseMatDescr_t csr_descr, const float* csr_val, const int* csr_row_ptr, const int* csr_col_ind, const hipsparseMatDescr_t bsr_descr, float* bsr_val, int* bsr_row_ptr, int* bsr_col_ind, int row_block_dim, int col_block_dim, void* p_buffer);
  // CHECK: status_t = hipsparseScsr2gebsr(handle_t, direction_t, m, n, matDescr_A, &csrSortedValA, &csrRowPtrA, &csrColIndA, matDescr_C, &fcsrSortedValC, &bsrSortedRowPtrC, &bsrSortedColIndC, rowBlockDimA, colBlockDimA, pBuffer);
  status_t = hipsparseScsr2gebsr(handle_t, direction_t, m, n, matDescr_A, &csrSortedValA, &csrRowPtrA, &csrColIndA, matDescr_C, &fcsrSortedValC, &bsrSortedRowPtrC, &bsrSortedColIndC, rowBlockDimA, colBlockDimA, pBuffer);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseXcsr2gebsrNnz(cusparseHandle_t handle, cusparseDirection_t dirA, int m, int n, const cusparseMatDescr_t descrA, const int* csrSortedRowPtrA, const int* csrSortedColIndA, const cusparseMatDescr_t descrC, int* bsrSortedRowPtrC, int rowBlockDim, int colBlockDim, int* nnzTotalDevHostPtr, void* pBuffer);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseXcsr2gebsrNnz(hipsparseHandle_t handle, hipsparseDirection_t dir, int m, int n, const hipsparseMatDescr_t csr_descr, const int* csr_row_ptr, const int* csr_col_ind, const hipsparseMatDescr_t bsr_descr, int* bsr_row_ptr, int row_block_dim, int col_block_dim, int* bsr_nnz_devhost, void* p_buffer);
  // CHECK: status_t = hipsparseXcsr2gebsrNnz(handle_t, direction_t, m, n, matDescr_A, &csrRowPtrA, &csrColIndA, matDescr_C, &bsrSortedRowPtrC, rowBlockDimA, colBlockDimA, &nnzTotalDevHostPtr, pBuffer);
  status_t = hipsparseXcsr2gebsrNnz(handle_t, direction_t, m, n, matDescr_A, &csrRowPtrA, &csrColIndA, matDescr_C, &bsrSortedRowPtrC, rowBlockDimA, colBlockDimA, &nnzTotalDevHostPtr, pBuffer);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseZcsr2gebsr_bufferSize(cusparseHandle_t handle, cusparseDirection_t dirA, int m, int n, const cusparseMatDescr_t descrA, const cuDoubleComplex* csrSortedValA, const int* csrSortedRowPtrA, const int* csrSortedColIndA, int rowBlockDim, int colBlockDim, int* pBufferSizeInBytes);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseZcsr2gebsr_bufferSize(hipsparseHandle_t handle, hipsparseDirection_t dir, int m, int n, const hipsparseMatDescr_t csr_descr, const hipDoubleComplex* csr_val, const int* csr_row_ptr, const int* csr_col_ind, int row_block_dim, int col_block_dim, size_t* p_buffer_size);
  // CHECK: status_t = hipsparseZcsr2gebsr_bufferSize(handle_t, direction_t, m, n, matDescr_A, &dComplexcsrSortedValA, &csrRowPtrA, &csrColIndA, rowBlockDimA, colBlockDimA, &bufferSizeInBytes);
  status_t = hipsparseZcsr2gebsr_bufferSize(handle_t, direction_t, m, n, matDescr_A, &dComplexcsrSortedValA, &csrRowPtrA, &csrColIndA, rowBlockDimA, colBlockDimA, &bufferSizeInBytes);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseCcsr2gebsr_bufferSize(cusparseHandle_t handle, cusparseDirection_t dirA, int m, int n, const cusparseMatDescr_t descrA, const cuComplex* csrSortedValA, const int* csrSortedRowPtrA, const int* csrSortedColIndA, int rowBlockDim, int colBlockDim, int* pBufferSizeInBytes);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseCcsr2gebsr_bufferSize(hipsparseHandle_t handle, hipsparseDirection_t dir, int m, int n, const hipsparseMatDescr_t csr_descr, const hipComplex* csr_val, const int* csr_row_ptr, const int* csr_col_ind, int row_block_dim, int col_block_dim, size_t* p_buffer_size);
 // CHECK: status_t = hipsparseCcsr2gebsr_bufferSize(handle_t, direction_t, m, n, matDescr_A, &complexcsrSortedValA, &csrRowPtrA, &csrColIndA, rowBlockDimA, colBlockDimA, &bufferSizeInBytes);
  status_t = hipsparseCcsr2gebsr_bufferSize(handle_t, direction_t, m, n, matDescr_A, &complexcsrSortedValA, &csrRowPtrA, &csrColIndA, rowBlockDimA, colBlockDimA, &bufferSizeInBytes);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseDcsr2gebsr_bufferSize(cusparseHandle_t handle, cusparseDirection_t dirA, int m, int n, const cusparseMatDescr_t descrA, const double* csrSortedValA, const int* csrSortedRowPtrA, const int* csrSortedColIndA, int rowBlockDim, int colBlockDim, int* pBufferSizeInBytes);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseDcsr2gebsr_bufferSize(hipsparseHandle_t handle, hipsparseDirection_t dir, int m, int n, const hipsparseMatDescr_t csr_descr, const double* csr_val, const int* csr_row_ptr, const int* csr_col_ind, int row_block_dim, int col_block_dim, size_t* p_buffer_size);
  // CHECK: status_t = hipsparseDcsr2gebsr_bufferSize(handle_t, direction_t, m, n, matDescr_A, &dcsrSortedValA, &csrRowPtrA, &csrColIndA, rowBlockDimA, colBlockDimA, &bufferSizeInBytes);
  status_t = hipsparseDcsr2gebsr_bufferSize(handle_t, direction_t, m, n, matDescr_A, &dcsrSortedValA, &csrRowPtrA, &csrColIndA, rowBlockDimA, colBlockDimA, &bufferSizeInBytes);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseScsr2gebsr_bufferSize(cusparseHandle_t handle, cusparseDirection_t dirA, int m, int n, const cusparseMatDescr_t descrA, const float* csrSortedValA, const int* csrSortedRowPtrA, const int* csrSortedColIndA, int rowBlockDim, int colBlockDim, int* pBufferSizeInBytes);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseScsr2gebsr_bufferSize(hipsparseHandle_t handle, hipsparseDirection_t dir, int m, int n, const hipsparseMatDescr_t csr_descr, const float* csr_val, const int* csr_row_ptr, const int* csr_col_ind, int row_block_dim, int col_block_dim, size_t* p_buffer_size);
  // CHECK: status_t = hipsparseScsr2gebsr_bufferSize(handle_t, direction_t, m, n, matDescr_A, &csrSortedValA, &csrRowPtrA, &csrColIndA, rowBlockDimA, colBlockDimA, &bufferSizeInBytes);
  status_t = hipsparseScsr2gebsr_bufferSize(handle_t, direction_t, m, n, matDescr_A, &csrSortedValA, &csrRowPtrA, &csrColIndA, rowBlockDimA, colBlockDimA, &bufferSizeInBytes);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseZcsr2bsr(cusparseHandle_t handle, cusparseDirection_t dirA, int m, int n, const cusparseMatDescr_t descrA, const cuDoubleComplex* csrSortedValA, const int* csrSortedRowPtrA, const int* csrSortedColIndA, int blockDim, const cusparseMatDescr_t descrC, cuDoubleComplex* bsrSortedValC, int* bsrSortedRowPtrC, int* bsrSortedColIndC);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseZcsr2bsr(hipsparseHandle_t handle, hipsparseDirection_t dirA, int m, int n, const hipsparseMatDescr_t descrA, const hipDoubleComplex* csrValA, const int* csrRowPtrA, const int* csrColIndA, int blockDim, const hipsparseMatDescr_t descrC, hipDoubleComplex* bsrValC, int* bsrRowPtrC, int* bsrColIndC);
  // CHECK: status_t = hipsparseZcsr2bsr(handle_t, direction_t, m, n, matDescr_A, &dComplexcsrSortedValA, &csrRowPtrA, &csrColIndA, blockDim, matDescr_C, &dComplexcsrSortedValC, &bsrSortedRowPtrC, &bsrSortedColIndC);
  status_t = hipsparseZcsr2bsr(handle_t, direction_t, m, n, matDescr_A, &dComplexcsrSortedValA, &csrRowPtrA, &csrColIndA, blockDim, matDescr_C, &dComplexcsrSortedValC, &bsrSortedRowPtrC, &bsrSortedColIndC);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseCcsr2bsr(cusparseHandle_t handle, cusparseDirection_t dirA, int m, int n, const cusparseMatDescr_t descrA, const cuComplex* csrSortedValA, const int* csrSortedRowPtrA, const int* csrSortedColIndA, int blockDim, const cusparseMatDescr_t descrC, cuComplex* bsrSortedValC, int* bsrSortedRowPtrC, int* bsrSortedColIndC);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseCcsr2bsr(hipsparseHandle_t handle, hipsparseDirection_t dirA, int m, int n, const hipsparseMatDescr_t descrA, const hipComplex* csrValA, const int* csrRowPtrA, const int* csrColIndA, int blockDim, const hipsparseMatDescr_t descrC, hipComplex* bsrValC, int* bsrRowPtrC, int* bsrColIndC);
  // CHECK: status_t = hipsparseCcsr2bsr(handle_t, direction_t, m, n, matDescr_A, &complexcsrSortedValA, &csrRowPtrA, &csrColIndA, blockDim, matDescr_C, &complexcsrSortedValC, &bsrSortedRowPtrC, &bsrSortedColIndC);
  status_t = hipsparseCcsr2bsr(handle_t, direction_t, m, n, matDescr_A, &complexcsrSortedValA, &csrRowPtrA, &csrColIndA, blockDim, matDescr_C, &complexcsrSortedValC, &bsrSortedRowPtrC, &bsrSortedColIndC);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseDcsr2bsr(cusparseHandle_t handle, cusparseDirection_t dirA, int m, int n, const cusparseMatDescr_t descrA, const double* csrSortedValA, const int* csrSortedRowPtrA, const int* csrSortedColIndA, int blockDim, const cusparseMatDescr_t descrC, double* bsrSortedValC, int* bsrSortedRowPtrC, int* bsrSortedColIndC);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseDcsr2bsr(hipsparseHandle_t handle, hipsparseDirection_t dirA, int m, int n,const hipsparseMatDescr_t descrA, const double* csrValA, const int* csrRowPtrA, const int* csrColIndA, int blockDim,const hipsparseMatDescr_t descrC, double* bsrValC, int* bsrRowPtrC, int* bsrColIndC);
  // CHECK: status_t = hipsparseDcsr2bsr(handle_t, direction_t, m, n, matDescr_A, &dcsrSortedValA, &csrRowPtrA, &csrColIndA, blockDim, matDescr_C, &dcsrSortedValC, &bsrSortedRowPtrC, &bsrSortedColIndC);
  status_t = hipsparseDcsr2bsr(handle_t, direction_t, m, n, matDescr_A, &dcsrSortedValA, &csrRowPtrA, &csrColIndA, blockDim, matDescr_C, &dcsrSortedValC, &bsrSortedRowPtrC, &bsrSortedColIndC);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseScsr2bsr(cusparseHandle_t handle, cusparseDirection_t dirA, int m, int n, const cusparseMatDescr_t descrA, const float* csrSortedValA, const int* csrSortedRowPtrA, const int* csrSortedColIndA, int blockDim, const cusparseMatDescr_t descrC, float* bsrSortedValC, int* bsrSortedRowPtrC, int* bsrSortedColIndC);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseScsr2bsr(hipsparseHandle_t handle, hipsparseDirection_t dirA, int m, int n, const hipsparseMatDescr_t descrA, const float* csrValA, const int* csrRowPtrA, const int* csrColIndA, int blockDim, const hipsparseMatDescr_t descrC, float* bsrValC, int* bsrRowPtrC, int* bsrColIndC);
  // CHECK: status_t = hipsparseScsr2bsr(handle_t, direction_t, m, n, matDescr_A, &csrSortedValA, &csrRowPtrA, &csrColIndA, blockDim, matDescr_C, &fcsrSortedValC, &bsrSortedRowPtrC, &bsrSortedColIndC);
  status_t = hipsparseScsr2bsr(handle_t, direction_t, m, n, matDescr_A, &csrSortedValA, &csrRowPtrA, &csrColIndA, blockDim, matDescr_C, &fcsrSortedValC, &bsrSortedRowPtrC, &bsrSortedColIndC);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseXcsr2bsrNnz(cusparseHandle_t handle, cusparseDirection_t dirA, int m, int n, const cusparseMatDescr_t descrA, const int* csrSortedRowPtrA, const int* csrSortedColIndA, int blockDim, const cusparseMatDescr_t descrC, int* bsrSortedRowPtrC, int* nnzTotalDevHostPtr);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseXcsr2bsrNnz(hipsparseHandle_t handle, hipsparseDirection_t dirA, int m, int n, const hipsparseMatDescr_t descrA, const int* csrRowPtrA, const int* csrColIndA, int blockDim, const hipsparseMatDescr_t descrC, int* bsrRowPtrC, int* bsrNnzb);
  // CHECK: status_t = hipsparseXcsr2bsrNnz(handle_t, direction_t, m, n, matDescr_A, &csrRowPtrA, &csrColIndA, blockDim, matDescr_C, &bsrSortedRowPtrC, &nnzTotalDevHostPtr);
  status_t = hipsparseXcsr2bsrNnz(handle_t, direction_t, m, n, matDescr_A, &csrRowPtrA, &csrColIndA, blockDim, matDescr_C, &bsrSortedRowPtrC, &nnzTotalDevHostPtr);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseZgebsr2gebsc(cusparseHandle_t handle, int mb, int nb, int nnzb, const cuDoubleComplex* bsrSortedVal, const int* bsrSortedRowPtr, const int* bsrSortedColInd, int rowBlockDim, int colBlockDim, cuDoubleComplex* bscVal, int* bscRowInd, int* bscColPtr, cusparseAction_t copyValues, cusparseIndexBase_t idxBase, void* pBuffer);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseZgebsr2gebsc(hipsparseHandle_t handle, int mb, int nb, int nnzb, const hipDoubleComplex* bsr_val, const int* bsr_row_ptr, const int* bsr_col_ind, int row_block_dim, int col_block_dim, hipDoubleComplex* bsc_val, int* bsc_row_ind, int* bsc_col_ptr, hipsparseAction_t copy_values, hipsparseIndexBase_t idx_base, void* temp_buffer);
  // CHECK: status_t = hipsparseZgebsr2gebsc(handle_t, mb, nb, nnzb, &dComplexbsrSortedVal, &bsrSortedRowPtr, &bsrSortedColInd, rowBlockDim, colBlockDim, &dComplexbscVal, &bscRowInd, &bscColPtr, copyValues, indexBase_t, pBuffer);
  status_t = hipsparseZgebsr2gebsc(handle_t, mb, nb, nnzb, &dComplexbsrSortedVal, &bsrSortedRowPtr, &bsrSortedColInd, rowBlockDim, colBlockDim, &dComplexbscVal, &bscRowInd, &bscColPtr, copyValues, indexBase_t, pBuffer);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseCgebsr2gebsc(cusparseHandle_t handle, int mb, int nb, int nnzb, const cuComplex* bsrSortedVal, const int* bsrSortedRowPtr, const int* bsrSortedColInd, int rowBlockDim, int colBlockDim, cuComplex* bscVal, int* bscRowInd, int* bscColPtr, cusparseAction_t copyValues, cusparseIndexBase_t idxBase, void* pBuffer);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseCgebsr2gebsc(hipsparseHandle_t handle, int mb, int nb, int nnzb, const hipComplex* bsr_val, const int* bsr_row_ptr, const int* bsr_col_ind, int row_block_dim, int col_block_dim, hipComplex* bsc_val, int* bsc_row_ind, int* bsc_col_ptr,hipsparseAction_t copy_values, hipsparseIndexBase_t idx_base, void* temp_buffer);
  // CHECK: status_t = hipsparseCgebsr2gebsc(handle_t, mb, nb, nnzb, &complexbsrSortedVal, &bsrSortedRowPtr, &bsrSortedColInd, rowBlockDim, colBlockDim, &complexbscVal, &bscRowInd, &bscColPtr, copyValues, indexBase_t, pBuffer);
  status_t = hipsparseCgebsr2gebsc(handle_t, mb, nb, nnzb, &complexbsrSortedVal, &bsrSortedRowPtr, &bsrSortedColInd, rowBlockDim, colBlockDim, &complexbscVal, &bscRowInd, &bscColPtr, copyValues, indexBase_t, pBuffer);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseDgebsr2gebsc(cusparseHandle_t handle, int mb, int nb, int nnzb, const double* bsrSortedVal, const int* bsrSortedRowPtr, const int* bsrSortedColInd, int rowBlockDim, int colBlockDim, double* bscVal, int* bscRowInd, int* bscColPtr, cusparseAction_t copyValues, cusparseIndexBase_t idxBase, void* pBuffer);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseDgebsr2gebsc(hipsparseHandle_t handle, int mb, int nb, int nnzb, const double* bsr_val, const int* bsr_row_ptr, const int* bsr_col_ind, int row_block_dim, int col_block_dim, double* bsc_val, int* bsc_row_ind, int* bsc_col_ptr, hipsparseAction_t copy_values, hipsparseIndexBase_t idx_base, void* temp_buffer);
  // CHECK: status_t = hipsparseDgebsr2gebsc(handle_t, mb, nb, nnzb, &dbsrSortedVal, &bsrSortedRowPtr, &bsrSortedColInd, rowBlockDim, colBlockDim, &dbscVal, &bscRowInd, &bscColPtr, copyValues, indexBase_t, pBuffer);
  status_t = hipsparseDgebsr2gebsc(handle_t, mb, nb, nnzb, &dbsrSortedVal, &bsrSortedRowPtr, &bsrSortedColInd, rowBlockDim, colBlockDim, &dbscVal, &bscRowInd, &bscColPtr, copyValues, indexBase_t, pBuffer);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseSgebsr2gebsc(cusparseHandle_t handle, int mb, int nb, int nnzb, const float* bsrSortedVal, const int* bsrSortedRowPtr, const int* bsrSortedColInd, int rowBlockDim, int colBlockDim, float* bscVal, int* bscRowInd, int* bscColPtr, cusparseAction_t copyValues, cusparseIndexBase_t idxBase, void* pBuffer);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseSgebsr2gebsc(hipsparseHandle_t handle, int mb, int nb, int nnzb, const float* bsr_val, const int* bsr_row_ptr, const int* bsr_col_ind, int row_block_dim, int col_block_dim, float* bsc_val, int* bsc_row_ind, int* bsc_col_ptr, hipsparseAction_t copy_values, hipsparseIndexBase_t idx_base, void* temp_buffer);
  // CHECK: status_t = hipsparseSgebsr2gebsc(handle_t, mb, nb, nnzb, &fbsrSortedVal, &bsrSortedRowPtr, &bsrSortedColInd, rowBlockDim, colBlockDim, &fbscVal, &bscRowInd, &bscColPtr, copyValues, indexBase_t, pBuffer);
  status_t = hipsparseSgebsr2gebsc(handle_t, mb, nb, nnzb, &fbsrSortedVal, &bsrSortedRowPtr, &bsrSortedColInd, rowBlockDim, colBlockDim, &fbscVal, &bscRowInd, &bscColPtr, copyValues, indexBase_t, pBuffer);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseZgebsr2gebsc_bufferSize(cusparseHandle_t handle, int mb, int nb, int nnzb, const cuDoubleComplex* bsrSortedVal, const int* bsrSortedRowPtr, const int* bsrSortedColInd, int rowBlockDim, int colBlockDim, int* pBufferSizeInBytes);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseZgebsr2gebsc_bufferSize(hipsparseHandle_t handle, int mb, int nb, int nnzb, const hipDoubleComplex* bsr_val, const int* bsr_row_ptr, const int* bsr_col_ind, int row_block_dim, int col_block_dim, size_t* p_buffer_size);
  // CHECK: status_t = hipsparseZgebsr2gebsc_bufferSize(handle_t, mb, nb, nnzb, &dComplexbsrSortedVal, &bsrSortedRowPtr, &bsrSortedColInd, rowBlockDim, colBlockDim, &bufferSizeInBytes);
  status_t = hipsparseZgebsr2gebsc_bufferSize(handle_t, mb, nb, nnzb, &dComplexbsrSortedVal, &bsrSortedRowPtr, &bsrSortedColInd, rowBlockDim, colBlockDim, &bufferSizeInBytes);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseCgebsr2gebsc_bufferSize(cusparseHandle_t handle, int mb, int nb, int nnzb, const cuComplex* bsrSortedVal, const int* bsrSortedRowPtr, const int* bsrSortedColInd, int rowBlockDim, int colBlockDim, int* pBufferSizeInBytes);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseCgebsr2gebsc_bufferSize(hipsparseHandle_t handle, int mb, int nb, int nnzb, const hipComplex* bsr_val, const int* bsr_row_ptr, const int* bsr_col_ind, int row_block_dim, int col_block_dim, size_t* p_buffer_size);
  // CHECK: status_t = hipsparseCgebsr2gebsc_bufferSize(handle_t, mb, nb, nnzb, &complexbsrSortedVal, &bsrSortedRowPtr, &bsrSortedColInd, rowBlockDim, colBlockDim, &bufferSizeInBytes);
  status_t = hipsparseCgebsr2gebsc_bufferSize(handle_t, mb, nb, nnzb, &complexbsrSortedVal, &bsrSortedRowPtr, &bsrSortedColInd, rowBlockDim, colBlockDim, &bufferSizeInBytes);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseDgebsr2gebsc_bufferSize(cusparseHandle_t handle, int mb, int nb, int nnzb, const double* bsrSortedVal, const int* bsrSortedRowPtr, const int* bsrSortedColInd, int rowBlockDim, int colBlockDim, int* pBufferSizeInBytes);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseDgebsr2gebsc_bufferSize(hipsparseHandle_t handle, int mb, int nb, int nnzb, const double* bsr_val, const int* bsr_row_ptr, const int* bsr_col_ind, int row_block_dim, int col_block_dim, size_t* p_buffer_size);
  // CHECK: status_t = hipsparseDgebsr2gebsc_bufferSize(handle_t, mb, nb, nnzb, &dbsrSortedVal, &bsrSortedRowPtr, &bsrSortedColInd, rowBlockDim, colBlockDim, &bufferSizeInBytes);
  status_t = hipsparseDgebsr2gebsc_bufferSize(handle_t, mb, nb, nnzb, &dbsrSortedVal, &bsrSortedRowPtr, &bsrSortedColInd, rowBlockDim, colBlockDim, &bufferSizeInBytes);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseSgebsr2gebsc_bufferSize(cusparseHandle_t handle, int mb, int nb, int nnzb, const float* bsrSortedVal, const int* bsrSortedRowPtr, const int* bsrSortedColInd, int rowBlockDim, int colBlockDim, int* pBufferSizeInBytes);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseSgebsr2gebsc_bufferSize(hipsparseHandle_t handle, int mb, int nb, int nnzb, const float* bsr_val, const int* bsr_row_ptr, const int* bsr_col_ind, int row_block_dim, int col_block_dim, size_t* p_buffer_size);
  // CHECK: status_t = hipsparseSgebsr2gebsc_bufferSize(handle_t, mb, nb, nnzb, &fbsrSortedVal, &bsrSortedRowPtr, &bsrSortedColInd, rowBlockDim, colBlockDim, &bufferSizeInBytes);
  status_t = hipsparseSgebsr2gebsc_bufferSize(handle_t, mb, nb, nnzb, &fbsrSortedVal, &bsrSortedRowPtr, &bsrSortedColInd, rowBlockDim, colBlockDim, &bufferSizeInBytes);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseXcsr2coo(cusparseHandle_t handle, const int* csrSortedRowPtr, int nnz, int m, int* cooRowInd, cusparseIndexBase_t idxBase);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseXcsr2coo(hipsparseHandle_t handle, const int* csrRowPtr, int nnz, int m, int* cooRowInd, hipsparseIndexBase_t idxBase);
  // CHECK: status_t = hipsparseXcsr2coo(handle_t, &csrSortedRowPtr, nnz, m, &icooRowInd, indexBase_t);
  status_t = hipsparseXcsr2coo(handle_t, &csrSortedRowPtr, nnz, m, &icooRowInd, indexBase_t);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseZnnz(cusparseHandle_t handle, cusparseDirection_t dirA, int m, int n, const cusparseMatDescr_t descrA, const cuDoubleComplex* A, int lda, int* nnzPerRowCol, int* nnzTotalDevHostPtr);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseZnnz(hipsparseHandle_t handle, hipsparseDirection_t dirA, int m, int n, const hipsparseMatDescr_t descrA, const hipDoubleComplex* A, int lda, int* nnzPerRowColumn, int* nnzTotalDevHostPtr);
  // CHECK: status_t = hipsparseZnnz(handle_t, direction_t, m, n, matDescr_A, &dcomplexA, lda, &nnzPerRowCol, &nnzTotalDevHostPtr);
  status_t = hipsparseZnnz(handle_t, direction_t, m, n, matDescr_A, &dcomplexA, lda, &nnzPerRowCol, &nnzTotalDevHostPtr);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseCnnz(cusparseHandle_t handle, cusparseDirection_t dirA, int m, int n, const cusparseMatDescr_t descrA, const cuComplex* A, int lda, int* nnzPerRowCol, int* nnzTotalDevHostPtr);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseCnnz(hipsparseHandle_t handle, hipsparseDirection_t dirA, int m, int n, const hipsparseMatDescr_t descrA, const hipComplex* A, int lda, int* nnzPerRowColumn, int* nnzTotalDevHostPtr);
  // CHECK: status_t = hipsparseCnnz(handle_t, direction_t, m, n, matDescr_A, &complexA, lda, &nnzPerRowCol, &nnzTotalDevHostPtr);
  status_t = hipsparseCnnz(handle_t, direction_t, m, n, matDescr_A, &complexA, lda, &nnzPerRowCol, &nnzTotalDevHostPtr);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseDnnz(cusparseHandle_t handle, cusparseDirection_t dirA, int m, int n, const cusparseMatDescr_t descrA, const double* A, int lda, int* nnzPerRowCol, int* nnzTotalDevHostPtr);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseDnnz(hipsparseHandle_t handle, hipsparseDirection_t dirA, int m, int n, const hipsparseMatDescr_t descrA, const double* A, int lda, int* nnzPerRowColumn, int* nnzTotalDevHostPtr);
  // CHECK: status_t = hipsparseDnnz(handle_t, direction_t, m, n, matDescr_A, &dA, lda, &nnzPerRowCol, &nnzTotalDevHostPtr);
  status_t = hipsparseDnnz(handle_t, direction_t, m, n, matDescr_A, &dA, lda, &nnzPerRowCol, &nnzTotalDevHostPtr);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseSnnz(cusparseHandle_t handle, cusparseDirection_t dirA, int m, int n, const cusparseMatDescr_t descrA, const float* A, int lda, int* nnzPerRowCol, int* nnzTotalDevHostPtr);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseSnnz(hipsparseHandle_t handle, hipsparseDirection_t dirA, int m, int n, const hipsparseMatDescr_t descrA, const float* A, int lda, int* nnzPerRowColumn, int* nnzTotalDevHostPtr);
  // CHECK: status_t = hipsparseSnnz(handle_t, direction_t, m, n, matDescr_A, &fA, lda, &nnzPerRowCol, &nnzTotalDevHostPtr);
  status_t = hipsparseSnnz(handle_t, direction_t, m, n, matDescr_A, &fA, lda, &nnzPerRowCol, &nnzTotalDevHostPtr);

#if CUDA_VERSION >= 8000
  // CHECK: hipDataType dataType_t;
  // CHECK-NEXT: hipDataType dataType;
  hipDataType dataType_t;
  hipDataType dataType;

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseZcsr2csr_compress(cusparseHandle_t handle, int m, int n, const cusparseMatDescr_t descrA, const cuDoubleComplex* csrSortedValA, const int* csrSortedColIndA, const int* csrSortedRowPtrA, int nnzA, const int* nnzPerRow, cuDoubleComplex* csrSortedValC, int* csrSortedColIndC, int* csrSortedRowPtrC, cuDoubleComplex tol);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseZcsr2csr_compress(hipsparseHandle_t handle, int m, int n, const hipsparseMatDescr_t descrA, const hipDoubleComplex* csrValA, const int* csrColIndA, const int* csrRowPtrA, int nnzA, const int* nnzPerRow, hipDoubleComplex* csrValC, int* csrColIndC, int* csrRowPtrC, hipDoubleComplex tol);
  // CHECK: status_t = hipsparseZcsr2csr_compress(handle_t, m, n, matDescr_A, &dComplexcsrSortedValA, &csrColIndA, &csrRowPtrA, nnza, &nnzPerRow, &dComplexcsrSortedValC, &csrColIndC, &csrRowPtrC, dcomplextol);
  status_t = hipsparseZcsr2csr_compress(handle_t, m, n, matDescr_A, &dComplexcsrSortedValA, &csrColIndA, &csrRowPtrA, nnza, &nnzPerRow, &dComplexcsrSortedValC, &csrColIndC, &csrRowPtrC, dcomplextol);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseCcsr2csr_compress(cusparseHandle_t handle, int m, int n, const cusparseMatDescr_t descrA, const cuComplex* csrSortedValA, const int* csrSortedColIndA, const int* csrSortedRowPtrA, int nnzA, const int* nnzPerRow, cuComplex* csrSortedValC, int* csrSortedColIndC, int* csrSortedRowPtrC, cuComplex tol);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseCcsr2csr_compress(hipsparseHandle_t handle, int m, int n, const hipsparseMatDescr_t descrA, const hipComplex* csrValA, const int* csrColIndA, const int* csrRowPtrA, int nnzA, const int* nnzPerRow, hipComplex* csrValC, int* csrColIndC, int* csrRowPtrC, hipComplex tol);
  // CHECK: status_t = hipsparseCcsr2csr_compress(handle_t, m, n, matDescr_A, &complexcsrSortedValA, &csrColIndA, &csrRowPtrA, nnza, &nnzPerRow, &complexcsrSortedValC, &csrColIndC, &csrRowPtrC, complextol);
  status_t = hipsparseCcsr2csr_compress(handle_t, m, n, matDescr_A, &complexcsrSortedValA, &csrColIndA, &csrRowPtrA, nnza, &nnzPerRow, &complexcsrSortedValC, &csrColIndC, &csrRowPtrC, complextol);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseDcsr2csr_compress(cusparseHandle_t handle, int m, int n, const cusparseMatDescr_t descrA, const double* csrSortedValA, const int* csrSortedColIndA, const int* csrSortedRowPtrA, int nnzA, const int* nnzPerRow, double* csrSortedValC, int* csrSortedColIndC, int* csrSortedRowPtrC, double tol);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseDcsr2csr_compress(hipsparseHandle_t handle, int m, int n, const hipsparseMatDescr_t descrA, const double* csrValA, const int* csrColIndA, const int* csrRowPtrA, int nnzA, const int* nnzPerRow, double* csrValC, int* csrColIndC, int* csrRowPtrC, double tol);
  // CHECK: status_t = hipsparseDcsr2csr_compress(handle_t, m, n, matDescr_A, &dcsrSortedValA, &csrColIndA, &csrRowPtrA, nnza, &nnzPerRow, &dcsrSortedValC, &csrColIndC, &csrRowPtrC, dtol);
  status_t = hipsparseDcsr2csr_compress(handle_t, m, n, matDescr_A, &dcsrSortedValA, &csrColIndA, &csrRowPtrA, nnza, &nnzPerRow, &dcsrSortedValC, &csrColIndC, &csrRowPtrC, dtol);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseScsr2csr_compress(cusparseHandle_t handle, int m, int n, const cusparseMatDescr_t descrA, const float* csrSortedValA, const int* csrSortedColIndA, const int* csrSortedRowPtrA, int nnzA, const int* nnzPerRow, float* csrSortedValC, int* csrSortedColIndC, int* csrSortedRowPtrC, float tol);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseScsr2csr_compress(hipsparseHandle_t handle, int m, int n, const hipsparseMatDescr_t descrA, const float* csrValA, const int* csrColIndA, const int* csrRowPtrA, int nnzA, const int* nnzPerRow, float* csrValC, int* csrColIndC, int* csrRowPtrC, float tol);
  // CHECK: status_t = hipsparseScsr2csr_compress(handle_t, m, n, matDescr_A, &csrSortedValA, &csrColIndA, &csrRowPtrA, nnza, &nnzPerRow, &fcsrSortedValC, &csrColIndC, &csrRowPtrC, ftol);
  status_t = hipsparseScsr2csr_compress(handle_t, m, n, matDescr_A, &csrSortedValA, &csrColIndA, &csrRowPtrA, nnza, &nnzPerRow, &fcsrSortedValC, &csrColIndC, &csrRowPtrC, ftol);

  // CUDA: CUSPARSE_DEPRECATED cusparseStatus_t CUSPARSEAPI cusparseZnnz_compress(cusparseHandle_t handle, int m, const cusparseMatDescr_t descr, const cuDoubleComplex* csrSortedValA, const int* csrSortedRowPtrA, int* nnzPerRow, int* nnzC, cuDoubleComplex tol);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseZnnz_compress(hipsparseHandle_t handle, int m, const hipsparseMatDescr_t descrA, const hipDoubleComplex* csrValA, const int* csrRowPtrA, int* nnzPerRow, int* nnzC, hipDoubleComplex tol);
  // CHECK: status_t = hipsparseZnnz_compress(handle_t, m, matDescr_A, &dComplexcsrSortedValA, &csrRowPtrA, &nnzPerRow, &nnzc, dcomplextol);
 status_t = hipsparseZnnz_compress(handle_t, m, matDescr_A, &dComplexcsrSortedValA, &csrRowPtrA, &nnzPerRow, &nnzc, dcomplextol);

  // CUDA: CUSPARSE_DEPRECATED cusparseStatus_t CUSPARSEAPI cusparseCnnz_compress(cusparseHandle_t handle, int m, const cusparseMatDescr_t descr, const cuComplex* csrSortedValA, const int* csrSortedRowPtrA, int* nnzPerRow, int* nnzC, cuComplex tol);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseCnnz_compress(hipsparseHandle_t handle, int m, const hipsparseMatDescr_t descrA, const hipComplex* csrValA, const int* csrRowPtrA, int* nnzPerRow, int* nnzC, hipComplex tol);
  // CHECK: status_t = hipsparseCnnz_compress(handle_t, m, matDescr_A, &complexcsrSortedValA, &csrRowPtrA, &nnzPerRow, &nnzc, complextol);
 status_t = hipsparseCnnz_compress(handle_t, m, matDescr_A, &complexcsrSortedValA, &csrRowPtrA, &nnzPerRow, &nnzc, complextol);

  // CUDA: CUSPARSE_DEPRECATED cusparseStatus_t CUSPARSEAPI cusparseDnnz_compress(cusparseHandle_t handle, int m, const cusparseMatDescr_t descr, const double* csrSortedValA, const int* csrSortedRowPtrA, int* nnzPerRow, int* nnzC, double tol);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseDnnz_compress(hipsparseHandle_t handle, int m, const hipsparseMatDescr_t descrA, const double* csrValA, const int* csrRowPtrA, int* nnzPerRow, int* nnzC, double tol);
  // CHECK: status_t = hipsparseDnnz_compress(handle_t, m, matDescr_A, &dcsrSortedValA, &csrRowPtrA, &nnzPerRow, &nnzc, dtol);
 status_t = hipsparseDnnz_compress(handle_t, m, matDescr_A, &dcsrSortedValA, &csrRowPtrA, &nnzPerRow, &nnzc, dtol);

  // CUDA: CUSPARSE_DEPRECATED cusparseStatus_t CUSPARSEAPI cusparseSnnz_compress(cusparseHandle_t handle, int m, const cusparseMatDescr_t descr, const float* csrSortedValA, const int* csrSortedRowPtrA, int* nnzPerRow, int* nnzC, float tol);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseSnnz_compress(hipsparseHandle_t handle, int m, const hipsparseMatDescr_t descrA, const float* csrValA, const int* csrRowPtrA, int* nnzPerRow, int* nnzC, float tol);
  // CHECK: status_t = hipsparseSnnz_compress(handle_t, m, matDescr_A, &csrSortedValA, &csrRowPtrA, &nnzPerRow, &nnzc, ftol);
 status_t = hipsparseSnnz_compress(handle_t, m, matDescr_A, &csrSortedValA, &csrRowPtrA, &nnzPerRow, &nnzc, ftol);
#endif

#if CUDA_VERSION >= 8000 && CUDA_VERSION < 12000
  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseCopyMatDescr(cusparseMatDescr_t dest, const cusparseMatDescr_t src);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseCopyMatDescr(hipsparseMatDescr_t dest, const hipsparseMatDescr_t src);
  // CHECK: status_t = hipsparseCopyMatDescr(matDescr_t, matDescr_t_2);
  status_t = cusparseCopyMatDescr(matDescr_t, matDescr_t_2);
#endif

#if CUDA_VERSION >= 9000
  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseDpruneCsr2csrByPercentage(cusparseHandle_t handle, int m, int n, int nnzA, const cusparseMatDescr_t descrA, const double* csrSortedValA, const int* csrSortedRowPtrA, const int* csrSortedColIndA, float percentage, const cusparseMatDescr_t descrC, double* csrSortedValC, const int* csrSortedRowPtrC, int* csrSortedColIndC, pruneInfo_t info, void* pBuffer);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseDpruneCsr2csrByPercentage(hipsparseHandle_t handle, int m, int n, int nnzA, const hipsparseMatDescr_t descrA, const double* csrValA, const int* csrRowPtrA, const int* csrColIndA, double percentage, const hipsparseMatDescr_t descrC, double* csrValC, const int* csrRowPtrC, int* csrColIndC, pruneInfo_t info, void* buffer);
  // CHECK: status_t = hipsparseDpruneCsr2csrByPercentage(handle_t, m, n, nnz, matDescr_A, &csrValA, &csrRowPtrA, &csrColIndA, percentage, matDescr_C, &csrValC, &csrRowPtrC, &csrColIndC, prune_info, pBuffer);
  status_t = hipsparseDpruneCsr2csrByPercentage(handle_t, m, n, nnz, matDescr_A, &csrValA, &csrRowPtrA, &csrColIndA, percentage, matDescr_C, &csrValC, &csrRowPtrC, &csrColIndC, prune_info, pBuffer);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseSpruneCsr2csrByPercentage(cusparseHandle_t handle, int m, int n, int nnzA, const cusparseMatDescr_t descrA, const float* csrSortedValA, const int* csrSortedRowPtrA, const int* csrSortedColIndA, float percentage, const cusparseMatDescr_t descrC, float* csrSortedValC, const int* csrSortedRowPtrC, int* csrSortedColIndC, pruneInfo_t info, void* pBuffer);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseSpruneCsr2csrByPercentage(hipsparseHandle_t handle, int m, int n, int nnzA, const hipsparseMatDescr_t descrA, const float* csrValA, const int* csrRowPtrA, const int* csrColIndA, float percentage, const hipsparseMatDescr_t descrC, float* csrValC, const int* csrRowPtrC, int* csrColIndC, pruneInfo_t info, void* buffer);
  // CHECK: status_t = hipsparseSpruneCsr2csrByPercentage(handle_t, m, n, nnz, matDescr_A, &fcsrValA, &csrRowPtrA, &csrColIndA, percentage, matDescr_C, &fcsrValC, &csrRowPtrC, &csrColIndC, prune_info, pBuffer);
  status_t = hipsparseSpruneCsr2csrByPercentage(handle_t, m, n, nnz, matDescr_A, &fcsrValA, &csrRowPtrA, &csrColIndA, percentage, matDescr_C, &fcsrValC, &csrRowPtrC, &csrColIndC, prune_info, pBuffer);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseDpruneCsr2csrNnzByPercentage(cusparseHandle_t handle, int m, int n, int nnzA, const cusparseMatDescr_t descrA, const double* csrSortedValA, const int* csrSortedRowPtrA, const int* csrSortedColIndA, float percentage, const cusparseMatDescr_t descrC, int* csrSortedRowPtrC, int* nnzTotalDevHostPtr, pruneInfo_t info, void* pBuffer);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseDpruneCsr2csrNnzByPercentage(hipsparseHandle_t handle, int m, int n, int nnzA, const hipsparseMatDescr_t descrA, const double* csrValA, const int* csrRowPtrA, const int* csrColIndA, double percentage, const hipsparseMatDescr_t descrC, int* csrRowPtrC, int* nnzTotalDevHostPtr, pruneInfo_t info, void* buffer);
  // CHECK: status_t = hipsparseDpruneCsr2csrNnzByPercentage(handle_t, m, n, nnz, matDescr_A, &dbsrSortedValA, &csrRowPtrA, &csrColIndA, percentage, matDescr_C, &csrRowPtrC, &csrColIndC, prune_info, pBuffer);
  status_t = hipsparseDpruneCsr2csrNnzByPercentage(handle_t, m, n, nnz, matDescr_A, &dbsrSortedValA, &csrRowPtrA, &csrColIndA, percentage, matDescr_C, &csrRowPtrC, &csrColIndC, prune_info, pBuffer);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseSpruneCsr2csrNnzByPercentage(cusparseHandle_t handle, int m, int n, int nnzA, const cusparseMatDescr_t descrA, const float* csrSortedValA, const int* csrSortedRowPtrA, const int* csrSortedColIndA, float percentage, const cusparseMatDescr_t descrC, int* csrSortedRowPtrC, int* nnzTotalDevHostPtr, pruneInfo_t info, void* pBuffer);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseSpruneCsr2csrNnzByPercentage(hipsparseHandle_t handle, int m, int n, int nnzA, const hipsparseMatDescr_t descrA, const float* csrValA, const int* csrRowPtrA, const int* csrColIndA, float percentage, const hipsparseMatDescr_t descrC, int* csrRowPtrC, int* nnzTotalDevHostPtr, pruneInfo_t info, void* buffer);
  // CHECK: status_t = hipsparseSpruneCsr2csrNnzByPercentage(handle_t, m, n, nnz, matDescr_A, &csrSortedValA, &csrRowPtrA, &csrColIndA, percentage, matDescr_C, &csrRowPtrC, &csrColIndC, prune_info, pBuffer);
  status_t = hipsparseSpruneCsr2csrNnzByPercentage(handle_t, m, n, nnz, matDescr_A, &csrSortedValA, &csrRowPtrA, &csrColIndA, percentage, matDescr_C, &csrRowPtrC, &csrColIndC, prune_info, pBuffer);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseDpruneCsr2csrByPercentage_bufferSizeExt(cusparseHandle_t handle, int m, int n, int nnzA, const cusparseMatDescr_t descrA, const double* csrSortedValA, const int* csrSortedRowPtrA, const int* csrSortedColIndA, float percentage, const cusparseMatDescr_t descrC, const double* csrSortedValC, const int* csrSortedRowPtrC, const int* csrSortedColIndC, pruneInfo_t info, size_t* pBufferSizeInBytes);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseDpruneCsr2csrByPercentage_bufferSizeExt(hipsparseHandle_t handle, int m, int n, int nnzA, const hipsparseMatDescr_t descrA, const double* csrValA, const int* csrRowPtrA, const int* csrColIndA, double percentage, const hipsparseMatDescr_t descrC, const double* csrValC, const int* csrRowPtrC, const int* csrColIndC, pruneInfo_t info, size_t* bufferSize);
  // CHECK: status_t = hipsparseDpruneCsr2csrByPercentage_bufferSizeExt(handle_t, m, n, nnz, matDescr_A, &dbsrSortedValA, &csrRowPtrA, &csrColIndA, percentage, matDescr_C, &dbsrSortedValC, &csrRowPtrC, &csrColIndC, prune_info, &bufferSize);
  status_t = hipsparseDpruneCsr2csrByPercentage_bufferSizeExt(handle_t, m, n, nnz, matDescr_A, &dbsrSortedValA, &csrRowPtrA, &csrColIndA, percentage, matDescr_C, &dbsrSortedValC, &csrRowPtrC, &csrColIndC, prune_info, &bufferSize);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseSpruneCsr2csrByPercentage_bufferSizeExt(cusparseHandle_t handle, int m, int n, int nnzA, const cusparseMatDescr_t descrA, const float* csrSortedValA, const int* csrSortedRowPtrA, const int* csrSortedColIndA, float percentage, const cusparseMatDescr_t descrC, const float* csrSortedValC, const int* csrSortedRowPtrC, const int* csrSortedColIndC, pruneInfo_t info, size_t* pBufferSizeInBytes);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseSpruneCsr2csrByPercentage_bufferSizeExt(hipsparseHandle_t handle, int m, int n, int nnzA, const hipsparseMatDescr_t descrA, const float* csrValA, const int* csrRowPtrA, const int* csrColIndA, float percentage, const hipsparseMatDescr_t descrC, const float* csrValC, const int* csrRowPtrC, const int* csrColIndC, pruneInfo_t info, size_t* bufferSize);
  // CHECK: status_t = hipsparseSpruneCsr2csrByPercentage_bufferSizeExt(handle_t, m, n, nnz, matDescr_A, &fbsrSortedValA, &csrRowPtrA, &csrColIndA, percentage, matDescr_C, &fbsrSortedValC, &csrRowPtrC, &csrColIndC, prune_info, &bufferSize);
  status_t = hipsparseSpruneCsr2csrByPercentage_bufferSizeExt(handle_t, m, n, nnz, matDescr_A, &fbsrSortedValA, &csrRowPtrA, &csrColIndA, percentage, matDescr_C, &fbsrSortedValC, &csrRowPtrC, &csrColIndC, prune_info, &bufferSize);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseDpruneCsr2csr(cusparseHandle_t handle, int m, int n, int nnzA, const cusparseMatDescr_t descrA, const double* csrSortedValA, const int* csrSortedRowPtrA, const int* csrSortedColIndA, const double* threshold, const cusparseMatDescr_t descrC, double* csrSortedValC, const int* csrSortedRowPtrC, int* csrSortedColIndC, void* pBuffer);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseDpruneCsr2csr(hipsparseHandle_t handle, int m, int n, int nnzA, const hipsparseMatDescr_t descrA, const double* csrValA, const int* csrRowPtrA, const int* csrColIndA, const double* threshold, const hipsparseMatDescr_t descrC, double* csrValC, const int* csrRowPtrC, int* csrColIndC, void* buffer);
  // CHECK: status_t = hipsparseDpruneCsr2csr(handle_t, m, n, nnz, matDescr_A, &dbsrSortedValA, &csrRowPtrA, &csrColIndA, &dthreshold, matDescr_C, &dbsrSortedValC, &csrRowPtrC, &csrColIndC, pBuffer);
  status_t = hipsparseDpruneCsr2csr(handle_t, m, n, nnz, matDescr_A, &dbsrSortedValA, &csrRowPtrA, &csrColIndA, &dthreshold, matDescr_C, &dbsrSortedValC, &csrRowPtrC, &csrColIndC, pBuffer);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseSpruneCsr2csr(cusparseHandle_t handle, int m, int n, int nnzA, const cusparseMatDescr_t descrA, const float* csrSortedValA, const int* csrSortedRowPtrA, const int* csrSortedColIndA, const float* threshold, const cusparseMatDescr_t descrC, float* csrSortedValC, const int* csrSortedRowPtrC, int* csrSortedColIndC, void* pBuffer);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseSpruneCsr2csr(hipsparseHandle_t handle, int m, int n, int nnzA, const hipsparseMatDescr_t descrA, const float* csrValA, const int* csrRowPtrA, const int* csrColIndA, const float* threshold, const hipsparseMatDescr_t descrC, float* csrValC, const int* csrRowPtrC, int* csrColIndC, void* buffer);
  // CHECK: status_t = hipsparseSpruneCsr2csr(handle_t, m, n, nnz, matDescr_A, &fbsrSortedValA, &csrRowPtrA, &csrColIndA, &fthreshold, matDescr_C, &fbsrSortedValC, &csrRowPtrC, &csrColIndC, pBuffer);
  status_t = hipsparseSpruneCsr2csr(handle_t, m, n, nnz, matDescr_A, &fbsrSortedValA, &csrRowPtrA, &csrColIndA, &fthreshold, matDescr_C, &fbsrSortedValC, &csrRowPtrC, &csrColIndC, pBuffer);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseDpruneCsr2csrNnz(cusparseHandle_t handle, int m, int n, int nnzA, const cusparseMatDescr_t descrA, const double* csrSortedValA, const int* csrSortedRowPtrA, const int* csrSortedColIndA, const double* threshold, const cusparseMatDescr_t descrC, int* csrSortedRowPtrC, int* nnzTotalDevHostPtr, void* pBuffer);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseDpruneCsr2csrNnz(hipsparseHandle_t handle, int m, int n, int nnzA, const hipsparseMatDescr_t descrA, const double* csrValA, const int* csrRowPtrA, const int* csrColIndA, const double* threshold, const hipsparseMatDescr_t descrC, int* csrRowPtrC, int* nnzTotalDevHostPtr, void* buffer);
  // CHECK: status_t = hipsparseDpruneCsr2csrNnz(handle_t, m, n, nnz, matDescr_A, &dbsrSortedValA, &csrRowPtrA, &csrColIndA, &dthreshold, matDescr_C, &csrRowPtrC, &nnzTotalDevHostPtr, pBuffer);
  status_t = hipsparseDpruneCsr2csrNnz(handle_t, m, n, nnz, matDescr_A, &dbsrSortedValA, &csrRowPtrA, &csrColIndA, &dthreshold, matDescr_C, &csrRowPtrC, &nnzTotalDevHostPtr, pBuffer);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseSpruneCsr2csrNnz(cusparseHandle_t handle, int m, int n, int nnzA, const cusparseMatDescr_t descrA, const float* csrSortedValA, const int* csrSortedRowPtrA, const int* csrSortedColIndA, const float* threshold, const cusparseMatDescr_t descrC, int* csrSortedRowPtrC, int* nnzTotalDevHostPtr, void* pBuffer);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseSpruneCsr2csrNnz(hipsparseHandle_t handle, int m, int n, int nnzA, const hipsparseMatDescr_t descrA, const float* csrValA, const int* csrRowPtrA, const int* csrColIndA, const float* threshold, const hipsparseMatDescr_t descrC, int* csrRowPtrC, int* nnzTotalDevHostPtr, void* buffer);
  // CHECK: status_t = hipsparseSpruneCsr2csrNnz(handle_t, m, n, nnz, matDescr_A, &csrSortedValA, &csrRowPtrA, &csrColIndA, &fthreshold, matDescr_C, &csrRowPtrC, &nnzTotalDevHostPtr, pBuffer);
  status_t = hipsparseSpruneCsr2csrNnz(handle_t, m, n, nnz, matDescr_A, &csrSortedValA, &csrRowPtrA, &csrColIndA, &fthreshold, matDescr_C, &csrRowPtrC, &nnzTotalDevHostPtr, pBuffer);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseDpruneCsr2csr_bufferSizeExt(cusparseHandle_t handle, int m, int n, int nnzA, const cusparseMatDescr_t descrA, const double* csrSortedValA, const int* csrSortedRowPtrA, const int* csrSortedColIndA, const double* threshold, const cusparseMatDescr_t descrC, const double* csrSortedValC, const int* csrSortedRowPtrC, const int* csrSortedColIndC, size_t* pBufferSizeInBytes);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseDpruneCsr2csr_bufferSizeExt(hipsparseHandle_t handle, int m, int n, int nnzA, const hipsparseMatDescr_t descrA, const double* csrValA, const int* csrRowPtrA, const int* csrColIndA, const double* threshold, const hipsparseMatDescr_t descrC, const double* csrValC, const int* csrRowPtrC, const int* csrColIndC, size_t* bufferSize);
  // CHECK: status_t = hipsparseDpruneCsr2csr_bufferSizeExt(handle_t, m, n, nnz, matDescr_A, &dbsrSortedValA, &csrRowPtrA, &csrColIndA, &dthreshold, matDescr_C, &dbsrSortedValC, &csrRowPtrC, &csrColIndC, &bufferSize);
  status_t = hipsparseDpruneCsr2csr_bufferSizeExt(handle_t, m, n, nnz, matDescr_A, &dbsrSortedValA, &csrRowPtrA, &csrColIndA, &dthreshold, matDescr_C, &dbsrSortedValC, &csrRowPtrC, &csrColIndC, &bufferSize);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseSpruneCsr2csr_bufferSizeExt(cusparseHandle_t handle, int m, int n, int nnzA, const cusparseMatDescr_t descrA, const float* csrSortedValA, const int* csrSortedRowPtrA, const int* csrSortedColIndA, const float* threshold, const cusparseMatDescr_t descrC, const float* csrSortedValC, const int* csrSortedRowPtrC, const int* csrSortedColIndC, size_t* pBufferSizeInBytes);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseSpruneCsr2csr_bufferSizeExt(hipsparseHandle_t handle, int m, int n, int nnzA, const hipsparseMatDescr_t descrA, const float* csrValA, const int* csrRowPtrA, const int* csrColIndA, const float* threshold, const hipsparseMatDescr_t descrC, const float* csrValC, const int* csrRowPtrC, const int* csrColIndC, size_t* bufferSize);
  // CHECK: status_t = hipsparseSpruneCsr2csr_bufferSizeExt(handle_t, m, n, nnz, matDescr_A, &fbsrSortedValA, &csrRowPtrA, &csrColIndA, &fthreshold, matDescr_C, &fbsrSortedValC, &csrRowPtrC, &csrColIndC, &bufferSize);
  status_t = hipsparseSpruneCsr2csr_bufferSizeExt(handle_t, m, n, nnz, matDescr_A, &fbsrSortedValA, &csrRowPtrA, &csrColIndA, &fthreshold, matDescr_C, &fbsrSortedValC, &csrRowPtrC, &csrColIndC, &bufferSize);

  // CUDA: CUSPARSE_DEPRECATED cusparseStatus_t CUSPARSEAPI cusparseDpruneDense2csrByPercentage(cusparseHandle_t handle, int m, int n, const double* A, int lda, float percentage, const cusparseMatDescr_t descrC, double* csrSortedValC, const int* csrSortedRowPtrC,int* csrSortedColIndC, pruneInfo_t info, void* pBuffer);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseDpruneDense2csrByPercentage(hipsparseHandle_t handle, int m, int n, const double* A, int lda, double percentage, const hipsparseMatDescr_t descr, double* csrVal, const int* csrRowPtr, int* csrColInd, pruneInfo_t info, void* buffer);
  // CHECK: status_t = hipsparseDpruneDense2csrByPercentage(handle_t, m, n, &dA, lda, percentage, matDescr_C, &dcsrSortedValC, &csrRowPtrC, &csrColIndC, prune_info, pBuffer);
  status_t = hipsparseDpruneDense2csrByPercentage(handle_t, m, n, &dA, lda, percentage, matDescr_C, &dcsrSortedValC, &csrRowPtrC, &csrColIndC, prune_info, pBuffer);

  // CUDA: CUSPARSE_DEPRECATED cusparseStatus_t CUSPARSEAPI cusparseSpruneDense2csrByPercentage(cusparseHandle_t handle, int m, int n, const float* A, int lda, float percentage, const cusparseMatDescr_t descrC, float* csrSortedValC, const int* csrSortedRowPtrC, int* csrSortedColIndC, pruneInfo_t info, void* pBuffer);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseSpruneDense2csrByPercentage(hipsparseHandle_t handle, int m, int n, const float* A, int lda, float percentage, const hipsparseMatDescr_t descr, float* csrVal, const int* csrRowPtr, int* csrColInd, pruneInfo_t info, void* buffer);
  // CHECK: status_t = hipsparseSpruneDense2csrByPercentage(handle_t, m, n, &fA, lda, fpercentage, matDescr_C, &fcsrSortedValC, &csrRowPtrC, &csrColIndC, prune_info, pBuffer);
  status_t = hipsparseSpruneDense2csrByPercentage(handle_t, m, n, &fA, lda, fpercentage, matDescr_C, &fcsrSortedValC, &csrRowPtrC, &csrColIndC, prune_info, pBuffer);

  // CUDA: CUSPARSE_DEPRECATED cusparseStatus_t CUSPARSEAPI cusparseDpruneDense2csrNnzByPercentage(cusparseHandle_t handle, int m, int n, const double* A, int lda, float percentage, const cusparseMatDescr_t descrC, int* csrRowPtrC, int* nnzTotalDevHostPtr, pruneInfo_t info, void* pBuffer);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseDpruneDense2csrNnzByPercentage(hipsparseHandle_t handle, int m, int n, const double* A, int lda, double percentage, const hipsparseMatDescr_t descr, int* csrRowPtr, int* nnzTotalDevHostPtr, pruneInfo_t info, void* buffer);
  // CHECK: status_t = hipsparseDpruneDense2csrNnzByPercentage(handle_t, m, n, &dA, lda, fpercentage, matDescr_C, &csrRowPtrC, &nnzTotalDevHostPtr, prune_info, pBuffer);
  status_t = hipsparseDpruneDense2csrNnzByPercentage(handle_t, m, n, &dA, lda, fpercentage, matDescr_C, &csrRowPtrC, &nnzTotalDevHostPtr, prune_info, pBuffer);

  // CUDA: CUSPARSE_DEPRECATED cusparseStatus_t CUSPARSEAPI cusparseSpruneDense2csrNnzByPercentage(cusparseHandle_t handle, int m, int n, const float* A, int lda, float percentage, const cusparseMatDescr_t descrC, int* csrRowPtrC, int* nnzTotalDevHostPtr, pruneInfo_t info, void* pBuffer);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseSpruneDense2csrNnzByPercentage(hipsparseHandle_t handle, int m, int n, const float* A, int lda, float percentage, const hipsparseMatDescr_t descr, int* csrRowPtr, int* nnzTotalDevHostPtr, pruneInfo_t info, void* buffer);
  // CHECK: status_t = hipsparseSpruneDense2csrNnzByPercentage(handle_t, m, n, &fA, lda, fpercentage, matDescr_C, &csrRowPtrC, &nnzTotalDevHostPtr, prune_info, pBuffer);
  status_t = hipsparseSpruneDense2csrNnzByPercentage(handle_t, m, n, &fA, lda, fpercentage, matDescr_C, &csrRowPtrC, &nnzTotalDevHostPtr, prune_info, pBuffer);

  // CUDA: CUSPARSE_DEPRECATED cusparseStatus_t CUSPARSEAPI cusparseDpruneDense2csrByPercentage_bufferSizeExt(cusparseHandle_t handle, int m, int n, const double* A, int lda, float percentage, const cusparseMatDescr_t descrC, const double* csrSortedValC, const int* csrSortedRowPtrC, const int* csrSortedColIndC, pruneInfo_t info, size_t* pBufferSizeInBytes);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseDpruneDense2csrByPercentage_bufferSizeExt(hipsparseHandle_t handle, int m, int n, const double* A, int lda, double percentage, const hipsparseMatDescr_t descr, const double* csrVal, const int* csrRowPtr, const int* csrColInd, pruneInfo_t info, size_t* bufferSize);
  // CHECK: status_t = hipsparseDpruneDense2csrByPercentage_bufferSizeExt(handle_t, m, n, &dA, lda, fpercentage, matDescr_C, &dcsrSortedValC, &csrRowPtrC, &csrColIndC, prune_info, &bufferSize);
  status_t = hipsparseDpruneDense2csrByPercentage_bufferSizeExt(handle_t, m, n, &dA, lda, fpercentage, matDescr_C, &dcsrSortedValC, &csrRowPtrC, &csrColIndC, prune_info, &bufferSize);

  // CUDA: CUSPARSE_DEPRECATED cusparseStatus_t CUSPARSEAPI cusparseSpruneDense2csrByPercentage_bufferSizeExt(cusparseHandle_t handle, int m, int n, const float* A, int lda, float percentage, const cusparseMatDescr_t descrC, const float* csrSortedValC, const int* csrSortedRowPtrC, const int* csrSortedColIndC, pruneInfo_t info, size_t* pBufferSizeInBytes);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseSpruneDense2csrByPercentage_bufferSizeExt(hipsparseHandle_t handle, int m, int n, const float* A, int lda, float percentage, const hipsparseMatDescr_t descr, const float* csrVal, const int* csrRowPtr, const int* csrColInd, pruneInfo_t info, size_t* bufferSize);
  // CHECK: status_t = hipsparseSpruneDense2csrByPercentage_bufferSizeExt(handle_t, m, n, &fA, lda, fpercentage, matDescr_C, &fcsrSortedValC, &csrRowPtrC, &csrColIndC, prune_info, &bufferSize);
  status_t = hipsparseSpruneDense2csrByPercentage_bufferSizeExt(handle_t, m, n, &fA, lda, fpercentage, matDescr_C, &fcsrSortedValC, &csrRowPtrC, &csrColIndC, prune_info, &bufferSize);

  // CUDA: CUSPARSE_DEPRECATED cusparseStatus_t CUSPARSEAPI cusparseDpruneDense2csr(cusparseHandle_t handle, int m, int n, const double* A, int lda, const double* threshold, const cusparseMatDescr_t descrC, double* csrSortedValC, const int* csrSortedRowPtrC, int* csrSortedColIndC, void* pBuffer);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseDpruneDense2csr(hipsparseHandle_t handle, int m, int n, const double* A, int lda, const double* threshold, const hipsparseMatDescr_t descr, double* csrVal, const int* csrRowPtr, int* csrColInd, void* buffer);
  // CHECK: status_t = hipsparseDpruneDense2csr(handle_t, m, n, &dA, lda, &dthreshold, matDescr_C, &dcsrSortedValC, &csrRowPtrC, &csrColIndC, pBuffer);
  status_t = hipsparseDpruneDense2csr(handle_t, m, n, &dA, lda, &dthreshold, matDescr_C, &dcsrSortedValC, &csrRowPtrC, &csrColIndC, pBuffer);

  // CUDA: CUSPARSE_DEPRECATED cusparseStatus_t CUSPARSEAPI cusparseSpruneDense2csr(cusparseHandle_t handle, int m, int n, const float* A, int lda, const float* threshold, const cusparseMatDescr_t descrC, float* csrSortedValC, const int* csrSortedRowPtrC, int* csrSortedColIndC, void* pBuffer);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseSpruneDense2csr(hipsparseHandle_t handle, int m, int n, const float* A, int lda, const float* threshold, const hipsparseMatDescr_t descr, float* csrVal, const int* csrRowPtr, int* csrColInd, void* buffer);
  // CHECK: status_t = hipsparseSpruneDense2csr(handle_t, m, n, &fA, lda, &fthreshold, matDescr_C, &fcsrSortedValC, &csrRowPtrC, &csrColIndC, pBuffer);
  status_t = hipsparseSpruneDense2csr(handle_t, m, n, &fA, lda, &fthreshold, matDescr_C, &fcsrSortedValC, &csrRowPtrC, &csrColIndC, pBuffer);

  // CUDA: CUSPARSE_DEPRECATED cusparseStatus_t CUSPARSEAPI cusparseDpruneDense2csrNnz(cusparseHandle_t handle, int m, int n, const double* A, int lda, const double* threshold, const cusparseMatDescr_t descrC, int* csrSortedRowPtrC, int* nnzTotalDevHostPtr, void* pBuffer);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseDpruneDense2csrNnz(hipsparseHandle_t handle, int m, int n, const double* A, int lda, const double* threshold, const hipsparseMatDescr_t descr, int* csrRowPtr, int* nnzTotalDevHostPtr, void* buffer);
  // CHECK: status_t = hipsparseDpruneDense2csrNnz(handle_t, m, n, &dA, lda, &dthreshold, matDescr_C, &csrRowPtrC, &nnzTotalDevHostPtr, pBuffer);
  status_t = hipsparseDpruneDense2csrNnz(handle_t, m, n, &dA, lda, &dthreshold, matDescr_C, &csrRowPtrC, &nnzTotalDevHostPtr, pBuffer);

  // CUDA: CUSPARSE_DEPRECATED cusparseStatus_t CUSPARSEAPI cusparseSpruneDense2csrNnz(cusparseHandle_t handle, int m, int n, const float* A, int lda, const float* threshold, const cusparseMatDescr_t descrC, int* csrRowPtrC, int* nnzTotalDevHostPtr, void* pBuffer);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseSpruneDense2csrNnz(hipsparseHandle_t handle, int m, int n, const float* A, int lda, const float* threshold, const hipsparseMatDescr_t descr, int* csrRowPtr, int* nnzTotalDevHostPtr, void* buffer);
  // CHECK: status_t = hipsparseSpruneDense2csrNnz(handle_t, m, n, &fA, lda, &fthreshold, matDescr_C, &csrRowPtrC, &nnzTotalDevHostPtr, pBuffer);
  status_t = hipsparseSpruneDense2csrNnz(handle_t, m, n, &fA, lda, &fthreshold, matDescr_C, &csrRowPtrC, &nnzTotalDevHostPtr, pBuffer);

  // CUDA: CUSPARSE_DEPRECATED cusparseStatus_t CUSPARSEAPI cusparseDpruneDense2csr_bufferSizeExt(cusparseHandle_t handle, int m, int n, const double* A, int lda, const double* threshold, const cusparseMatDescr_t descrC, const double* csrSortedValC, const int* csrSortedRowPtrC, const int* csrSortedColIndC, size_t* pBufferSizeInBytes);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseDpruneDense2csr_bufferSizeExt(hipsparseHandle_t handle, int m, int n, const double* A, int lda, const double* threshold, const hipsparseMatDescr_t descr, const double* csrVal, const int* csrRowPtr, const int* csrColInd, size_t* bufferSize);
  // CHECK: status_t = hipsparseDpruneDense2csr_bufferSizeExt(handle_t, m, n, &dA, lda, &dthreshold, matDescr_C, &dcsrSortedValC, &csrRowPtrC, &csrColIndC, &bufferSize);
  status_t = hipsparseDpruneDense2csr_bufferSizeExt(handle_t, m, n, &dA, lda, &dthreshold, matDescr_C, &dcsrSortedValC, &csrRowPtrC, &csrColIndC, &bufferSize);

  // CUDA: CUSPARSE_DEPRECATED cusparseStatus_t CUSPARSEAPI cusparseSpruneDense2csr_bufferSizeExt(cusparseHandle_t handle, int m, int n, const float* A, int lda, const float* threshold, const cusparseMatDescr_t descrC, const float* csrSortedValC, const int* csrSortedRowPtrC, const int* csrSortedColIndC, size_t* pBufferSizeInBytes);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseSpruneDense2csr_bufferSizeExt(hipsparseHandle_t handle, int m, int n, const float* A, int lda, const float* threshold, const hipsparseMatDescr_t descr,const float* csrVal, const int* csrRowPtr, const int* csrColInd, size_t* bufferSize);
  // CHECK: status_t = hipsparseSpruneDense2csr_bufferSizeExt(handle_t, m, n, &fA, lda, &fthreshold, matDescr_C, &fcsrSortedValC, &csrRowPtrC, &csrColIndC, &bufferSize);
  status_t = hipsparseSpruneDense2csr_bufferSizeExt(handle_t, m, n, &fA, lda, &fthreshold, matDescr_C, &fcsrSortedValC, &csrRowPtrC, &csrColIndC, &bufferSize);
#endif

#if CUDA_VERSION >= 9020
  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseZgpsvInterleavedBatch(cusparseHandle_t handle, int algo, int m, cuDoubleComplex* ds, cuDoubleComplex* dl, cuDoubleComplex* d, cuDoubleComplex* du, cuDoubleComplex* dw, cuDoubleComplex* x, int batchCount, void* pBuffer);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseZgpsvInterleavedBatch(hipsparseHandle_t handle, int algo, int m, hipDoubleComplex* ds, hipDoubleComplex* dl, hipDoubleComplex* d, hipDoubleComplex* du, hipDoubleComplex* dw, hipDoubleComplex* x, int batchCount, void* pBuffer);
  // CHECK: status_t = hipsparseZgpsvInterleavedBatch(handle_t, algo, m, &dcomplexds, &dcomplexdl, &dcomplexd, &dcomplexdu, &dcomplexdw, &dcomplexx, batchCount, pBuffer);
  status_t = hipsparseZgpsvInterleavedBatch(handle_t, algo, m, &dcomplexds, &dcomplexdl, &dcomplexd, &dcomplexdu, &dcomplexdw, &dcomplexx, batchCount, pBuffer);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseCgpsvInterleavedBatch(cusparseHandle_t handle, int algo, int m, cuComplex* ds, cuComplex* dl, cuComplex* d, cuComplex* du, cuComplex* dw, cuComplex* x, int batchCount, void* pBuffer);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseCgpsvInterleavedBatch(hipsparseHandle_t handle, int algo, int m, hipComplex* ds, hipComplex* dl, hipComplex* d, hipComplex* du, hipComplex* dw, hipComplex* x, int batchCount, void* pBuffer);
  // CHECK: status_t = hipsparseCgpsvInterleavedBatch(handle_t, algo, m, &complexds, &complexdl, &complexd, &complexdu, &complexdw, &complexx, batchCount, pBuffer);
  status_t = hipsparseCgpsvInterleavedBatch(handle_t, algo, m, &complexds, &complexdl, &complexd, &complexdu, &complexdw, &complexx, batchCount, pBuffer);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseDgpsvInterleavedBatch(cusparseHandle_t handle, int algo, int m, double* ds, double* dl, double* d, double* du, double* dw, double* x, int batchCount, void* pBuffer);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseDgpsvInterleavedBatch(hipsparseHandle_t handle, int algo, int m, double* ds, double* dl, double* d, double* du, double* dw, double* x, int batchCount, void* pBuffer);
  // CHECK: status_t = hipsparseDgpsvInterleavedBatch(handle_t, algo, m, &dds, &ddl, &dd, &ddu, &ddw, &dx, batchCount, pBuffer);
  status_t = hipsparseDgpsvInterleavedBatch(handle_t, algo, m, &dds, &ddl, &dd, &ddu, &ddw, &dx, batchCount, pBuffer);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseSgpsvInterleavedBatch(cusparseHandle_t handle, int algo, int m, float* ds, float* dl, float* d, float* du, float* dw, float* x, int batchCount, void* pBuffer);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseSgpsvInterleavedBatch(hipsparseHandle_t handle, int algo, int m, float* ds, float* dl, float* d, float* du, float* dw, float* x, int batchCount, void* pBuffer);
  // CHECK: status_t = hipsparseSgpsvInterleavedBatch(handle_t, algo, m, &fds, &fdl, &fd, &fdu, &fdw, &fx, batchCount, pBuffer);
  status_t = hipsparseSgpsvInterleavedBatch(handle_t, algo, m, &fds, &fdl, &fd, &fdu, &fdw, &fx, batchCount, pBuffer);
#endif

#if CUDA_VERSION >= 10010
  // CHECK: hipsparseCsr2CscAlg_t Csr2CscAlg_t;
  // CHECK-NEXT: hipsparseCsr2CscAlg_t CSR2CSC_ALG1 = HIPSPARSE_CSR2CSC_ALG1;
  hipsparseCsr2CscAlg_t Csr2CscAlg_t;
  hipsparseCsr2CscAlg_t CSR2CSC_ALG1 = HIPSPARSE_CSR2CSC_ALG1;
#endif

#if (CUDA_VERSION >= 10010 && CUDA_VERSION < 11000 && !defined(_WIN32)) || CUDA_VERSION >= 11000
  // CHECK: hipsparseSpMatDescr_t spMatDescr_t, matC;
  hipsparseSpMatDescr_t spMatDescr_t, matC;

  // CHECK: hipsparseDnMatDescr_t dnMatDescr_t, matA, matB;
  hipsparseDnMatDescr_t dnMatDescr_t, matA, matB;

  // CHECK: hipsparseIndexType_t indexType_t;
  // CHECK-NEXT: hipsparseIndexType_t csrRowOffsetsType;
  // CHECK-NEXT: hipsparseIndexType_t cscColOffsetsType;
  // CHECK-NEXT: hipsparseIndexType_t cscRowIndType;
  // CHECK-NEXT: hipsparseIndexType_t csrColIndType;
  // CHECK-NEXT: hipsparseIndexType_t ellIdxType;
  // CHECK-NEXT: hipsparseIndexType_t INDEX_16U = HIPSPARSE_INDEX_16U;
  // CHECK-NEXT: hipsparseIndexType_t INDEX_32I = HIPSPARSE_INDEX_32I;
  // CHECK-NEXT: hipsparseIndexType_t INDEX_64I = HIPSPARSE_INDEX_64I;
  hipsparseIndexType_t indexType_t;
  hipsparseIndexType_t csrRowOffsetsType;
  hipsparseIndexType_t cscColOffsetsType;
  hipsparseIndexType_t cscRowIndType;
  hipsparseIndexType_t csrColIndType;
  hipsparseIndexType_t ellIdxType;
  hipsparseIndexType_t INDEX_16U = HIPSPARSE_INDEX_16U;
  hipsparseIndexType_t INDEX_32I = HIPSPARSE_INDEX_32I;
  hipsparseIndexType_t INDEX_64I = HIPSPARSE_INDEX_64I;

  // CHECK: hipsparseFormat_t format_t;
  // CHECK-NEXT: hipsparseFormat_t FORMAT_CSR = HIPSPARSE_FORMAT_CSR;
  // CHECK-NEXT: hipsparseFormat_t FORMAT_CSC = HIPSPARSE_FORMAT_CSC;
  // CHECK-NEXT: hipsparseFormat_t FORMAT_CSO = HIPSPARSE_FORMAT_COO;
  hipsparseFormat_t format_t;
  hipsparseFormat_t FORMAT_CSR = HIPSPARSE_FORMAT_CSR;
  hipsparseFormat_t FORMAT_CSC = HIPSPARSE_FORMAT_CSC;
  hipsparseFormat_t FORMAT_CSO = HIPSPARSE_FORMAT_COO;

  // CHECK: hipsparseOrder_t order_t;
  // CHECK-NEXT: hipsparseOrder_t ORDER_COL = HIPSPARSE_ORDER_COL;
  // CHECK-NEXT: hipsparseOrder_t ORDER_ROW = HIPSPARSE_ORDER_ROW;
  hipsparseOrder_t order_t;
  hipsparseOrder_t ORDER_COL = HIPSPARSE_ORDER_COL;
  hipsparseOrder_t ORDER_ROW = HIPSPARSE_ORDER_ROW;

  // CHECK: hipsparseSpMMAlg_t spMMAlg_t;
  hipsparseSpMMAlg_t spMMAlg_t;

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseCreateCoo(cusparseSpMatDescr_t* spMatDescr, int64_t ows, int64_t cols, int64_t nnz, void* cooRowInd, void* cooColInd, void* cooValues, cusparseIndexType_t cooIdxType, cusparseIndexBase_t idxBase, cudaDataType valueType);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseCreateCoo(hipsparseSpMatDescr_t* spMatDescr, int64_t rows, int64_t cols, int64_t nnz, void* cooRowInd, void* cooColInd, void* cooValues, hipsparseIndexType_t cooIdxType, hipsparseIndexBase_t idxBase, hipDataType valueType);
  // CHECK: status_t = hipsparseCreateCoo(&spMatDescr_t, rows, cols, nnz, cooRowInd, cooColInd, cooValues, indexType_t, indexBase_t, dataType);
  status_t = hipsparseCreateCoo(&spMatDescr_t, rows, cols, nnz, cooRowInd, cooColInd, cooValues, indexType_t, indexBase_t, dataType);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseDestroySpMat(cusparseConstSpMatDescr_t spMatDescr);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseDestroySpMat(hipsparseSpMatDescr_t spMatDescr);
  // CHECK: status_t = hipsparseDestroySpMat(spMatDescr_t);
  status_t = hipsparseDestroySpMat(spMatDescr_t);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseCooGet(cusparseSpMatDescr_t spMatDescr, int64_t* rows, int64_t* cols, int64_t* nnz, void** cooRowInd, void** cooColInd, void** cooValues, cusparseIndexType_t* idxType, cusparseIndexBase_t* idxBase, cudaDataType* valueType);
  // HIP: hipsparseStatus_t hipsparseCooGet(const hipsparseSpMatDescr_t spMatDescr, int64_t* rows, int64_t* cols, int64_t* nnz, void** cooRowInd, void** cooColInd, void** cooValues, hipsparseIndexType_t* idxType, hipsparseIndexBase_t* idxBase, hipDataType* valueType);
  // CHECK: status_t = hipsparseCooGet(spMatDescr_t, &rows, &cols, &nnz, &cooRowInd, &cooColInd, &cooValues, &indexType_t, &indexBase_t, &dataType);
  status_t = hipsparseCooGet(spMatDescr_t, &rows, &cols, &nnz, &cooRowInd, &cooColInd, &cooValues, &indexType_t, &indexBase_t, &dataType);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseSpMatGetFormat(cusparseConstSpMatDescr_t spMatDescr, cusparseFormat_t* format);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseSpMatGetFormat(const hipsparseSpMatDescr_t spMatDescr, hipsparseFormat_t* format);
  // CHECK: status_t = hipsparseSpMatGetFormat(spMatDescr_t, &format_t);
  status_t = hipsparseSpMatGetFormat(spMatDescr_t, &format_t);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseSpMatGetIndexBase(cusparseConstSpMatDescr_t spMatDescr, cusparseIndexBase_t* idxBase);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseSpMatGetIndexBase(const hipsparseSpMatDescr_t spMatDescr, hipsparseIndexBase_t* idxBase);
  // CHECK: status_t = hipsparseSpMatGetIndexBase(spMatDescr_t, &indexBase_t);
  status_t = hipsparseSpMatGetIndexBase(spMatDescr_t, &indexBase_t);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseCreateDnMat(cusparseDnMatDescr_t* dnMatDescr, int64_t rows, int64_t cols, int64_t ld, void* values, cudaDataType valueType, cusparseOrder_t order);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseCreateDnMat(hipsparseDnMatDescr_t* dnMatDescr, int64_t rows, int64_t cols, int64_t ld, void* values, hipDataType valueType, hipsparseOrder_t order);
  // CHECK: status_t = hipsparseCreateDnMat(&dnMatDescr_t, rows, cols, ld, values, dataType, order_t);
  status_t = hipsparseCreateDnMat(&dnMatDescr_t, rows, cols, ld, values, dataType, order_t);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseDestroyDnMat(cusparseConstDnMatDescr_t dnMatDescr);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseDestroyDnMat(hipsparseDnMatDescr_t dnMatDescr);
  // CHECK: status_t = hipsparseDestroyDnMat(dnMatDescr_t);
  status_t = hipsparseDestroyDnMat(dnMatDescr_t);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseDnMatGet(cusparseDnMatDescr_t dnMatDescr, int64_t* rows, int64_t* cols, int64_t* ld, void** values, cudaDataType* type, cusparseOrder_t* order);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseDnMatGet(const hipsparseDnMatDescr_t dnMatDescr, int64_t* rows, int64_t* cols, int64_t* ld, void** values, hipDataType* valueType, hipsparseOrder_t* order);
  // CHECK: status_t = hipsparseDnMatGet(dnMatDescr_t, &rows, &cols, &ld, &values, &dataType, &order_t);
  status_t = hipsparseDnMatGet(dnMatDescr_t, &rows, &cols, &ld, &values, &dataType, &order_t);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseDnMatGetStridedBatch(cusparseConstDnMatDescr_t dnMatDescr, int* batchCount, int64_t* batchStride);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseDnMatGetStridedBatch(hipsparseDnMatDescr_t dnMatDescr, int* batchCount, int64_t* batchStride);
  // CHECK: status_t = hipsparseDnMatGetStridedBatch(dnMatDescr_t, &batchCount, &batchStride);
  status_t = hipsparseDnMatGetStridedBatch(dnMatDescr_t, &batchCount, &batchStride);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseDnMatSetStridedBatch(cusparseDnMatDescr_t dnMatDescr, int batchCount, int64_t batchStride);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseDnMatSetStridedBatch(hipsparseDnMatDescr_t dnMatDescr, int batchCount, int64_t batchStride);
  // CHECK: status_t = hipsparseDnMatSetStridedBatch(dnMatDescr_t, batchCount, batchStride);
  status_t = hipsparseDnMatSetStridedBatch(dnMatDescr_t, batchCount, batchStride);
#endif

#if (CUDA_VERSION >= 10010 && CUDA_VERSION < 11000 && !defined(_WIN32)) || (CUDA_VERSION >= 11000 && CUDA_VERSION < 12000)
  // CHECK: hipsparseSpMMAlg_t COOMM_ALG1 = HIPSPARSE_COOMM_ALG1;
  // CHECK-NEXT: hipsparseSpMMAlg_t COOMM_ALG2 = HIPSPARSE_COOMM_ALG2;
  // CHECK-NEXT: hipsparseSpMMAlg_t COOMM_ALG3 = HIPSPARSE_COOMM_ALG3;
  cusparseSpMMAlg_t COOMM_ALG1 = CUSPARSE_COOMM_ALG1;
  cusparseSpMMAlg_t COOMM_ALG2 = CUSPARSE_COOMM_ALG2;
  cusparseSpMMAlg_t COOMM_ALG3 = CUSPARSE_COOMM_ALG3;
#endif

#if CUDA_VERSION >= 10010 && CUDA_VERSION < 12000
  // CHECK: hipsparseCsr2CscAlg_t CSR2CSC_ALG2 = HIPSPARSE_CSR2CSC_ALG2;
  cusparseCsr2CscAlg_t CSR2CSC_ALG2 = CUSPARSE_CSR2CSC_ALG2;
#endif

#if (CUDA_VERSION >= 10020 && CUDA_VERSION < 11000 && !defined(_WIN32)) || CUDA_VERSION >= 11000
  // CHECK: hipsparseSpVecDescr_t spVecDescr_t;
  hipsparseSpVecDescr_t spVecDescr_t;

  // CHECK: hipsparseDnVecDescr_t dnVecDescr_t, vecX, vecY;
  hipsparseDnVecDescr_t dnVecDescr_t, vecX, vecY;

  // CHECK: hipsparseSpMVAlg_t spMVAlg_t;
  hipsparseSpMVAlg_t spMVAlg_t;

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseCreateSpVec(cusparseSpVecDescr_t* spVecDescr, int64_t size, int64_t nnz, void* indices, void* values, cusparseIndexType_t idxType, cusparseIndexBase_t idxBase, cudaDataType valueType);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseCreateSpVec(hipsparseSpVecDescr_t* spVecDescr, int64_t size, int64_t nnz, void* indices, void* values, hipsparseIndexType_t idxType, hipsparseIndexBase_t idxBase, hipDataType valueType);
  // CHECK: status_t = hipsparseCreateSpVec(&spVecDescr_t, size, nnz, indices, values, indexType_t, indexBase_t, dataType);
  status_t = hipsparseCreateSpVec(&spVecDescr_t, size, nnz, indices, values, indexType_t, indexBase_t, dataType);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseDestroySpVec(cusparseConstSpVecDescr_t spVecDescr);
  // HIP: hipsparseStatus_t hipsparseDestroySpVec(hipsparseSpVecDescr_t spVecDescr);
  // CHECK: status_t = hipsparseDestroySpVec(spVecDescr_t);
  status_t = hipsparseDestroySpVec(spVecDescr_t);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseSpVecGet(cusparseSpVecDescr_t spVecDescr, int64_t* size, int64_t* nnz, void** indices, void** values, cusparseIndexType_t* idxType, cusparseIndexBase_t* idxBase, cudaDataType* valueType);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseSpVecGet(const hipsparseSpVecDescr_t spVecDescr, int64_t* size, int64_t* nnz, void** indices, void** values, hipsparseIndexType_t* idxType, hipsparseIndexBase_t* idxBase, hipDataType* valueType);
  // CHECK: status_t = hipsparseSpVecGet(spVecDescr_t, &size, &nnz, &indices, &values, &indexType_t, &indexBase_t, &dataType);
  status_t = hipsparseSpVecGet(spVecDescr_t, &size, &nnz, &indices, &values, &indexType_t, &indexBase_t, &dataType);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseSpVecGetIndexBase(cusparseConstSpVecDescr_t spVecDescr, cusparseIndexBase_t* idxBase);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseSpVecGetIndexBase(const hipsparseSpVecDescr_t spVecDescr, hipsparseIndexBase_t* idxBase);
  // CHECK: status_t = hipsparseSpVecGetIndexBase(spVecDescr_t, &indexBase_t);
  status_t = hipsparseSpVecGetIndexBase(spVecDescr_t, &indexBase_t);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseSpVecGetValues(cusparseSpVecDescr_t spVecDescr, void** values);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseSpVecGetValues(const hipsparseSpVecDescr_t spVecDescr, void** values);
  // CHECK: status_t = hipsparseSpVecGetValues(spVecDescr_t, &values);
  status_t = hipsparseSpVecGetValues(spVecDescr_t, &values);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseSpVecSetValues(cusparseSpVecDescr_t spVecDescr, void* values);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseSpVecSetValues(hipsparseSpVecDescr_t spVecDescr, void* values);
  // CHECK: status_t = hipsparseSpVecSetValues(spVecDescr_t, values);
  status_t = hipsparseSpVecSetValues(spVecDescr_t, values);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseCreateCsr(cusparseSpMatDescr_t* spMatDescr, int64_t rows, int64_t cols, int64_t nnz, void* csrRowOffsets, void* csrColInd, void* csrValues, cusparseIndexType_t csrRowOffsetsType, cusparseIndexType_t csrColIndType, cusparseIndexBase_t idxBase, cudaDataType valueType);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseCreateCsr(hipsparseSpMatDescr_t* spMatDescr, int64_t rows, int64_t cols, int64_t nnz, void* csrRowOffsets, void* csrColInd, void* csrValues, hipsparseIndexType_t csrRowOffsetsType, hipsparseIndexType_t csrColIndType, hipsparseIndexBase_t idxBase, hipDataType valueType);
  // CHECK: status_t = hipsparseCreateCsr(&spMatDescr_t, rows, cols, nnz, csrRowOffsets, csrColInd, csrValues, csrRowOffsetsType, csrColIndType, indexBase_t, dataType);
  status_t = hipsparseCreateCsr(&spMatDescr_t, rows, cols, nnz, csrRowOffsets, csrColInd, csrValues, csrRowOffsetsType, csrColIndType, indexBase_t, dataType);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseCsrGet(cusparseSpMatDescr_t spMatDescr, int64_t* rows, int64_t* cols, int64_t* nnz, void** csrRowOffsets, void** csrColInd, void** csrValues, cusparseIndexType_t* csrRowOffsetsType, cusparseIndexType_t* csrColIndType, cusparseIndexBase_t* idxBase, cudaDataType* valueType);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseCsrGet(const hipsparseSpMatDescr_t spMatDescr, int64_t* rows, int64_t* cols, int64_t* nnz, void** csrRowOffsets, void** csrColInd, void** csrValues, hipsparseIndexType_t* csrRowOffsetsType, hipsparseIndexType_t* csrColIndType, hipsparseIndexBase_t* idxBase, hipDataType* valueType);
  // CHECK: status_t = hipsparseCsrGet(spMatDescr_t, &rows, &cols, &nnz, &csrRowOffsets, &csrColInd, &csrValues, &csrRowOffsetsType, &csrColIndType, &indexBase_t, &dataType);
  status_t = hipsparseCsrGet(spMatDescr_t, &rows, &cols, &nnz, &csrRowOffsets, &csrColInd, &csrValues, &csrRowOffsetsType, &csrColIndType, &indexBase_t, &dataType);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseSpMatGetValues(cusparseSpMatDescr_t spMatDescr, void** values);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseSpMatGetValues(hipsparseSpMatDescr_t spMatDescr, void** values);
  // CHECK: status_t = hipsparseSpMatGetValues(spMatDescr_t, &values);
  status_t = hipsparseSpMatGetValues(spMatDescr_t, &values);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseSpMatSetValues(cusparseSpMatDescr_t spMatDescr, void* values);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseSpMatSetValues(hipsparseSpMatDescr_t spMatDescr, void* values);
  // CHECK: status_t = hipsparseSpMatSetValues(spMatDescr_t, values);
  status_t = hipsparseSpMatSetValues(spMatDescr_t, values);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseSpMatGetStridedBatch(cusparseConstSpMatDescr_t spMatDescr, int* batchCount);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseSpMatGetStridedBatch(hipsparseSpMatDescr_t spMatDescr, int* batchCount);
  // CHECK: status_t = hipsparseSpMatGetStridedBatch(spMatDescr_t, &batchCount);
  status_t = hipsparseSpMatGetStridedBatch(spMatDescr_t, &batchCount);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseCreateDnVec(cusparseDnVecDescr_t* dnVecDescr, int64_t size, void* values, cudaDataType valueType);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseCreateDnVec(hipsparseDnVecDescr_t* dnVecDescr, int64_t size, void* values, hipDataType valueType);
  // CHECK: status_t = hipsparseCreateDnVec(&dnVecDescr_t, size, values, dataType);
  status_t = hipsparseCreateDnVec(&dnVecDescr_t, size, values, dataType);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseDestroyDnVec(cusparseConstDnVecDescr_t dnVecDescr);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseDestroyDnVec(hipsparseDnVecDescr_t dnVecDescr);
  // CHECK: status_t = hipsparseDestroyDnVec(dnVecDescr_t);
  status_t = hipsparseDestroyDnVec(dnVecDescr_t);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseDnVecGet(cusparseDnVecDescr_t dnVecDescr, int64_t* size, void** values, cudaDataType* valueType);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseDnVecGet(const hipsparseDnVecDescr_t dnVecDescr, int64_t* size, void** values, hipDataType* valueType);
  // CHECK: status_t = hipsparseDnVecGet(dnVecDescr_t, &size, &values, &dataType);
  status_t = hipsparseDnVecGet(dnVecDescr_t, &size, &values, &dataType);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseDnVecGetValues(cusparseDnVecDescr_t dnVecDescr, void** values);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseDnVecGetValues(const hipsparseDnVecDescr_t dnVecDescr, void** values);
  // CHECK: status_t = hipsparseDnVecGetValues(dnVecDescr_t, &values);
  status_t = hipsparseDnVecGetValues(dnVecDescr_t, &values);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseDnVecSetValues(cusparseDnVecDescr_t dnVecDescr, void* values);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseDnVecSetValues(hipsparseDnVecDescr_t dnVecDescr, void* values);
  // CHECK: status_t = hipsparseDnVecSetValues(dnVecDescr_t, values);
  status_t = hipsparseDnVecSetValues(dnVecDescr_t, values);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseDnMatGetValues(cusparseDnMatDescr_t dnMatDescr, void** values);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseDnMatGetValues(const hipsparseDnMatDescr_t dnMatDescr, void** values);
  // CHECK: status_t = hipsparseDnMatGetValues(dnMatDescr_t, &values);
  status_t = hipsparseDnMatGetValues(dnMatDescr_t, &values);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseDnMatSetValues(cusparseDnMatDescr_t dnMatDescr, void* values);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseDnMatSetValues(hipsparseDnMatDescr_t dnMatDescr, void* values);
  // CHECK: status_t = hipsparseDnMatSetValues(dnMatDescr_t, values);
  status_t = hipsparseDnMatSetValues(dnMatDescr_t, values);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseSpMV(cusparseHandle_t handle, cusparseOperation_t opA, const void* alpha, cusparseConstSpMatDescr_t matA, cusparseConstDnVecDescr_t vecX, const void* beta, cusparseDnVecDescr_t vecY, cudaDataType computeType, cusparseSpMVAlg_t alg, void* externalBuffer);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseSpMV(hipsparseHandle_t handle, hipsparseOperation_t opA, const void* alpha, const hipsparseSpMatDescr_t matA, const hipsparseDnVecDescr_t vecX, const void* beta, const hipsparseDnVecDescr_t vecY, hipDataType computeType, hipsparseSpMVAlg_t alg, void* externalBuffer);
  // CHECK: status_t = hipsparseSpMV(handle_t, opA, alpha, spMatDescr_t, vecX, beta, vecY, dataType, spMVAlg_t, tempBuffer);
  status_t = hipsparseSpMV(handle_t, opA, alpha, spMatDescr_t, vecX, beta, vecY, dataType, spMVAlg_t, tempBuffer);
#endif

#if CUDA_VERSION >= 10020
  // CHECK: hipsparseStatus_t STATUS_NOT_SUPPORTED = HIPSPARSE_STATUS_NOT_SUPPORTED;
  hipsparseStatus_t STATUS_NOT_SUPPORTED = HIPSPARSE_STATUS_NOT_SUPPORTED;
#endif

#if (CUDA_VERSION >= 10020 && CUDA_VERSION < 11000 && !defined(_WIN32)) || (CUDA_VERSION >= 11000 && CUDA_VERSION < 12000)
  // CHECK: hipsparseFormat_t FORMAT_COO_AOS = HIPSPARSE_FORMAT_COO_AOS;
  cusparseFormat_t FORMAT_COO_AOS = CUSPARSE_FORMAT_COO_AOS;

  // CHECK: hipsparseSpMVAlg_t MV_ALG_DEFAULT = HIPSPARSE_MV_ALG_DEFAULT;
  cusparseSpMVAlg_t MV_ALG_DEFAULT = CUSPARSE_MV_ALG_DEFAULT;

  // CHECK: hipsparseSpMVAlg_t COOMV_ALG = HIPSPARSE_COOMV_ALG;
  // CHECK-NEXT: hipsparseSpMVAlg_t CSRMV_ALG1 = HIPSPARSE_CSRMV_ALG1;
  // CHECK-NEXT: hipsparseSpMVAlg_t CSRMV_ALG2 = HIPSPARSE_CSRMV_ALG2;
  cusparseSpMVAlg_t COOMV_ALG = CUSPARSE_COOMV_ALG;
  cusparseSpMVAlg_t CSRMV_ALG1 = CUSPARSE_CSRMV_ALG1;
  cusparseSpMVAlg_t CSRMV_ALG2 = CUSPARSE_CSRMV_ALG2;

  // CHECK: hipsparseSpMMAlg_t MM_ALG_DEFAULT = HIPSPARSE_MM_ALG_DEFAULT;
  // CHECK: hipsparseSpMMAlg_t CSRMM_ALG1 = HIPSPARSE_CSRMM_ALG1;
  cusparseSpMMAlg_t MM_ALG_DEFAULT = CUSPARSE_MM_ALG_DEFAULT;
  cusparseSpMMAlg_t CSRMM_ALG1 = CUSPARSE_CSRMM_ALG1;

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseCreateCooAoS(cusparseSpMatDescr_t* spMatDescr, int64_t rows, int64_t cols, int64_t nnz, void* cooInd, void* cooValues, cusparseIndexType_t cooIdxType, cusparseIndexBase_t idxBase, cudaDataType valueType);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseCreateCooAoS(hipsparseSpMatDescr_t* spMatDescr, int64_t rows, int64_t cols, int64_t nnz, void* cooInd, void* cooValues, hipsparseIndexType_t cooIdxType, hipsparseIndexBase_t idxBase, hipDataType valueType);
  // CHECK: status_t = hipsparseCreateCooAoS(&spMatDescr_t, rows, cols, nnz, cooColInd, cooValues, indexType_t, indexBase_t, dataType);
  status_t = cusparseCreateCooAoS(&spMatDescr_t, rows, cols, nnz, cooColInd, cooValues, indexType_t, indexBase_t, dataType);

  // CUDA: CUSPARSE_DEPRECATED(cusparseCooGet) cusparseStatus_t CUSPARSEAPI cusparseCooAoSGet(cusparseSpMatDescr_t spMatDescr, int64_t* rows, int64_t* cols, int64_t* nnz, void** cooInd, void** cooValues, cusparseIndexType_t* idxType, cusparseIndexBase_t* idxBase, cudaDataType* valueType);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseCooAoSGet(const hipsparseSpMatDescr_t spMatDescr, int64_t* rows, int64_t* cols, int64_t* nnz, void** cooInd, void** cooValues, hipsparseIndexType_t* idxType, hipsparseIndexBase_t* idxBase, hipDataType* valueType);
  // CHECK: status_t = hipsparseCooAoSGet(spMatDescr_t, &rows, &cols, &nnz, &cooColInd, &cooValues, &indexType_t, &indexBase_t, &dataType);
  status_t = cusparseCooAoSGet(spMatDescr_t, &rows, &cols, &nnz, &cooColInd, &cooValues, &indexType_t, &indexBase_t, &dataType);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseSpMatSetStridedBatch(cusparseSpMatDescr_t spMatDescr, int batchCount);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseSpMatSetStridedBatch(hipsparseSpMatDescr_t spMatDescr, int batchCount);
  // CHECK: status_t = hipsparseSpMatSetStridedBatch(spMatDescr_t, batchCount);
  status_t = cusparseSpMatSetStridedBatch(spMatDescr_t, batchCount);
#endif

#if CUDA_VERSION < 11000
  // CHECK: hipsparseHybMat_t hybMat_t;
  cusparseHybMat_t hybMat_t;

  // CHECK: hipsparseHybPartition_t hybPartition_t;
  // CHECK-NEXT: hipsparseHybPartition_t HYB_PARTITION_AUTO = HIPSPARSE_HYB_PARTITION_AUTO;
  // CHECK-NEXT: hipsparseHybPartition_t HYB_PARTITION_USER = HIPSPARSE_HYB_PARTITION_USER;
  // CHECK-NEXT: hipsparseHybPartition_t HYB_PARTITION_MAX = HIPSPARSE_HYB_PARTITION_MAX;
  cusparseHybPartition_t hybPartition_t;
  cusparseHybPartition_t HYB_PARTITION_AUTO = CUSPARSE_HYB_PARTITION_AUTO;
  cusparseHybPartition_t HYB_PARTITION_USER = CUSPARSE_HYB_PARTITION_USER;
  cusparseHybPartition_t HYB_PARTITION_MAX = CUSPARSE_HYB_PARTITION_MAX;

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseCreateHybMat(cusparseHybMat_t* hybA);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseCreateHybMat(hipsparseHybMat_t* hybA);
  // CHECK: status_t = hipsparseCreateHybMat(&hybMat_t);
  status_t = cusparseCreateHybMat(&hybMat_t);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseDestroyHybMat(cusparseHybMat_t hybA);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseDestroyHybMat(hipsparseHybMat_t hybA);
  // CHECK: status_t = hipsparseDestroyHybMat(hybMat_t);
  status_t = cusparseDestroyHybMat(hybMat_t);

  // CUDA: CUSPARSE_DEPRECATED cusparseStatus_t CUSPARSEAPI cusparseZhyb2csr(cusparseHandle_t handle, const cusparseMatDescr_t descrA, const cusparseHybMat_t hybA, cuDoubleComplex* csrSortedValA, int* csrSortedRowPtrA, int* csrSortedColIndA);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseZhyb2csr(hipsparseHandle_t handle, const hipsparseMatDescr_t descrA, const hipsparseHybMat_t hybA, hipDoubleComplex* csrSortedValA, int* csrSortedRowPtrA, int* csrSortedColIndA);
  // CHECK: status_t = hipsparseZhyb2csr(handle_t, matDescr_t, hybMat_t, &dComplexbsrSortedValA, &csrRowPtrA, &csrColIndA);
  status_t = cusparseZhyb2csr(handle_t, matDescr_t, hybMat_t, &dComplexbsrSortedValA, &csrRowPtrA, &csrColIndA);

  // CUDA: CUSPARSE_DEPRECATED cusparseStatus_t CUSPARSEAPI cusparseChyb2csr(cusparseHandle_t handle, const cusparseMatDescr_t descrA, const cusparseHybMat_t hybA, cuComplex* csrSortedValA, int* csrSortedRowPtrA, int* csrSortedColIndA);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseChyb2csr(hipsparseHandle_t handle, const hipsparseMatDescr_t descrA, const hipsparseHybMat_t hybA, hipComplex* csrSortedValA, int* csrSortedRowPtrA, int* csrSortedColIndA);
  // CHECK: status_t = hipsparseChyb2csr(handle_t, matDescr_t, hybMat_t, &complex, &csrRowPtrA, &csrColIndA);
  status_t = cusparseChyb2csr(handle_t, matDescr_t, hybMat_t, &complex, &csrRowPtrA, &csrColIndA);

  // CUDA: CUSPARSE_DEPRECATED cusparseStatus_t CUSPARSEAPI cusparseDhyb2csr(cusparseHandle_t handle, const cusparseMatDescr_t descrA, const cusparseHybMat_t hybA, double* csrSortedValA, int* csrSortedRowPtrA, int* csrSortedColIndA);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseDhyb2csr(hipsparseHandle_t handle, const hipsparseMatDescr_t descrA, const hipsparseHybMat_t hybA, double* csrSortedValA, int* csrSortedRowPtrA, int* csrSortedColIndA);
  // CHECK: status_t = hipsparseDhyb2csr(handle_t, matDescr_t, hybMat_t, &csrValA, &csrRowPtrA, &csrColIndA);
  status_t = cusparseDhyb2csr(handle_t, matDescr_t, hybMat_t, &csrValA, &csrRowPtrA, &csrColIndA);

  // CUDA: CUSPARSE_DEPRECATED cusparseStatus_t CUSPARSEAPI cusparseShyb2csr(cusparseHandle_t handle, const cusparseMatDescr_t descrA, const cusparseHybMat_t hybA, float* csrSortedValA, int* csrSortedRowPtrA, int* csrSortedColIndA);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseShyb2csr(hipsparseHandle_t handle, const hipsparseMatDescr_t descrA, const hipsparseHybMat_t hybA, float* csrSortedValA, int* csrSortedRowPtrA, int* csrSortedColIndA);
  // CHECK: status_t = hipsparseShyb2csr(handle_t, matDescr_t, hybMat_t, &fcsrValA, &csrRowPtrA, &csrColIndA);
  status_t = cusparseShyb2csr(handle_t, matDescr_t, hybMat_t, &fcsrValA, &csrRowPtrA, &csrColIndA);

  // CUDA: CUSPARSE_DEPRECATED cusparseStatus_t CUSPARSEAPI cusparseZcsr2hyb(cusparseHandle_t handle, int m, int n, const cusparseMatDescr_t descrA, const cuDoubleComplex* csrSortedValA, const int* csrSortedRowPtrA, const int* csrSortedColIndA, cusparseHybMat_t hybA, int userEllWidth, cusparseHybPartition_t   partitionType);
  // HIP: DEPRECATED_CUDA_10000("The routine will be removed in CUDA 11") HIPSPARSE_EXPORT hipsparseStatus_t hipsparseZcsr2hyb(hipsparseHandle_t handle, int m, int n, const hipsparseMatDescr_t descrA, const hipDoubleComplex* csrSortedValA, const int* csrSortedRowPtrA, const int* csrSortedColIndA, hipsparseHybMat_t hybA, int userEllWidth, hipsparseHybPartition_t partitionType);
  // CHECK: status_t = hipsparseZcsr2hyb(handle_t, m, n, matDescr_A, &dComplexcsrSortedValA, &csrRowPtrA, &csrColIndA, hybMat_t, userEllWidth, hybPartition_t);
  status_t = cusparseZcsr2hyb(handle_t, m, n, matDescr_A, &dComplexcsrSortedValA, &csrRowPtrA, &csrColIndA, hybMat_t, userEllWidth, hybPartition_t);

  // CUDA: CUSPARSE_DEPRECATED cusparseStatus_t CUSPARSEAPI cusparseCcsr2hyb(cusparseHandle_t handle, int m, int n, const cusparseMatDescr_t descrA, const cuComplex* csrSortedValA, const int* csrSortedRowPtrA, const int* csrSortedColIndA, cusparseHybMat_t hybA, int userEllWidth, cusparseHybPartition_t partitionType);
  // HIP: DEPRECATED_CUDA_10000("The routine will be removed in CUDA 11") HIPSPARSE_EXPORT hipsparseStatus_t hipsparseCcsr2hyb(hipsparseHandle_t handle, int m, int n, const hipsparseMatDescr_t descrA, const hipComplex* csrSortedValA, const int* csrSortedRowPtrA, const int* csrSortedColIndA, hipsparseHybMat_t hybA, int userEllWidth, hipsparseHybPartition_t partitionType);
  // CHECK: status_t = hipsparseCcsr2hyb(handle_t, m, n, matDescr_A, &complexcsrSortedValA, &csrRowPtrA, &csrColIndA, hybMat_t, userEllWidth, hybPartition_t);
  status_t = cusparseCcsr2hyb(handle_t, m, n, matDescr_A, &complexcsrSortedValA, &csrRowPtrA, &csrColIndA, hybMat_t, userEllWidth, hybPartition_t);

  // CUDA: CUSPARSE_DEPRECATED cusparseStatus_t CUSPARSEAPI cusparseDcsr2hyb(cusparseHandle_t handle, int m, int n, const cusparseMatDescr_t descrA, const double* csrSortedValA, const int* csrSortedRowPtrA, const int* csrSortedColIndA, cusparseHybMat_t hybA, int userEllWidth, cusparseHybPartition_t partitionType);
  // HIP: DEPRECATED_CUDA_10000("The routine will be removed in CUDA 11") HIPSPARSE_EXPORT hipsparseStatus_t hipsparseDcsr2hyb(hipsparseHandle_t handle, int m, int n, const hipsparseMatDescr_t descrA, const double* csrSortedValA, const int* csrSortedRowPtrA, const int* csrSortedColIndA, hipsparseHybMat_t hybA, int userEllWidth, hipsparseHybPartition_t partitionType);
  // CHECK: status_t = hipsparseDcsr2hyb(handle_t, m, n, matDescr_A, &dcsrSortedValA, &csrRowPtrA, &csrColIndA, hybMat_t, userEllWidth, hybPartition_t);
  status_t = cusparseDcsr2hyb(handle_t, m, n, matDescr_A, &dcsrSortedValA, &csrRowPtrA, &csrColIndA, hybMat_t, userEllWidth, hybPartition_t);

  // CUDA: CUSPARSE_DEPRECATED cusparseStatus_t CUSPARSEAPI cusparseScsr2hyb(cusparseHandle_t handle, int m, int n, const cusparseMatDescr_t descrA, const float* csrSortedValA, const int* csrSortedRowPtrA, const int* csrSortedColIndA, cusparseHybMat_t hybA, int userEllWidth, cusparseHybPartition_t partitionType);
  // HIP: DEPRECATED_CUDA_10000("The routine will be removed in CUDA 11") HIPSPARSE_EXPORT hipsparseStatus_t hipsparseScsr2hyb(hipsparseHandle_t handle, int m, int n, const hipsparseMatDescr_t descrA, const float* csrSortedValA, const int* csrSortedRowPtrA, const int* csrSortedColIndA, hipsparseHybMat_t hybA, int userEllWidth, hipsparseHybPartition_t partitionType);
  // CHECK: status_t = hipsparseScsr2hyb(handle_t, m, n, matDescr_A, &csrSortedValA, &csrRowPtrA, &csrColIndA, hybMat_t, userEllWidth, hybPartition_t);
  status_t = cusparseScsr2hyb(handle_t, m, n, matDescr_A, &csrSortedValA, &csrRowPtrA, &csrColIndA, hybMat_t, userEllWidth, hybPartition_t);

  // CUDA: CUSPARSE_DEPRECATED_HINT(cusparseCsr2cscEx2) cusparseStatus_t CUSPARSEAPI cusparseZcsr2csc(cusparseHandle_t handle, int m, int n, int nnz, const cuDoubleComplex* csrSortedVal, const int* csrSortedRowPtr, const int* csrSortedColInd, cuDoubleComplex* cscSortedVal, int* cscSortedRowInd, int* cscSortedColPtr, cusparseAction_t copyValues, cusparseIndexBase_t idxBase);
  // HIP: DEPRECATED_CUDA_10000("The routine will be removed in CUDA 11") HIPSPARSE_EXPORT hipsparseStatus_t hipsparseZcsr2csc(hipsparseHandle_t handle, int m, int n, int nnz, const hipDoubleComplex* csrSortedVal, const int* csrSortedRowPtr, const int* csrSortedColInd, hipDoubleComplex* cscSortedVal, int* cscSortedRowInd, int* cscSortedColPtr, hipsparseAction_t copyValues, hipsparseIndexBase_t idxBase);
  // CHECK: status_t = hipsparseZcsr2csc(handle_t, m, n, nnz, &dComplexcsrSortedValA, &csrRowPtrA, &csrColIndA, &dComplexcscSortedVal, &csrSortedRowPtr, &csrSortedColInd, copyValues, indexBase_t);
  status_t = cusparseZcsr2csc(handle_t, m, n, nnz, &dComplexcsrSortedValA, &csrRowPtrA, &csrColIndA, &dComplexcscSortedVal, &csrSortedRowPtr, &csrSortedColInd, copyValues, indexBase_t);

  // CUDA: CUSPARSE_DEPRECATED_HINT(cusparseCsr2cscEx2) cusparseStatus_t CUSPARSEAPI cusparseCcsr2csc(cusparseHandle_t handle, int m, int n, int nnz, const cuComplex* csrSortedVal, const int* csrSortedRowPtr, const int* csrSortedColInd, cuComplex* cscSortedVal, int* cscSortedRowInd, int* cscSortedColPtr,cusparseAction_t copyValues, cusparseIndexBase_t idxBase);
  // HIP: DEPRECATED_CUDA_10000("The routine will be removed in CUDA 11") HIPSPARSE_EXPORT hipsparseStatus_t hipsparseCcsr2csc(hipsparseHandle_t handle, int m, int n, int nnz, const hipComplex* csrSortedVal, const int* csrSortedRowPtr, const int* csrSortedColInd, hipComplex* cscSortedVal, int* cscSortedRowInd, int* cscSortedColPtr, hipsparseAction_t copyValues, hipsparseIndexBase_t idxBase);
  // CHECK: status_t = hipsparseCcsr2csc(handle_t, m, n, nnz, &complexcsrSortedValA, &csrRowPtrA, &csrColIndA, &ñomplexcscSortedVal, &csrSortedRowPtr, &csrSortedColInd, copyValues, indexBase_t);
  status_t = cusparseCcsr2csc(handle_t, m, n, nnz, &complexcsrSortedValA, &csrRowPtrA, &csrColIndA, &ñomplexcscSortedVal, &csrSortedRowPtr, &csrSortedColInd, copyValues, indexBase_t);

  // CUDA: CUSPARSE_DEPRECATED_HINT(cusparseCsr2cscEx2) cusparseStatus_t CUSPARSEAPI cusparseDcsr2csc(cusparseHandle_t handle, int m, int n, int nnz, const double* csrSortedVal, const int* csrSortedRowPtr, const int* csrSortedColInd, double* cscSortedVal, int* cscSortedRowInd, int* cscSortedColPtr, cusparseAction_t copyValues, cusparseIndexBase_t idxBase);
  // HIP: DEPRECATED_CUDA_10000("The routine will be removed in CUDA 11") HIPSPARSE_EXPORT hipsparseStatus_t hipsparseDcsr2csc(hipsparseHandle_t handle, int m, int n, int nnz, const double* csrSortedVal, const int* csrSortedRowPtr, const int* csrSortedColInd, double* cscSortedVal, int* cscSortedRowInd, int* cscSortedColPtr, hipsparseAction_t copyValues, hipsparseIndexBase_t idxBase);
  // CHECK: status_t = hipsparseDcsr2csc(handle_t, m, n, nnz, &dcsrSortedVal, &csrRowPtrA, &csrColIndA, &dcscSortedVal, &csrSortedRowPtr, &csrSortedColInd, copyValues, indexBase_t);
  status_t = cusparseDcsr2csc(handle_t, m, n, nnz, &dcsrSortedVal, &csrRowPtrA, &csrColIndA, &dcscSortedVal, &csrSortedRowPtr, &csrSortedColInd, copyValues, indexBase_t);

  // CUDA: CUSPARSE_DEPRECATED_HINT(cusparseCsr2cscEx2) cusparseStatus_t CUSPARSEAPI cusparseScsr2csc(cusparseHandle_t handle, int m, int n, int nnz, const float* csrSortedVal, const int* csrSortedRowPtr, const int* csrSortedColInd, float* cscSortedVal, int* cscSortedRowInd, int* cscSortedColPtr, cusparseAction_t copyValues, cusparseIndexBase_t idxBase);
  // HIP: DEPRECATED_CUDA_10000("The routine will be removed in CUDA 11") HIPSPARSE_EXPORT hipsparseStatus_t hipsparseScsr2csc(hipsparseHandle_t handle, int m, int n, int nnz, const float* csrSortedVal, const int* csrSortedRowPtr, const int* csrSortedColInd, float* cscSortedVal, int* cscSortedRowInd, int* cscSortedColPtr, hipsparseAction_t copyValues, hipsparseIndexBase_t idxBase);
  // CHECK: status_t = hipsparseScsr2csc(handle_t, m, n, nnz, &csrSortedVal, &csrRowPtrA, &csrColIndA, &cscSortedVal, &csrSortedRowPtr, &csrSortedColInd, copyValues, indexBase_t);
  status_t = cusparseScsr2csc(handle_t, m, n, nnz, &csrSortedVal, &csrRowPtrA, &csrColIndA, &cscSortedVal, &csrSortedRowPtr, &csrSortedColInd, copyValues, indexBase_t);
#endif

#if CUDA_VERSION >= 11000
  // CHECK: hipsparseStatus_t STATUS_INSUFFICIENT_RESOURCES = HIPSPARSE_STATUS_INSUFFICIENT_RESOURCES;
  hipsparseStatus_t STATUS_INSUFFICIENT_RESOURCES = HIPSPARSE_STATUS_INSUFFICIENT_RESOURCES;

  // CHECK: hipsparseSpGEMMAlg_t spGEMMAlg_t;
  // CHECK-NEXT: hipsparseSpGEMMAlg_t SPGEMM_DEFAULT = HIPSPARSE_SPGEMM_DEFAULT;
  hipsparseSpGEMMAlg_t spGEMMAlg_t;
  hipsparseSpGEMMAlg_t SPGEMM_DEFAULT = HIPSPARSE_SPGEMM_DEFAULT;

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseCsrSetPointers(cusparseSpMatDescr_t spMatDescr, void* csrRowOffsets, void* csrColInd, void* csrValues);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseCsrSetPointers(hipsparseSpMatDescr_t spMatDescr, void* csrRowOffsets, void* csrColInd, void* csrValues);
  // CHECK: status_t = hipsparseCsrSetPointers(spMatDescr_t, csrRowOffsets, csrColInd, csrValues);
  status_t = hipsparseCsrSetPointers(spMatDescr_t, csrRowOffsets, csrColInd, csrValues);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseSpMatGetSize(cusparseConstSpMatDescr_t spMatDescr, int64_t* rows, int64_t* cols, int64_t* nnz);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseSpMatGetSize(hipsparseSpMatDescr_t spMatDescr, int64_t* rows, int64_t* cols, int64_t* nnz);
  // CHECK: status_t = hipsparseSpMatGetSize(spMatDescr_t, &rows, &cols, &nnz);
  status_t = hipsparseSpMatGetSize(spMatDescr_t, &rows, &cols, &nnz);
#endif

#if CUDA_VERSION >= 11000 && CUSPARSE_VERSION >= 11100
  // CHECK: hipsparseSpMMAlg_t SPMM_ALG_DEFAULT = HIPSPARSE_SPMM_ALG_DEFAULT;
  // CHECK-NEXT: hipsparseSpMMAlg_t SPMM_COO_ALG1 = HIPSPARSE_SPMM_COO_ALG1;
  // CHECK-NEXT: hipsparseSpMMAlg_t SPMM_COO_ALG2 = HIPSPARSE_SPMM_COO_ALG2;
  // CHECK-NEXT: hipsparseSpMMAlg_t SPMM_COO_ALG3 = HIPSPARSE_SPMM_COO_ALG3;
  // CHECK-NEXT: hipsparseSpMMAlg_t SPMM_CSR_ALG1 = HIPSPARSE_SPMM_CSR_ALG1;
  // CHECK-NEXT: hipsparseSpMMAlg_t SPMM_COO_ALG4 = HIPSPARSE_SPMM_COO_ALG4;
  // CHECK-NEXT: hipsparseSpMMAlg_t SPMM_CSR_ALG2 = HIPSPARSE_SPMM_CSR_ALG2;
  hipsparseSpMMAlg_t SPMM_ALG_DEFAULT = HIPSPARSE_SPMM_ALG_DEFAULT;
  hipsparseSpMMAlg_t SPMM_COO_ALG1 = HIPSPARSE_SPMM_COO_ALG1;
  hipsparseSpMMAlg_t SPMM_COO_ALG2 = HIPSPARSE_SPMM_COO_ALG2;
  hipsparseSpMMAlg_t SPMM_COO_ALG3 = HIPSPARSE_SPMM_COO_ALG3;
  hipsparseSpMMAlg_t SPMM_CSR_ALG1 = HIPSPARSE_SPMM_CSR_ALG1;
  hipsparseSpMMAlg_t SPMM_COO_ALG4 = HIPSPARSE_SPMM_COO_ALG4;
  hipsparseSpMMAlg_t SPMM_CSR_ALG2 = HIPSPARSE_SPMM_CSR_ALG2;

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseCooSetStridedBatch(cusparseSpMatDescr_t spMatDescr, int batchCount, int64_t batchStride);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseCooSetStridedBatch(hipsparseSpMatDescr_t spMatDescr, int batchCount, int64_t batchStride);
  // CHECK: status_t = hipsparseCooSetStridedBatch(spMatDescr_t, batchCount, batchStride);
  status_t = hipsparseCooSetStridedBatch(spMatDescr_t, batchCount, batchStride);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseCsrSetStridedBatch(cusparseSpMatDescr_t spMatDescr, int batchCount, int64_t offsetsBatchStride, int64_t columnsValuesBatchStride);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseCsrSetStridedBatch(hipsparseSpMatDescr_t spMatDescr, int batchCount, int64_t offsetsBatchStride, int64_t columnsValuesBatchStride);
  // CHECK: status_t = hipsparseCsrSetStridedBatch(spMatDescr_t, batchCount, offsetsBatchStride, columnsValuesBatchStride);
  status_t = hipsparseCsrSetStridedBatch(spMatDescr_t, batchCount, offsetsBatchStride, columnsValuesBatchStride);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseRot(cusparseHandle_t handle, const void* c_coeff, const void* s_coeff, cusparseSpVecDescr_t vecX, cusparseDnVecDescr_t vecY);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseRot(hipsparseHandle_t handle, const void* c_coeff, const void* s_coeff, hipsparseSpVecDescr_t vecX, hipsparseDnVecDescr_t vecY);
  // CHECK: status_t = hipsparseRot(handle_t, c_coeff, s_coeff, spVecDescr_t, vecY);
  status_t = hipsparseRot(handle_t, c_coeff, s_coeff, spVecDescr_t, vecY);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseScatter(cusparseHandle_t handle, cusparseConstSpVecDescr_t vecX, cusparseDnVecDescr_t vecY);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseScatter(hipsparseHandle_t handle, hipsparseSpVecDescr_t vecX, hipsparseDnVecDescr_t vecY);
  // CHECK: status_t = hipsparseScatter(handle_t, spVecDescr_t, vecY);
  status_t = hipsparseScatter(handle_t, spVecDescr_t, vecY);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseGather(cusparseHandle_t handle, cusparseConstDnVecDescr_t vecY, cusparseSpVecDescr_t vecX);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseGather(hipsparseHandle_t handle, hipsparseDnVecDescr_t vecY, hipsparseSpVecDescr_t vecX);
  // CHECK: status_t = hipsparseGather(handle_t, vecY, spVecDescr_t);
  status_t = hipsparseGather(handle_t, vecY, spVecDescr_t);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseAxpby(cusparseHandle_t handle, const void* alpha, cusparseConstSpVecDescr_t vecX, const void* beta, cusparseDnVecDescr_t vecY);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseAxpby(hipsparseHandle_t handle, const void* alpha, hipsparseSpVecDescr_t vecX, const void* beta, hipsparseDnVecDescr_t vecY);
  // CHECK: status_t = hipsparseAxpby(handle_t, alpha, spVecDescr_t, beta, vecY);
  status_t = hipsparseAxpby(handle_t, alpha, spVecDescr_t, beta, vecY);
#endif

#if CUDA_VERSION >= 11010 && CUSPARSE_VERSION >= 11300
  // CHECK: hipsparseSparseToDenseAlg_t sparseToDenseAlg_t;
  // CHECK-NEXT: hipsparseSparseToDenseAlg_t SPARSETODENSE_ALG_DEFAULT = HIPSPARSE_SPARSETODENSE_ALG_DEFAULT;
  hipsparseSparseToDenseAlg_t sparseToDenseAlg_t;
  hipsparseSparseToDenseAlg_t SPARSETODENSE_ALG_DEFAULT = HIPSPARSE_SPARSETODENSE_ALG_DEFAULT;

  // CHECK: hipsparseDenseToSparseAlg_t denseToSparseAlg_t;
  // CHECK-NEXT: hipsparseDenseToSparseAlg_t DENSETOSPARSE_ALG_DEFAULT = HIPSPARSE_DENSETOSPARSE_ALG_DEFAULT;
  hipsparseDenseToSparseAlg_t denseToSparseAlg_t;
  hipsparseDenseToSparseAlg_t DENSETOSPARSE_ALG_DEFAULT = HIPSPARSE_DENSETOSPARSE_ALG_DEFAULT;

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseCreateCsc(cusparseSpMatDescr_t* spMatDescr, int64_t rows, int64_t cols, int64_t nnz, void* cscColOffsets, void* cscRowInd, void* cscValues, cusparseIndexType_t cscColOffsetsType, cusparseIndexType_t cscRowIndType, cusparseIndexBase_t idxBase, cudaDataType valueType);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseCreateCsc(hipsparseSpMatDescr_t* spMatDescr, int64_t rows, int64_t cols, int64_t nnz, void* cscColOffsets, void* cscRowInd, void* cscValues, hipsparseIndexType_t cscColOffsetsType, hipsparseIndexType_t cscRowIndType, hipsparseIndexBase_t idxBase, hipDataType valueType);
  // CHECK: status_t = hipsparseCreateCsc(&spMatDescr_t, rows, cols, nnz, cscColOffsets, cscRowInd, cscValues, cscColOffsetsType, csrColIndType, indexBase_t, dataType);
  status_t = hipsparseCreateCsc(&spMatDescr_t, rows, cols, nnz, cscColOffsets, cscRowInd, cscValues, cscColOffsetsType, csrColIndType, indexBase_t, dataType);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseCooSetPointers(cusparseSpMatDescr_t spMatDescr, void* cooRows, void* cooColumns, void* cooValues);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseCooSetPointers(hipsparseSpMatDescr_t spMatDescr, void* cooRowInd, void* cooColInd, void* cooValues);
  // CHECK: status_t = hipsparseCooSetPointers(spMatDescr_t, cooRows, cooColumns, cooValues);
  status_t = hipsparseCooSetPointers(spMatDescr_t, cooRows, cooColumns, cooValues);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseCscSetPointers(cusparseSpMatDescr_t spMatDescr, void* cscColOffsets, void* cscRowInd, void* cscValues);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseCscSetPointers(hipsparseSpMatDescr_t spMatDescr, void* cscColOffsets, void* cscRowInd, void* cscValues);
  // CHECK: status_t = hipsparseCscSetPointers(spMatDescr_t, cscColOffsets, cscRowInd, cscValues);
  status_t = hipsparseCscSetPointers(spMatDescr_t, cscColOffsets, cscRowInd, cscValues);
#endif

#if CUDA_VERSION >= 11020 && CUSPARSE_VERSION >= 11400
  // CHECK: hipsparseFormat_t FORMAT_BLOCKED_ELL = HIPSPARSE_FORMAT_BLOCKED_ELL;
  hipsparseFormat_t FORMAT_BLOCKED_ELL = HIPSPARSE_FORMAT_BLOCKED_ELL;

  // CHECK: hipsparseSpMVAlg_t SPMV_ALG_DEFAULT = HIPSPARSE_SPMV_ALG_DEFAULT;
  // CHECK-NEXT: hipsparseSpMVAlg_t SPMV_COO_ALG1 = HIPSPARSE_SPMV_COO_ALG1;
  // CHECK-NEXT: hipsparseSpMVAlg_t SPMV_COO_ALG2 = HIPSPARSE_SPMV_COO_ALG2;
  // CHECK-NEXT: hipsparseSpMVAlg_t SPMV_CSR_ALG1 = HIPSPARSE_SPMV_CSR_ALG1;
  // CHECK-NEXT: hipsparseSpMVAlg_t SPMV_CSR_ALG2 = HIPSPARSE_SPMV_CSR_ALG2;
  hipsparseSpMVAlg_t SPMV_ALG_DEFAULT = HIPSPARSE_SPMV_ALG_DEFAULT;
  hipsparseSpMVAlg_t SPMV_COO_ALG1 = HIPSPARSE_SPMV_COO_ALG1;
  hipsparseSpMVAlg_t SPMV_COO_ALG2 = HIPSPARSE_SPMV_COO_ALG2;
  hipsparseSpMVAlg_t SPMV_CSR_ALG1 = HIPSPARSE_SPMV_CSR_ALG1;
  hipsparseSpMVAlg_t SPMV_CSR_ALG2 = HIPSPARSE_SPMV_CSR_ALG2;

  // CHECK: hipsparseSpMMAlg_t SPMM_CSR_ALG3 = HIPSPARSE_SPMM_CSR_ALG3;
  // CHECK-NEXT: hipsparseSpMMAlg_t SPMM_BLOCKED_ELL_ALG1 = HIPSPARSE_SPMM_BLOCKED_ELL_ALG1;
  hipsparseSpMMAlg_t SPMM_CSR_ALG3 = HIPSPARSE_SPMM_CSR_ALG3;
  hipsparseSpMMAlg_t SPMM_BLOCKED_ELL_ALG1 = HIPSPARSE_SPMM_BLOCKED_ELL_ALG1;

  // CHECK: hipsparseSDDMMAlg_t sDDMMAlg_t;
  // CHECK-NEXT: hipsparseSDDMMAlg_t SDDMM_ALG_DEFAULT = HIPSPARSE_SDDMM_ALG_DEFAULT;
  hipsparseSDDMMAlg_t sDDMMAlg_t;
  hipsparseSDDMMAlg_t SDDMM_ALG_DEFAULT = HIPSPARSE_SDDMM_ALG_DEFAULT;

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseCreateBlockedEll(cusparseSpMatDescr_t* spMatDescr, int64_t rows, int64_t cols, int64_t ellBlockSize, int64_t ellCols, void* ellColInd, void* ellValue, cusparseIndexType_t ellIdxType, cusparseIndexBase_t idxBase, cudaDataType valueType);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseCreateBlockedEll(hipsparseSpMatDescr_t* spMatDescr, int64_t rows, int64_t cols, int64_t ellBlockSize, int64_t ellCols, void* ellColInd, void* ellValue, hipsparseIndexType_t ellIdxType, hipsparseIndexBase_t idxBase, hipDataType valueType);
  // CHECK: status_t = hipsparseCreateBlockedEll(&spMatDescr_t, rows, cols, ellBlockSize, ellCols, ellColInd, ellValue, ellIdxType, indexBase_t, dataType);
  status_t = hipsparseCreateBlockedEll(&spMatDescr_t, rows, cols, ellBlockSize, ellCols, ellColInd, ellValue, ellIdxType, indexBase_t, dataType);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseBlockedEllGet(cusparseSpMatDescr_t spMatDescr, int64_t* rows, int64_t* cols, int64_t* ellBlockSize, int64_t* ellCols, void** ellColInd, void** ellValue, cusparseIndexType_t* ellIdxType, cusparseIndexBase_t* idxBase, cudaDataType* valueType);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseBlockedEllGet(const hipsparseSpMatDescr_t spMatDescr, int64_t* rows, int64_t* cols, int64_t* ellBlockSize, int64_t* ellCols, void** ellColInd, void** ellValue, hipsparseIndexType_t* ellIdxType, hipsparseIndexBase_t* idxBase, hipDataType* valueType);
  // CHECK: status_t = hipsparseBlockedEllGet(spMatDescr_t, &rows, &cols, &ellBlockSize, &ellCols, &ellColInd, &ellValue, &ellIdxType, &indexBase_t, &dataType);
  status_t = hipsparseBlockedEllGet(spMatDescr_t, &rows, &cols, &ellBlockSize, &ellCols, &ellColInd, &ellValue, &ellIdxType, &indexBase_t, &dataType);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseSDDMM_preprocess(cusparseHandle_t handle, cusparseOperation_t opA, cusparseOperation_t opB, const void* alpha, cusparseConstDnMatDescr_t matA, cusparseConstDnMatDescr_t matB, const void* beta, cusparseSpMatDescr_t matC, cudaDataType computeType, cusparseSDDMMAlg_t alg, void* externalBuffer);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseSDDMM_preprocess(hipsparseHandle_t handle, hipsparseOperation_t opA, hipsparseOperation_t opB, const void* alpha, const hipsparseDnMatDescr_t A, const hipsparseDnMatDescr_t B, const void* beta, hipsparseSpMatDescr_t C, hipDataType computeType, hipsparseSDDMMAlg_t alg, void* tempBuffer);
  // CHECK: status_t = hipsparseSDDMM_preprocess(handle_t, opA, opB, alpha, matA, matB, beta, matC, dataType, sDDMMAlg_t, tempBuffer);
 status_t = hipsparseSDDMM_preprocess(handle_t, opA, opB, alpha, matA, matB, beta, matC, dataType, sDDMMAlg_t, tempBuffer);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseSDDMM_bufferSize(cusparseHandle_t handle, cusparseOperation_t opA, cusparseOperation_t opB, const void* alpha, cusparseConstDnMatDescr_t matA, cusparseConstDnMatDescr_t matB, const void* beta, cusparseSpMatDescr_t matC, cudaDataType computeType, cusparseSDDMMAlg_t alg, size_t* bufferSize);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseSDDMM_bufferSize(hipsparseHandle_t handle, hipsparseOperation_t opA, hipsparseOperation_t opB, const void* alpha, const hipsparseDnMatDescr_t A, const hipsparseDnMatDescr_t B, const void* beta, hipsparseSpMatDescr_t C, hipDataType computeType, hipsparseSDDMMAlg_t alg, size_t* bufferSize);
  // CHECK: status_t = hipsparseSDDMM_bufferSize(handle_t, opA, opB, alpha, matA, matB, beta, matC, dataType, sDDMMAlg_t, &bufferSize);
 status_t = hipsparseSDDMM_bufferSize(handle_t, opA, opB, alpha, matA, matB, beta, matC, dataType, sDDMMAlg_t, &bufferSize);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseSDDMM(cusparseHandle_t handle, cusparseOperation_t opA, cusparseOperation_t opB, const void* alpha, cusparseConstDnMatDescr_t matA, cusparseConstDnMatDescr_t matB, const void* beta, cusparseSpMatDescr_t matC, cudaDataType computeType, cusparseSDDMMAlg_t alg, void* externalBuffer);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseSDDMM(hipsparseHandle_t handle, hipsparseOperation_t opA, hipsparseOperation_t opB, const void* alpha, const hipsparseDnMatDescr_t A, const hipsparseDnMatDescr_t B, const void* beta, hipsparseSpMatDescr_t C, hipDataType computeType, hipsparseSDDMMAlg_t alg, void* tempBuffer);
  // CHECK: status_t = hipsparseSDDMM(handle_t, opA, opB, alpha, matA, matB, beta, matC, dataType, sDDMMAlg_t, tempBuffer);
 status_t = hipsparseSDDMM(handle_t, opA, opB, alpha, matA, matB, beta, matC, dataType, sDDMMAlg_t, tempBuffer);
#endif

#if CUDA_VERSION >= 11030
  // CHECK: hipsparseSpMatAttribute_t spMatAttribute_t;
  // CHECK-NEXT: hipsparseSpMatAttribute_t SPMAT_FILL_MODE = HIPSPARSE_SPMAT_FILL_MODE;
  // CHECK-NEXT: hipsparseSpMatAttribute_t SPMAT_DIAG_TYPE = HIPSPARSE_SPMAT_DIAG_TYPE;
  hipsparseSpMatAttribute_t spMatAttribute_t;
  hipsparseSpMatAttribute_t SPMAT_FILL_MODE = HIPSPARSE_SPMAT_FILL_MODE;
  hipsparseSpMatAttribute_t SPMAT_DIAG_TYPE = HIPSPARSE_SPMAT_DIAG_TYPE;

  // CHECK: hipsparseSpSVAlg_t spSVAlg_t;
  // CHECK-NEXT: hipsparseSpSVAlg_t SPSV_ALG_DEFAULT = HIPSPARSE_SPSV_ALG_DEFAULT;
  hipsparseSpSVAlg_t spSVAlg_t;
  hipsparseSpSVAlg_t SPSV_ALG_DEFAULT = HIPSPARSE_SPSV_ALG_DEFAULT;

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseSpMatGetAttribute(cusparseConstSpMatDescr_t spMatDescr, cusparseSpMatAttribute_t attribute, void* data, size_t dataSize);
  // HIP: hipsparseStatus_t hipsparseSpMatGetAttribute(hipsparseSpMatDescr_t spMatDescr, hipsparseSpMatAttribute_t attribute, void* data, size_t dataSize);
  // CHECK: status_t = hipsparseSpMatGetAttribute(spMatDescr_t, spMatAttribute_t, &data, dataSize);
  status_t = hipsparseSpMatGetAttribute(spMatDescr_t, spMatAttribute_t, &data, dataSize);

  // CUDA: cusparseStatus_t CUSPARSEAPI cusparseSpMatSetAttribute(cusparseSpMatDescr_t spMatDescr, cusparseSpMatAttribute_t attribute, void* data, size_t dataSize);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseSpMatSetAttribute(hipsparseSpMatDescr_t spMatDescr, hipsparseSpMatAttribute_t attribute, const void* data, size_t dataSize);
  // CHECK: status_t = hipsparseSpMatSetAttribute(spMatDescr_t, spMatAttribute_t, &data, dataSize);
  status_t = hipsparseSpMatSetAttribute(spMatDescr_t, spMatAttribute_t, &data, dataSize);
#endif

#if CUDA_VERSION >= 11030 && CUSPARSE_VERSION >= 11600
  // CHECK: hipsparseSpSMAlg_t spSMAlg_t;
  // CHECK-NEXT: hipsparseSpSMAlg_t SPSM_ALG_DEFAULT = HIPSPARSE_SPSM_ALG_DEFAULT;
  hipsparseSpSMAlg_t spSMAlg_t;
  hipsparseSpSMAlg_t SPSM_ALG_DEFAULT = HIPSPARSE_SPSM_ALG_DEFAULT;

  // CHECK: hipsparseSpGEMMAlg_t SPGEMM_CSR_ALG_DETERMINITIC = HIPSPARSE_SPGEMM_CSR_ALG_DETERMINISTIC;
  // CHECK-NEXT: hipsparseSpGEMMAlg_t SPGEMM_CSR_ALG_NONDETERMINITIC = HIPSPARSE_SPGEMM_CSR_ALG_NONDETERMINISTIC;
  hipsparseSpGEMMAlg_t SPGEMM_CSR_ALG_DETERMINITIC = HIPSPARSE_SPGEMM_CSR_ALG_DETERMINISTIC;
  hipsparseSpGEMMAlg_t SPGEMM_CSR_ALG_NONDETERMINITIC = HIPSPARSE_SPGEMM_CSR_ALG_NONDETERMINISTIC;
#endif

#if CUDA_VERSION < 12000
  // CUDA: CUSPARSE_DEPRECATED(cusparseSparseToDense) cusparseStatus_t CUSPARSEAPI cusparseZcsc2dense(cusparseHandle_t handle, int m, int n, const cusparseMatDescr_t descrA, const cuDoubleComplex* cscSortedValA, const int* cscSortedRowIndA, const int* cscSortedColPtrA, cuDoubleComplex* A, int lda);
  // HIP: DEPRECATED_CUDA_11000("The routine will be removed in CUDA 12") HIPSPARSE_EXPORT hipsparseStatus_t hipsparseZcsc2dense(hipsparseHandle_t handle, int m, int n, const hipsparseMatDescr_t descr, const hipDoubleComplex* csc_val, const int* csc_row_ind, const int* csc_col_ptr, hipDoubleComplex* A, int ld);
  // CHECK: status_t = hipsparseZcsc2dense(handle_t, m, n, matDescr_A, &dComplexcscSortedVal, &csrSortedRowPtr, &csrSortedColInd, &dcomplexA, lda);
  status_t = cusparseZcsc2dense(handle_t, m, n, matDescr_A, &dComplexcscSortedVal, &csrSortedRowPtr, &csrSortedColInd, &dcomplexA, lda);

  // CUDA: CUSPARSE_DEPRECATED(cusparseSparseToDense) cusparseStatus_t CUSPARSEAPI cusparseCcsc2dense(cusparseHandle_t handle, int m, int n, const cusparseMatDescr_t descrA, const cuComplex* cscSortedValA, const int* cscSortedRowIndA, const int* cscSortedColPtrA, cuComplex* A, int lda);
  // HIP: DEPRECATED_CUDA_11000("The routine will be removed in CUDA 12") HIPSPARSE_EXPORT hipsparseStatus_t hipsparseCcsc2dense(hipsparseHandle_t handle, int m, int n, const hipsparseMatDescr_t descr, const hipComplex* csc_val, const int* csc_row_ind, const int* csc_col_ptr, hipComplex* A, int ld);
  // CHECK: status_t = hipsparseCcsc2dense(handle_t, m, n, matDescr_A, &complexcscSortedVal, &csrSortedRowPtr, &csrSortedColInd, &complexA, lda);
  status_t = cusparseCcsc2dense(handle_t, m, n, matDescr_A, &complexcscSortedVal, &csrSortedRowPtr, &csrSortedColInd, &complexA, lda);

  // CUDA: CUSPARSE_DEPRECATED(cusparseSparseToDense) cusparseStatus_t CUSPARSEAPI cusparseDcsc2dense(cusparseHandle_t handle, int m, int n, const cusparseMatDescr_t descrA, const double* cscSortedValA, const int* cscSortedRowIndA, const int* cscSortedColPtrA, double* A, int lda);
  // HIP: DEPRECATED_CUDA_11000("The routine will be removed in CUDA 12") HIPSPARSE_EXPORT hipsparseStatus_t hipsparseDcsc2dense(hipsparseHandle_t handle, int m, int n, const hipsparseMatDescr_t descr, const double* csc_val, const int* csc_row_ind, const int* csc_col_ptr, double* A, int ld);
  // CHECK: status_t = hipsparseDcsc2dense(handle_t, m, n, matDescr_A, &dcscSortedVal, &csrSortedRowPtr, &csrSortedColInd, &dA, lda);
  status_t = cusparseDcsc2dense(handle_t, m, n, matDescr_A, &dcscSortedVal, &csrSortedRowPtr, &csrSortedColInd, &dA, lda);

  // CUDA: CUSPARSE_DEPRECATED(cusparseSparseToDense) cusparseStatus_t CUSPARSEAPI cusparseScsc2dense(cusparseHandle_t handle, int m, int n, const cusparseMatDescr_t descrA, const float* cscSortedValA, const int* cscSortedRowIndA, const int* cscSortedColPtrA, float* A, int lda);
  // HIP: DEPRECATED_CUDA_11000("The routine will be removed in CUDA 12") HIPSPARSE_EXPORT hipsparseStatus_t hipsparseScsc2dense(hipsparseHandle_t handle, int m, int n, const hipsparseMatDescr_t descr, const float* csc_val, const int* csc_row_ind, const int* csc_col_ptr, float* A, int ld);
  // CHECK: status_t = hipsparseScsc2dense(handle_t, m, n, matDescr_A, &cscSortedVal, &csrSortedRowPtr, &csrSortedColInd, &fA, lda);
  status_t = cusparseScsc2dense(handle_t, m, n, matDescr_A, &cscSortedVal, &csrSortedRowPtr, &csrSortedColInd, &fA, lda);

  // CUDA: CUSPARSE_DEPRECATED(cusparseSparseToDense) cusparseStatus_t CUSPARSEAPI cusparseZcsr2dense(cusparseHandle_t handle, int m, int n, const cusparseMatDescr_t descrA, const cuDoubleComplex* csrSortedValA, const int* csrSortedRowPtrA, const int* csrSortedColIndA, cuDoubleComplex* A, int lda);
  // HIP: DEPRECATED_CUDA_11000("The routine will be removed in CUDA 12") HIPSPARSE_EXPORT hipsparseStatus_t hipsparseZcsr2dense(hipsparseHandle_t handle, int m, int n, const hipsparseMatDescr_t descr, const hipDoubleComplex* csr_val, const int* csr_row_ptr, const int* csr_col_ind, hipDoubleComplex* A, int ld);
  // CHECK: status_t = hipsparseZcsr2dense(handle_t, m, n, matDescr_A, &dComplexcscSortedVal, &csrSortedRowPtr, &csrSortedColInd, &dcomplexA, lda);
  status_t = cusparseZcsr2dense(handle_t, m, n, matDescr_A, &dComplexcscSortedVal, &csrSortedRowPtr, &csrSortedColInd, &dcomplexA, lda);

  // CUDA: CUSPARSE_DEPRECATED(cusparseSparseToDense) cusparseStatus_t CUSPARSEAPI cusparseCcsr2dense(cusparseHandle_t handle, int m, int n, const cusparseMatDescr_t descrA, const cuComplex* csrSortedValA, const int* csrSortedRowPtrA, const int* csrSortedColIndA, cuComplex* A, int lda);
  // HIP: DEPRECATED_CUDA_11000("The routine will be removed in CUDA 12") HIPSPARSE_EXPORT hipsparseStatus_t hipsparseCcsr2dense(hipsparseHandle_t handle, int m, int n, const hipsparseMatDescr_t descr, const hipComplex* csr_val, const int* csr_row_ptr, const int* csr_col_ind, hipComplex* A, int ld);
  // CHECK: status_t = hipsparseCcsr2dense(handle_t, m, n, matDescr_A, &complexcscSortedVal, &csrSortedRowPtr, &csrSortedColInd, &complexA, lda);
  status_t = cusparseCcsr2dense(handle_t, m, n, matDescr_A, &complexcscSortedVal, &csrSortedRowPtr, &csrSortedColInd, &complexA, lda);

  // CUDA: CUSPARSE_DEPRECATED(cusparseSparseToDense) cusparseStatus_t CUSPARSEAPI cusparseDcsr2dense(cusparseHandle_t handle, int m, int n, const cusparseMatDescr_t descrA, const double* csrSortedValA, const int* csrSortedRowPtrA, const int* csrSortedColIndA, double* A, int lda);
  // HIP: DEPRECATED_CUDA_11000("The routine will be removed in CUDA 12") HIPSPARSE_EXPORT hipsparseStatus_t hipsparseDcsr2dense(hipsparseHandle_t handle, int m, int n, const hipsparseMatDescr_t descr, const double* csr_val, const int* csr_row_ptr, const int* csr_col_ind, double* A, int ld);
  // CHECK: status_t = hipsparseDcsr2dense(handle_t, m, n, matDescr_A, &dcscSortedVal, &csrSortedRowPtr, &csrSortedColInd, &dA, lda);
  status_t = cusparseDcsr2dense(handle_t, m, n, matDescr_A, &dcscSortedVal, &csrSortedRowPtr, &csrSortedColInd, &dA, lda);

  // CUDA: CUSPARSE_DEPRECATED(cusparseSparseToDense) cusparseStatus_t CUSPARSEAPI cusparseScsr2dense(cusparseHandle_t handle, int m, int n, const cusparseMatDescr_t descrA, const float* csrSortedValA, const int* csrSortedRowPtrA, const int* csrSortedColIndA, float* A, int lda);
  // HIP: DEPRECATED_CUDA_11000("The routine will be removed in CUDA 12") HIPSPARSE_EXPORT hipsparseStatus_t hipsparseScsr2dense(hipsparseHandle_t handle, int m, int n, const hipsparseMatDescr_t descr, const float* csr_val, const int* csr_row_ptr, const int* csr_col_ind, float* A, int ld);
  // CHECK: status_t = hipsparseScsr2dense(handle_t, m, n, matDescr_A, &cscSortedVal, &csrSortedRowPtr, &csrSortedColInd, &fA, lda);
  status_t = cusparseScsr2dense(handle_t, m, n, matDescr_A, &cscSortedVal, &csrSortedRowPtr, &csrSortedColInd, &fA, lda);

  // CUDA: CUSPARSE_DEPRECATED(cusparseDenseToSparse) cusparseStatus_t CUSPARSEAPI cusparseZdense2csc(cusparseHandle_t handle, int m, int n, const cusparseMatDescr_t descrA, const cuDoubleComplex* A, int lda, const int* nnzPerCol, cuDoubleComplex* cscSortedValA, int* cscSortedRowIndA, int* cscSortedColPtrA);
  // HIP: DEPRECATED_CUDA_11000("The routine will be removed in CUDA 12") HIPSPARSE_EXPORT hipsparseStatus_t hipsparseZdense2csc(hipsparseHandle_t handle, int m, int n, const hipsparseMatDescr_t descr, const hipDoubleComplex* A, int ld, const int* nnz_per_columns, hipDoubleComplex* csc_val, int* csc_row_ind, int* csc_col_ptr);
  // CHECK: status_t = hipsparseZdense2csc(handle_t, m, n, matDescr_A, &dcomplexA, lda, &nnzPerCol, &dComplexcscSortedVal, &csrSortedRowPtr, &csrSortedColInd);
  status_t = cusparseZdense2csc(handle_t, m, n, matDescr_A, &dcomplexA, lda, &nnzPerCol, &dComplexcscSortedVal, &csrSortedRowPtr, &csrSortedColInd);

  // CUDA: CUSPARSE_DEPRECATED(cusparseDenseToSparse) cusparseStatus_t CUSPARSEAPI cusparseCdense2csc(cusparseHandle_t handle, int m, int n, const cusparseMatDescr_t descrA, const cuComplex* A, int lda, const int* nnzPerCol, cuComplex* cscSortedValA, int* cscSortedRowIndA, int* cscSortedColPtrA);
  // HIP: DEPRECATED_CUDA_11000("The routine will be removed in CUDA 12") HIPSPARSE_EXPORT hipsparseStatus_t hipsparseCdense2csc(hipsparseHandle_t handle, int m, int n, const hipsparseMatDescr_t descr, const hipComplex* A, int ld, const int* nnz_per_columns, hipComplex* csc_val, int* csc_row_ind, int* csc_col_ptr);
  // CHECK: status_t = hipsparseCdense2csc(handle_t, m, n, matDescr_A, &complexA, lda, &nnzPerCol, &complexcscSortedVal, &csrSortedRowPtr, &csrSortedColInd);
  status_t = cusparseCdense2csc(handle_t, m, n, matDescr_A, &complexA, lda, &nnzPerCol, &complexcscSortedVal, &csrSortedRowPtr, &csrSortedColInd);

  // CUDA: CUSPARSE_DEPRECATED(cusparseDenseToSparse) cusparseStatus_t CUSPARSEAPI cusparseDdense2csc(cusparseHandle_t handle, int m, int n, const cusparseMatDescr_t descrA, const double* A, int lda, const int* nnzPerCol, double* cscSortedValA, int* cscSortedRowIndA, int* cscSortedColPtrA);
  // HIP: DEPRECATED_CUDA_11000("The routine will be removed in CUDA 12") HIPSPARSE_EXPORT hipsparseStatus_t hipsparseDdense2csc(hipsparseHandle_t handle, int m, int n, const hipsparseMatDescr_t descr, const double* A, int ld, const int* nnz_per_columns, double* csc_val, int* csc_row_ind, int* csc_col_ptr);
  // CHECK: status_t = hipsparseDdense2csc(handle_t, m, n, matDescr_A, &dA, lda, &nnzPerCol, &dcscSortedVal, &csrSortedRowPtr, &csrSortedColInd);
  status_t = cusparseDdense2csc(handle_t, m, n, matDescr_A, &dA, lda, &nnzPerCol, &dcscSortedVal, &csrSortedRowPtr, &csrSortedColInd);

  // CUDA: CUSPARSE_DEPRECATED(cusparseDenseToSparse) cusparseStatus_t CUSPARSEAPI cusparseSdense2csc(cusparseHandle_t handle, int m, int n, const cusparseMatDescr_t descrA, const float* A, int lda, const int* nnzPerCol, float* cscSortedValA, int* cscSortedRowIndA, int* cscSortedColPtrA);
  // HIP: DEPRECATED_CUDA_11000("The routine will be removed in CUDA 12") HIPSPARSE_EXPORT hipsparseStatus_t hipsparseSdense2csc(hipsparseHandle_t handle, int m, int n, const hipsparseMatDescr_t descr, const float* A, int ld, const int* nnz_per_columns, float* csc_val, int* csc_row_ind, int* csc_col_ptr);
  // CHECK: status_t = hipsparseSdense2csc(handle_t, m, n, matDescr_A, &fA, lda, &nnzPerCol, &cscSortedVal, &csrSortedRowPtr, &csrSortedColInd);
  status_t = cusparseSdense2csc(handle_t, m, n, matDescr_A, &fA, lda, &nnzPerCol, &cscSortedVal, &csrSortedRowPtr, &csrSortedColInd);

  // CUDA: CUSPARSE_DEPRECATED(cusparseDenseToSparse) cusparseStatus_t CUSPARSEAPI cusparseZdense2csr(cusparseHandle_t handle, int m, int n, const cusparseMatDescr_t descrA, const cuDoubleComplex* A, int lda, const int* nnzPerRow, cuDoubleComplex* csrSortedValA, int* csrSortedRowPtrA, int* csrSortedColIndA);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseZdense2csr(hipsparseHandle_t handle, int m, int n, const hipsparseMatDescr_t descr, const hipDoubleComplex* A, int ld, const int* nnz_per_rows, hipDoubleComplex* csr_val, int* csr_row_ptr, int* csr_col_ind);
  // CHECK: status_t = hipsparseZdense2csr(handle_t, m, n, matDescr_A, &dcomplexA, lda, &nnzPerRow, &dComplexcsrSortedValA, &csrRowPtrA, &csrColIndA);
  status_t = cusparseZdense2csr(handle_t, m, n, matDescr_A, &dcomplexA, lda, &nnzPerRow, &dComplexcsrSortedValA, &csrRowPtrA, &csrColIndA);

  // CUDA: CUSPARSE_DEPRECATED(cusparseDenseToSparse) cusparseStatus_t CUSPARSEAPI cusparseCdense2csr(cusparseHandle_t handle, int m, int n, const cusparseMatDescr_t descrA, const cuComplex* A, int lda, const int* nnzPerRow, cuComplex* csrSortedValA, int* csrSortedRowPtrA, int* csrSortedColIndA);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseCdense2csr(hipsparseHandle_t handle, int m, int n, const hipsparseMatDescr_t descr, const hipComplex* A, int ld, const int* nnz_per_rows, hipComplex* csr_val, int* csr_row_ptr, int* csr_col_ind);
  // CHECK: status_t = hipsparseCdense2csr(handle_t, m, n, matDescr_A, &complexA, lda, &nnzPerRow, &complexcsrSortedValA, &csrRowPtrA, &csrColIndA);
  status_t = cusparseCdense2csr(handle_t, m, n, matDescr_A, &complexA, lda, &nnzPerRow, &complexcsrSortedValA, &csrRowPtrA, &csrColIndA);

  // CUDA: CUSPARSE_DEPRECATED(cusparseDenseToSparse) cusparseStatus_t CUSPARSEAPI cusparseDdense2csr(cusparseHandle_t handle, int m, int n, const cusparseMatDescr_t descrA, const double* A, int lda, const int* nnzPerRow, double* csrSortedValA, int* csrSortedRowPtrA, int* csrSortedColIndA);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseDdense2csr(hipsparseHandle_t handle, int m, int n, const hipsparseMatDescr_t descr, const double* A, int ld, const int* nnz_per_rows, double* csr_val, int* csr_row_ptr, int* csr_col_ind);
  // CHECK: status_t = hipsparseDdense2csr(handle_t, m, n, matDescr_A, &dA, lda, &nnzPerRow, &dcsrSortedValA, &csrRowPtrA, &csrColIndA);
  status_t = cusparseDdense2csr(handle_t, m, n, matDescr_A, &dA, lda, &nnzPerRow, &dcsrSortedValA, &csrRowPtrA, &csrColIndA);

  // CUDA: CUSPARSE_DEPRECATED(cusparseDenseToSparse) cusparseStatus_t CUSPARSEAPI cusparseSdense2csr(cusparseHandle_t handle, int m, int n, const cusparseMatDescr_t descrA, const float* A, int lda, const int* nnzPerRow, float* csrSortedValA, int* csrSortedRowPtrA, int* csrSortedColIndA);
  // HIP: HIPSPARSE_EXPORT hipsparseStatus_t hipsparseSdense2csr(hipsparseHandle_t handle, int m, int n, const hipsparseMatDescr_t descr, const float* A, int ld, const int* nnz_per_rows, float* csr_val, int* csr_row_ptr, int* csr_col_ind);
  // CHECK: status_t = hipsparseSdense2csr(handle_t, m, n, matDescr_A, &fA, lda, &nnzPerRow, &csrSortedValA, &csrRowPtrA, &csrColIndA);
  status_t = cusparseSdense2csr(handle_t, m, n, matDescr_A, &fA, lda, &nnzPerRow, &csrSortedValA, &csrRowPtrA, &csrColIndA);
#endif

#if CUDA_VERSION >= 12000
  // CHECK: hipsparseCsr2CscAlg_t CSR2CSC_ALG_DEFAULT = HIPSPARSE_CSR2CSC_ALG_DEFAULT;
  hipsparseCsr2CscAlg_t CSR2CSC_ALG_DEFAULT = HIPSPARSE_CSR2CSC_ALG_DEFAULT;

  // CHECK: hipsparseSpGEMMAlg_t SPGEMM_ALG1 = HIPSPARSE_SPGEMM_ALG1;
  // CHECK: hipsparseSpGEMMAlg_t SPGEMM_ALG2 = HIPSPARSE_SPGEMM_ALG2;
  // CHECK: hipsparseSpGEMMAlg_t SPGEMM_ALG3 = HIPSPARSE_SPGEMM_ALG3;
  hipsparseSpGEMMAlg_t SPGEMM_ALG1 = HIPSPARSE_SPGEMM_ALG1;
  hipsparseSpGEMMAlg_t SPGEMM_ALG2 = HIPSPARSE_SPGEMM_ALG2;
  hipsparseSpGEMMAlg_t SPGEMM_ALG3 = HIPSPARSE_SPGEMM_ALG3;
#endif

  return 0;
}
